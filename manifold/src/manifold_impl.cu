#include "hip/hip_runtime.h"
// Copyright 2021 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/adjacent_difference.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>
#include <thrust/gather.h>
#include <thrust/logical.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>

#include <algorithm>
#include <map>
#include <set>

#include "connected_components.cuh"
#include "manifold_impl.cuh"
#include "polygon.h"

namespace {
using namespace manifold;

constexpr uint32_t kNoCode = 0xFFFFFFFFu;

__host__ __device__ glm::vec3 SafeNormalize(glm::vec3 v) {
  v = glm::normalize(v);
  return isfinite(v.x) ? v : glm::vec3(0);
}

__host__ __device__ glm::vec3 OrthogonalTo(glm::vec3 in, glm::vec3 ref) {
  in -= glm::dot(in, ref) * ref;
  return in;
}

__host__ __device__ int NextHalfedge(int current) {
  ++current;
  if (current % 3 == 0) current -= 3;
  return current;
}

/**
 * The total number of verts if a triangle is subdivided naturally such that
 * each edge has edgeVerts verts along it (edgeVerts >= -1).
 */
__host__ __device__ int VertsPerTri(int edgeVerts) {
  return (edgeVerts * edgeVerts + edgeVerts) / 2;
}

__host__ __device__ void AtomicAddVec3(glm::vec3& target,
                                       const glm::vec3& add) {
  for (int i : {0, 1, 2}) {
#ifdef __CUDA_ARCH__
    atomicAdd(&target[i], add[i]);
#else
#pragma omp atomic
    target[i] += add[i];
#endif
  }
}

/**
 * By using the closest axis-aligned projection to the normal instead of a
 * projection along the normal, we avoid introducing any rounding error.
 */
__host__ __device__ glm::mat3x2 GetAxisAlignedProjection(glm::vec3 normal) {
  glm::vec3 absNormal = glm::abs(normal);
  float xyzMax;
  glm::mat2x3 projection;
  if (absNormal.z > absNormal.x && absNormal.z > absNormal.y) {
    projection = glm::mat2x3(1.0f, 0.0f, 0.0f,  //
                             0.0f, 1.0f, 0.0f);
    xyzMax = normal.z;
  } else if (absNormal.y > absNormal.x) {
    projection = glm::mat2x3(0.0f, 0.0f, 1.0f,  //
                             1.0f, 0.0f, 0.0f);
    xyzMax = normal.y;
  } else {
    projection = glm::mat2x3(0.0f, 1.0f, 0.0f,  //
                             0.0f, 0.0f, 1.0f);
    xyzMax = normal.x;
  }
  if (xyzMax < 0) projection[0] *= -1.0f;
  return glm::transpose(projection);
}

struct Barycentric {
  int tri;
  glm::vec3 uvw;
};

struct Normalize {
  __host__ __device__ void operator()(glm::vec3& v) {
    v = glm::normalize(v);
    if (isnan(v.x)) v = glm::vec3(0.0);
  }
};

/**
 * This is a temporary edge strcture which only stores edges forward and
 * references the halfedge it was created from.
 */
struct TmpEdge {
  int first, second, halfedgeIdx;

  __host__ __device__ TmpEdge() {}
  __host__ __device__ TmpEdge(int start, int end, int idx) {
    first = glm::min(start, end);
    second = glm::max(start, end);
    halfedgeIdx = idx;
  }

  __host__ __device__ bool operator<(const TmpEdge& other) const {
    return first == other.first ? second < other.second : first < other.first;
  }
};

struct Halfedge2Tmp {
  __host__ __device__ void operator()(
      thrust::tuple<TmpEdge&, const Halfedge&, int> inout) {
    const Halfedge& halfedge = thrust::get<1>(inout);
    int idx = thrust::get<2>(inout);
    if (!halfedge.IsForward()) idx = -1;

    thrust::get<0>(inout) = TmpEdge(halfedge.startVert, halfedge.endVert, idx);
  }
};

struct TmpInvalid {
  __host__ __device__ bool operator()(const TmpEdge& edge) {
    return edge.halfedgeIdx < 0;
  }
};

VecDH<TmpEdge> CreateTmpEdges(const VecDH<Halfedge>& halfedge) {
  VecDH<TmpEdge> edges(halfedge.size());
  thrust::for_each_n(zip(edges.beginD(), halfedge.beginD(), countAt(0)),
                     edges.size(), Halfedge2Tmp());
  int numEdge = thrust::remove_if(edges.beginD(), edges.endD(), TmpInvalid()) -
                edges.beginD();
  ALWAYS_ASSERT(numEdge == halfedge.size() / 2, topologyErr, "Not oriented!");
  edges.resize(numEdge);
  return edges;
}

struct ReindexEdge {
  const TmpEdge* edges;

  __host__ __device__ void operator()(int& edge) {
    edge = edges[edge].halfedgeIdx;
  }
};

struct ReindexHalfedge {
  int* half2Edge;

  __host__ __device__ void operator()(thrust::tuple<int, TmpEdge> in) {
    const int edge = thrust::get<0>(in);
    const int halfedge = thrust::get<1>(in).halfedgeIdx;

    half2Edge[halfedge] = edge;
  }
};

struct EdgeVerts {
  glm::vec3* vertPos;
  const int startIdx;
  const int n;

  __host__ __device__ void operator()(thrust::tuple<int, TmpEdge> in) {
    int edge = thrust::get<0>(in);
    TmpEdge edgeVerts = thrust::get<1>(in);

    float invTotal = 1.0f / n;
    for (int i = 1; i < n; ++i)
      vertPos[startIdx + (n - 1) * edge + i - 1] =
          (float(n - i) * vertPos[edgeVerts.first] +
           float(i) * vertPos[edgeVerts.second]) *
          invTotal;
  }
};

struct InteriorVerts {
  glm::vec3* vertPos;
  glm::vec3* uvw;
  BaryRef* triBary;
  const int startIdx;
  const int n;
  const Halfedge* halfedge;

  __host__ __device__ void operator()(int tri) {
    const float invTotal = 1.0f / n;
    int posTri = tri * n * n;
    int posBary = tri * VertsPerTri(n + 1);
    int pos = startIdx + tri * VertsPerTri(n - 2);
    for (int i = 0; i <= n; ++i)
      for (int j = 0; j <= n - i; ++j) {
        const int k = n - i - j;
        const float u = invTotal * j;
        const float v = invTotal * k;
        const float w = invTotal * i;
        const int first = posBary;
        uvw[posBary++] = {u, v, w};
        if (j == n - i) continue;

        // The three retained verts are denoted -1. uvw entries are added for
        // them out of laziness of indexing only.
        const int a = (k == n) ? -1 : first;
        const int b = (i == n - 1) ? -1 : first + n - i + 1;
        const int c = (j == n - 1) ? -1 : first + 1;
        triBary[posTri++] = {tri, {c, a, b}};
        if (j < n - 1 - i) {
          int d = b + 1;
          triBary[posTri++] = {tri, {b, d, c}};
        }

        if (i == 0 || j == 0 || k == 0) continue;

        vertPos[pos++] = u * vertPos[halfedge[3 * tri].startVert] +      //
                         v * vertPos[halfedge[3 * tri + 1].startVert] +  //
                         w * vertPos[halfedge[3 * tri + 2].startVert];
      }
  }
};

struct SplitTris {
  glm::ivec3* triVerts;
  const Halfedge* halfedge;
  const int* half2Edge;
  const int edgeIdx;
  const int triIdx;
  const int n;

  __host__ __device__ int EdgeVert(int i, int inHalfedge) const {
    bool forward = halfedge[inHalfedge].IsForward();
    int edge = forward ? half2Edge[inHalfedge]
                       : half2Edge[halfedge[inHalfedge].pairedHalfedge];
    return edgeIdx + (n - 1) * edge + (forward ? i - 1 : n - 1 - i);
  }

  __host__ __device__ int TriVert(int i, int j, int tri) const {
    --i;
    --j;
    int m = n - 2;
    int vertsPerTri = (m * m + m) / 2;
    int vertOffset = (i * (2 * m - i + 1)) / 2 + j;
    return triIdx + vertsPerTri * tri + vertOffset;
  }

  __host__ __device__ int Vert(int i, int j, int tri) const {
    bool edge0 = i == 0;
    bool edge1 = j == 0;
    bool edge2 = j == n - i;
    if (edge0) {
      if (edge1)
        return halfedge[3 * tri + 1].startVert;
      else if (edge2)
        return halfedge[3 * tri].startVert;
      else
        return EdgeVert(n - j, 3 * tri);
    } else if (edge1) {
      if (edge2)
        return halfedge[3 * tri + 2].startVert;
      else
        return EdgeVert(i, 3 * tri + 1);
    } else if (edge2)
      return EdgeVert(j, 3 * tri + 2);
    else
      return TriVert(i, j, tri);
  }

  __host__ __device__ void operator()(int tri) {
    int pos = n * n * tri;
    for (int i = 0; i < n; ++i) {
      for (int j = 0; j < n - i; ++j) {
        int a = Vert(i, j, tri);
        int b = Vert(i + 1, j, tri);
        int c = Vert(i, j + 1, tri);
        triVerts[pos++] = glm::ivec3(c, a, b);
        if (j < n - 1 - i) {
          int d = Vert(i + 1, j + 1, tri);
          triVerts[pos++] = glm::ivec3(b, d, c);
        }
      }
    }
  }
};

struct SmoothBezier {
  const glm::vec3* vertPos;
  const glm::vec3* triNormal;
  const glm::vec3* vertNormal;
  const Halfedge* halfedge;

  __host__ __device__ void operator()(
      thrust::tuple<glm::vec4&, Halfedge> inOut) {
    glm::vec4& tangent = thrust::get<0>(inOut);
    const Halfedge edge = thrust::get<1>(inOut);

    const glm::vec3 startV = vertPos[edge.startVert];
    const glm::vec3 edgeVec = vertPos[edge.endVert] - startV;
    const glm::vec3 edgeNormal =
        (triNormal[edge.face] + triNormal[halfedge[edge.pairedHalfedge].face]) /
        2.0f;
    glm::vec3 dir = glm::normalize(glm::cross(glm::cross(edgeNormal, edgeVec),
                                              vertNormal[edge.startVert]));

    const float weight = glm::abs(glm::dot(dir, glm::normalize(edgeVec)));
    // Quadratic weighted bezier for circular interpolation
    const glm::vec4 bz2 =
        weight *
        glm::vec4(startV + dir * glm::length(edgeVec) / (2 * weight), 1.0f);
    // Equivalent cubic weighted bezier
    const glm::vec4 bz3 = glm::mix(glm::vec4(startV, 1.0f), bz2, 2 / 3.0f);
    // Convert from homogeneous form to geometric form
    tangent = glm::vec4(glm::vec3(bz3) / bz3.w - startV, bz3.w);
  }
};

struct TriBary2Vert {
  Barycentric* vertBary;
  int* lock;
  const glm::vec3* uvw;
  const Halfedge* halfedge;

  __host__ __device__ void operator()(thrust::tuple<BaryRef, int> in) {
    const BaryRef baryRef = thrust::get<0>(in);
    const int tri = thrust::get<1>(in);

    for (int i : {0, 1, 2}) {
      int vert = halfedge[3 * tri + i].startVert;
      if (AtomicAdd(lock[vert], 1) != 0) continue;

      const int idx = baryRef.vertBary[i];
      glm::vec3 bary(0);
      if (idx < 0)
        bary[i] = 1;
      else
        bary = uvw[idx];
      vertBary[vert] = {baryRef.tri, bary};
    }
  }
};

struct InterpTri {
  const Halfedge* halfedge;
  const glm::vec4* halfedgeTangent;
  const glm::vec3* vertPos;

  __host__ __device__ glm::vec4 Homogeneous(glm::vec4 v) const {
    v.x *= v.w;
    v.y *= v.w;
    v.z *= v.w;
    return v;
  }

  __host__ __device__ glm::vec4 Homogeneous(glm::vec3 v) const {
    return glm::vec4(v, 1.0f);
  }

  __host__ __device__ glm::vec3 HNormalize(glm::vec4 v) const {
    return glm::vec3(v) / v.w;
  }

  __host__ __device__ glm::vec4 Bezier(glm::vec3 point,
                                       glm::vec4 tangent) const {
    return Homogeneous(glm::vec4(point, 0) + tangent);
  }

  __host__ __device__ glm::mat2x4 CubicBezier2Linear(glm::vec4 p0, glm::vec4 p1,
                                                     glm::vec4 p2, glm::vec4 p3,
                                                     float x) const {
    glm::mat2x4 out;
    glm::vec4 p12 = glm::mix(p1, p2, x);
    out[0] = glm::mix(glm::mix(p0, p1, x), p12, x);
    out[1] = glm::mix(p12, glm::mix(p2, p3, x), x);
    return out;
  }

  __host__ __device__ glm::vec3 BezierPoint(glm::mat2x4 points, float x) const {
    return HNormalize(glm::mix(points[0], points[1], x));
  }

  __host__ __device__ glm::vec3 BezierTangent(glm::mat2x4 points) const {
    return glm::normalize(HNormalize(points[1]) - HNormalize(points[0]));
  }

  __host__ __device__ void operator()(
      thrust::tuple<glm::vec3&, Barycentric> inOut) {
    glm::vec3& pos = thrust::get<0>(inOut);
    const int tri = thrust::get<1>(inOut).tri;
    const glm::vec3 uvw = thrust::get<1>(inOut).uvw;

    glm::vec4 posH(0);
    const glm::mat3 corners = {vertPos[halfedge[3 * tri].startVert],
                               vertPos[halfedge[3 * tri + 1].startVert],
                               vertPos[halfedge[3 * tri + 2].startVert]};

    for (const int i : {0, 1, 2}) {
      if (uvw[i] == 1) {
        pos = glm::vec3(corners[i]);
        return;
      }
    }

    const glm::mat3x4 tangentR = {halfedgeTangent[3 * tri],
                                  halfedgeTangent[3 * tri + 1],
                                  halfedgeTangent[3 * tri + 2]};
    const glm::mat3x4 tangentL = {
        halfedgeTangent[halfedge[3 * tri + 2].pairedHalfedge],
        halfedgeTangent[halfedge[3 * tri].pairedHalfedge],
        halfedgeTangent[halfedge[3 * tri + 1].pairedHalfedge]};

    for (const int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      const int k = (i + 2) % 3;
      const float x = uvw[k] / (1 - uvw[i]);

      const glm::mat2x4 bez = CubicBezier2Linear(
          Homogeneous(corners[j]), Bezier(corners[j], tangentR[j]),
          Bezier(corners[k], tangentL[k]), Homogeneous(corners[k]), x);
      const glm::vec3 end = BezierPoint(bez, x);
      const glm::vec3 tangent = BezierTangent(bez);

      const glm::vec3 jBitangent = SafeNormalize(OrthogonalTo(
          glm::vec3(tangentL[j]), SafeNormalize(glm::vec3(tangentR[j]))));
      const glm::vec3 kBitangent = SafeNormalize(OrthogonalTo(
          glm::vec3(tangentR[k]), -SafeNormalize(glm::vec3(tangentL[k]))));
      const glm::vec3 normal = SafeNormalize(
          glm::cross(glm::mix(jBitangent, kBitangent, x), tangent));
      const glm::vec3 delta = OrthogonalTo(
          glm::mix(glm::vec3(tangentL[j]), glm::vec3(tangentR[k]), x), normal);
      const float deltaW = glm::mix(tangentL[j].w, tangentR[k].w, x);

      const glm::mat2x4 bez1 = CubicBezier2Linear(
          Homogeneous(end), Homogeneous(glm::vec4(end + delta, deltaW)),
          Bezier(corners[i], glm::mix(tangentR[i], tangentL[i], x)),
          Homogeneous(corners[i]), uvw[i]);
      const glm::vec3 p = BezierPoint(bez1, uvw[i]);
      float w = uvw[j] * uvw[j] * uvw[k] * uvw[k];
      posH += Homogeneous(glm::vec4(p, w));
    }
    pos = HNormalize(posH);
  }
};

struct FaceAreaVolume {
  const Halfedge* halfedges;
  const glm::vec3* vertPos;
  const float precision;

  __host__ __device__ thrust::pair<float, float> operator()(int face) {
    float perimeter = 0;
    glm::vec3 edge[3];
    for (int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      edge[i] = vertPos[halfedges[3 * face + j].startVert] -
                vertPos[halfedges[3 * face + i].startVert];
      perimeter += glm::length(edge[i]);
    }
    glm::vec3 crossP = glm::cross(edge[0], edge[1]);

    float area = glm::length(crossP);
    float volume = glm::dot(crossP, vertPos[halfedges[3 * face].startVert]);

    return area > perimeter * precision
               ? thrust::make_pair(area / 2.0f, volume / 6.0f)
               : thrust::make_pair(0.0f, 0.0f);
  }
};

struct Extrema : public thrust::binary_function<Halfedge, Halfedge, Halfedge> {
  __host__ __device__ void MakeForward(Halfedge& a) {
    if (!a.IsForward()) {
      int tmp = a.startVert;
      a.startVert = a.endVert;
      a.endVert = tmp;
    }
  }

  __host__ __device__ int MaxOrMinus(int a, int b) {
    return glm::min(a, b) < 0 ? -1 : glm::max(a, b);
  }

  __host__ __device__ Halfedge operator()(Halfedge a, Halfedge b) {
    MakeForward(a);
    MakeForward(b);
    a.startVert = glm::min(a.startVert, b.startVert);
    a.endVert = glm::max(a.endVert, b.endVert);
    a.face = MaxOrMinus(a.face, b.face);
    a.pairedHalfedge = MaxOrMinus(a.pairedHalfedge, b.pairedHalfedge);
    return a;
  }
};

struct PosMin
    : public thrust::binary_function<glm::vec3, glm::vec3, glm::vec3> {
  __host__ __device__ glm::vec3 operator()(glm::vec3 a, glm::vec3 b) {
    if (isnan(a.x)) return b;
    if (isnan(b.x)) return a;
    return glm::min(a, b);
  }
};

struct PosMax
    : public thrust::binary_function<glm::vec3, glm::vec3, glm::vec3> {
  __host__ __device__ glm::vec3 operator()(glm::vec3 a, glm::vec3 b) {
    if (isnan(a.x)) return b;
    if (isnan(b.x)) return a;
    return glm::max(a, b);
  }
};

struct SumPair : public thrust::binary_function<thrust::pair<float, float>,
                                                thrust::pair<float, float>,
                                                thrust::pair<float, float>> {
  __host__ __device__ thrust::pair<float, float> operator()(
      thrust::pair<float, float> a, thrust::pair<float, float> b) {
    a.first += b.first;
    a.second += b.second;
    return a;
  }
};

struct Transform4x3 {
  const glm::mat4x3 transform;

  __host__ __device__ void operator()(glm::vec3& position) {
    position = transform * glm::vec4(position, 1.0f);
  }
};

struct TransformNormals {
  const glm::mat3 transform;

  __host__ __device__ void operator()(glm::vec3& normal) {
    normal = glm::normalize(transform * normal);
    if (isnan(normal.x)) normal = glm::vec3(0.0f);
  }
};

__host__ __device__ uint32_t SpreadBits3(uint32_t v) {
  v = 0xFF0000FFu & (v * 0x00010001u);
  v = 0x0F00F00Fu & (v * 0x00000101u);
  v = 0xC30C30C3u & (v * 0x00000011u);
  v = 0x49249249u & (v * 0x00000005u);
  return v;
}

__host__ __device__ uint32_t MortonCode(glm::vec3 position, Box bBox) {
  // Unreferenced vertices are marked NaN, and this will sort them to the end
  // (the Morton code only uses the first 30 of 32 bits).
  if (isnan(position.x)) return kNoCode;

  glm::vec3 xyz = (position - bBox.min) / (bBox.max - bBox.min);
  xyz = glm::min(glm::vec3(1023.0f), glm::max(glm::vec3(0.0f), 1024.0f * xyz));
  uint32_t x = SpreadBits3(static_cast<uint32_t>(xyz.x));
  uint32_t y = SpreadBits3(static_cast<uint32_t>(xyz.y));
  uint32_t z = SpreadBits3(static_cast<uint32_t>(xyz.z));
  return x * 4 + y * 2 + z;
}

struct Morton {
  const Box bBox;

  __host__ __device__ void operator()(
      thrust::tuple<uint32_t&, const glm::vec3&> inout) {
    glm::vec3 position = thrust::get<1>(inout);
    thrust::get<0>(inout) = MortonCode(position, bBox);
  }
};

struct FaceMortonBox {
  const Halfedge* halfedge;
  const glm::vec3* vertPos;
  const Box bBox;

  __host__ __device__ void operator()(
      thrust::tuple<uint32_t&, Box&, int> inout) {
    uint32_t& mortonCode = thrust::get<0>(inout);
    Box& faceBox = thrust::get<1>(inout);
    int face = thrust::get<2>(inout);

    // Removed tris are marked by all halfedges having pairedHalfedge = -1, and
    // this will sort them to the end (the Morton code only uses the first 30 of
    // 32 bits).
    if (halfedge[3 * face].pairedHalfedge < 0) {
      mortonCode = kNoCode;
      return;
    }

    glm::vec3 center(0.0f);

    for (const int i : {0, 1, 2}) {
      const glm::vec3 pos = vertPos[halfedge[3 * face + i].startVert];
      center += pos;
      faceBox.Union(pos);
    }
    center /= 3;

    mortonCode = MortonCode(center, bBox);
  }
};

struct Reindex {
  const int* indexInv;

  __host__ __device__ void operator()(Halfedge& edge) {
    if (edge.startVert < 0) return;
    edge.startVert = indexInv[edge.startVert];
    edge.endVert = indexInv[edge.endVert];
  }
};

template <typename T>
void Permute(VecDH<T>& inOut, const VecDH<int>& new2Old) {
  VecDH<T> tmp(inOut);
  inOut.resize(new2Old.size());
  thrust::gather(new2Old.beginD(), new2Old.endD(), tmp.beginD(),
                 inOut.beginD());
}

template void Permute<BaryRef>(VecDH<BaryRef>&, const VecDH<int>&);
template void Permute<glm::vec3>(VecDH<glm::vec3>&, const VecDH<int>&);

struct ReindexFace {
  Halfedge* halfedge;
  glm::vec4* halfedgeTangent;
  const Halfedge* oldHalfedge;
  const glm::vec4* oldHalfedgeTangent;
  const int* faceNew2Old;
  const int* faceOld2New;

  __host__ __device__ void operator()(int newFace) {
    const int oldFace = faceNew2Old[newFace];
    for (const int i : {0, 1, 2}) {
      const int oldEdge = 3 * oldFace + i;
      Halfedge edge = oldHalfedge[oldEdge];
      edge.face = newFace;
      const int pairedFace = oldHalfedge[edge.pairedHalfedge].face;
      const int offset = edge.pairedHalfedge - 3 * pairedFace;
      edge.pairedHalfedge = 3 * faceOld2New[pairedFace] + offset;
      const int newEdge = 3 * newFace + i;
      halfedge[newEdge] = edge;
      if (oldHalfedgeTangent != nullptr) {
        halfedgeTangent[newEdge] = oldHalfedgeTangent[oldEdge];
      }
    }
  }
};

struct AssignNormals {
  glm::vec3* vertNormal;
  const glm::vec3* vertPos;
  const Halfedge* halfedges;
  const float precision;
  const bool calculateTriNormal;

  __host__ __device__ void operator()(thrust::tuple<glm::vec3&, int> in) {
    glm::vec3& triNormal = thrust::get<0>(in);
    const int face = thrust::get<1>(in);

    glm::ivec3 triVerts;
    for (int i : {0, 1, 2}) triVerts[i] = halfedges[3 * face + i].startVert;

    glm::vec3 edge[3];
    glm::vec3 edgeLength;
    float perimeter = 0;
    for (int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      edge[i] = vertPos[triVerts[j]] - vertPos[triVerts[i]];
      edgeLength[i] = glm::length(edge[i]);
      perimeter += edgeLength[i];
    }

    if (calculateTriNormal) {
      triNormal = glm::normalize(glm::cross(edge[0], edge[1]));
      if (isnan(triNormal.x)) triNormal = glm::vec3(0, 0, 1);
    }

    // corner angles
    glm::vec3 phi;
    for (int i : {0, 1, 2}) edge[i] /= edgeLength[i];
    phi[0] = glm::acos(-glm::dot(edge[2], edge[0]));
    phi[1] = glm::acos(-glm::dot(edge[0], edge[1]));
    phi[2] = glm::pi<float>() - phi[0] - phi[1];

    // assign weighted sum
    for (int i : {0, 1, 2}) {
      AtomicAddVec3(vertNormal[triVerts[i]], phi[i] * triNormal);
    }
  }
};

struct Tri2Halfedges {
  Halfedge* halfedges;
  TmpEdge* edges;

  __host__ __device__ void operator()(
      thrust::tuple<int, const glm::ivec3&> in) {
    const int tri = thrust::get<0>(in);
    const glm::ivec3& triVerts = thrust::get<1>(in);
    for (const int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      const int edge = 3 * tri + i;
      halfedges[edge] = {triVerts[i], triVerts[j], -1, tri};
      edges[edge] = TmpEdge(triVerts[i], triVerts[j], edge);
    }
  }
};

struct LinkHalfedges {
  Halfedge* halfedges;
  const TmpEdge* edges;

  __host__ __device__ void operator()(int k) {
    const int i = 2 * k;
    const int j = i + 1;
    const int pair0 = edges[i].halfedgeIdx;
    const int pair1 = edges[j].halfedgeIdx;
    if (halfedges[pair0].startVert != halfedges[pair1].endVert ||
        halfedges[pair0].endVert != halfedges[pair1].startVert ||
        halfedges[pair0].face == halfedges[pair1].face)
      printf("Not manifold!\n");
    halfedges[pair0].pairedHalfedge = pair1;
    halfedges[pair1].pairedHalfedge = pair0;
  }
};

struct SwapHalfedges {
  Halfedge* halfedges;
  const TmpEdge* edges;

  __host__ void operator()(int k) {
    const int i = 2 * k;
    const int j = i - 2;
    const TmpEdge thisEdge = edges[i];
    const TmpEdge lastEdge = edges[j];
    if (thisEdge.first == lastEdge.first &&
        thisEdge.second == lastEdge.second) {
      const int swap0idx = thisEdge.halfedgeIdx;
      Halfedge& swap0 = halfedges[swap0idx];
      const int swap1idx = swap0.pairedHalfedge;
      Halfedge& swap1 = halfedges[swap1idx];

      const int next0idx = swap0idx + ((swap0idx + 1) % 3 == 0 ? -2 : 1);
      const int next1idx = swap1idx + ((swap1idx + 1) % 3 == 0 ? -2 : 1);
      Halfedge& next0 = halfedges[next0idx];
      Halfedge& next1 = halfedges[next1idx];

      next0.startVert = swap0.endVert = next1.endVert;
      swap0.pairedHalfedge = next1.pairedHalfedge;
      halfedges[swap0.pairedHalfedge].pairedHalfedge = swap0idx;

      next1.startVert = swap1.endVert = next0.endVert;
      swap1.pairedHalfedge = next0.pairedHalfedge;
      halfedges[swap1.pairedHalfedge].pairedHalfedge = swap1idx;

      next0.pairedHalfedge = next1idx;
      next1.pairedHalfedge = next0idx;
    }
  }
};

struct InitializeBaryRef {
  __host__ __device__ void operator()(thrust::tuple<BaryRef&, int> inOut) {
    BaryRef& baryRef = thrust::get<0>(inOut);
    int tri = thrust::get<1>(inOut);

    baryRef.tri = tri;
    baryRef.vertBary = {-1, -1, -1};
  }
};

struct MarkShortEdge {
  const glm::vec3* vertPos;
  const float precision;

  __host__ __device__ void operator()(Halfedge& edge) {
    const glm::vec3 delta = vertPos[edge.endVert] - vertPos[edge.startVert];
    if (glm::dot(delta, delta) < precision * precision) edge.face = -1;
  }
};

struct MarkColinearEdge {
  bool* marked;
  const glm::vec3* vertPos;
  const glm::vec3* faceNormal;
  const Halfedge* halfedge;
  const float precision;

  __host__ __device__ void operator()(Halfedge& edge) {
    if (edge.pairedHalfedge < 0) return;
    const glm::vec3 delta = vertPos[edge.endVert] - vertPos[edge.startVert];

    // If all triangles touching startVert are coplanar with the edge, then
    // startVert can be moved to endVert and merged without altering the
    // geometry.
    int start = edge.pairedHalfedge;
    int current = NextHalfedge(start);
    current = halfedge[current].pairedHalfedge;
    while (current != start) {
      current = NextHalfedge(current);
      if (glm::abs(glm::dot(delta, faceNormal[current / 3])) > precision)
        return;
      current = halfedge[current].pairedHalfedge;
    }
    edge.face = -1;
    *marked = true;
  }
};

struct CollapseEdge {
  bool& collapsed;
  VecH<Halfedge>& halfedge;
  VecH<glm::vec3>& vertPos;
  const VecH<glm::vec3>& triNormal;
  const bool shortEdge;

  __host__ glm::ivec3 TriOf(int edge) const {
    glm::ivec3 triEdge;
    triEdge[0] = edge;
    triEdge[1] = edge + ((edge + 1) % 3 == 0 ? -2 : 1);
    triEdge[2] = triEdge[1] + ((triEdge[1] + 1) % 3 == 0 ? -2 : 1);
    return triEdge;
  }

  __host__ void UnmarkEdge(int edge) {
    if (!shortEdge) halfedge[edge].face = edge / 3;
  }

  __host__ bool CCW2Normal(const glm::vec3& a, const glm::vec3& b,
                           const glm::vec3& normal) const {
    return glm::dot(glm::cross(a, b), normal) >= 0;
  }

  // Traverses CW around startEdge.endVert from startEdge to endEdge
  // (edgeEdge.endVert must == startEdge.endVert), updating each edge to point
  // to vert instead.
  __host__ void UpdateVert(int vert, int startEdge, int endEdge) {
    while (startEdge != endEdge) {
      halfedge[startEdge].endVert = vert;
      startEdge = NextHalfedge(startEdge);
      halfedge[startEdge].startVert = vert;
      startEdge = halfedge[startEdge].pairedHalfedge;
    }
  }

  // In the event that the edge collapse would create a non-manifold edge,
  // instead we duplicate the two verts and attach the manifolds the other way
  // across this edge.
  __host__ void FormLoop(int current, int end) {
    int startVert = vertPos.size();
    vertPos.push_back(vertPos[halfedge[current].startVert]);
    int endVert = vertPos.size();
    vertPos.push_back(vertPos[halfedge[current].endVert]);

    int oldMatch = halfedge[current].pairedHalfedge;
    int newMatch = halfedge[end].pairedHalfedge;

    UpdateVert(startVert, oldMatch, newMatch);
    UpdateVert(endVert, end, current);

    halfedge[current].pairedHalfedge = newMatch;
    halfedge[newMatch].pairedHalfedge = current;
    halfedge[end].pairedHalfedge = oldMatch;
    halfedge[oldMatch].pairedHalfedge = end;
  }

  __host__ void CollapseTri(const glm::ivec3& triEdge) {
    int pair1 = halfedge[triEdge[1]].pairedHalfedge;
    int pair2 = halfedge[triEdge[2]].pairedHalfedge;
    halfedge[pair1].pairedHalfedge = pair2;
    halfedge[pair2].pairedHalfedge = pair1;
    for (int i : {0, 1, 2}) {
      halfedge[triEdge[i]] = {-1, -1, -1, -1};
    }
  }

  __host__ void operator()(int edge) {
    const Halfedge toRemove = halfedge[edge];
    if (toRemove.face >= 0 || toRemove.pairedHalfedge < 0) return;

    const glm::ivec3 tri0edge = TriOf(edge);
    const glm::ivec3 tri1edge = TriOf(toRemove.pairedHalfedge);

    if (halfedge[tri0edge[1]].endVert == halfedge[tri1edge[1]].endVert) {
      // Remove disconnected triangles
      for (int i : {0, 1, 2}) {
        vertPos[halfedge[tri0edge[i]].startVert] = glm::vec3(0.0f / 0.0f);
        halfedge[tri0edge[i]] = {-1, -1, -1, -1};
        halfedge[tri1edge[i]] = {-1, -1, -1, -1};
      }
      return;
    }

    std::vector<int> edges;
    int current = halfedge[tri0edge[1]].pairedHalfedge;
    while (current != tri1edge[2]) {
      UnmarkEdge(current);
      current = NextHalfedge(current);
      edges.push_back(current);
      UnmarkEdge(current);
      current = halfedge[current].pairedHalfedge;
    }

    const int endVert = toRemove.endVert;
    int start = halfedge[tri1edge[1]].pairedHalfedge;
    if (!shortEdge) {
      current = start;
      glm::vec3 lastEdge =
          vertPos[halfedge[tri1edge[1]].endVert] - vertPos[endVert];
      while (current != tri0edge[2]) {
        current = NextHalfedge(current);
        glm::vec3 thisEdge =
            vertPos[halfedge[current].endVert] - vertPos[endVert];
        if (!CCW2Normal(thisEdge, lastEdge, triNormal[current / 3])) {
          UnmarkEdge(edge);
          return;
        }
        lastEdge = thisEdge;
        current = halfedge[current].pairedHalfedge;
      }
    }

    // Remove toRemove.startVert and replace with endVert.
    vertPos[toRemove.startVert] = glm::vec3(0.0f / 0.0f);
    CollapseTri(tri1edge);

    current = start;
    while (current != tri0edge[2]) {
      UnmarkEdge(current);
      current = NextHalfedge(current);
      UnmarkEdge(current);
      const int vert = halfedge[current].endVert;
      const int next = halfedge[current].pairedHalfedge;
      for (int i = 0; i < edges.size(); ++i) {
        if (vert == halfedge[edges[i]].endVert) {
          FormLoop(edges[i], current);
          start = next;
          edges.resize(i);
          break;
        }
      }
      current = next;
    }

    UpdateVert(endVert, start, tri0edge[2]);
    CollapseTri(tri0edge);
    collapsed = true;
  }
};

struct EdgeBox {
  const glm::vec3* vertPos;

  __host__ __device__ void operator()(
      thrust::tuple<Box&, const TmpEdge&> inout) {
    const TmpEdge& edge = thrust::get<1>(inout);
    thrust::get<0>(inout) = Box(vertPos[edge.first], vertPos[edge.second]);
  }
};

struct CheckManifold {
  const Halfedge* halfedges;

  __host__ __device__ bool operator()(int edge) {
    bool good = true;
    const Halfedge halfedge = halfedges[edge];
    const Halfedge paired = halfedges[halfedge.pairedHalfedge];
    good &= paired.pairedHalfedge == edge;
    good &= halfedge.startVert != halfedge.endVert;
    good &= halfedge.startVert == paired.endVert;
    good &= halfedge.endVert == paired.startVert;
    return good;
  }
};

struct NoDuplicates {
  const Halfedge* halfedges;

  __host__ __device__ bool operator()(int edge) {
    return halfedges[edge].startVert != halfedges[edge + 1].startVert ||
           halfedges[edge].endVert != halfedges[edge + 1].endVert;
  }
};

struct CheckCCW {
  const Halfedge* halfedges;
  const glm::vec3* vertPos;
  const glm::vec3* triNormal;
  const float precision;

  __host__ __device__ bool operator()(int face) {
    const glm::mat3x2 projection = GetAxisAlignedProjection(triNormal[face]);
    glm::vec2 v[3];
    for (int i : {0, 1, 2})
      v[i] = projection * vertPos[halfedges[3 * face + i].startVert];
    int ccw = CCW(v[0], v[1], v[2], 2 * precision);
    if (ccw < 0) {
      glm::vec2 v1 = v[1] - v[0];
      glm::vec2 v2 = v[2] - v[0];
      float area = v1.x * v2.y - v1.y * v2.x;
      float base = glm::sqrt(glm::max(glm::dot(v1, v1), glm::dot(v2, v2)));
      printf(
          "Tri %d does not match normal, height = %g, base = %g\n"
          "normal = %g, %g, %g\n",
          face, area / base, base, triNormal[face].x, triNormal[face].y,
          triNormal[face].z);
    }
    return ccw >= 0;
  }
};

}  // namespace

namespace manifold {

/**
 * Create a manifold from an input triangle Mesh. Will throw if the Mesh is not
 * manifold. TODO: update halfedgeTangent during CollapseDegenerates.
 */
Manifold::Impl::Impl(const Mesh& mesh)
    : vertPos_(mesh.vertPos), halfedgeTangent_(mesh.halfedgeTangent) {
  CheckDevice();
  CalculateBBox();
  SetPrecision();
  CreateAndFixHalfedges(mesh.triVerts);
  CalculateNormals();
  CollapseDegenerates();
  Finish();
}

/**
 * Create eiter a unit tetrahedron, cube or octahedron. The cube is in the first
 * octant, while the others are symmetric about the origin.
 */
Manifold::Impl::Impl(Shape shape) {
  std::vector<glm::vec3> vertPos;
  std::vector<glm::ivec3> triVerts;
  switch (shape) {
    case Shape::TETRAHEDRON:
      vertPos = {{-1.0f, -1.0f, 1.0f},
                 {-1.0f, 1.0f, -1.0f},
                 {1.0f, -1.0f, -1.0f},
                 {1.0f, 1.0f, 1.0f}};
      triVerts = {{2, 0, 1}, {0, 3, 1}, {2, 3, 0}, {3, 2, 1}};
      break;
    case Shape::CUBE:
      vertPos = {{0.0f, 0.0f, 0.0f},  //
                 {1.0f, 0.0f, 0.0f},  //
                 {1.0f, 1.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},  //
                 {1.0f, 0.0f, 1.0f},  //
                 {1.0f, 1.0f, 1.0f},  //
                 {0.0f, 1.0f, 1.0f}};
      triVerts = {{0, 2, 1}, {0, 3, 2},  //
                  {4, 5, 6}, {4, 6, 7},  //
                  {0, 1, 5}, {0, 5, 4},  //
                  {1, 2, 6}, {1, 6, 5},  //
                  {2, 3, 7}, {2, 7, 6},  //
                  {3, 0, 4}, {3, 4, 7}};
      break;
    case Shape::OCTAHEDRON:
      vertPos = {{1.0f, 0.0f, 0.0f},   //
                 {-1.0f, 0.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},   //
                 {0.0f, -1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},   //
                 {0.0f, 0.0f, -1.0f}};
      triVerts = {{0, 2, 4}, {1, 5, 3},  //
                  {2, 1, 4}, {3, 5, 0},  //
                  {1, 3, 4}, {0, 5, 2},  //
                  {3, 0, 4}, {2, 5, 1}};
      break;
    default:
      throw userErr("Unrecognized shape!");
  }
  vertPos_ = vertPos;
  CreateAndFixHalfedges(triVerts);
  Finish();
}

/**
 * Create the halfedge_ data structure from an input triVerts array like Mesh.
 */
void Manifold::Impl::CreateHalfedges(const VecDH<glm::ivec3>& triVerts) {
  const int numTri = triVerts.size();
  halfedge_.resize(3 * numTri);
  VecDH<TmpEdge> edge(3 * numTri);
  thrust::for_each_n(zip(countAt(0), triVerts.beginD()), numTri,
                     Tri2Halfedges({halfedge_.ptrD(), edge.ptrD()}));
  thrust::sort(edge.beginD(), edge.endD());
  thrust::for_each_n(countAt(0), halfedge_.size() / 2,
                     LinkHalfedges({halfedge_.ptrD(), edge.cptrD()}));
  if (meshRelation_.triBary.size() != numTri) {
    meshRelation_.triBary.resize(numTri);
    thrust::for_each_n(zip(meshRelation_.triBary.beginD(), countAt(0)), numTri,
                       InitializeBaryRef());
  }
}

/**
 * Create the halfedge_ data structure from an input triVerts array like Mesh.
 * Check that the input is an even-manifold, and if it is not 2-manifold,
 * perform edge swaps until it is. This is a host function.
 */
void Manifold::Impl::CreateAndFixHalfedges(const VecDH<glm::ivec3>& triVerts) {
  const int numTri = triVerts.size();
  halfedge_.resize(3 * numTri);
  VecDH<TmpEdge> edge(3 * numTri);
  thrust::for_each_n(zip(countAt(0), triVerts.begin()), numTri,
                     Tri2Halfedges({halfedge_.ptrH(), edge.ptrH()}));
  // Stable sort is required here so that halfedges from the same face are
  // paired together (the triangles were created in face order). In some
  // degenerate situations the triangulator can add the same internal edge in
  // two different faces, causing this edge to not be 2-manifold. We detect this
  // and fix it by swapping one of the identical edges, so it is important that
  // we have the edges paired according to their face.
  std::stable_sort(edge.begin(), edge.end());
  thrust::for_each_n(thrust::host, countAt(0), halfedge_.size() / 2,
                     LinkHalfedges({halfedge_.ptrH(), edge.cptrH()}));
  thrust::for_each(thrust::host, countAt(1), countAt(halfedge_.size() / 2),
                   SwapHalfedges({halfedge_.ptrH(), edge.cptrH()}));
  meshRelation_.triBary.resize(numTri);
  thrust::for_each_n(zip(meshRelation_.triBary.begin(), countAt(0)), numTri,
                     InitializeBaryRef());
}

/**
 * Collapses degenerate triangles by removing edges shorter than precision_ and
 * edges that are colinear whose collapse does not generate a geometric change.
 * Rather than actually removing them, this step merely marks them for removal,
 * by setting vertPos to NaN and halfedge to -1.
 *
 * TODO: remove colinear edge collapse, since this could easily conflict with
 * mesh property junctions; consider test Boolean.FaceUnion if the two cubes
 * were different colors - we are losing the verts that denote the boundary.
 */
void Manifold::Impl::CollapseDegenerates() {
  // Short edge collapse is commented out because it was causing a test to fail,
  // but only on the GPU. Colinear removal is broader, so it should still
  // accomplish the same thing, but with some additional computation. I'm
  // leaving this here because I'd like to understand why it gets the wrong
  // result and why it's not any faster.

  // thrust::for_each(halfedge_.beginD(), halfedge_.endD(),
  //                  MarkShortEdge({vertPos_.cptrD(), precision_}));
  bool collapsed = false;
  // thrust::for_each_n(thrust::host, countAt(0), halfedge_.size(),
  //                    CollapseEdge({collapsed, halfedge_.H(), vertPos_.H(),
  //                                  faceNormal_.H(), true}));

  VecDH<bool> marked(1);
  while (1) {
    marked.H()[0] = false;
    thrust::for_each(
        halfedge_.beginD(), halfedge_.endD(),
        MarkColinearEdge({marked.ptrD(), vertPos_.cptrD(), faceNormal_.cptrD(),
                          halfedge_.cptrD(), precision_}));
    if (!marked.H()[0]) break;

    collapsed = false;
    thrust::for_each_n(thrust::host, countAt(0), halfedge_.size(),
                       CollapseEdge({collapsed, halfedge_.H(), vertPos_.H(),
                                     faceNormal_.H(), false}));
    if (!collapsed) break;
  }
}

/**
 * Once halfedge_ has been filled in, this function can be called to create the
 * rest of the internal data structures. This function also removes the verts
 * and halfedges flagged for removal (NaN verts and -1 halfedges).
 */
void Manifold::Impl::Finish() {
  if (halfedge_.size() == 0) return;

  CalculateBBox();
  SetPrecision(precision_);
  if (!bBox_.isFinite()) {
    vertPos_.resize(0);
    halfedge_.resize(0);
    faceNormal_.resize(0);
    return;
  }

  SortVerts();
  VecDH<Box> faceBox;
  VecDH<uint32_t> faceMorton;
  GetFaceBoxMorton(faceBox, faceMorton);
  SortFaces(faceBox, faceMorton);
  if (halfedge_.size() == 0) return;

  ALWAYS_ASSERT(halfedge_.size() % 6 == 0, topologyErr,
                "Not an even number of faces after sorting faces!");
  Halfedge extrema = {0, 0, 0, 0};
  extrema =
      thrust::reduce(halfedge_.beginD(), halfedge_.endD(), extrema, Extrema());

  ALWAYS_ASSERT(extrema.startVert >= 0, topologyErr,
                "Vertex index is negative!");
  ALWAYS_ASSERT(extrema.endVert < NumVert(), topologyErr,
                "Vertex index exceeds number of verts!");
  ALWAYS_ASSERT(extrema.face >= 0, topologyErr, "Face index is negative!");
  ALWAYS_ASSERT(extrema.face < NumTri(), topologyErr,
                "Face index exceeds number of faces!");
  ALWAYS_ASSERT(extrema.pairedHalfedge >= 0, topologyErr,
                "Halfedge index is negative!");
  ALWAYS_ASSERT(extrema.pairedHalfedge < 2 * NumEdge(), topologyErr,
                "Halfedge index exceeds number of halfedges!");

  CalculateNormals();
  collider_ = Collider(faceBox, faceMorton);
}

/**
 * Does a full recalculation of the face bounding boxes, including updating the
 * collider, but does not resort the faces.
 */
void Manifold::Impl::Update() {
  CalculateBBox();
  VecDH<Box> faceBox;
  VecDH<uint32_t> faceMorton;
  GetFaceBoxMorton(faceBox, faceMorton);
  collider_.UpdateBoxes(faceBox);
}

void Manifold::Impl::ApplyTransform() const {
  // This const_cast is here because these operations cancel out, leaving the
  // state conceptually unchanged. This enables lazy transformation evaluation.
  const_cast<Impl*>(this)->ApplyTransform();
}

/**
 * Bake the manifold's transform into its vertices. This function allows lazy
 * evaluation, which is important because often several transforms are applied
 * between operations.
 */
void Manifold::Impl::ApplyTransform() {
  if (transform_ == glm::mat4x3(1.0f)) return;
  thrust::for_each(vertPos_.beginD(), vertPos_.endD(),
                   Transform4x3({transform_}));

  glm::mat3 normalTransform =
      glm::inverse(glm::transpose(glm::mat3(transform_)));
  thrust::for_each(faceNormal_.beginD(), faceNormal_.endD(),
                   TransformNormals({normalTransform}));
  thrust::for_each(vertNormal_.beginD(), vertNormal_.endD(),
                   TransformNormals({normalTransform}));
  // This optimization does a cheap collider update if the transform is
  // axis-aligned.
  if (!collider_.Transform(transform_)) Update();

  const float oldScale = bBox_.Scale();
  transform_ = glm::mat4x3(1.0f);
  CalculateBBox();

  const float newScale = bBox_.Scale();
  precision_ *= glm::max(1.0f, newScale / oldScale) *
                glm::max(glm::length(transform_[0]),
                         glm::max(glm::length(transform_[1]),
                                  glm::length(transform_[2])));

  // Maximum of inherited precision loss and translational precision loss.
  SetPrecision(precision_);
}

/**
 * Triangulates the faces. In this case, the halfedge_ vector is not yet a set
 * of triangles as required by this data structure, but is instead a set of
 * general faces with the input faceEdge vector having length of the number of
 * faces + 1. The values are indicies into the halfedge_ vector for the first
 * edge of each face, with the final value being the length of the halfedge_
 * vector itself. Upon return, halfedge_ has been lengthened and properly
 * represents the mesh as a set of triangles as usual. In this process the
 * faceNormal_ values are retained, repeated as necessary.
 */
void Manifold::Impl::Face2Tri(const VecDH<int>& faceEdge) {
  VecDH<glm::ivec3> triVertsOut;
  VecDH<glm::vec3> triNormalOut;

  VecH<glm::ivec3>& triVerts = triVertsOut.H();
  VecH<glm::vec3>& triNormal = triNormalOut.H();
  const VecH<glm::vec3>& vertPos = vertPos_.H();
  const VecH<int>& face = faceEdge.H();
  const VecH<Halfedge>& halfedge = halfedge_.H();
  const VecH<glm::vec3>& faceNormal = faceNormal_.H();

  for (int i = 0; i < face.size() - 1; ++i) {
    const int edge = face[i];
    const int lastEdge = face[i + 1];
    const int numEdge = lastEdge - edge;
    ALWAYS_ASSERT(numEdge >= 3, topologyErr, "face has less than three edges.");
    const glm::vec3 normal = faceNormal[i];

    if (numEdge == 3) {  // Single triangle
      glm::ivec3 tri(halfedge[edge].startVert, halfedge[edge + 1].startVert,
                     halfedge[edge + 2].startVert);
      glm::ivec3 ends(halfedge[edge].endVert, halfedge[edge + 1].endVert,
                      halfedge[edge + 2].endVert);
      if (ends[0] == tri[2]) {
        std::swap(tri[1], tri[2]);
        std::swap(ends[1], ends[2]);
      }
      ALWAYS_ASSERT(ends[0] == tri[1] && ends[1] == tri[2] && ends[2] == tri[0],
                    topologyErr, "These 3 edges do not form a triangle!");

      triVerts.push_back(tri);
      triNormal.push_back(normal);
    } else if (numEdge == 4) {  // Pair of triangles
      const glm::mat3x2 projection = GetAxisAlignedProjection(normal);
      auto triCCW = [&projection, &vertPos, this](const glm::ivec3 tri) {
        return CCW(projection * vertPos[tri[0]], projection * vertPos[tri[1]],
                   projection * vertPos[tri[2]], precision_) >= 0;
      };

      glm::ivec3 tri0(halfedge[edge].startVert, halfedge[edge].endVert, -1);
      glm::ivec3 tri1(-1, -1, tri0[0]);
      for (const int i : {1, 2, 3}) {
        if (halfedge[edge + i].startVert == tri0[1]) {
          tri0[2] = halfedge[edge + i].endVert;
          tri1[0] = tri0[2];
        }
        if (halfedge[edge + i].endVert == tri0[0]) {
          tri1[1] = halfedge[edge + i].startVert;
        }
      }
      ALWAYS_ASSERT(glm::all(glm::greaterThanEqual(tri0, glm::ivec3(0))) &&
                        glm::all(glm::greaterThanEqual(tri1, glm::ivec3(0))),
                    topologyErr, "non-manifold quad!");
      bool firstValid = triCCW(tri0) && triCCW(tri1);
      tri0[2] = tri1[1];
      tri1[2] = tri0[1];
      bool secondValid = triCCW(tri0) && triCCW(tri1);

      if (!secondValid) {
        tri0[2] = tri1[0];
        tri1[2] = tri0[0];
      } else if (firstValid) {
        glm::vec3 firstCross = vertPos[tri0[0]] - vertPos[tri1[0]];
        glm::vec3 secondCross = vertPos[tri0[1]] - vertPos[tri1[1]];
        if (glm::dot(firstCross, firstCross) <
            glm::dot(secondCross, secondCross)) {
          tri0[2] = tri1[0];
          tri1[2] = tri0[0];
        }
      }

      triVerts.push_back(tri0);
      triNormal.push_back(normal);
      triVerts.push_back(tri1);
      triNormal.push_back(normal);
    } else {  // General triangulation
      const glm::mat3x2 projection = GetAxisAlignedProjection(normal);

      Polygons polys;
      try {
        polys = Face2Polygons(i, projection, face);
      } catch (const std::exception& e) {
        std::cout << e.what() << std::endl;
        for (int edge = face[i]; edge < face[i + 1]; ++edge)
          std::cout << "halfedge: " << edge << ", " << halfedge[edge]
                    << std::endl;
        throw;
      }

      std::vector<glm::ivec3> newTris = Triangulate(polys, precision_);

      for (auto tri : newTris) {
        triVerts.push_back(tri);
        triNormal.push_back(normal);
      }
    }
  }
  faceNormal_ = triNormalOut;
  CreateAndFixHalfedges(triVertsOut);
}

void Manifold::Impl::CreateTangents(
    const std::vector<Smoothness>& sharpenedEdges) {
  const int numHalfedge = halfedge_.size();
  halfedgeTangent_.resize(numHalfedge);

  thrust::for_each_n(zip(halfedgeTangent_.beginD(), halfedge_.cbeginD()),
                     numHalfedge,
                     SmoothBezier({vertPos_.cptrD(), faceNormal_.cptrD(),
                                   vertNormal_.cptrD(), halfedge_.cptrD()}));

  if (!sharpenedEdges.empty()) {
    const VecH<Halfedge>& halfedge = halfedge_.H();
    const VecH<BaryRef>& triBary = meshRelation_.triBary.H();

    std::vector<int> oldHalfedge2New(halfedge.size());
    for (int tri = 0; tri < NumTri(); ++tri) {
      int oldTri = triBary[tri].tri;
      for (int i : {0, 1, 2}) oldHalfedge2New[3 * oldTri + i] = 3 * tri + i;
    }

    using Pair = std::pair<Smoothness, Smoothness>;
    // Fill in missing pairs with default smoothness = 1.
    std::map<int, Pair> edges;
    for (Smoothness edge : sharpenedEdges) {
      if (edge.smoothness == 1) continue;
      edge.halfedge = oldHalfedge2New[edge.halfedge];
      int pair = halfedge[edge.halfedge].pairedHalfedge;
      if (edges.find(pair) == edges.end()) {
        edges[edge.halfedge] = {edge, {pair, 1}};
      } else {
        edges[pair].second = edge;
      }
    }

    std::map<int, std::vector<Pair>> vertTangents;
    for (const auto value : edges) {
      const Pair edge = value.second;
      vertTangents[halfedge[edge.first.halfedge].startVert].push_back(edge);
      vertTangents[halfedge[edge.second.halfedge].startVert].push_back(
          {edge.second, edge.first});
    }

    VecH<glm::vec4>& tangent = halfedgeTangent_.H();
    for (const auto& value : vertTangents) {
      const std::vector<Pair>& vert = value.second;
      // Sharp edges that end are smooth at their terminal vert.
      if (vert.size() == 1) continue;
      if (vert.size() == 2) {  // Make continuous edge
        const int first = vert[0].first.halfedge;
        const int second = vert[1].first.halfedge;
        const glm::vec3 newTangent = glm::normalize(glm::vec3(tangent[first]) -
                                                    glm::vec3(tangent[second]));
        tangent[first] =
            glm::vec4(glm::length(glm::vec3(tangent[first])) * newTangent,
                      tangent[first].w);
        tangent[second] =
            glm::vec4(-glm::length(glm::vec3(tangent[second])) * newTangent,
                      tangent[second].w);

        auto SmoothHalf = [&](int first, int last, float smoothness) {
          int current = NextHalfedge(halfedge[first].pairedHalfedge);
          while (current != last) {
            const float cosBeta = glm::dot(
                newTangent, glm::normalize(glm::vec3(tangent[current])));
            const float factor =
                (1 - smoothness) * cosBeta * cosBeta + smoothness;
            tangent[current] = glm::vec4(factor * glm::vec3(tangent[current]),
                                         tangent[current].w);
            current = NextHalfedge(halfedge[current].pairedHalfedge);
          }
        };

        SmoothHalf(first, second,
                   (vert[0].second.smoothness + vert[1].first.smoothness) / 2);
        SmoothHalf(second, first,
                   (vert[1].second.smoothness + vert[0].first.smoothness) / 2);

      } else {  // Sharpen vertex uniformly
        float smoothness = 0;
        for (const Pair pair : vert) {
          smoothness += pair.first.smoothness;
          smoothness += pair.second.smoothness;
        }
        smoothness /= 2 * vert.size();

        const int start = vert[0].first.halfedge;
        int current = start;
        do {
          tangent[current] = glm::vec4(smoothness * glm::vec3(tangent[current]),
                                       tangent[current].w);
          current = NextHalfedge(halfedge[current].pairedHalfedge);
        } while (current != start);
      }
    }
  }
}

/**
 * Split each edge into n pieces and sub-triangulate each triangle accordingly.
 * This function doesn't run Finish(), as that is expensive and it'll need to be
 * run after the new vertices have moved, which is a likely scenario after
 * refinement (smoothing).
 */
void Manifold::Impl::Subdivide(int n) {
  int numVert = NumVert();
  int numEdge = NumEdge();
  int numTri = NumTri();
  // Append new verts
  int vertsPerEdge = n - 1;
  int triVertStart = numVert + numEdge * vertsPerEdge;
  vertPos_.resize(triVertStart + numTri * VertsPerTri(n - 2));
  meshRelation_.barycentric.resize(numTri * VertsPerTri(n + 1));
  meshRelation_.triBary.resize(n * n * numTri);
  VecDH<TmpEdge> edges = CreateTmpEdges(halfedge_);
  VecDH<int> half2Edge(2 * numEdge);
  thrust::for_each_n(zip(countAt(0), edges.beginD()), numEdge,
                     ReindexHalfedge({half2Edge.ptrD()}));
  thrust::for_each_n(zip(countAt(0), edges.beginD()), numEdge,
                     EdgeVerts({vertPos_.ptrD(), numVert, n}));
  thrust::for_each_n(
      countAt(0), numTri,
      InteriorVerts({vertPos_.ptrD(), meshRelation_.barycentric.ptrD(),
                     meshRelation_.triBary.ptrD(), triVertStart, n,
                     halfedge_.ptrD()}));
  // Create subtriangles
  VecDH<glm::ivec3> triVerts(n * n * numTri);
  thrust::for_each_n(countAt(0), numTri,
                     SplitTris({triVerts.ptrD(), halfedge_.cptrD(),
                                half2Edge.cptrD(), numVert, triVertStart, n}));
  CreateHalfedges(triVerts);
}

void Manifold::Impl::Refine(int n) {
  Manifold::Impl old = *this;
  Subdivide(n);

  if (old.halfedgeTangent_.size() == old.halfedge_.size()) {
    VecDH<Barycentric> vertBary(NumVert());
    VecDH<int> lock(NumVert(), 0);
    thrust::for_each_n(
        zip(meshRelation_.triBary.beginD(), countAt(0)), NumTri(),
        TriBary2Vert({vertBary.ptrD(), lock.ptrD(),
                      meshRelation_.barycentric.cptrD(), halfedge_.cptrD()}));

    thrust::for_each_n(
        zip(vertPos_.beginD(), vertBary.beginD()), NumVert(),
        InterpTri({old.halfedge_.cptrD(), old.halfedgeTangent_.cptrD(),
                   old.vertPos_.cptrD()}));
  }

  halfedgeTangent_.resize(0);
  Finish();
}

/**
 * Returns true if this manifold is in fact an oriented 2-manifold and all of
 * the data structures are consistent.
 */
bool Manifold::Impl::IsManifold() const {
  if (halfedge_.size() == 0) return true;
  bool isManifold = thrust::all_of(countAt(0), countAt(halfedge_.size()),
                                   CheckManifold({halfedge_.cptrD()}));
  if (!isManifold) std::cout << "not manifold!" << std::endl;
  VecDH<Halfedge> halfedge(halfedge_);
  thrust::sort(halfedge.beginD(), halfedge.endD());
  isManifold &= thrust::all_of(countAt(0), countAt(2 * NumEdge() - 1),
                               NoDuplicates({halfedge.cptrD()}));
  if (!isManifold) std::cout << "not 2-manifold!" << std::endl;
  return isManifold;
}

/**
 * Returns true if all triangles are CCW relative to their triNormals_.
 */
bool Manifold::Impl::MatchesTriNormals() const {
  if (halfedge_.size() == 0 || faceNormal_.size() != NumTri()) return true;
  return thrust::all_of(thrust::device, countAt(0), countAt(NumTri()),
                        CheckCCW({halfedge_.cptrD(), vertPos_.cptrD(),
                                  faceNormal_.cptrD(), precision_}));
}

/**
 * Returns the surface area and volume of the manifold in a Properties
 * structure. These properties are clamped to zero for a given face if they are
 * within rounding tolerance. This means degenerate manifolds can by identified
 * by testing these properties as == 0.
 */
Properties Manifold::Impl::GetProperties() const {
  if (halfedge_.size() == 0) return {0, 0};
  ApplyTransform();
  thrust::pair<float, float> areaVolume = thrust::transform_reduce(
      countAt(0), countAt(NumTri()),
      FaceAreaVolume({halfedge_.cptrD(), vertPos_.cptrD(), precision_}),
      thrust::make_pair(0.0f, 0.0f), SumPair());
  return {areaVolume.first, areaVolume.second};
}

/**
 * Calculates the bounding box of the entire manifold, which is stored
 * internally to short-cut Boolean operations and to serve as the precision
 * range for Morton code calculation.
 */
void Manifold::Impl::CalculateBBox() {
  bBox_.min = thrust::reduce(vertPos_.beginD(), vertPos_.endD(),
                             glm::vec3(1 / 0.0f), PosMin());
  bBox_.max = thrust::reduce(vertPos_.beginD(), vertPos_.endD(),
                             glm::vec3(-1 / 0.0f), PosMax());
}

/**
 * Sets the precision based on the bounding box, and limits its minimum value by
 * the optional input.
 */
void Manifold::Impl::SetPrecision(float minPrecision) {
  precision_ = glm::max(minPrecision, kTolerance * bBox_.Scale());
  if (!glm::isfinite(precision_)) precision_ = -1;
}

/**
 * Sorts the vertices according to their Morton code.
 */
void Manifold::Impl::SortVerts() {
  VecDH<uint32_t> vertMorton(NumVert());
  thrust::for_each_n(zip(vertMorton.beginD(), vertPos_.cbeginD()), NumVert(),
                     Morton({bBox_}));

  VecDH<int> vertNew2Old(NumVert());
  thrust::sequence(vertNew2Old.beginD(), vertNew2Old.endD());
  thrust::sort_by_key(vertMorton.beginD(), vertMorton.endD(),
                      zip(vertPos_.beginD(), vertNew2Old.beginD()));

  ReindexVerts(vertNew2Old, NumVert());

  // Verts were flagged for removal with NaNs and assigned kNoCode to sort them
  // to the end, which allows them to be removed.
  const int newNumVert =
      thrust::find(vertMorton.beginD(), vertMorton.endD(), kNoCode) -
      vertMorton.beginD();
  vertPos_.resize(newNumVert);
}

/**
 * Updates the halfedges to point to new vert indices based on a mapping,
 * vertNew2Old. This may be a subset, so the total number of original verts is
 * also given.
 */
void Manifold::Impl::ReindexVerts(const VecDH<int>& vertNew2Old,
                                  int oldNumVert) {
  VecDH<int> vertOld2New(oldNumVert);
  thrust::scatter(countAt(0), countAt(NumVert()), vertNew2Old.beginD(),
                  vertOld2New.beginD());
  thrust::for_each(halfedge_.beginD(), halfedge_.endD(),
                   Reindex({vertOld2New.cptrD()}));
}

/**
 * Fills the faceBox and faceMorton input with the bounding boxes and Morton
 * codes of the faces, respectively. The Morton code is based on the center of
 * the bounding box.
 */
void Manifold::Impl::GetFaceBoxMorton(VecDH<Box>& faceBox,
                                      VecDH<uint32_t>& faceMorton) const {
  faceBox.resize(NumTri());
  faceMorton.resize(NumTri());
  thrust::for_each_n(
      zip(faceMorton.beginD(), faceBox.beginD(), countAt(0)), NumTri(),
      FaceMortonBox({halfedge_.cptrD(), vertPos_.cptrD(), bBox_}));
}

/**
 * Sorts the faces of this manifold according to their input Morton code. The
 * bounding box and Morton code arrays are also sorted accordingly.
 */
void Manifold::Impl::SortFaces(VecDH<Box>& faceBox,
                               VecDH<uint32_t>& faceMorton) {
  VecDH<int> faceNew2Old(NumTri());
  thrust::sequence(faceNew2Old.beginD(), faceNew2Old.endD());

  thrust::sort_by_key(faceMorton.beginD(), faceMorton.endD(),
                      zip(faceBox.beginD(), faceNew2Old.beginD()));

  // Tris were flagged for removal with pairedHalfedge = -1 and assigned kNoCode
  // to sort them to the end, which allows them to be removed.
  const int newNumTri =
      thrust::find(faceMorton.beginD(), faceMorton.endD(), kNoCode) -
      faceMorton.beginD();
  faceBox.resize(newNumTri);
  faceMorton.resize(newNumTri);
  faceNew2Old.resize(newNumTri);

  GatherFaces(faceNew2Old);
}

/**
 * Creates the halfedge_ vector for this manifold by copying a set of faces from
 * another manifold, given by oldHalfedge. Input faceNew2Old defines the old
 * faces to gather into this.
 */
void Manifold::Impl::GatherFaces(const VecDH<int>& faceNew2Old) {
  const int numTri = faceNew2Old.size();
  Permute(meshRelation_.triBary, faceNew2Old);

  if (faceNormal_.size() == NumTri()) Permute(faceNormal_, faceNew2Old);

  VecDH<Halfedge> oldHalfedge(halfedge_);
  VecDH<glm::vec4> oldHalfedgeTangent(halfedgeTangent_);
  VecDH<int> faceOld2New(oldHalfedge.size() / 3);
  thrust::scatter(countAt(0), countAt(numTri), faceNew2Old.beginD(),
                  faceOld2New.beginD());

  halfedge_.resize(3 * numTri);
  if (oldHalfedgeTangent.size() != 0) halfedgeTangent_.resize(3 * numTri);
  thrust::for_each_n(
      countAt(0), numTri,
      ReindexFace({halfedge_.ptrD(), halfedgeTangent_.ptrD(),
                   oldHalfedge.cptrD(), oldHalfedgeTangent.cptrD(),
                   faceNew2Old.cptrD(), faceOld2New.cptrD()}));
}

void Manifold::Impl::GatherFaces(const Impl& old,
                                 const VecDH<int>& faceNew2Old) {
  const int numTri = faceNew2Old.size();
  meshRelation_.triBary.resize(numTri);
  thrust::gather(faceNew2Old.beginD(), faceNew2Old.endD(),
                 old.meshRelation_.triBary.beginD(),
                 meshRelation_.triBary.beginD());

  if (old.faceNormal_.size() == old.NumTri()) {
    faceNormal_.resize(numTri);
    thrust::gather(faceNew2Old.beginD(), faceNew2Old.endD(),
                   old.faceNormal_.beginD(), faceNormal_.beginD());
  }

  VecDH<int> faceOld2New(old.NumTri());
  thrust::scatter(countAt(0), countAt(numTri), faceNew2Old.beginD(),
                  faceOld2New.beginD());

  halfedge_.resize(3 * numTri);
  if (old.halfedgeTangent_.size() != 0) halfedgeTangent_.resize(3 * numTri);
  thrust::for_each_n(
      countAt(0), numTri,
      ReindexFace({halfedge_.ptrD(), halfedgeTangent_.ptrD(),
                   old.halfedge_.cptrD(), old.halfedgeTangent_.cptrD(),
                   faceNew2Old.cptrD(), faceOld2New.cptrD()}));
}

/**
 * If face normals are already present, this function uses them to compute
 * vertex normals (angle-weighted pseudo-normals); otherwise it also computes
 * the face normals. Face normals are only calculated when needed because nearly
 * degenerate faces will accrue rounding error, while the Boolean can retain
 * their original normal, which is more accurate and can help with merging
 * coplanar faces.
 *
 * If the face normals have been invalidated by an operation like Warp(), ensure
 * you do faceNormal_.resize(0) before calling this function to force
 * recalculation.
 */
void Manifold::Impl::CalculateNormals() {
  vertNormal_.resize(NumVert());
  thrust::fill(vertNormal_.beginD(), vertNormal_.endD(), glm::vec3(0));
  bool calculateTriNormal = false;
  if (faceNormal_.size() != NumTri()) {
    faceNormal_.resize(NumTri());
    calculateTriNormal = true;
  }
  thrust::for_each_n(
      zip(faceNormal_.beginD(), countAt(0)), NumTri(),
      AssignNormals({vertNormal_.ptrD(), vertPos_.cptrD(), halfedge_.cptrD(),
                     precision_, calculateTriNormal}));
  thrust::for_each(vertNormal_.beginD(), vertNormal_.endD(), Normalize());
}

/**
 * Returns a sparse array of the bounding box overlaps between the edges of the
 * input manifold, Q and the faces of this manifold. Returned indices only
 * point to forward halfedges.
 */
SparseIndices Manifold::Impl::EdgeCollisions(const Impl& Q) const {
  VecDH<TmpEdge> edges = CreateTmpEdges(Q.halfedge_);
  const int numEdge = edges.size();
  VecDH<Box> QedgeBB(numEdge);
  thrust::for_each_n(zip(QedgeBB.beginD(), edges.cbeginD()), numEdge,
                     EdgeBox({Q.vertPos_.cptrD()}));

  SparseIndices q1p2 = collider_.Collisions(QedgeBB);

  thrust::for_each(q1p2.beginD(0), q1p2.endD(0), ReindexEdge({edges.cptrD()}));
  return q1p2;
}

/**
 * Returns a sparse array of the input vertices that project inside the XY
 * bounding boxes of the faces of this manifold.
 */
SparseIndices Manifold::Impl::VertexCollisionsZ(
    const VecDH<glm::vec3>& vertsIn) const {
  return collider_.Collisions(vertsIn);
}

/**
 * For the input face index, return a set of 2D polygons formed by the input
 * projection of the vertices.
 */
Polygons Manifold::Impl::Face2Polygons(int face, glm::mat3x2 projection,
                                       const VecH<int>& faceEdge) const {
  const VecH<glm::vec3>& vertPos = vertPos_.H();
  const VecH<Halfedge>& halfedge = halfedge_.H();
  const int firstEdge = faceEdge[face];
  const int lastEdge = faceEdge[face + 1];

  std::map<int, int> vert_edge;
  for (int edge = firstEdge; edge < lastEdge; ++edge) {
    ALWAYS_ASSERT(
        vert_edge.emplace(std::make_pair(halfedge[edge].startVert, edge))
            .second,
        topologyErr, "face has duplicate vertices.");
  }

  Polygons polys;
  int startEdge = 0;
  int thisEdge = startEdge;
  while (1) {
    if (thisEdge == startEdge) {
      if (vert_edge.empty()) break;
      startEdge = vert_edge.begin()->second;
      thisEdge = startEdge;
      polys.push_back({});
    }
    int vert = halfedge[thisEdge].startVert;
    polys.back().push_back({projection * vertPos[vert], vert});
    const auto result = vert_edge.find(halfedge[thisEdge].endVert);
    ALWAYS_ASSERT(result != vert_edge.end(), topologyErr, "nonmanifold edge");
    thisEdge = result->second;
    vert_edge.erase(result);
  }
  return polys;
}
}  // namespace manifold