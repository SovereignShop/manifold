#include "hip/hip_runtime.h"
// Copyright 2019 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/count.h>
#include <thrust/gather.h>
#include <thrust/logical.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <algorithm>

#include "connected_components.cuh"
#include "manifold_impl.cuh"
#include "polygon.h"

namespace {
using namespace manifold;

constexpr float kTolerance = 1e-5;

struct NormalizeTo {
  float length;
  __host__ __device__ void operator()(glm::vec3& v) {
    v = length * glm::normalize(v);
    if (isnan(v.x)) v = glm::vec3(0.0);
  }
};

struct SplitEdges {
  glm::vec3* vertPos;
  const int startIdx;
  const int n;

  __host__ __device__ void operator()(thrust::tuple<int, EdgeVertsD> in) {
    int edge = thrust::get<0>(in);
    EdgeVertsD edgeVerts = thrust::get<1>(in);

    float invTotal = 1.0f / n;
    for (int i = 1; i < n; ++i)
      vertPos[startIdx + (n - 1) * edge + i - 1] =
          (float(n - i) * vertPos[edgeVerts.first] +
           float(i) * vertPos[edgeVerts.second]) *
          invTotal;
  }
};

struct InteriorVerts {
  glm::vec3* vertPos;
  const int startIdx;
  const int n;

  __host__ __device__ void operator()(thrust::tuple<int, glm::ivec3> in) {
    int tri = thrust::get<0>(in);
    glm::ivec3 triVerts = thrust::get<1>(in);

    int vertsPerTri = ((n - 2) * (n - 2) + (n - 2)) / 2;
    float invTotal = 1.0f / n;
    int pos = startIdx + vertsPerTri * tri;
    for (int i = 1; i < n - 1; ++i)
      for (int j = 1; j < n - i; ++j)
        vertPos[pos++] = (float(i) * vertPos[triVerts[2]] +  //
                          float(j) * vertPos[triVerts[0]] +  //
                          float(n - i - j) * vertPos[triVerts[1]]) *
                         invTotal;
  }
};

struct SplitTris {
  glm::ivec3* triVerts;
  const int edgeIdx;
  const int triIdx;
  const int n;

  __host__ __device__ int EdgeVert(int i, EdgeIdx edge) const {
    return edgeIdx + (n - 1) * edge.Idx() +
           (edge.Dir() > 0 ? i - 1 : n - 1 - i);
  }

  __host__ __device__ int TriVert(int i, int j, int tri) const {
    --i;
    --j;
    int m = n - 2;
    int vertsPerTri = (m * m + m) / 2;
    int vertOffset = (i * (2 * m - i + 1)) / 2 + j;
    return triIdx + vertsPerTri * tri + vertOffset;
  }

  __host__ __device__ int Vert(int i, int j, int tri, glm::ivec3 triVert,
                               TriEdges triEdge) const {
    bool edge0 = i == 0;
    bool edge1 = j == 0;
    bool edge2 = j == n - i;
    if (edge0) {
      if (edge1)
        return triVert[1];
      else if (edge2)
        return triVert[0];
      else
        return EdgeVert(n - j, triEdge[0]);
    } else if (edge1) {
      if (edge2)
        return triVert[2];
      else
        return EdgeVert(i, triEdge[1]);
    } else if (edge2)
      return EdgeVert(j, triEdge[2]);
    else
      return TriVert(i, j, tri);
  }

  __host__ __device__ void operator()(
      thrust::tuple<int, glm::ivec3, TriEdges> in) {
    int tri = thrust::get<0>(in);
    glm::ivec3 triVert = thrust::get<1>(in);
    TriEdges triEdge = thrust::get<2>(in);

    int pos = n * n * tri;
    for (int i = 0; i < n; ++i) {
      for (int j = 0; j < n - i; ++j) {
        int a = Vert(i, j, tri, triVert, triEdge);
        int b = Vert(i + 1, j, tri, triVert, triEdge);
        int c = Vert(i, j + 1, tri, triVert, triEdge);
        triVerts[pos++] = glm::ivec3(a, b, c);
        if (j < n - 1 - i) {
          int d = Vert(i + 1, j + 1, tri, triVert, triEdge);
          triVerts[pos++] = glm::ivec3(b, d, c);
        }
      }
    }
  }
};

__host__ __device__ void AtomicAddFloat(float& target, float add) {
#ifdef __CUDA_ARCH__
  atomicAdd(&target, add);
#else
#pragma omp atomic
  target += add;
#endif
}

struct AreaVolume {
  float* surfaceArea;
  float* volume;
  const int* vertLabel;
  const glm::vec3* vertPos;

  __host__ __device__ void operator()(const glm::ivec3& triVerts) {
    glm::vec3 edge[3];
    float perimeter = 0.0f;
    for (int i : {0, 1, 2}) {
      edge[i] = vertPos[triVerts[(i + 1) % 3]] - vertPos[triVerts[i]];
      perimeter += glm::length(edge[i]);
    }
    glm::vec3 crossP = glm::cross(edge[0], edge[1]);
    float area = glm::length(crossP) / 2.0f;
    if (area > perimeter * kTolerance) {
      int comp = vertLabel[triVerts[0]];
      AtomicAddFloat(surfaceArea[comp], area);
      AtomicAddFloat(volume[comp],
                     glm::dot(crossP, vertPos[triVerts[0]]) / 6.0f);
    }
  }
};

struct ClampVolume {
  __host__ __device__ void operator()(thrust::tuple<float&, float> inOut) {
    float& volume = thrust::get<0>(inOut);
    float surfaceArea = thrust::get<1>(inOut);

    if (glm::abs(volume) < surfaceArea * kTolerance) volume = 0.0f;
  }
};

struct NonZero {
  __host__ __device__ bool operator()(float val) { return val != 0.0f; }
};

struct RemoveVert {
  const float* volume;

  __host__ __device__ bool operator()(thrust::tuple<int, int, glm::vec3> in) {
    int vertLabel = thrust::get<0>(in);
    return volume[vertLabel] == 0.0f;
  }
};

struct RemoveTri {
  __host__ __device__ bool operator()(thrust::tuple<glm::ivec3, glm::vec3> in) {
    const glm::ivec3& triVerts = thrust::get<0>(in);
    return triVerts[0] < 0;
  }
};

struct IdxMin
    : public thrust::binary_function<glm::ivec3, glm::ivec3, glm::ivec3> {
  __host__ __device__ int min3(glm::ivec3 a) {
    return glm::min(a.x, glm::min(a.y, a.z));
  }
  __host__ __device__ glm::ivec3 operator()(glm::ivec3 a, glm::ivec3 b) {
    return glm::ivec3(glm::min(min3(a), min3(b)));
  }
};

struct IdxMax
    : public thrust::binary_function<glm::ivec3, glm::ivec3, glm::ivec3> {
  __host__ __device__ int max3(glm::ivec3 a) {
    return glm::max(a.x, glm::max(a.y, a.z));
  }
  __host__ __device__ glm::ivec3 operator()(glm::ivec3 a, glm::ivec3 b) {
    return glm::ivec3(glm::max(max3(a), max3(b)));
  }
};

struct PosMin
    : public thrust::binary_function<glm::vec3, glm::vec3, glm::vec3> {
  __host__ __device__ glm::vec3 operator()(glm::vec3 a, glm::vec3 b) {
    return glm::min(a, b);
  }
};

struct PosMax
    : public thrust::binary_function<glm::vec3, glm::vec3, glm::vec3> {
  __host__ __device__ glm::vec3 operator()(glm::vec3 a, glm::vec3 b) {
    return glm::max(a, b);
  }
};

struct Transform {
  const glm::mat4x3 transform;

  __host__ __device__ void operator()(glm::vec3& position) {
    position = transform * glm::vec4(position, 1.0f);
  }
};

struct TransformNormals {
  const glm::mat3 transform;

  __host__ __device__ void operator()(glm::vec3& normal) {
    normal = glm::normalize(transform * normal);
    if (isnan(normal.x)) normal = glm::vec3(0.0f);
  }
};

__host__ __device__ uint32_t SpreadBits3(uint32_t v) {
  v = 0xFF0000FFu & (v * 0x00010001u);
  v = 0x0F00F00Fu & (v * 0x00000101u);
  v = 0xC30C30C3u & (v * 0x00000011u);
  v = 0x49249249u & (v * 0x00000005u);
  return v;
}

__host__ __device__ uint32_t MortonCode(glm::vec3 position, Box bBox) {
  glm::vec3 xyz = (position - bBox.min) / (bBox.max - bBox.min);
  xyz = glm::min(glm::vec3(1023.0f), glm::max(glm::vec3(0.0f), 1024.0f * xyz));
  uint32_t x = SpreadBits3(static_cast<uint32_t>(xyz.x));
  uint32_t y = SpreadBits3(static_cast<uint32_t>(xyz.y));
  uint32_t z = SpreadBits3(static_cast<uint32_t>(xyz.z));
  return x * 4 + y * 2 + z;
}

struct Morton {
  const Box bBox;

  __host__ __device__ void operator()(
      thrust::tuple<uint32_t&, const glm::vec3&> inout) {
    glm::vec3 position = thrust::get<1>(inout);
    thrust::get<0>(inout) = MortonCode(position, bBox);
  }
};

struct Reindex {
  const int* indexInv_;

  __host__ __device__ void operator()(glm::ivec3& triVerts) {
    for (int i : {0, 1, 2}) triVerts[i] = indexInv_[triVerts[i]];
  }
};

struct TriMortonBox {
  const glm::vec3* vertPos;
  const Box bBox;

  __host__ __device__ void operator()(
      thrust::tuple<uint32_t&, Box&, const glm::ivec3&> inout) {
    uint32_t& mortonCode = thrust::get<0>(inout);
    Box& triBox = thrust::get<1>(inout);
    const glm::ivec3& triVerts = thrust::get<2>(inout);

    glm::vec3 center =
        (vertPos[triVerts[0]] + vertPos[triVerts[1]] + vertPos[triVerts[2]]) /
        3.0f;
    mortonCode = MortonCode(center, bBox);
    triBox = Box(vertPos[triVerts[0]], vertPos[triVerts[1]]);
    triBox.Union(vertPos[triVerts[2]]);
  }
};

__host__ __device__ void AtomicAddVec3(glm::vec3& target,
                                       const glm::vec3& add) {
  for (int i : {0, 1, 2}) {
#ifdef __CUDA_ARCH__
    atomicAdd(&target[i], add[i]);
#else
#pragma omp atomic
    target[i] += add[i];
#endif
  }
}

struct AssignNormals {
  glm::vec3* vertNormal;
  const glm::vec3* vertPos;
  const bool calculateTriNormal;

  __host__ __device__ void operator()(
      thrust::tuple<glm::vec3&, const glm::ivec3&, const TriEdges&> in) {
    glm::vec3& triNormal = thrust::get<0>(in);
    const glm::ivec3& triVerts = thrust::get<1>(in);
    const TriEdges& triEdges = thrust::get<2>(in);

    glm::vec3 v0 = vertPos[triVerts[0]];
    glm::vec3 v1 = vertPos[triVerts[1]];
    glm::vec3 v2 = vertPos[triVerts[2]];
    // edge vectors
    glm::vec3 e01 = glm::normalize(v1 - v0);
    glm::vec3 e12 = glm::normalize(v2 - v1);
    glm::vec3 e20 = glm::normalize(v0 - v2);

    if (calculateTriNormal) {
      triNormal = glm::normalize(glm::cross(e01, e12));
      if (isnan(triNormal.x)) triNormal = glm::vec3(0.0);
    }
    // corner angles
    glm::vec3 phi;
    phi[0] = glm::acos(-glm::dot(e01, e12));
    phi[1] = glm::acos(-glm::dot(e12, e20));
    phi[2] = glm::pi<float>() - phi[0] - phi[1];
    // assign weighted sum
    for (int i : {0, 1, 2}) {
      AtomicAddVec3(vertNormal[triVerts[i]],
                    glm::max(phi[i], kTolerance) * triNormal);
    }
  }
};

struct TmpEdge {
  int first, second, halfedgeIdx;

  __host__ __device__ TmpEdge() {}
  __host__ __device__ TmpEdge(int start, int end, int idx) {
    first = glm::min(start, end);
    second = glm::max(start, end);
    halfedgeIdx = idx;
  }

  __host__ __device__ bool operator<(const TmpEdge& other) const {
    return first == other.first ? second < other.second : first < other.first;
  }
};

struct Tri2Halfedges {
  Halfedge* halfedges;
  TmpEdge* edges;

  __host__ __device__ void operator()(
      thrust::tuple<int, const glm::ivec3&> in) {
    const int tri = thrust::get<0>(in);
    const glm::ivec3& triVerts = thrust::get<1>(in);
    for (const int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      const int edge = 3 * tri + i;
      halfedges[edge] = {triVerts[i], triVerts[j], -1, tri};
      edges[edge] = TmpEdge(triVerts[i], triVerts[j], edge);
    }
  }
};

struct LinkHalfedges {
  Halfedge* halfedges;
  const TmpEdge* edges;

  __host__ __device__ void operator()(int k) {
    const int i = 2 * k;
    const int j = i + 1;
    const int pair0 = edges[i].halfedgeIdx;
    const int pair1 = edges[j].halfedgeIdx;
    if (halfedges[pair0].startVert != halfedges[pair1].endVert ||
        halfedges[pair0].endVert != halfedges[pair1].startVert ||
        halfedges[pair0].face == halfedges[pair1].face)
      printf("Not manifold!\n");
    halfedges[pair0].pairedHalfedge = pair1;
    halfedges[pair1].pairedHalfedge = pair0;
  }
};

struct MakeHalfedges {
  int i, j;

  __host__ __device__ void operator()(
      thrust::tuple<TriEdges&, int&, EdgeVertsD&, const glm::ivec3&> inout) {
    const glm::ivec3& in = thrust::get<3>(inout);
    int V1 = in[i];
    int V2 = in[j];
    TriEdges& triEdges = thrust::get<0>(inout);
    int& dir = thrust::get<1>(inout);
    EdgeVertsD& edgeVerts = thrust::get<2>(inout);
    if (V1 < V2) {  // forward
      dir = 1;
      edgeVerts = thrust::make_pair(V1, V2);
    } else if (V1 > V2) {  // backward
      dir = -1;
      edgeVerts = thrust::make_pair(V2, V1);
    } else {
      dir = 0;
      edgeVerts = thrust::make_pair(V2, V1);
    }
    triEdges[i] = EdgeIdx(0, dir);
  }
};

struct AssignEdges {
  int i;

  __host__ __device__ void operator()(thrust::tuple<TriEdges&, int> inout) {
    int idx2 = thrust::get<1>(inout);
    TriEdges& triEdges = thrust::get<0>(inout);
    triEdges[i] = EdgeIdx(idx2 / 2, triEdges[i].Dir());
  }
};

struct OpposedDir {
  __host__ __device__ bool operator()(int a, int b) const {
    return a * b == -1;
  }
};

struct LinkEdges2Tris {
  EdgeTrisD* edgeTris;

  __host__ __device__ void operator()(thrust::tuple<int, TriEdges> in) {
    const int tri = thrust::get<0>(in);
    const TriEdges triEdges = thrust::get<1>(in);
    for (int i : {0, 1, 2}) {
      if (triEdges[i].Dir() > 0)
        edgeTris[triEdges[i].Idx()].left = tri;
      else
        edgeTris[triEdges[i].Idx()].right = tri;
    }
  }
};

struct Halfedge2Tmp {
  __host__ __device__ void operator()(
      thrust::tuple<TmpEdge&, const Halfedge&, int> inout) {
    const Halfedge& halfedge = thrust::get<1>(inout);
    int idx = thrust::get<2>(inout);
    if (halfedge.startVert > halfedge.endVert) idx = -1;

    thrust::get<0>(inout) = TmpEdge(halfedge.startVert, halfedge.endVert, idx);
  }
};

struct TmpInvalid {
  __host__ __device__ bool operator()(const TmpEdge& edge) {
    return edge.halfedgeIdx < 0;
  }
};

struct EdgeBox {
  const glm::vec3* vertPos;

  __host__ __device__ void operator()(
      thrust::tuple<Box&, const TmpEdge&> inout) {
    const TmpEdge& edge = thrust::get<1>(inout);
    thrust::get<0>(inout) = Box(vertPos[edge.first], vertPos[edge.second]);
  }
};

struct ReindexEdge {
  const TmpEdge* edges;

  __host__ __device__ void operator()(int& edge) {
    edge = edges[edge].halfedgeIdx;
  }
};

struct CheckTris {
  const EdgeVertsD* edgeVerts;

  __host__ __device__ bool operator()(thrust::tuple<glm::ivec3, TriEdges> in) {
    const glm::ivec3& triVerts = thrust::get<0>(in);
    const TriEdges& triEdges = thrust::get<1>(in);
    bool good = true;
    for (int i : {0, 1, 2}) {
      int j = (i + 1) % 3;
      if (triEdges[i].Dir() > 0) {
        good &= triVerts[i] == edgeVerts[triEdges[i].Idx()].first;
        good &= triVerts[j] == edgeVerts[triEdges[i].Idx()].second;
      } else {
        good &= triVerts[i] == edgeVerts[triEdges[i].Idx()].second;
        good &= triVerts[j] == edgeVerts[triEdges[i].Idx()].first;
      }
    }
    return good;
  }
};
}  // namespace

namespace manifold {

Manifold::Impl::Impl(const Mesh& manifold)
    : vertPos_(manifold.vertPos), triVerts_(manifold.triVerts) {
  CheckDevice();
  Finish();
}

Manifold::Impl::Impl(Shape shape) {
  std::vector<glm::vec3> vertPos;
  std::vector<glm::ivec3> triVerts;
  switch (shape) {
    case Shape::TETRAHEDRON:
      vertPos = {{-1.0f, -1.0f, 1.0f},
                 {-1.0f, 1.0f, -1.0f},
                 {1.0f, -1.0f, -1.0f},
                 {1.0f, 1.0f, 1.0f}};
      triVerts = {{2, 0, 1}, {0, 3, 1}, {2, 3, 0}, {3, 2, 1}};
      break;
    case Shape::CUBE:
      vertPos = {{0.0f, 0.0f, 0.0f},  //
                 {1.0f, 0.0f, 0.0f},  //
                 {1.0f, 1.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},  //
                 {1.0f, 0.0f, 1.0f},  //
                 {1.0f, 1.0f, 1.0f},  //
                 {0.0f, 1.0f, 1.0f}};
      triVerts = {{0, 2, 1}, {0, 3, 2},  //
                  {4, 5, 6}, {4, 6, 7},  //
                  {0, 1, 5}, {0, 5, 4},  //
                  {1, 2, 6}, {1, 6, 5},  //
                  {2, 3, 7}, {2, 7, 6},  //
                  {3, 0, 4}, {3, 4, 7}};
      break;
    case Shape::OCTAHEDRON:
      vertPos = {{1.0f, 0.0f, 0.0f},   //
                 {-1.0f, 0.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},   //
                 {0.0f, -1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},   //
                 {0.0f, 0.0f, -1.0f}};
      triVerts = {{0, 2, 4}, {1, 5, 3},  //
                  {2, 1, 4}, {3, 5, 0},  //
                  {1, 3, 4}, {0, 5, 2},  //
                  {3, 0, 4}, {2, 5, 1}};
      break;
    default:
      throw logicErr("Unrecognized shape!");
  }
  vertPos_ = vertPos;
  triVerts_ = triVerts;
  Finish();
}

void Manifold::Impl::RemoveChaff() {
  CreateEdges();
  int n_comp = ConnectedComponents(vertLabel_, NumVert(), halfedge_);

  VecDH<float> surfaceArea(n_comp), volume(n_comp);
  thrust::for_each_n(triVerts_.beginD(), NumTri(),
                     AreaVolume({surfaceArea.ptrD(), volume.ptrD(),
                                 vertLabel_.cptrD(), vertPos_.cptrD()}));
  thrust::for_each_n(zip(volume.beginD(), surfaceArea.beginD()), n_comp,
                     ClampVolume());
  numLabel_ = thrust::count_if(volume.beginD(), volume.endD(), NonZero());

  VecDH<int> newVert2Old(NumVert());
  thrust::sequence(newVert2Old.begin(), newVert2Old.end());
  auto begin =
      zip(vertLabel_.beginD(), newVert2Old.beginD(), vertPos_.beginD());
  int newNumVert =
      thrust::remove_if(
          begin, zip(vertLabel_.endD(), newVert2Old.endD(), vertPos_.endD()),
          RemoveVert({volume.cptrD()})) -
      begin;

  VecDH<int> oldVert2New(NumVert(), -1);
  vertPos_.resize(newNumVert);
  vertLabel_.resize(newNumVert);
  thrust::scatter(thrust::make_counting_iterator(0),
                  thrust::make_counting_iterator(newNumVert),
                  newVert2Old.beginD(), oldVert2New.beginD());

  thrust::for_each(triVerts_.beginD(), triVerts_.endD(),
                   Reindex({oldVert2New.cptrD()}));

  auto start = zip(triVerts_.beginD(), triNormal_.beginD());
  int newNumTri =
      thrust::remove_if(start, zip(triVerts_.endD(), triNormal_.endD()),
                        RemoveTri()) -
      start;
  triVerts_.resize(newNumTri);
  triNormal_.resize(newNumTri);
}

void Manifold::Impl::Finish() {
  if (triVerts_.size() == 0) return;
  ALWAYS_ASSERT(thrust::reduce(triVerts_.beginD(), triVerts_.endD(),
                               glm::ivec3(std::numeric_limits<int>::max()),
                               IdxMin())[0] >= 0,
                runtimeErr, "Negative vertex index!");
  ALWAYS_ASSERT(thrust::reduce(triVerts_.beginD(), triVerts_.endD(),
                               glm::ivec3(-1), IdxMax())[0] < NumVert(),
                runtimeErr, "Vertex index exceeds number of verts!");
  if (vertLabel_.size() != NumVert()) {
    vertLabel_.resize(NumVert());
    numLabel_ = 1;
    thrust::fill(vertLabel_.beginD(), vertLabel_.endD(), 0);
  }
  CalculateBBox();
  SortVerts();
  VecDH<Box> triBox;
  VecDH<uint32_t> triMorton;
  GetTriBoxMorton(triBox, triMorton);
  SortTris(triBox, triMorton);
  CreateEdges();
  CreateHalfedges(triVerts_);
  CalculateNormals();
  collider_ = Collider(triBox, triMorton);
}

void Manifold::Impl::CreateHalfedges(const VecDH<glm::ivec3>& triVerts) {
  const int numTri = triVerts.size();
  halfedge_.resize(3 * numTri);
  VecDH<TmpEdge> edge(3 * numTri);
  thrust::for_each_n(zip(thrust::make_counting_iterator(0), triVerts.beginD()),
                     numTri, Tri2Halfedges({halfedge_.ptrD(), edge.ptrD()}));
  thrust::sort(edge.beginD(), edge.endD());
  thrust::for_each_n(thrust::make_counting_iterator(0), halfedge_.size() / 2,
                     LinkHalfedges({halfedge_.ptrD(), edge.cptrD()}));
}

void Manifold::Impl::Update() {
  CalculateBBox();
  VecDH<Box> triBox;
  VecDH<uint32_t> triMorton;
  GetTriBoxMorton(triBox, triMorton);
  collider_.UpdateBoxes(triBox);
}

void Manifold::Impl::ApplyTransform() const {
  // This const_cast is here because these operations cancel out, leaving the
  // state conceptually unchanged. This enables lazy transformation evaluation.
  const_cast<Impl*>(this)->ApplyTransform();
}

void Manifold::Impl::ApplyTransform() {
  if (transform_ == glm::mat4x3(1.0f)) return;
  thrust::for_each(vertPos_.beginD(), vertPos_.endD(), Transform({transform_}));

  glm::mat3 normalTransform =
      glm::inverse(glm::transpose(glm::mat3(transform_)));
  thrust::for_each(triNormal_.beginD(), triNormal_.endD(),
                   TransformNormals({normalTransform}));
  thrust::for_each(vertNormal_.beginD(), vertNormal_.endD(),
                   TransformNormals({normalTransform}));
  // This optimization does a cheap collider update if the transform is
  // axis-aligned.
  if (!collider_.Transform(transform_)) Update();
  transform_ = glm::mat4x3(1.0f);
  CalculateBBox();
}

bool Manifold::Impl::Tri2Face() const {
  // This const_cast is here because this operation tweaks the internal data
  // structure, but does not change what it represents.
  return const_cast<Impl*>(this)->Tri2Face();
}

bool Manifold::Impl::Tri2Face() {
  if (face_.size() != 0 || halfedge_.size() % 3 != 0) return false;
  face_.resize(halfedge_.size() / 3 + 1);
  thrust::sequence(face_.beginD(), face_.endD(), 0, 3);
  return true;
}

void Manifold::Impl::Refine(int n) {
  // This function doesn't run Finish(), as that is expensive and it'll need to
  // be run after the new vertices have moved, which is a likely scenario after
  // refinement (smoothing).
  int numVert = NumVert();
  int numEdge = NumEdge();
  int numTri = NumTri();
  // Append new verts
  int vertsPerEdge = n - 1;
  int vertsPerTri = ((n - 2) * (n - 2) + (n - 2)) / 2;
  int triVertStart = numVert + numEdge * vertsPerEdge;
  vertPos_.resize(triVertStart + numTri * vertsPerTri);
  thrust::for_each_n(
      zip(thrust::make_counting_iterator(0), edgeVerts_.beginD()), numEdge,
      SplitEdges({vertPos_.ptrD(), numVert, n}));
  thrust::for_each_n(zip(thrust::make_counting_iterator(0), triVerts_.beginD()),
                     numTri, InteriorVerts({vertPos_.ptrD(), triVertStart, n}));
  // Create subtriangles
  VecDH<glm::ivec3> inTri(triVerts_);
  triVerts_.resize(n * n * numTri);
  thrust::for_each_n(zip(thrust::make_counting_iterator(0), inTri.beginD(),
                         triEdges_.beginD()),
                     numTri,
                     SplitTris({triVerts_.ptrD(), numVert, triVertStart, n}));
}

bool Manifold::Impl::IsValid() const {
  return thrust::all_of(zip(triVerts_.beginD(), triEdges_.beginD()),
                        zip(triVerts_.endD(), triEdges_.endD()),
                        CheckTris({edgeVerts_.ptrD()}));
}

void Manifold::Impl::CalculateBBox() {
  bBox_.min = thrust::reduce(vertPos_.begin(), vertPos_.end(),
                             glm::vec3(1 / 0.0f), PosMin());
  bBox_.max = thrust::reduce(vertPos_.begin(), vertPos_.end(),
                             glm::vec3(-1 / 0.0f), PosMax());
  ALWAYS_ASSERT(bBox_.isFinite(), runtimeErr,
                "Input vertices are not all finite!");
}

void Manifold::Impl::SortVerts() {
  VecDH<uint32_t> vertMorton(NumVert());
  thrust::for_each_n(zip(vertMorton.beginD(), vertPos_.cbeginD()), NumVert(),
                     Morton({bBox_}));

  VecDH<int> vertNew2Old(NumVert());
  thrust::sequence(vertNew2Old.beginD(), vertNew2Old.endD());
  thrust::sort_by_key(
      vertMorton.beginD(), vertMorton.endD(),
      zip(vertPos_.beginD(), vertLabel_.beginD(), vertNew2Old.beginD()));

  VecDH<int> vertOld2New(NumVert());
  thrust::scatter(thrust::make_counting_iterator(0),
                  thrust::make_counting_iterator(NumVert()),
                  vertNew2Old.beginD(), vertOld2New.beginD());
  thrust::for_each(triVerts_.beginD(), triVerts_.endD(),
                   Reindex({vertOld2New.cptrD()}));
}

void Manifold::Impl::CreateEdges() {
  VecDH<EdgeVertsD> halfEdgeVerts(NumTri() * 3);
  VecDH<int> dir(NumTri() * 3);
  edgeVerts_.resize(halfEdgeVerts.size() / 2);
  triEdges_.resize(NumTri());
  edgeTris_.resize(NumEdge());
  for (int i : {0, 1, 2}) {
    int j = (i + 1) % 3;
    int start = i * NumTri();
    thrust::for_each_n(zip(triEdges_.beginD(), dir.beginD() + start,
                           halfEdgeVerts.beginD() + start, triVerts_.cbeginD()),
                       NumTri(), MakeHalfedges({i, j}));
  }
  SortHalfedges(halfEdgeVerts, dir);
  strided_range<VecDH<EdgeVertsD>::IterD> edgeVerts(halfEdgeVerts.beginD(),
                                                    halfEdgeVerts.endD(), 2);
  thrust::copy(edgeVerts.begin(), edgeVerts.end(), edgeVerts_.beginD());

  thrust::for_each_n(zip(thrust::make_counting_iterator(0), triEdges_.beginD()),
                     NumTri(), LinkEdges2Tris({edgeTris_.ptrD()}));
  // verify
  strided_range<VecDH<EdgeVertsD>::IterD> edgesOdd(halfEdgeVerts.beginD() + 1,
                                                   halfEdgeVerts.endD(), 2);
  ALWAYS_ASSERT(
      thrust::equal(edgeVerts.begin(), edgeVerts.end(), edgesOdd.begin()),
      runtimeErr, "Manifold is not manifold!");
  strided_range<VecDH<int>::IterD> dir1(dir.beginD(), dir.endD(), 2);
  strided_range<VecDH<int>::IterD> dir2(dir.beginD() + 1, dir.endD(), 2);
  ALWAYS_ASSERT(
      thrust::equal(dir1.begin(), dir1.end(), dir2.begin(), OpposedDir()),
      runtimeErr, "Manifold is not oriented!");
}

void Manifold::Impl::SortHalfedges(VecDH<EdgeVertsD>& halfEdgeVerts,
                                   VecDH<int>& dir) {
  VecDH<int> halfedgeNew2Old(NumTri() * 3);
  thrust::sequence(halfedgeNew2Old.beginD(), halfedgeNew2Old.endD());
  thrust::sort_by_key(halfEdgeVerts.beginD(), halfEdgeVerts.endD(),
                      zip(dir.beginD(), halfedgeNew2Old.beginD()));

  VecDH<int> halfedgeOld2New(NumTri() * 3);
  thrust::scatter(thrust::make_counting_iterator(0),
                  thrust::make_counting_iterator((int)halfedgeNew2Old.size()),
                  halfedgeNew2Old.beginD(), halfedgeOld2New.beginD());
  // assign edge idx to triEdges_ (assumes edge dir is already assigned)
  for (int i : {0, 1, 2}) {
    int start = i * NumTri();
    thrust::for_each_n(
        zip(triEdges_.beginD(), halfedgeOld2New.cbeginD() + start), NumTri(),
        AssignEdges({i}));
  }
}

void Manifold::Impl::GetTriBoxMorton(VecDH<Box>& triBox,
                                     VecDH<uint32_t>& triMorton) const {
  triBox.resize(NumTri());
  triMorton.resize(NumTri());
  thrust::for_each_n(
      zip(triMorton.beginD(), triBox.beginD(), triVerts_.cbeginD()), NumTri(),
      TriMortonBox({vertPos_.cptrD(), bBox_}));
}

void Manifold::Impl::SortTris(VecDH<Box>& triBox, VecDH<uint32_t>& triMorton) {
  if (triNormal_.size() == NumTri()) {
    thrust::sort_by_key(
        triMorton.beginD(), triMorton.endD(),
        zip(triBox.beginD(), triVerts_.beginD(), triNormal_.beginD()));
  } else {
    thrust::sort_by_key(triMorton.beginD(), triMorton.endD(),
                        zip(triBox.beginD(), triVerts_.beginD()));
  }
}

void Manifold::Impl::CalculateNormals() {
  vertNormal_.resize(NumVert());
  bool calculateTriNormal = false;
  if (triNormal_.size() != NumTri()) {
    triNormal_.resize(NumTri());
    calculateTriNormal = true;
  }
  thrust::for_each_n(
      zip(triNormal_.beginD(), triVerts_.beginD(), triEdges_.beginD()),
      NumTri(), AssignNormals({vertNormal_.ptrD(), vertPos_.cptrD(),
                               calculateTriNormal}));
  thrust::for_each(vertNormal_.begin(), vertNormal_.end(), NormalizeTo({1.0}));
}

SparseIndices Manifold::Impl::EdgeCollisions(const Impl& Q) const {
  VecDH<TmpEdge> edges(Q.halfedge_.size());
  thrust::for_each_n(zip(edges.beginD(), Q.halfedge_.beginD(),
                         thrust::make_counting_iterator(0)),
                     edges.size(), Halfedge2Tmp());
  int numEdge = thrust::remove_if(edges.beginD(), edges.endD(), TmpInvalid()) -
                edges.beginD();
  ALWAYS_ASSERT(numEdge == Q.NumEdge(), runtimeErr, "Not oriented!");
  edges.resize(numEdge);

  VecDH<Box> QedgeBB(numEdge);
  thrust::for_each_n(zip(QedgeBB.beginD(), edges.cbeginD()), numEdge,
                     EdgeBox({Q.vertPos_.cptrD()}));

  SparseIndices p2q1 = collider_.Collisions(QedgeBB);

  thrust::for_each(p2q1.beginD(1), p2q1.endD(1), ReindexEdge({edges.cptrD()}));
  return p2q1;
}

SparseIndices Manifold::Impl::VertexCollisionsZ(
    const VecDH<glm::vec3>& vertsIn) const {
  return collider_.Collisions(vertsIn);
}
}  // namespace manifold