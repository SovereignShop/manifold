#include "hip/hip_runtime.h"
// Copyright 2019 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/adjacent_difference.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>
#include <thrust/gather.h>
#include <thrust/logical.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>

#include <algorithm>
#include <map>

#include "connected_components.cuh"
#include "manifold_impl.cuh"
#include "polygon.h"

namespace {
using namespace manifold;

/**
 * Represents the uncertainty of the vertices (greater than or equal to
 * worst-case floating-point precision). Used to determine when face surface
 * area or volume is small enough to clamp to zero. TODO: this should be based
 * on the bounding box, and probably passed through Boolean operations. It
 * should also be passed into the Polygon triangulator, where it is more
 * important.
 */
constexpr float kTolerance = 1e-5;

struct Normalize {
  __host__ __device__ void operator()(glm::vec3& v) {
    v = glm::normalize(v);
    if (isnan(v.x)) v = glm::vec3(0.0);
  }
};

/**
 * This is a temporary edge strcture which only stores edges forward and
 * references the halfedge it was created from.
 */
struct TmpEdge {
  int first, second, halfedgeIdx;

  __host__ __device__ TmpEdge() {}
  __host__ __device__ TmpEdge(int start, int end, int idx) {
    first = glm::min(start, end);
    second = glm::max(start, end);
    halfedgeIdx = idx;
  }

  __host__ __device__ bool operator<(const TmpEdge& other) const {
    return first == other.first ? second < other.second : first < other.first;
  }
};

struct Halfedge2Tmp {
  __host__ __device__ void operator()(
      thrust::tuple<TmpEdge&, const Halfedge&, int> inout) {
    const Halfedge& halfedge = thrust::get<1>(inout);
    int idx = thrust::get<2>(inout);
    if (!halfedge.IsForward()) idx = -1;

    thrust::get<0>(inout) = TmpEdge(halfedge.startVert, halfedge.endVert, idx);
  }
};

struct TmpInvalid {
  __host__ __device__ bool operator()(const TmpEdge& edge) {
    return edge.halfedgeIdx < 0;
  }
};

VecDH<TmpEdge> CreateTmpEdges(const VecDH<Halfedge>& halfedge) {
  VecDH<TmpEdge> edges(halfedge.size());
  thrust::for_each_n(zip(edges.beginD(), halfedge.beginD(), countAt(0)),
                     edges.size(), Halfedge2Tmp());
  int numEdge = thrust::remove_if(edges.beginD(), edges.endD(), TmpInvalid()) -
                edges.beginD();
  ALWAYS_ASSERT(numEdge == halfedge.size() / 2, topologyErr, "Not oriented!");
  edges.resize(numEdge);
  return edges;
}

struct ReindexEdge {
  const TmpEdge* edges;

  __host__ __device__ void operator()(int& edge) {
    edge = edges[edge].halfedgeIdx;
  }
};

struct ReindexHalfedge {
  int* half2Edge;

  __host__ __device__ void operator()(thrust::tuple<int, TmpEdge> in) {
    const int edge = thrust::get<0>(in);
    const int halfedge = thrust::get<1>(in).halfedgeIdx;

    half2Edge[halfedge] = edge;
  }
};

struct SplitEdges {
  glm::vec3* vertPos;
  const int startIdx;
  const int n;

  __host__ __device__ void operator()(thrust::tuple<int, TmpEdge> in) {
    int edge = thrust::get<0>(in);
    TmpEdge edgeVerts = thrust::get<1>(in);

    float invTotal = 1.0f / n;
    for (int i = 1; i < n; ++i)
      vertPos[startIdx + (n - 1) * edge + i - 1] =
          (float(n - i) * vertPos[edgeVerts.first] +
           float(i) * vertPos[edgeVerts.second]) *
          invTotal;
  }
};

struct InteriorVerts {
  glm::vec3* vertPos;
  const int startIdx;
  const int n;
  const Halfedge* halfedge;

  __host__ __device__ void operator()(int tri) {
    int vertsPerTri = ((n - 2) * (n - 2) + (n - 2)) / 2;
    float invTotal = 1.0f / n;
    int pos = startIdx + vertsPerTri * tri;
    for (int i = 1; i < n - 1; ++i)
      for (int j = 1; j < n - i; ++j)
        vertPos[pos++] =
            (float(i) * vertPos[halfedge[3 * tri + 2].startVert] +  //
             float(j) * vertPos[halfedge[3 * tri].startVert] +      //
             float(n - i - j) * vertPos[halfedge[3 * tri + 1].startVert]) *
            invTotal;
  }
};

struct SplitTris {
  glm::ivec3* triVerts;
  const Halfedge* halfedge;
  const int* half2Edge;
  const int edgeIdx;
  const int triIdx;
  const int n;

  __host__ __device__ int EdgeVert(int i, int inHalfedge) const {
    bool forward = halfedge[inHalfedge].IsForward();
    int edge = forward ? half2Edge[inHalfedge]
                       : half2Edge[halfedge[inHalfedge].pairedHalfedge];
    return edgeIdx + (n - 1) * edge + (forward ? i - 1 : n - 1 - i);
  }

  __host__ __device__ int TriVert(int i, int j, int tri) const {
    --i;
    --j;
    int m = n - 2;
    int vertsPerTri = (m * m + m) / 2;
    int vertOffset = (i * (2 * m - i + 1)) / 2 + j;
    return triIdx + vertsPerTri * tri + vertOffset;
  }

  __host__ __device__ int Vert(int i, int j, int tri) const {
    bool edge0 = i == 0;
    bool edge1 = j == 0;
    bool edge2 = j == n - i;
    if (edge0) {
      if (edge1)
        return halfedge[3 * tri + 1].startVert;
      else if (edge2)
        return halfedge[3 * tri].startVert;
      else
        return EdgeVert(n - j, 3 * tri);
    } else if (edge1) {
      if (edge2)
        return halfedge[3 * tri + 2].startVert;
      else
        return EdgeVert(i, 3 * tri + 1);
    } else if (edge2)
      return EdgeVert(j, 3 * tri + 2);
    else
      return TriVert(i, j, tri);
  }

  __host__ __device__ void operator()(int tri) {
    int pos = n * n * tri;
    for (int i = 0; i < n; ++i) {
      for (int j = 0; j < n - i; ++j) {
        int a = Vert(i, j, tri);
        int b = Vert(i + 1, j, tri);
        int c = Vert(i, j + 1, tri);
        triVerts[pos++] = glm::ivec3(a, b, c);
        if (j < n - 1 - i) {
          int d = Vert(i + 1, j + 1, tri);
          triVerts[pos++] = glm::ivec3(b, d, c);
        }
      }
    }
  }
};

struct FaceAreaVolume {
  const Halfedge* halfedges;
  const glm::vec3* vertPos;

  __host__ __device__ thrust::pair<float, float> operator()(int face) {
    float perimeter = 0;
    glm::vec3 edge[3];
    for (int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      edge[i] = vertPos[halfedges[3 * face + j].startVert] -
                vertPos[halfedges[3 * face + i].startVert];
      perimeter += glm::length(edge[i]);
    }
    glm::vec3 crossP = glm::cross(edge[0], edge[1]);

    float area = glm::length(crossP);
    float volume = glm::dot(crossP, vertPos[halfedges[3 * face].startVert]);

    return area > perimeter * kTolerance
               ? thrust::make_pair(area / 2.0f, volume / 6.0f)
               : thrust::make_pair(0.0f, 0.0f);
  }
};

struct Extrema : public thrust::binary_function<Halfedge, Halfedge, Halfedge> {
  __host__ __device__ void MakeForward(Halfedge& a) {
    if (!a.IsForward()) {
      int tmp = a.startVert;
      a.startVert = a.endVert;
      a.endVert = tmp;
    }
  }

  __host__ __device__ int MaxOrMinus(int a, int b) {
    return glm::min(a, b) < 0 ? -1 : glm::max(a, b);
  }

  __host__ __device__ Halfedge operator()(Halfedge a, Halfedge b) {
    MakeForward(a);
    MakeForward(b);
    a.startVert = glm::min(a.startVert, b.startVert);
    a.endVert = glm::max(a.endVert, b.endVert);
    a.face = MaxOrMinus(a.face, b.face);
    a.pairedHalfedge = MaxOrMinus(a.pairedHalfedge, b.pairedHalfedge);
    return a;
  }
};

struct PosMin
    : public thrust::binary_function<glm::vec3, glm::vec3, glm::vec3> {
  __host__ __device__ glm::vec3 operator()(glm::vec3 a, glm::vec3 b) {
    return glm::min(a, b);
  }
};

struct PosMax
    : public thrust::binary_function<glm::vec3, glm::vec3, glm::vec3> {
  __host__ __device__ glm::vec3 operator()(glm::vec3 a, glm::vec3 b) {
    return glm::max(a, b);
  }
};

struct SumPair : public thrust::binary_function<thrust::pair<float, float>,
                                                thrust::pair<float, float>,
                                                thrust::pair<float, float>> {
  __host__ __device__ thrust::pair<float, float> operator()(
      thrust::pair<float, float> a, thrust::pair<float, float> b) {
    a.first += b.first;
    a.second += b.second;
    return a;
  }
};

struct Transform {
  const glm::mat4x3 transform;

  __host__ __device__ void operator()(glm::vec3& position) {
    position = transform * glm::vec4(position, 1.0f);
  }
};

struct TransformNormals {
  const glm::mat3 transform;

  __host__ __device__ void operator()(glm::vec3& normal) {
    normal = glm::normalize(transform * normal);
    if (isnan(normal.x)) normal = glm::vec3(0.0f);
  }
};

__host__ __device__ uint32_t SpreadBits3(uint32_t v) {
  v = 0xFF0000FFu & (v * 0x00010001u);
  v = 0x0F00F00Fu & (v * 0x00000101u);
  v = 0xC30C30C3u & (v * 0x00000011u);
  v = 0x49249249u & (v * 0x00000005u);
  return v;
}

__host__ __device__ uint32_t MortonCode(glm::vec3 position, Box bBox) {
  glm::vec3 xyz = (position - bBox.min) / (bBox.max - bBox.min);
  xyz = glm::min(glm::vec3(1023.0f), glm::max(glm::vec3(0.0f), 1024.0f * xyz));
  uint32_t x = SpreadBits3(static_cast<uint32_t>(xyz.x));
  uint32_t y = SpreadBits3(static_cast<uint32_t>(xyz.y));
  uint32_t z = SpreadBits3(static_cast<uint32_t>(xyz.z));
  return x * 4 + y * 2 + z;
}

struct Morton {
  const Box bBox;

  __host__ __device__ void operator()(
      thrust::tuple<uint32_t&, const glm::vec3&> inout) {
    glm::vec3 position = thrust::get<1>(inout);
    thrust::get<0>(inout) = MortonCode(position, bBox);
  }
};

struct FaceMortonBox {
  const Halfedge* halfedge;
  const glm::vec3* vertPos;
  const Box bBox;

  __host__ __device__ void operator()(
      thrust::tuple<uint32_t&, Box&, int> inout) {
    uint32_t& mortonCode = thrust::get<0>(inout);
    Box& faceBox = thrust::get<1>(inout);
    int face = thrust::get<2>(inout);

    glm::vec3 center(0.0f);

    for (const int i : {0, 1, 2}) {
      const glm::vec3 pos = vertPos[halfedge[3 * face + i].startVert];
      center += pos;
      faceBox.Union(pos);
    }
    center /= 3;

    mortonCode = MortonCode(center, bBox);
  }
};

struct Reindex {
  const int* indexInv;

  __host__ __device__ void operator()(Halfedge& edge) {
    edge.startVert = indexInv[edge.startVert];
    edge.endVert = indexInv[edge.endVert];
  }
};

struct ReindexFace {
  Halfedge* halfedge;
  const Halfedge* oldHalfedge;
  const int* faceNew2Old;
  const int* faceOld2New;

  __host__ __device__ void operator()(int newFace) {
    const int oldFace = faceNew2Old[newFace];
    for (const int i : {0, 1, 2}) {
      Halfedge edge = oldHalfedge[3 * oldFace + i];
      edge.face = newFace;
      const int pairedFace = oldHalfedge[edge.pairedHalfedge].face;
      const int offset = edge.pairedHalfedge - 3 * pairedFace;
      edge.pairedHalfedge = 3 * faceOld2New[pairedFace] + offset;
      halfedge[3 * newFace + i] = edge;
    }
  }
};

__host__ __device__ void AtomicAddVec3(glm::vec3& target,
                                       const glm::vec3& add) {
  for (int i : {0, 1, 2}) {
#ifdef __CUDA_ARCH__
    atomicAdd(&target[i], add[i]);
#else
#pragma omp atomic
    target[i] += add[i];
#endif
  }
}

struct AssignNormals {
  glm::vec3* vertNormal;
  const glm::vec3* vertPos;
  const Halfedge* halfedges;
  const bool calculateTriNormal;

  __host__ __device__ void operator()(thrust::tuple<glm::vec3&, int> in) {
    glm::vec3& triNormal = thrust::get<0>(in);
    const int face = thrust::get<1>(in);

    glm::ivec3 triVerts;
    for (int i : {0, 1, 2}) triVerts[i] = halfedges[3 * face + i].startVert;

    glm::vec3 edge[3];
    glm::vec3 edgeLength;
    float perimeter = 0;
    for (int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      edge[i] = vertPos[triVerts[j]] - vertPos[triVerts[i]];
      edgeLength[i] = glm::length(edge[i]);
      perimeter += edgeLength[i];
    }
    glm::vec3 crossP = glm::cross(edge[0], edge[1]);

    const bool isDegenerate = glm::length(crossP) <= perimeter * kTolerance;

    if (calculateTriNormal) {
      triNormal = isDegenerate ? glm::vec3(0)
                               : glm::normalize(glm::cross(edge[0], edge[1]));
    }

    // corner angles
    glm::vec3 phi;
    if (isDegenerate) {
      phi = glm::vec3(kTolerance);
    } else {
      for (int i : {0, 1, 2}) edge[i] /= edgeLength[i];
      phi[0] = glm::acos(-glm::dot(edge[2], edge[0]));
      phi[1] = glm::acos(-glm::dot(edge[0], edge[1]));
      phi[2] = glm::pi<float>() - phi[0] - phi[1];
    }

    // assign weighted sum
    for (int i : {0, 1, 2}) {
      AtomicAddVec3(vertNormal[triVerts[i]], phi[i] * triNormal);
    }
  }
};

struct Tri2Halfedges {
  Halfedge* halfedges;
  TmpEdge* edges;

  __host__ __device__ void operator()(
      thrust::tuple<int, const glm::ivec3&> in) {
    const int tri = thrust::get<0>(in);
    const glm::ivec3& triVerts = thrust::get<1>(in);
    for (const int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      const int edge = 3 * tri + i;
      halfedges[edge] = {triVerts[i], triVerts[j], -1, tri};
      edges[edge] = TmpEdge(triVerts[i], triVerts[j], edge);
    }
  }
};

struct LinkHalfedges {
  Halfedge* halfedges;
  const TmpEdge* edges;

  __host__ __device__ void operator()(int k) {
    const int i = 2 * k;
    const int j = i + 1;
    const int pair0 = edges[i].halfedgeIdx;
    const int pair1 = edges[j].halfedgeIdx;
    if (halfedges[pair0].startVert != halfedges[pair1].endVert ||
        halfedges[pair0].endVert != halfedges[pair1].startVert ||
        halfedges[pair0].face == halfedges[pair1].face)
      printf("Not manifold!\n");
    halfedges[pair0].pairedHalfedge = pair1;
    halfedges[pair1].pairedHalfedge = pair0;
  }
};

struct SwapHalfedges {
  Halfedge* halfedges;
  const TmpEdge* edges;

  __host__ void operator()(int k) {
    const int i = 2 * k;
    const int j = i - 2;
    const TmpEdge thisEdge = edges[i];
    const TmpEdge lastEdge = edges[j];
    if (thisEdge.first == lastEdge.first &&
        thisEdge.second == lastEdge.second) {
      const int swap0idx = thisEdge.halfedgeIdx;
      Halfedge& swap0 = halfedges[swap0idx];
      const int swap1idx = swap0.pairedHalfedge;
      Halfedge& swap1 = halfedges[swap1idx];

      const int next0idx = swap0idx + ((swap0idx + 1) % 3 == 0 ? -2 : 1);
      const int next1idx = swap1idx + ((swap1idx + 1) % 3 == 0 ? -2 : 1);
      Halfedge& next0 = halfedges[next0idx];
      Halfedge& next1 = halfedges[next1idx];

      next0.startVert = swap0.endVert = next1.endVert;
      swap0.pairedHalfedge = next1.pairedHalfedge;
      halfedges[swap0.pairedHalfedge].pairedHalfedge = swap0idx;

      next1.startVert = swap1.endVert = next0.endVert;
      swap1.pairedHalfedge = next0.pairedHalfedge;
      halfedges[swap1.pairedHalfedge].pairedHalfedge = swap1idx;

      next0.pairedHalfedge = next1idx;
      next1.pairedHalfedge = next0idx;
    }
  }
};

struct EdgeBox {
  const glm::vec3* vertPos;

  __host__ __device__ void operator()(
      thrust::tuple<Box&, const TmpEdge&> inout) {
    const TmpEdge& edge = thrust::get<1>(inout);
    thrust::get<0>(inout) = Box(vertPos[edge.first], vertPos[edge.second]);
  }
};

struct CheckManifold {
  const Halfedge* halfedges;

  __host__ __device__ bool operator()(int edge) {
    bool good = true;
    const Halfedge halfedge = halfedges[edge];
    const Halfedge paired = halfedges[halfedge.pairedHalfedge];
    good &= paired.pairedHalfedge == edge;
    good &= halfedge.startVert != halfedge.endVert;
    good &= halfedge.startVert == paired.endVert;
    good &= halfedge.endVert == paired.startVert;
    return good;
  }
};

struct NoDuplicates {
  const Halfedge* halfedges;

  __host__ __device__ bool operator()(int edge) {
    return halfedges[edge].startVert != halfedges[edge + 1].startVert ||
           halfedges[edge].endVert != halfedges[edge + 1].endVert;
  }
};

/**
 * By using the closest axis-aligned projection to the normal instead of a
 * projection along the normal, we avoid introducing any rounding error.
 */
glm::mat3x2 GetAxisAlignedProjection(glm::vec3 normal) {
  glm::vec3 absNormal = glm::abs(normal);
  float xyzMax;
  glm::mat2x3 projection;
  if (absNormal.z > absNormal.x && absNormal.z > absNormal.y) {
    projection = glm::mat2x3(1.0f, 0.0f, 0.0f,  //
                             0.0f, 1.0f, 0.0f);
    xyzMax = normal.z;
  } else if (absNormal.y > absNormal.x) {
    projection = glm::mat2x3(0.0f, 0.0f, 1.0f,  //
                             1.0f, 0.0f, 0.0f);
    xyzMax = normal.y;
  } else {
    projection = glm::mat2x3(0.0f, 1.0f, 0.0f,  //
                             0.0f, 0.0f, 1.0f);
    xyzMax = normal.x;
  }
  if (xyzMax < 0) projection[0] *= -1.0f;
  return glm::transpose(projection);
}
}  // namespace

namespace manifold {

/**
 * Create a manifold from an input triangle Mesh. Will throw if the Mesh is not
 * manifold.
 */
Manifold::Impl::Impl(const Mesh& manifold) : vertPos_(manifold.vertPos) {
  CheckDevice();
  CreateAndFixHalfedges(manifold.triVerts);
  Finish();
}

/**
 * Create eiter a unit tetrahedron, cube or octahedron. The cube is in the first
 * octant, while the others are symmetric about the origin.
 */
Manifold::Impl::Impl(Shape shape) {
  std::vector<glm::vec3> vertPos;
  std::vector<glm::ivec3> triVerts;
  switch (shape) {
    case Shape::TETRAHEDRON:
      vertPos = {{-1.0f, -1.0f, 1.0f},
                 {-1.0f, 1.0f, -1.0f},
                 {1.0f, -1.0f, -1.0f},
                 {1.0f, 1.0f, 1.0f}};
      triVerts = {{2, 0, 1}, {0, 3, 1}, {2, 3, 0}, {3, 2, 1}};
      break;
    case Shape::CUBE:
      vertPos = {{0.0f, 0.0f, 0.0f},  //
                 {1.0f, 0.0f, 0.0f},  //
                 {1.0f, 1.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},  //
                 {1.0f, 0.0f, 1.0f},  //
                 {1.0f, 1.0f, 1.0f},  //
                 {0.0f, 1.0f, 1.0f}};
      triVerts = {{0, 2, 1}, {0, 3, 2},  //
                  {4, 5, 6}, {4, 6, 7},  //
                  {0, 1, 5}, {0, 5, 4},  //
                  {1, 2, 6}, {1, 6, 5},  //
                  {2, 3, 7}, {2, 7, 6},  //
                  {3, 0, 4}, {3, 4, 7}};
      break;
    case Shape::OCTAHEDRON:
      vertPos = {{1.0f, 0.0f, 0.0f},   //
                 {-1.0f, 0.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},   //
                 {0.0f, -1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},   //
                 {0.0f, 0.0f, -1.0f}};
      triVerts = {{0, 2, 4}, {1, 5, 3},  //
                  {2, 1, 4}, {3, 5, 0},  //
                  {1, 3, 4}, {0, 5, 2},  //
                  {3, 0, 4}, {2, 5, 1}};
      break;
    default:
      throw userErr("Unrecognized shape!");
  }
  vertPos_ = vertPos;
  CreateAndFixHalfedges(triVerts);
  Finish();
}

/**
 * Create the halfedge_ data structure from an input triVerts array like Mesh.
 */
void Manifold::Impl::CreateHalfedges(const VecDH<glm::ivec3>& triVerts) {
  const int numTri = triVerts.size();
  halfedge_.resize(3 * numTri);
  VecDH<TmpEdge> edge(3 * numTri);
  thrust::for_each_n(zip(countAt(0), triVerts.beginD()), numTri,
                     Tri2Halfedges({halfedge_.ptrD(), edge.ptrD()}));
  thrust::sort(edge.beginD(), edge.endD());
  thrust::for_each_n(countAt(0), halfedge_.size() / 2,
                     LinkHalfedges({halfedge_.ptrD(), edge.cptrD()}));
}

/**
 * Create the halfedge_ data structure from an input triVerts array like Mesh.
 * Check that the input is an even-manifold, and if it is not 2-manifold,
 * perform edge swaps until it is. This is a host function.
 */
void Manifold::Impl::CreateAndFixHalfedges(const VecDH<glm::ivec3>& triVerts) {
  const int numTri = triVerts.size();
  halfedge_.resize(3 * numTri);
  VecDH<TmpEdge> edge(3 * numTri);
  thrust::for_each_n(zip(countAt(0), triVerts.begin()), numTri,
                     Tri2Halfedges({halfedge_.ptrH(), edge.ptrH()}));
  // Stable sort is required here so that halfedges from the same face are
  // paired together (the triangles were created in face order). In some
  // degenerate situations the triangulator can add the same internal edge in
  // two different faces, causing this edge to not be 2-manifold. We detect this
  // and fix it by swapping one of the identical edges, so it is important that
  // we have the edges paired according to their face.
  std::stable_sort(edge.begin(), edge.end());
  thrust::for_each_n(thrust::host, countAt(0), halfedge_.size() / 2,
                     LinkHalfedges({halfedge_.ptrH(), edge.cptrH()}));
  thrust::for_each(thrust::host, countAt(1), countAt(halfedge_.size() / 2),
                   SwapHalfedges({halfedge_.ptrH(), edge.cptrH()}));
}

void Manifold::Impl::SplitNonmanifoldVerts() {
  // halfedge_.Dump();
  const VecH<Halfedge>& halfedge = halfedge_.H();
  VecH<Halfedge> sorted = halfedge;
  VecH<int> sorted2non(halfedge.size());
  thrust::sequence(sorted2non.begin(), sorted2non.end());
  thrust::sort_by_key(sorted.begin(), sorted.end(), sorted2non.begin(),
                      [](const Halfedge& a, const Halfedge& b) {
                        return a.startVert == b.startVert
                                   ? a.endVert < b.endVert
                                   : a.startVert < b.startVert;
                      });
  int numVert = NumVert();
  int edge = 0;
  for (int i = 0; i < numVert; ++i) {
    Halfedge start = sorted[edge];
    if (i != start.startVert)
      std::cout << i << " != " << start.startVert << std::endl;
    int numEdge = 1;
    while (sorted[edge + numEdge].startVert == i) {
      ++numEdge;
    }
    const int first = sorted2non[edge];
    int current = first;
    // std::cout << numEdge << std::endl;
    for (int numAround = 0; numAround < numEdge; ++numAround) {
      // std::cout << halfedge[current] << std::endl;
      current = halfedge[current].pairedHalfedge + 1;
      if (current % 3 == 0) current -= 3;
      if (current == first && numAround != numEdge - 1)
        std::cout << "cycled in " << numAround + 1 << " when there are "
                  << numEdge << " edges total!" << std::endl;
    }
    if (current != first) std::cout << "did not cycle!" << std::endl;
    edge += numEdge;
  }
}

/**
 * Once halfedge_ has been filled in, this function can be called to create the
 * rest of the internal data structures.
 */
void Manifold::Impl::Finish() {
  if (halfedge_.size() == 0) return;
  Halfedge extrema = {0, 0, 0, 0};
  extrema =
      thrust::reduce(halfedge_.beginD(), halfedge_.endD(), extrema, Extrema());

  ALWAYS_ASSERT(extrema.startVert >= 0, topologyErr,
                "Vertex index is negative!");
  ALWAYS_ASSERT(extrema.endVert < NumVert(), topologyErr,
                "Vertex index exceeds number of verts!");
  ALWAYS_ASSERT(extrema.face >= 0, topologyErr, "Face index is negative!");
  ALWAYS_ASSERT(extrema.face < NumTri(), topologyErr,
                "Face index exceeds number of faces!");
  ALWAYS_ASSERT(extrema.pairedHalfedge >= 0, topologyErr,
                "Halfedge index is negative!");
  ALWAYS_ASSERT(extrema.pairedHalfedge < 2 * NumEdge(), topologyErr,
                "Halfedge index exceeds number of halfedges!");

  CalculateBBox();
  SortVerts();
  VecDH<Box> faceBox;
  VecDH<uint32_t> faceMorton;
  GetFaceBoxMorton(faceBox, faceMorton);
  SortFaces(faceBox, faceMorton);
  CalculateNormals();
  collider_ = Collider(faceBox, faceMorton);
}

/**
 * Does a full recalculation of the face bounding boxes, including updating the
 * collider, but does not resort the faces.
 */
void Manifold::Impl::Update() {
  CalculateBBox();
  VecDH<Box> faceBox;
  VecDH<uint32_t> faceMorton;
  GetFaceBoxMorton(faceBox, faceMorton);
  collider_.UpdateBoxes(faceBox);
}

void Manifold::Impl::ApplyTransform() const {
  // This const_cast is here because these operations cancel out, leaving the
  // state conceptually unchanged. This enables lazy transformation evaluation.
  const_cast<Impl*>(this)->ApplyTransform();
}

/**
 * Bake the manifold's transform into its vertices. This function allows lazy
 * evaluation, which is important because often several transforms are applied
 * between operations.
 */
void Manifold::Impl::ApplyTransform() {
  if (transform_ == glm::mat4x3(1.0f)) return;
  thrust::for_each(vertPos_.beginD(), vertPos_.endD(), Transform({transform_}));

  glm::mat3 normalTransform =
      glm::inverse(glm::transpose(glm::mat3(transform_)));
  thrust::for_each(faceNormal_.beginD(), faceNormal_.endD(),
                   TransformNormals({normalTransform}));
  thrust::for_each(vertNormal_.beginD(), vertNormal_.endD(),
                   TransformNormals({normalTransform}));
  // This optimization does a cheap collider update if the transform is
  // axis-aligned.
  if (!collider_.Transform(transform_)) Update();
  transform_ = glm::mat4x3(1.0f);
  CalculateBBox();
}

/**
 * Triangulates the faces. In this case, the halfedge_ vector is not yet a set
 * of triangles as required by this data structure, but is instead a set of
 * general faces with the input faceEdge vector having length of the number of
 * faces + 1. The values are indicies into the halfedge_ vector for the first
 * edge of each face, with the final value being the length of the halfedge_
 * vector itself. Upon return, halfedge_ has been lengthened and properly
 * represents the mesh as a set of triangles as usual. In this process the
 * faceNormal_ values are retained, repeated as necessary.
 */
void Manifold::Impl::Face2Tri(const VecDH<int>& faceEdge) {
  VecDH<glm::ivec3> triVertsOut;
  VecDH<glm::vec3> triNormalOut;

  VecH<glm::ivec3>& triVerts = triVertsOut.H();
  VecH<glm::vec3>& triNormal = triNormalOut.H();
  const VecH<glm::vec3>& vertPos = vertPos_.H();
  const VecH<int>& face = faceEdge.H();
  const VecH<Halfedge>& halfedge = halfedge_.H();
  const VecH<glm::vec3>& faceNormal = faceNormal_.H();

  for (int i = 0; i < face.size() - 1; ++i) {
    const int edge = face[i];
    const int lastEdge = face[i + 1];
    const int numEdge = lastEdge - edge;
    ALWAYS_ASSERT(numEdge >= 3, topologyErr, "face has less than three edges.");
    const glm::vec3 normal = faceNormal[i];

    if (numEdge == 3) {  // Single triangle
      glm::ivec3 tri(halfedge[edge].startVert, halfedge[edge + 1].startVert,
                     halfedge[edge + 2].startVert);
      glm::ivec3 ends(halfedge[edge].endVert, halfedge[edge + 1].endVert,
                      halfedge[edge + 2].endVert);
      if (ends[0] == tri[2]) {
        std::swap(tri[1], tri[2]);
        std::swap(ends[1], ends[2]);
      }
      ALWAYS_ASSERT(ends[0] == tri[1] && ends[1] == tri[2] && ends[2] == tri[0],
                    topologyErr, "These 3 edges do not form a triangle!");

      triVerts.push_back(tri);
      triNormal.push_back(normal);
    } else if (numEdge == 4) {  // Pair of triangles
      const glm::mat3x2 projection = GetAxisAlignedProjection(normal);
      auto triCCW = [&projection, &vertPos](const glm::ivec3 tri) {
        return CCW(projection * vertPos[tri[0]], projection * vertPos[tri[1]],
                   projection * vertPos[tri[2]]) >= 0;
      };

      glm::ivec3 tri0(halfedge[edge].startVert, halfedge[edge].endVert, -1);
      glm::ivec3 tri1(-1, -1, tri0[0]);
      for (const int i : {1, 2, 3}) {
        if (halfedge[edge + i].startVert == tri0[1]) {
          tri0[2] = halfedge[edge + i].endVert;
          tri1[0] = tri0[2];
        }
        if (halfedge[edge + i].endVert == tri0[0]) {
          tri1[1] = halfedge[edge + i].startVert;
        }
      }
      ALWAYS_ASSERT(glm::all(glm::greaterThanEqual(tri0, glm::ivec3(0))) &&
                        glm::all(glm::greaterThanEqual(tri1, glm::ivec3(0))),
                    topologyErr, "non-manifold quad!");
      bool firstValid = triCCW(tri0) && triCCW(tri1);
      tri0[2] = tri1[1];
      tri1[2] = tri0[1];
      bool secondValid = triCCW(tri0) && triCCW(tri1);

      if (!secondValid) {
        tri0[2] = tri1[0];
        tri1[2] = tri0[0];
      } else if (firstValid) {
        glm::vec3 firstCross = vertPos[tri0[0]] - vertPos[tri1[0]];
        glm::vec3 secondCross = vertPos[tri0[1]] - vertPos[tri1[1]];
        if (glm::dot(firstCross, firstCross) <
            glm::dot(secondCross, secondCross)) {
          tri0[2] = tri1[0];
          tri1[2] = tri0[0];
        }
      }

      triVerts.push_back(tri0);
      triNormal.push_back(normal);
      triVerts.push_back(tri1);
      triNormal.push_back(normal);
    } else {  // General triangulation
      const glm::mat3x2 projection = GetAxisAlignedProjection(normal);

      Polygons polys;
      try {
        polys = Face2Polygons(i, projection, face);
      } catch (const std::exception& e) {
        std::cout << e.what() << std::endl;
        for (int edge = face[i]; edge < face[i + 1]; ++edge)
          std::cout << "halfedge: " << edge << ", " << halfedge[edge]
                    << std::endl;
        throw;
      }

      std::vector<glm::ivec3> newTris = Triangulate(polys);

      for (auto tri : newTris) {
        triVerts.push_back(tri);
        triNormal.push_back(normal);
      }
    }
  }
  faceNormal_ = triNormalOut;
  CreateAndFixHalfedges(triVertsOut);
}

/**
 * Split each edge into n pieces and sub-triangulate each triangle accordingly.
 * This function doesn't run Finish(), as that is expensive and it'll need to be
 * run after the new vertices have moved, which is a likely scenario after
 * refinement (smoothing).
 */
void Manifold::Impl::Refine(int n) {
  int numVert = NumVert();
  int numEdge = NumEdge();
  int numTri = NumTri();
  // Append new verts
  int vertsPerEdge = n - 1;
  int vertsPerTri = ((n - 2) * (n - 2) + (n - 2)) / 2;
  int triVertStart = numVert + numEdge * vertsPerEdge;
  vertPos_.resize(triVertStart + numTri * vertsPerTri);
  VecDH<TmpEdge> edges = CreateTmpEdges(halfedge_);
  VecDH<int> half2Edge(2 * numEdge);
  thrust::for_each_n(zip(countAt(0), edges.beginD()), numEdge,
                     ReindexHalfedge({half2Edge.ptrD()}));
  thrust::for_each_n(zip(countAt(0), edges.beginD()), numEdge,
                     SplitEdges({vertPos_.ptrD(), numVert, n}));
  thrust::for_each_n(
      countAt(0), numTri,
      InteriorVerts({vertPos_.ptrD(), triVertStart, n, halfedge_.ptrD()}));
  // Create subtriangles
  VecDH<glm::ivec3> triVerts(n * n * numTri);
  thrust::for_each_n(countAt(0), numTri,
                     SplitTris({triVerts.ptrD(), halfedge_.cptrD(),
                                half2Edge.cptrD(), numVert, triVertStart, n}));
  CreateHalfedges(triVerts);
}

/**
 * Returns true if this manifold is in fact an oriented 2-manifold and all of
 * the data structures are consistent.
 */
bool Manifold::Impl::IsManifold() const {
  if (halfedge_.size() == 0) return true;
  bool isManifold = thrust::all_of(countAt(0), countAt(halfedge_.size()),
                                   CheckManifold({halfedge_.cptrD()}));
  if (!isManifold) std::cout << "not manifold!" << std::endl;
  VecDH<Halfedge> halfedge(halfedge_);
  thrust::sort(halfedge.beginD(), halfedge.endD());
  isManifold &= thrust::all_of(countAt(0), countAt(2 * NumEdge() - 1),
                               NoDuplicates({halfedge.cptrD()}));
  if (!isManifold) std::cout << "not 2-manifold!" << std::endl;
  return isManifold;
}

/**
 * Returns the surface area and volume of the manifold in a Properties
 * structure. These properties are clamped to zero for a given face if they are
 * within rounding tolerance. This means degenerate manifolds can by identified
 * by testing these properties as == 0.
 */
Manifold::Properties Manifold::Impl::GetProperties() const {
  if (halfedge_.size() == 0) return {0, 0};
  ApplyTransform();
  thrust::pair<float, float> areaVolume = thrust::transform_reduce(
      countAt(0), countAt(NumTri()),
      FaceAreaVolume({halfedge_.cptrD(), vertPos_.cptrD()}),
      thrust::make_pair(0.0f, 0.0f), SumPair());
  return {areaVolume.first, areaVolume.second};
}

/**
 * Calculates the bounding box of the entire manifold, which is stored
 * internally to short-cut Boolean operations and to serve as the precision
 * range for Morton code calculation.
 */
void Manifold::Impl::CalculateBBox() {
  bBox_.min = thrust::reduce(vertPos_.begin(), vertPos_.end(),
                             glm::vec3(1 / 0.0f), PosMin());
  bBox_.max = thrust::reduce(vertPos_.begin(), vertPos_.end(),
                             glm::vec3(-1 / 0.0f), PosMax());
  ALWAYS_ASSERT(bBox_.isFinite(), topologyErr,
                "Input vertices are not all finite!");
}

/**
 * Sorts the vertices according to their Morton code.
 */
void Manifold::Impl::SortVerts() {
  VecDH<uint32_t> vertMorton(NumVert());
  thrust::for_each_n(zip(vertMorton.beginD(), vertPos_.cbeginD()), NumVert(),
                     Morton({bBox_}));

  VecDH<int> vertNew2Old(NumVert());
  thrust::sequence(vertNew2Old.beginD(), vertNew2Old.endD());
  thrust::sort_by_key(vertMorton.beginD(), vertMorton.endD(),
                      zip(vertPos_.beginD(), vertNew2Old.beginD()));

  ReindexVerts(vertNew2Old, NumVert());
}

/**
 * Updates the halfedges to point to new vert indices based on a mapping,
 * vertNew2Old. This may be a subset, so the total number of original verts is
 * also given.
 */
void Manifold::Impl::ReindexVerts(const VecDH<int>& vertNew2Old,
                                  int oldNumVert) {
  VecDH<int> vertOld2New(oldNumVert);
  thrust::scatter(countAt(0), countAt(NumVert()), vertNew2Old.beginD(),
                  vertOld2New.beginD());
  thrust::for_each(halfedge_.beginD(), halfedge_.endD(),
                   Reindex({vertOld2New.cptrD()}));
}

/**
 * Fills the faceBox and faceMorton input with the bounding boxes and Morton
 * codes of the faces, respectively. The Morton code is based on the center of
 * the bounding box.
 */
void Manifold::Impl::GetFaceBoxMorton(VecDH<Box>& faceBox,
                                      VecDH<uint32_t>& faceMorton) const {
  faceBox.resize(NumTri());
  faceMorton.resize(NumTri());
  thrust::for_each_n(
      zip(faceMorton.beginD(), faceBox.beginD(), countAt(0)), NumTri(),
      FaceMortonBox({halfedge_.cptrD(), vertPos_.cptrD(), bBox_}));
}

/**
 * Sorts the faces of this manifold according to their input Morton code. The
 * bounding box and Morton code arrays are also sorted accordingly.
 */
void Manifold::Impl::SortFaces(VecDH<Box>& faceBox,
                               VecDH<uint32_t>& faceMorton) {
  VecDH<int> faceNew2Old(NumTri());
  thrust::sequence(faceNew2Old.beginD(), faceNew2Old.endD());

  if (faceNormal_.size() == NumTri()) {
    thrust::sort_by_key(
        faceMorton.beginD(), faceMorton.endD(),
        zip(faceBox.beginD(), faceNew2Old.beginD(), faceNormal_.beginD()));
  } else {
    thrust::sort_by_key(faceMorton.beginD(), faceMorton.endD(),
                        zip(faceBox.beginD(), faceNew2Old.beginD()));
  }

  VecDH<Halfedge> oldHalfedge = halfedge_;
  GatherFaces(oldHalfedge, faceNew2Old);
}

/**
 * Creates the halfedge_ vector for this manifold by copying a set of faces from
 * another manifold, given by oldHalfedge. Input faceNew2Old defines the old
 * faces to gather into this.
 */
void Manifold::Impl::GatherFaces(const VecDH<Halfedge>& oldHalfedge,
                                 const VecDH<int>& faceNew2Old) {
  const int numTri = faceNew2Old.size();
  VecDH<int> faceOld2New(oldHalfedge.size() / 3);
  thrust::scatter(countAt(0), countAt(numTri), faceNew2Old.beginD(),
                  faceOld2New.beginD());

  halfedge_.resize(3 * numTri);
  thrust::for_each_n(countAt(0), numTri,
                     ReindexFace({halfedge_.ptrD(), oldHalfedge.cptrD(),
                                  faceNew2Old.cptrD(), faceOld2New.cptrD()}));
}

/**
 * If face normals are already present, this function uses them to compute
 * vertex normals (angle-weighted pseudo-normals); otherwise it also computes
 * the face normals. Face normals are only calculated when needed because nearly
 * degenerate faces will accrue rounding error, while the Boolean can retain
 * their original normal, which is more accurate and can help with merging
 * coplanar faces.
 *
 * If the face normals have been invalidated by an operation like Warp(), ensure
 * you do faceNormal_.resize(0) before calling this function to force
 * recalculation.
 */
void Manifold::Impl::CalculateNormals() {
  vertNormal_.resize(NumVert(), glm::vec3(0.0f));
  bool calculateTriNormal = false;
  if (faceNormal_.size() != NumTri()) {
    faceNormal_.resize(NumTri());
    calculateTriNormal = true;
  }
  thrust::for_each_n(zip(faceNormal_.beginD(), countAt(0)), NumTri(),
                     AssignNormals({vertNormal_.ptrD(), vertPos_.cptrD(),
                                    halfedge_.cptrD(), calculateTriNormal}));
  thrust::for_each(vertNormal_.begin(), vertNormal_.end(), Normalize());
}

/**
 * Returns a sparse array of the bounding box overlaps between the edges of the
 * input manifold, Q and the faces of this manifold. Returned indices only
 * point to forward halfedges.
 */
SparseIndices Manifold::Impl::EdgeCollisions(const Impl& Q) const {
  VecDH<TmpEdge> edges = CreateTmpEdges(Q.halfedge_);
  const int numEdge = edges.size();
  VecDH<Box> QedgeBB(numEdge);
  thrust::for_each_n(zip(QedgeBB.beginD(), edges.cbeginD()), numEdge,
                     EdgeBox({Q.vertPos_.cptrD()}));

  SparseIndices q1p2 = collider_.Collisions(QedgeBB);

  thrust::for_each(q1p2.beginD(0), q1p2.endD(0), ReindexEdge({edges.cptrD()}));
  return q1p2;
}

/**
 * Returns a sparse array of the input vertices that project inside the XY
 * bounding boxes of the faces of this manifold.
 */
SparseIndices Manifold::Impl::VertexCollisionsZ(
    const VecDH<glm::vec3>& vertsIn) const {
  return collider_.Collisions(vertsIn);
}

/**
 * For the input face index, return a set of 2D polygons formed by the input
 * projection of the vertices.
 */
Polygons Manifold::Impl::Face2Polygons(int face, glm::mat3x2 projection,
                                       const VecH<int>& faceEdge) const {
  const VecH<glm::vec3>& vertPos = vertPos_.H();
  const VecH<Halfedge>& halfedge = halfedge_.H();
  const int firstEdge = faceEdge[face];
  const int lastEdge = faceEdge[face + 1];

  std::map<int, int> vert_edge;
  for (int edge = firstEdge; edge < lastEdge; ++edge) {
    ALWAYS_ASSERT(
        vert_edge.emplace(std::make_pair(halfedge[edge].startVert, edge))
            .second,
        topologyErr, "face has duplicate vertices.");
  }

  Polygons polys;
  int startEdge = 0;
  int thisEdge = startEdge;
  while (1) {
    if (thisEdge == startEdge) {
      if (vert_edge.empty()) break;
      startEdge = vert_edge.begin()->second;
      thisEdge = startEdge;
      polys.push_back({});
    }
    int vert = halfedge[thisEdge].startVert;
    polys.back().push_back({projection * vertPos[vert], vert});
    const auto result = vert_edge.find(halfedge[thisEdge].endVert);
    ALWAYS_ASSERT(result != vert_edge.end(), topologyErr, "nonmanifold edge");
    thisEdge = result->second;
    vert_edge.erase(result);
  }
  return polys;
}
}  // namespace manifold