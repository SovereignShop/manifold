#include "hip/hip_runtime.h"
// Copyright 2019 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/adjacent_difference.h>
#include <thrust/count.h>
#include <thrust/gather.h>
#include <thrust/logical.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>

#include <algorithm>
#include <map>

#include "connected_components.cuh"
#include "manifold_impl.cuh"
#include "polygon.h"

namespace {
using namespace manifold;

/**
 * Represents the uncertainty of the vertices (greater than or equal to
 * worst-case floating-point precision). Used to determine when face surface
 * area or volume is small enough to clamp to zero. TODO: this should be based
 * on the bounding box, and probably passed through Boolean operations. It
 * should also be passed into the Polygon triangulator, where it is more
 * important.
 */
constexpr float kTolerance = 1e-5;

struct NormalizeTo {
  float length;
  __host__ __device__ void operator()(glm::vec3& v) {
    v = length * glm::normalize(v);
    if (isnan(v.x)) v = glm::vec3(0.0);
  }
};

/**
 * This is a temporary edge strcture which only stores edges forward and
 * references the halfedge it was created from.
 */
struct TmpEdge {
  int first, second, halfedgeIdx;

  __host__ __device__ TmpEdge() {}
  __host__ __device__ TmpEdge(int start, int end, int idx) {
    first = glm::min(start, end);
    second = glm::max(start, end);
    halfedgeIdx = idx;
  }

  __host__ __device__ bool operator<(const TmpEdge& other) const {
    return first == other.first ? second < other.second : first < other.first;
  }
};

struct Halfedge2Tmp {
  __host__ __device__ void operator()(
      thrust::tuple<TmpEdge&, const Halfedge&, int> inout) {
    const Halfedge& halfedge = thrust::get<1>(inout);
    int idx = thrust::get<2>(inout);
    if (!halfedge.IsForward()) idx = -1;

    thrust::get<0>(inout) = TmpEdge(halfedge.startVert, halfedge.endVert, idx);
  }
};

struct TmpInvalid {
  __host__ __device__ bool operator()(const TmpEdge& edge) {
    return edge.halfedgeIdx < 0;
  }
};

VecDH<TmpEdge> CreateTmpEdges(const VecDH<Halfedge>& halfedge) {
  VecDH<TmpEdge> edges(halfedge.size());
  thrust::for_each_n(
      zip(edges.beginD(), halfedge.beginD(), thrust::make_counting_iterator(0)),
      edges.size(), Halfedge2Tmp());
  int numEdge = thrust::remove_if(edges.beginD(), edges.endD(), TmpInvalid()) -
                edges.beginD();
  ALWAYS_ASSERT(numEdge == halfedge.size() / 2, runtimeErr, "Not oriented!");
  edges.resize(numEdge);
  return edges;
}

struct ReindexEdge {
  const TmpEdge* edges;

  __host__ __device__ void operator()(int& edge) {
    edge = edges[edge].halfedgeIdx;
  }
};

struct ReindexHalfedge {
  int* half2Edge;

  __host__ __device__ void operator()(thrust::tuple<int, TmpEdge> in) {
    const int edge = thrust::get<0>(in);
    const int halfedge = thrust::get<1>(in).halfedgeIdx;

    half2Edge[halfedge] = edge;
  }
};

struct SplitEdges {
  glm::vec3* vertPos;
  const int startIdx;
  const int n;

  __host__ __device__ void operator()(thrust::tuple<int, TmpEdge> in) {
    int edge = thrust::get<0>(in);
    TmpEdge edgeVerts = thrust::get<1>(in);

    float invTotal = 1.0f / n;
    for (int i = 1; i < n; ++i)
      vertPos[startIdx + (n - 1) * edge + i - 1] =
          (float(n - i) * vertPos[edgeVerts.first] +
           float(i) * vertPos[edgeVerts.second]) *
          invTotal;
  }
};

struct InteriorVerts {
  glm::vec3* vertPos;
  const int startIdx;
  const int n;
  const Halfedge* halfedge;

  __host__ __device__ void operator()(int tri) {
    int vertsPerTri = ((n - 2) * (n - 2) + (n - 2)) / 2;
    float invTotal = 1.0f / n;
    int pos = startIdx + vertsPerTri * tri;
    for (int i = 1; i < n - 1; ++i)
      for (int j = 1; j < n - i; ++j)
        vertPos[pos++] =
            (float(i) * vertPos[halfedge[3 * tri + 2].startVert] +  //
             float(j) * vertPos[halfedge[3 * tri].startVert] +      //
             float(n - i - j) * vertPos[halfedge[3 * tri + 1].startVert]) *
            invTotal;
  }
};

struct SplitTris {
  glm::ivec3* triVerts;
  const Halfedge* halfedge;
  const int* half2Edge;
  const int edgeIdx;
  const int triIdx;
  const int n;

  __host__ __device__ int EdgeVert(int i, int inHalfedge) const {
    bool forward = halfedge[inHalfedge].IsForward();
    int edge = forward ? half2Edge[inHalfedge]
                       : half2Edge[halfedge[inHalfedge].pairedHalfedge];
    return edgeIdx + (n - 1) * edge + (forward ? i - 1 : n - 1 - i);
  }

  __host__ __device__ int TriVert(int i, int j, int tri) const {
    --i;
    --j;
    int m = n - 2;
    int vertsPerTri = (m * m + m) / 2;
    int vertOffset = (i * (2 * m - i + 1)) / 2 + j;
    return triIdx + vertsPerTri * tri + vertOffset;
  }

  __host__ __device__ int Vert(int i, int j, int tri) const {
    bool edge0 = i == 0;
    bool edge1 = j == 0;
    bool edge2 = j == n - i;
    if (edge0) {
      if (edge1)
        return halfedge[3 * tri + 1].startVert;
      else if (edge2)
        return halfedge[3 * tri].startVert;
      else
        return EdgeVert(n - j, 3 * tri);
    } else if (edge1) {
      if (edge2)
        return halfedge[3 * tri + 2].startVert;
      else
        return EdgeVert(i, 3 * tri + 1);
    } else if (edge2)
      return EdgeVert(j, 3 * tri + 2);
    else
      return TriVert(i, j, tri);
  }

  __host__ __device__ void operator()(int tri) {
    int pos = n * n * tri;
    for (int i = 0; i < n; ++i) {
      for (int j = 0; j < n - i; ++j) {
        int a = Vert(i, j, tri);
        int b = Vert(i + 1, j, tri);
        int c = Vert(i, j + 1, tri);
        triVerts[pos++] = glm::ivec3(a, b, c);
        if (j < n - 1 - i) {
          int d = Vert(i + 1, j + 1, tri);
          triVerts[pos++] = glm::ivec3(b, d, c);
        }
      }
    }
  }
};

struct FaceAreaVolume {
  const Halfedge* halfedges;
  const glm::vec3* vertPos;

  __host__ __device__ thrust::pair<float, float> operator()(int face) {
    float perimeter = 0.0f;
    float area = 0.0f;
    float volume = 0.0f;

    glm::vec3 edge[3];
    for (int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      edge[i] = vertPos[halfedges[3 * face + j].startVert] -
                vertPos[halfedges[3 * face + i].startVert];
      perimeter += glm::length(edge[i]);
    }
    glm::vec3 crossP = glm::cross(edge[0], edge[1]);

    area += glm::length(crossP);
    volume += glm::dot(crossP, vertPos[halfedges[3 * face].startVert]);

    return area > perimeter * kTolerance
               ? thrust::make_pair(area / 2.0f, volume / 6.0f)
               : thrust::make_pair(0.0f, 0.0f);
  }
};

struct Extrema : public thrust::binary_function<Halfedge, Halfedge, Halfedge> {
  __host__ __device__ void MakeForward(Halfedge& a) {
    if (!a.IsForward()) {
      int tmp = a.startVert;
      a.startVert = a.endVert;
      a.endVert = tmp;
    }
  }

  __host__ __device__ int MaxOrMinus(int a, int b) {
    return glm::min(a, b) < 0 ? -1 : glm::max(a, b);
  }

  __host__ __device__ Halfedge operator()(Halfedge a, Halfedge b) {
    MakeForward(a);
    MakeForward(b);
    a.startVert = glm::min(a.startVert, b.startVert);
    a.endVert = glm::max(a.endVert, b.endVert);
    a.face = MaxOrMinus(a.face, b.face);
    a.pairedHalfedge = MaxOrMinus(a.pairedHalfedge, b.pairedHalfedge);
    return a;
  }
};

struct PosMin
    : public thrust::binary_function<glm::vec3, glm::vec3, glm::vec3> {
  __host__ __device__ glm::vec3 operator()(glm::vec3 a, glm::vec3 b) {
    return glm::min(a, b);
  }
};

struct PosMax
    : public thrust::binary_function<glm::vec3, glm::vec3, glm::vec3> {
  __host__ __device__ glm::vec3 operator()(glm::vec3 a, glm::vec3 b) {
    return glm::max(a, b);
  }
};

struct SumPair : public thrust::binary_function<thrust::pair<float, float>,
                                                thrust::pair<float, float>,
                                                thrust::pair<float, float>> {
  __host__ __device__ thrust::pair<float, float> operator()(
      thrust::pair<float, float> a, thrust::pair<float, float> b) {
    a.first += b.first;
    a.second += b.second;
    return a;
  }
};

struct Transform {
  const glm::mat4x3 transform;

  __host__ __device__ void operator()(glm::vec3& position) {
    position = transform * glm::vec4(position, 1.0f);
  }
};

struct TransformNormals {
  const glm::mat3 transform;

  __host__ __device__ void operator()(glm::vec3& normal) {
    normal = glm::normalize(transform * normal);
    if (isnan(normal.x)) normal = glm::vec3(0.0f);
  }
};

__host__ __device__ uint32_t SpreadBits3(uint32_t v) {
  v = 0xFF0000FFu & (v * 0x00010001u);
  v = 0x0F00F00Fu & (v * 0x00000101u);
  v = 0xC30C30C3u & (v * 0x00000011u);
  v = 0x49249249u & (v * 0x00000005u);
  return v;
}

__host__ __device__ uint32_t MortonCode(glm::vec3 position, Box bBox) {
  glm::vec3 xyz = (position - bBox.min) / (bBox.max - bBox.min);
  xyz = glm::min(glm::vec3(1023.0f), glm::max(glm::vec3(0.0f), 1024.0f * xyz));
  uint32_t x = SpreadBits3(static_cast<uint32_t>(xyz.x));
  uint32_t y = SpreadBits3(static_cast<uint32_t>(xyz.y));
  uint32_t z = SpreadBits3(static_cast<uint32_t>(xyz.z));
  return x * 4 + y * 2 + z;
}

struct Morton {
  const Box bBox;

  __host__ __device__ void operator()(
      thrust::tuple<uint32_t&, const glm::vec3&> inout) {
    glm::vec3 position = thrust::get<1>(inout);
    thrust::get<0>(inout) = MortonCode(position, bBox);
  }
};

struct FaceMortonBox {
  const Halfedge* halfedge;
  const glm::vec3* vertPos;
  const Box bBox;

  __host__ __device__ void operator()(
      thrust::tuple<uint32_t&, Box&, int> inout) {
    uint32_t& mortonCode = thrust::get<0>(inout);
    Box& faceBox = thrust::get<1>(inout);
    int face = thrust::get<2>(inout);

    glm::vec3 center(0.0f);

    for (const int i : {0, 1, 2}) {
      const glm::vec3 pos = vertPos[halfedge[3 * face + i].startVert];
      center += pos;
      faceBox.Union(pos);
    }
    center /= 3;

    mortonCode = MortonCode(center, bBox);
  }
};

struct Reindex {
  const int* indexInv;

  __host__ __device__ void operator()(Halfedge& edge) {
    edge.startVert = indexInv[edge.startVert];
    edge.endVert = indexInv[edge.endVert];
  }
};

struct ReindexFace {
  Halfedge* halfedge;
  const int* faceEdge;
  const Halfedge* oldHalfedge;
  const int* oldFaceEdge;
  const int* faceNew2Old;
  const int* faceOld2New;

  __host__ __device__ void operator()(thrust::tuple<int, int> in) {
    const int newFace = thrust::get<0>(in);
    int outEdge = thrust::get<1>(in);

    const int oldFace = faceNew2Old[newFace];
    int iEdge = oldFaceEdge[oldFace];
    const int end = oldFaceEdge[oldFace + 1];
    while (iEdge < end) {
      Halfedge edge = oldHalfedge[iEdge++];
      edge.face = newFace;
      const int pairedFace = oldHalfedge[edge.pairedHalfedge].face;
      const int offset = edge.pairedHalfedge - oldFaceEdge[pairedFace];
      edge.pairedHalfedge = faceEdge[faceOld2New[pairedFace]] + offset;
      halfedge[outEdge++] = edge;
    }
  }
};

__host__ __device__ void AtomicAddVec3(glm::vec3& target,
                                       const glm::vec3& add) {
  for (int i : {0, 1, 2}) {
#ifdef __CUDA_ARCH__
    atomicAdd(&target[i], add[i]);
#else
#pragma omp atomic
    target[i] += add[i];
#endif
  }
}

struct AssignNormals {
  glm::vec3* vertNormal;
  const glm::vec3* vertPos;
  const Halfedge* halfedges;
  const bool calculateTriNormal;

  __host__ __device__ void operator()(thrust::tuple<glm::vec3&, int> in) {
    glm::vec3& triNormal = thrust::get<0>(in);
    const int face = thrust::get<1>(in);

    glm::ivec3 triVerts(halfedges[3 * face].startVert,
                        halfedges[3 * face].endVert,
                        halfedges[3 * face + 1].endVert);
    glm::vec3 v0 = vertPos[triVerts[0]];
    glm::vec3 v1 = vertPos[triVerts[1]];
    glm::vec3 v2 = vertPos[triVerts[2]];
    // edge vectors
    glm::vec3 e01 = glm::normalize(v1 - v0);
    glm::vec3 e12 = glm::normalize(v2 - v1);
    glm::vec3 e20 = glm::normalize(v0 - v2);

    if (calculateTriNormal) {
      triNormal = glm::normalize(glm::cross(e01, e12));
      if (isnan(triNormal.x)) triNormal = glm::vec3(0.0);
    }

    // corner angles
    glm::vec3 phi;
    phi[0] = glm::acos(-glm::dot(e01, e12));
    phi[1] = glm::acos(-glm::dot(e12, e20));
    phi[2] = glm::pi<float>() - phi[0] - phi[1];
    // assign weighted sum
    for (int i : {0, 1, 2}) {
      if (isnan(phi[i])) phi[i] = 0;
      AtomicAddVec3(vertNormal[triVerts[i]],
                    glm::max(phi[i], kTolerance) * triNormal);
    }
  }
};

struct Tri2Halfedges {
  Halfedge* halfedges;
  TmpEdge* edges;

  __host__ __device__ void operator()(
      thrust::tuple<int, const glm::ivec3&> in) {
    const int tri = thrust::get<0>(in);
    const glm::ivec3& triVerts = thrust::get<1>(in);
    for (const int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      const int edge = 3 * tri + i;
      halfedges[edge] = {triVerts[i], triVerts[j], -1, tri};
      edges[edge] = TmpEdge(triVerts[i], triVerts[j], edge);
    }
  }
};

struct LinkHalfedges {
  Halfedge* halfedges;
  const TmpEdge* edges;

  __host__ __device__ void operator()(int k) {
    const int i = 2 * k;
    const int j = i + 1;
    const int pair0 = edges[i].halfedgeIdx;
    const int pair1 = edges[j].halfedgeIdx;
    if (halfedges[pair0].startVert != halfedges[pair1].endVert ||
        halfedges[pair0].endVert != halfedges[pair1].startVert ||
        halfedges[pair0].face == halfedges[pair1].face)
      printf("Not manifold!\n");
    halfedges[pair0].pairedHalfedge = pair1;
    halfedges[pair1].pairedHalfedge = pair0;
  }
};

struct EdgeBox {
  const glm::vec3* vertPos;

  __host__ __device__ void operator()(
      thrust::tuple<Box&, const TmpEdge&> inout) {
    const TmpEdge& edge = thrust::get<1>(inout);
    thrust::get<0>(inout) = Box(vertPos[edge.first], vertPos[edge.second]);
  }
};

struct CheckManifold {
  const Halfedge* halfedges;

  __host__ __device__ bool operator()(int face) {
    bool good = true;
    for (const int i : {0, 1, 2}) {
      const int edge = 3 * face + i;
      const Halfedge halfedge = halfedges[edge];
      const Halfedge paired = halfedges[halfedge.pairedHalfedge];
      good &= halfedge.face == face;
      good &= paired.pairedHalfedge == edge;
      good &= halfedge.startVert != halfedge.endVert;
      good &= halfedge.startVert == paired.endVert;
      good &= halfedge.endVert == paired.startVert;
    }
    return good;
  }
};

struct NoDuplicates {
  const Halfedge* halfedges;

  __host__ __device__ bool operator()(int edge) {
    return halfedges[edge].startVert != halfedges[edge + 1].startVert ||
           halfedges[edge].endVert != halfedges[edge + 1].endVert;
  }
};

/**
 * By using the closest axis-aligned projection to the normal instead of a
 * projection along the normal, we avoid introducing any rounding error.
 */
glm::mat3x2 GetAxisAlignedProjection(glm::vec3 normal) {
  glm::vec3 absNormal = glm::abs(normal);
  float xyzMax;
  glm::mat2x3 projection;
  if (absNormal.z > absNormal.x && absNormal.z > absNormal.y) {
    projection = glm::mat2x3(1.0f, 0.0f, 0.0f,  //
                             0.0f, 1.0f, 0.0f);
    xyzMax = normal.z;
  } else if (absNormal.y > absNormal.x) {
    projection = glm::mat2x3(0.0f, 0.0f, 1.0f,  //
                             1.0f, 0.0f, 0.0f);
    xyzMax = normal.y;
  } else {
    projection = glm::mat2x3(0.0f, 1.0f, 0.0f,  //
                             0.0f, 0.0f, 1.0f);
    xyzMax = normal.x;
  }
  if (xyzMax < 0) projection[0] *= -1.0f;
  return glm::transpose(projection);
}
}  // namespace

namespace manifold {

/**
 * Create a manifold from an input triangle Mesh. Will throw if the Mesh is not
 * manifold.
 */
Manifold::Impl::Impl(const Mesh& manifold) : vertPos_(manifold.vertPos) {
  CheckDevice();
  CreateHalfedges(manifold.triVerts);
  Finish();
}

/**
 * Create eiter a unit tetrahedron, cube or octahedron. The cube is in the first
 * octant, while the others are symmetric about the origin.
 */
Manifold::Impl::Impl(Shape shape) {
  std::vector<glm::vec3> vertPos;
  std::vector<glm::ivec3> triVerts;
  switch (shape) {
    case Shape::TETRAHEDRON:
      vertPos = {{-1.0f, -1.0f, 1.0f},
                 {-1.0f, 1.0f, -1.0f},
                 {1.0f, -1.0f, -1.0f},
                 {1.0f, 1.0f, 1.0f}};
      triVerts = {{2, 0, 1}, {0, 3, 1}, {2, 3, 0}, {3, 2, 1}};
      break;
    case Shape::CUBE:
      vertPos = {{0.0f, 0.0f, 0.0f},  //
                 {1.0f, 0.0f, 0.0f},  //
                 {1.0f, 1.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},  //
                 {1.0f, 0.0f, 1.0f},  //
                 {1.0f, 1.0f, 1.0f},  //
                 {0.0f, 1.0f, 1.0f}};
      triVerts = {{0, 2, 1}, {0, 3, 2},  //
                  {4, 5, 6}, {4, 6, 7},  //
                  {0, 1, 5}, {0, 5, 4},  //
                  {1, 2, 6}, {1, 6, 5},  //
                  {2, 3, 7}, {2, 7, 6},  //
                  {3, 0, 4}, {3, 4, 7}};
      break;
    case Shape::OCTAHEDRON:
      vertPos = {{1.0f, 0.0f, 0.0f},   //
                 {-1.0f, 0.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},   //
                 {0.0f, -1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},   //
                 {0.0f, 0.0f, -1.0f}};
      triVerts = {{0, 2, 4}, {1, 5, 3},  //
                  {2, 1, 4}, {3, 5, 0},  //
                  {1, 3, 4}, {0, 5, 2},  //
                  {3, 0, 4}, {2, 5, 1}};
      break;
    default:
      throw logicErr("Unrecognized shape!");
  }
  vertPos_ = vertPos;
  CreateHalfedges(triVerts);
  Finish();
}

/**
 * Create the halfedge_ data structure from an input triVerts array like Mesh.
 */
void Manifold::Impl::CreateHalfedges(const VecDH<glm::ivec3>& triVerts) {
  const int numTri = triVerts.size();
  faceEdge_.resize(0);
  halfedge_.resize(3 * numTri);
  VecDH<TmpEdge> edge(3 * numTri);
  thrust::for_each_n(zip(thrust::make_counting_iterator(0), triVerts.beginD()),
                     numTri, Tri2Halfedges({halfedge_.ptrD(), edge.ptrD()}));
  thrust::sort(edge.beginD(), edge.endD());
  thrust::for_each_n(thrust::make_counting_iterator(0), halfedge_.size() / 2,
                     LinkHalfedges({halfedge_.ptrD(), edge.cptrD()}));
  Tri2Face();
}

/**
 * Calculate vertLabels_ by running connected components on the halfedges. This
 * operation is a bit slow and currently CPU-only. Note: by operating on
 * halfedges, connectivity can be broken by faces that are polygons with holes
 * (no edges to attach one polygon to the other, even though they are part of
 * the same face). This style of labeling is consistent with what is needed in
 * the Boolean operation. To separate manifolds topologically, it is best to
 * first triangulate them.
 */
void Manifold::Impl::LabelVerts() {
  numLabel_ = ConnectedComponents(vertLabel_, NumVert(), halfedge_);
}

/**
 * Once halfedge_ and faceEdge_ have been filled in, this function can be called
 * to create the rest of the internal data structures. If vertLabel_ hasn't been
 * filled in, it is assumed the object is simply-connected and numLabel_ is set
 * to 1.
 */
void Manifold::Impl::Finish() {
  if (halfedge_.size() == 0) return;
  Halfedge extrema = {0, 0, 0, 0};
  extrema =
      thrust::reduce(halfedge_.beginD(), halfedge_.endD(), extrema, Extrema());

  ALWAYS_ASSERT(extrema.startVert >= 0, runtimeErr,
                "Vertex index is negative!");
  ALWAYS_ASSERT(extrema.endVert < NumVert(), runtimeErr,
                "Vertex index exceeds number of verts!");
  ALWAYS_ASSERT(extrema.face >= 0, runtimeErr, "Face index is negative!");
  ALWAYS_ASSERT(extrema.face < NumFace(), runtimeErr,
                "Face index exceeds number of faces!");
  ALWAYS_ASSERT(extrema.pairedHalfedge >= 0, runtimeErr,
                "Halfedge index is negative!");
  ALWAYS_ASSERT(extrema.pairedHalfedge < 2 * NumEdge(), runtimeErr,
                "Halfedge index exceeds number of halfedges!");
  ALWAYS_ASSERT(faceEdge_.H().front() == 0, runtimeErr,
                "Faces do not start at zero!");
  ALWAYS_ASSERT(faceEdge_.H().back() == 2 * NumEdge(), runtimeErr,
                "Faces do not end at halfedge length!");

  if (vertLabel_.size() != NumVert()) {
    vertLabel_.resize(NumVert());
    numLabel_ = 1;
    thrust::fill(vertLabel_.beginD(), vertLabel_.endD(), 0);
  }
  CalculateBBox();
  SortVerts();
  VecDH<Box> faceBox;
  VecDH<uint32_t> faceMorton;
  GetFaceBoxMorton(faceBox, faceMorton);
  SortFaces(faceBox, faceMorton);
  CalculateNormals();
  collider_ = Collider(faceBox, faceMorton);
}

/**
 * Does a full recalculation of the face bounding boxes, including updating the
 * collider, but does not resort the faces.
 */
void Manifold::Impl::Update() {
  CalculateBBox();
  VecDH<Box> faceBox;
  VecDH<uint32_t> faceMorton;
  GetFaceBoxMorton(faceBox, faceMorton);
  collider_.UpdateBoxes(faceBox);
}

void Manifold::Impl::ApplyTransform() const {
  // This const_cast is here because these operations cancel out, leaving the
  // state conceptually unchanged. This enables lazy transformation evaluation.
  const_cast<Impl*>(this)->ApplyTransform();
}

/**
 * Bake the manifold's transform into its vertices. This function allows lazy
 * evaluation, which is important because often several transforms are applied
 * between operations.
 */
void Manifold::Impl::ApplyTransform() {
  if (transform_ == glm::mat4x3(1.0f)) return;
  thrust::for_each(vertPos_.beginD(), vertPos_.endD(), Transform({transform_}));

  glm::mat3 normalTransform =
      glm::inverse(glm::transpose(glm::mat3(transform_)));
  thrust::for_each(faceNormal_.beginD(), faceNormal_.endD(),
                   TransformNormals({normalTransform}));
  thrust::for_each(vertNormal_.beginD(), vertNormal_.endD(),
                   TransformNormals({normalTransform}));
  // This optimization does a cheap collider update if the transform is
  // axis-aligned.
  if (!collider_.Transform(transform_)) Update();
  transform_ = glm::mat4x3(1.0f);
  CalculateBBox();
}

/**
 * This fills in the nextHalfedge_ vector indicating how the halfedges connect
 * to each other going CCW around a face. This data cannot be stored by simply
 * sorting the halfedges, as the faces may be polygons with holes.
 *
 * TODO: This function is slow and should be moved from CPU to GPU.
 */
VecH<int> Manifold::Impl::AssembleFaces(const VecH<int>& faceEdge) const {
  VecH<int> nextHalfedge(halfedge_.size());
  const VecH<Halfedge>& halfedge = halfedge_.H();

  for (int face = 0; face < NumFace(); ++face) {
    int edge = faceEdge[face];
    const int nEdge = faceEdge[face + 1] - edge;
    ALWAYS_ASSERT(nEdge >= 3, runtimeErr, "face has less than three edges.");
    if (nEdge == 3) {
      const bool forward =
          halfedge[edge].endVert == halfedge[edge + 1].startVert;
      const int edge1 = edge + (forward ? 1 : 2);
      const int edge2 = edge + (forward ? 2 : 1);
      ALWAYS_ASSERT(halfedge[edge].endVert == halfedge[edge1].startVert &&
                        halfedge[edge1].endVert == halfedge[edge2].startVert &&
                        halfedge[edge2].endVert == halfedge[edge].startVert,
                    runtimeErr, "triangle does not assemble.");
      nextHalfedge[edge] = edge1;
      nextHalfedge[edge1] = edge2;
      nextHalfedge[edge2] = edge;
      continue;
    }
    std::map<int, int> vert_edge;
    for (; edge < faceEdge[face + 1]; ++edge) {
      ALWAYS_ASSERT(
          vert_edge.emplace(std::make_pair(halfedge[edge].startVert, edge))
              .second,
          runtimeErr, "face has duplicate vertices.");
    }

    int startEdge = 0;
    int thisEdge = startEdge;
    while (1) {
      if (thisEdge == startEdge) {
        if (vert_edge.empty()) break;
        startEdge = vert_edge.begin()->second;
        thisEdge = startEdge;
      }
      const auto result = vert_edge.find(halfedge[thisEdge].endVert);
      ALWAYS_ASSERT(result != vert_edge.end(), runtimeErr, "nonmanifold edge");
      nextHalfedge[thisEdge] = result->second;
      thisEdge = result->second;
      vert_edge.erase(result);
    }
  }
  return nextHalfedge;
}

bool Manifold::Impl::Tri2Face() const {
  // This const_cast is here because this operation tweaks the internal data
  // structure, but does not change what it represents.
  return const_cast<Impl*>(this)->Tri2Face();
}

/**
 * Fills in the faceEdge_ structure for the situation where the halfedges
 * correspond to triVerts entries.
 */
bool Manifold::Impl::Tri2Face() {
  if (faceEdge_.size() != 0 || halfedge_.size() % 3 != 0) return false;
  faceEdge_.resize(halfedge_.size() / 3 + 1);
  thrust::sequence(faceEdge_.beginD(), faceEdge_.endD(), 0, 3);
  return true;
}

/**
 * Triangulates the faces. It is possible, but rare, that this function can
 * also add vertices. This never happens for geometrically valid manifolds.
 */
bool Manifold::Impl::Face2Tri(const VecDH<int>& faceEdge) {
  if (faceEdge.size() == 0 && halfedge_.size() % 3 == 0) return false;
  VecDH<glm::ivec3> triVertsOut;
  VecDH<glm::vec3> triNormalOut;

  VecH<glm::ivec3>& triVerts = triVertsOut.H();
  VecH<glm::vec3>& triNormal = triNormalOut.H();
  VecH<glm::vec3>& vertPos = vertPos_.H();
  const VecH<int>& face = faceEdge.H();
  const VecH<Halfedge>& halfedge = halfedge_.H();
  const VecH<glm::vec3>& faceNormal = faceNormal_.H();
  const VecH<int> nextHalfedge = AssembleFaces(face);

  for (int i = 0; i < face.size() - 1; ++i) {
    const int edge = face[i];
    const int lastEdge = face[i + 1];
    const int numEdge = lastEdge - edge;
    ALWAYS_ASSERT(numEdge >= 3, logicErr, "face has less than three edges.");
    const glm::vec3 normal = faceNormal[i];

    if (numEdge == 3) {  // Special case to increase performance
      glm::ivec3 tri(halfedge[edge].startVert, halfedge[edge + 1].startVert,
                     halfedge[edge + 2].startVert);
      glm::ivec3 ends(halfedge[edge].endVert, halfedge[edge + 1].endVert,
                      halfedge[edge + 2].endVert);
      if (ends[0] == tri[2]) {
        std::swap(tri[1], tri[2]);
        std::swap(ends[1], ends[2]);
      }
      ALWAYS_ASSERT(ends[0] == tri[1] && ends[1] == tri[2] && ends[2] == tri[0],
                    runtimeErr, "These 3 edges do not form a triangle!");

      triVerts.push_back(tri);
      triNormal.push_back(normal);
    } else {  // General triangulation
      const glm::mat3x2 projection = GetAxisAlignedProjection(normal);
      Polygons polys = Face2Polygons(i, projection, face, nextHalfedge);

      std::vector<glm::ivec3> newTris = Triangulate(polys);

      for (auto tri : newTris) {
        triVerts.push_back(tri);
        triNormal.push_back(normal);
      }
    }
  }
  faceNormal_ = triNormalOut;
  CreateHalfedges(triVertsOut);
  return true;
}

/**
 * Triangulate the manifold, then split each resulting edge into n pieces and
 * sub-triangulate each triangle accordingly. This function doesn't run
 * Finish(), as that is expensive and it'll need to be run after the new
 * vertices have moved, which is a likely scenario after refinement (smoothing).
 */
void Manifold::Impl::Refine(int n) {
  int numVert = NumVert();
  int numEdge = NumEdge();
  int numTri = NumFace();
  // Append new verts
  int vertsPerEdge = n - 1;
  int vertsPerTri = ((n - 2) * (n - 2) + (n - 2)) / 2;
  int triVertStart = numVert + numEdge * vertsPerEdge;
  vertPos_.resize(triVertStart + numTri * vertsPerTri);
  VecDH<TmpEdge> edges = CreateTmpEdges(halfedge_);
  VecDH<int> half2Edge(2 * numEdge);
  thrust::for_each_n(zip(thrust::make_counting_iterator(0), edges.beginD()),
                     numEdge, ReindexHalfedge({half2Edge.ptrD()}));
  thrust::for_each_n(zip(thrust::make_counting_iterator(0), edges.beginD()),
                     numEdge, SplitEdges({vertPos_.ptrD(), numVert, n}));
  thrust::for_each_n(
      thrust::make_counting_iterator(0), numTri,
      InteriorVerts({vertPos_.ptrD(), triVertStart, n, halfedge_.ptrD()}));
  // Create subtriangles
  VecDH<glm::ivec3> triVerts(n * n * numTri);
  thrust::for_each_n(thrust::make_counting_iterator(0), numTri,
                     SplitTris({triVerts.ptrD(), halfedge_.cptrD(),
                                half2Edge.cptrD(), numVert, triVertStart, n}));
  CreateHalfedges(triVerts);
}

/**
 * Returns true if this manifold is in fact an oriented 2-manifold and all of
 * the data structures are consistent.
 */
bool Manifold::Impl::IsManifold() const {
  if (halfedge_.size() == 0) return true;
  bool isManifold = thrust::all_of(thrust::make_counting_iterator(0),
                                   thrust::make_counting_iterator(NumFace()),
                                   CheckManifold({halfedge_.cptrD()}));

  VecDH<Halfedge> halfedge(halfedge_);
  thrust::sort(halfedge.beginD(), halfedge.endD());
  isManifold &=
      thrust::all_of(thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator(2 * NumEdge() - 1),
                     NoDuplicates({halfedge.cptrD()}));
  return isManifold;
}

/**
 * Returns the surface area and volume of the manifold in a Properties
 * structure. These properties are clamped to zero for a given face if they are
 * within rounding tolerance. This means degenerate manifolds can by identified
 * by testing these properties as == 0.
 */
Manifold::Properties Manifold::Impl::GetProperties() const {
  if (halfedge_.size() == 0) return {0, 0};
  ApplyTransform();
  thrust::pair<float, float> areaVolume = thrust::transform_reduce(
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(NumFace()),
      FaceAreaVolume({halfedge_.cptrD(), vertPos_.cptrD()}),
      thrust::make_pair(0.0f, 0.0f), SumPair());
  return {areaVolume.first, areaVolume.second};
}

/**
 * Calculates the bounding box of the entire manifold, which is stored
 * internally to short-cut Boolean operations and to serve as the precision
 * range for Morton code calulation.
 */
void Manifold::Impl::CalculateBBox() {
  bBox_.min = thrust::reduce(vertPos_.begin(), vertPos_.end(),
                             glm::vec3(1 / 0.0f), PosMin());
  bBox_.max = thrust::reduce(vertPos_.begin(), vertPos_.end(),
                             glm::vec3(-1 / 0.0f), PosMax());
  ALWAYS_ASSERT(bBox_.isFinite(), runtimeErr,
                "Input vertices are not all finite!");
}

/**
 * Sorts the vertices according to their Morton code.
 */
void Manifold::Impl::SortVerts() {
  VecDH<uint32_t> vertMorton(NumVert());
  thrust::for_each_n(zip(vertMorton.beginD(), vertPos_.cbeginD()), NumVert(),
                     Morton({bBox_}));

  VecDH<int> vertNew2Old(NumVert());
  thrust::sequence(vertNew2Old.beginD(), vertNew2Old.endD());
  thrust::sort_by_key(
      vertMorton.beginD(), vertMorton.endD(),
      zip(vertPos_.beginD(), vertLabel_.beginD(), vertNew2Old.beginD()));

  ReindexVerts(vertNew2Old, NumVert());
}

/**
 * Updates the halfedges to point to new vert indices based on a mapping,
 * vertNew2Old. This may be a subset, so the total number of original verts is
 * also given.
 */
void Manifold::Impl::ReindexVerts(const VecDH<int>& vertNew2Old,
                                  int oldNumVert) {
  VecDH<int> vertOld2New(oldNumVert);
  thrust::scatter(thrust::make_counting_iterator(0),
                  thrust::make_counting_iterator(NumVert()),
                  vertNew2Old.beginD(), vertOld2New.beginD());
  thrust::for_each(halfedge_.beginD(), halfedge_.endD(),
                   Reindex({vertOld2New.cptrD()}));
}

/**
 * Fills the faceBox and faceMorton input with the bounding boxes and Morton
 * codes of the faces, respectively. The Morton code is based on the center of
 * the bounding box.
 */
void Manifold::Impl::GetFaceBoxMorton(VecDH<Box>& faceBox,
                                      VecDH<uint32_t>& faceMorton) const {
  faceBox.resize(NumFace());
  faceMorton.resize(NumFace());
  thrust::for_each_n(
      zip(faceMorton.beginD(), faceBox.beginD(),
          thrust::make_counting_iterator(0)),
      NumFace(), FaceMortonBox({halfedge_.cptrD(), vertPos_.cptrD(), bBox_}));
}

/**
 * Sorts the faces of this manifold according to their input Morton code. The
 * bounding box and Morton code arrays are also sorted accordingly.
 */
void Manifold::Impl::SortFaces(VecDH<Box>& faceBox,
                               VecDH<uint32_t>& faceMorton) {
  VecDH<int> faceNew2Old(NumFace());
  thrust::sequence(faceNew2Old.beginD(), faceNew2Old.endD());

  VecDH<int> faceSize = FaceSize();

  if (faceNormal_.size() == NumFace()) {
    thrust::sort_by_key(faceMorton.beginD(), faceMorton.endD(),
                        zip(faceBox.beginD(), faceNew2Old.beginD(),
                            faceSize.beginD() + 1, faceNormal_.beginD()));
  } else {
    thrust::sort_by_key(
        faceMorton.beginD(), faceMorton.endD(),
        zip(faceBox.beginD(), faceNew2Old.beginD(), faceSize.beginD() + 1));
  }

  VecDH<Halfedge> oldHalfedge = halfedge_;
  VecDH<int> oldFaceEdge = faceEdge_;
  GatherFaces(oldHalfedge, oldFaceEdge, faceNew2Old, faceSize);
}

VecDH<int> Manifold::Impl::FaceSize() const {
  VecDH<int> faceSize(faceEdge_.size());
  thrust::adjacent_difference(faceEdge_.beginD(), faceEdge_.endD(),
                              faceSize.beginD());
  return faceSize;
}

/**
 * Creates the halfedge_ and faceEdge_ vectors for this manifold by copying a
 * set of faces from another manifold, given by oldHalfedge and oldFaceEdge.
 * Input faceNew2Old defines the old faces to gather into this, while
 * newFaceSize is the same length as faceNew2Old and contains the sizes of the
 * faces to be copied.
 */
void Manifold::Impl::GatherFaces(const VecDH<Halfedge>& oldHalfedge,
                                 const VecDH<int>& oldFaceEdge,
                                 const VecDH<int>& faceNew2Old,
                                 const VecDH<int>& newFaceSize) {
  faceEdge_.resize(faceNew2Old.size() + 1);

  VecDH<int> faceOld2New(oldFaceEdge.size() - 1);
  thrust::scatter(thrust::make_counting_iterator(0),
                  thrust::make_counting_iterator(NumFace()),
                  faceNew2Old.beginD(), faceOld2New.beginD());

  thrust::inclusive_scan(newFaceSize.beginD() + 1, newFaceSize.endD(),
                         faceEdge_.beginD() + 1);

  halfedge_.resize(faceEdge_.H().back());
  thrust::for_each_n(zip(thrust::make_counting_iterator(0), faceEdge_.beginD()),
                     NumFace(),
                     ReindexFace({halfedge_.ptrD(), faceEdge_.cptrD(),
                                  oldHalfedge.cptrD(), oldFaceEdge.cptrD(),
                                  faceNew2Old.cptrD(), faceOld2New.cptrD()}));
}

/**
 * If face normals are already present, this function uses them to compute
 * vertex normals (angle-weighted pseudo-normals); otherwise it also computes
 * the face normals as well. Face normals are only calculated when needed
 * because nearly degenerate faces will accrue rounding error, while the Boolean
 * can retain their original normal, which is more accurate and can help with
 * merging coplanar faces.
 *
 * If the face normals have been invalidated by an operation like Warp(), ensure
 * you do faceNormal_.resize(0) before calling this function to force
 * recalculation.
 */
void Manifold::Impl::CalculateNormals() {
  vertNormal_.resize(NumVert(), glm::vec3(0.0f));
  bool calculateTriNormal = false;
  if (faceNormal_.size() != NumFace()) {
    faceNormal_.resize(NumFace());
    calculateTriNormal = true;
  }
  thrust::for_each_n(
      zip(faceNormal_.beginD(), thrust::make_counting_iterator(0)), NumFace(),
      AssignNormals({vertNormal_.ptrD(), vertPos_.cptrD(), halfedge_.cptrD(),
                     calculateTriNormal}));
  thrust::for_each(vertNormal_.begin(), vertNormal_.end(), NormalizeTo({1.0}));
}

/**
 * Returns a sparse array of the bounding box overlaps between the edges of the
 * input manifold, Q and the faces of this manifold. Returned indices only
 * point to forward halfedges.
 */
SparseIndices Manifold::Impl::EdgeCollisions(const Impl& Q) const {
  VecDH<TmpEdge> edges = CreateTmpEdges(Q.halfedge_);
  const int numEdge = edges.size();
  VecDH<Box> QedgeBB(numEdge);
  thrust::for_each_n(zip(QedgeBB.beginD(), edges.cbeginD()), numEdge,
                     EdgeBox({Q.vertPos_.cptrD()}));

  SparseIndices q1p2 = collider_.Collisions(QedgeBB);

  thrust::for_each(q1p2.beginD(0), q1p2.endD(0), ReindexEdge({edges.cptrD()}));
  return q1p2;
}

/**
 * Returns a sparse array of the input vertices that project inside the XY
 * bounding boxes of the faces of this manifold.
 */
SparseIndices Manifold::Impl::VertexCollisionsZ(
    const VecDH<glm::vec3>& vertsIn) const {
  return collider_.Collisions(vertsIn);
}

/**
 * For the input face index, return a set of 2D polygons formed by the input
 * projection of the vertices.
 */
Polygons Manifold::Impl::Face2Polygons(int face, glm::mat3x2 projection,
                                       const VecH<int>& faceEdge,
                                       const VecH<int>& nextHalfedge) const {
  const VecH<Halfedge>& halfedge = halfedge_.H();
  const VecH<glm::vec3>& vertPos = vertPos_.H();
  const int firstEdge = faceEdge[face];
  const int lastEdge = faceEdge[face + 1];

  Polygons polys;
  std::vector<bool> visited(lastEdge - firstEdge, false);
  int startEdge = firstEdge;
  int thisEdge = firstEdge;
  while (1) {
    if (thisEdge == startEdge) {
      auto next = std::find(visited.begin(), visited.end(), false);
      if (next == visited.end()) break;
      startEdge = next - visited.begin() + firstEdge;
      thisEdge = startEdge;
      polys.push_back({});
    }
    int vert = halfedge[thisEdge].startVert;
    polys.back().push_back({projection * vertPos[vert], vert,
                            halfedge[halfedge[thisEdge].pairedHalfedge].face});
    visited[thisEdge - firstEdge] = true;
    thisEdge = nextHalfedge[thisEdge];
  }
  return polys;
}
}  // namespace manifold