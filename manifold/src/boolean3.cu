#include "hip/hip_runtime.h"
// Copyright 2019 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_math_constants.h>
#include <thrust/binary_search.h>
#include <thrust/count.h>
#include <thrust/gather.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/logical.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>
#include <thrust/unique.h>

#include <algorithm>
#include <map>

#include "boolean3.cuh"
#include "connected_components.cuh"
#include "polygon.h"

constexpr bool kVerbose = false;

using namespace thrust::placeholders;

namespace {
using namespace manifold;

// These two functions (Interpolate and Intersect) are the only places where
// floating-point operations take place in the whole Boolean function. These are
// carefully designed to minimize rounding error and to eliminate it at edge
// cases to ensure consistency.

__host__ __device__ glm::vec2 Interpolate(glm::vec3 pL, glm::vec3 pR, float x) {
  float dxL = x - pL.x;
  float dxR = x - pR.x;
  bool useL = fabs(dxL) < fabs(dxR);
  float lambda = (useL ? dxL : dxR) / (pR.x - pL.x);
  if (!isfinite(lambda)) return glm::vec2(pL.y, pL.z);
  glm::vec2 yz;
  yz[0] = (useL ? pL.y : pR.y) + lambda * (pR.y - pL.y);
  yz[1] = (useL ? pL.z : pR.z) + lambda * (pR.z - pL.z);
  return yz;
}

__host__ __device__ glm::vec4 Intersect(const glm::vec3 &pL,
                                        const glm::vec3 &pR,
                                        const glm::vec3 &qL,
                                        const glm::vec3 &qR) {
  float dyL = qL.y - pL.y;
  float dyR = qR.y - pR.y;
  bool useL = fabs(dyL) < fabs(dyR);
  float dx = pR.x - pL.x;
  float lambda = (useL ? dyL : dyR) / (dyL - dyR);
  if (!isfinite(lambda)) lambda = 0.0f;
  glm::vec4 xyzz;
  xyzz.x = (useL ? pL.x : pR.x) + lambda * dx;
  float pDy = pR.y - pL.y;
  float qDy = qR.y - qL.y;
  bool useP = fabs(pDy) < fabs(qDy);
  xyzz.y = (useL ? (useP ? pL.y : qL.y) : (useP ? pR.y : qR.y)) +
           lambda * (useP ? pDy : qDy);
  xyzz.z = (useL ? pL.z : pR.z) + lambda * (pR.z - pL.z);
  xyzz.w = (useL ? qL.z : qR.z) + lambda * (qR.z - qL.z);
  return xyzz;
}

struct MarkEdgeVerts {
  int *verts;
  const Halfedge *halfedges;

  __host__ __device__ void operator()(int edge) {
    int vert = halfedges[edge].startVert;
    verts[vert] = vert;
    vert = halfedges[edge].endVert;
    verts[vert] = vert;
  }
};

struct MarkFaceVerts {
  int *verts;
  const Halfedge *halfedges;
  const int *faces;

  __host__ __device__ void operator()(int face) {
    int edge = faces[face];
    const int lastEdge = faces[face + 1];
    while (edge < lastEdge) {
      int vert = halfedges[edge++].startVert;
      verts[vert] = vert;
    }
  }
};

SparseIndices Filter02(const Manifold::Impl &inP, const Manifold::Impl &inQ,
                       const VecDH<int> &edges, const VecDH<int> &faces) {
  // find inP's involved vertices from edges & faces
  VecDH<int> p0(inP.NumVert(), -1);
  // We keep the verts unique by marking the ones we want to keep
  // with their own index in parallel (collisions don't matter because any given
  // element is always being written with the same value). Any that are still
  // initialized to -1 are not involved and can be removed.
  thrust::for_each_n(edges.beginD(), edges.size(),
                     MarkEdgeVerts({p0.ptrD(), inP.halfedge_.cptrD()}));

  thrust::for_each_n(
      faces.beginD(), faces.size(),
      MarkFaceVerts({p0.ptrD(), inP.halfedge_.cptrD(), inP.faceEdge_.cptrD()}));

  // find one vertex from each connected component of inP (in case it has no
  // intersections)
  VecDH<int> vertLabels;
  int n_comp = ConnectedComponents(vertLabels, inP.NumVert(), inP.halfedge_);
  for (int i = 0; i < n_comp; ++i) {
    int vert = thrust::find(vertLabels.beginD(), vertLabels.endD(), i) -
               vertLabels.beginD();
    p0.H()[vert] = vert;
  }

  p0.resize(thrust::remove(p0.beginD(), p0.endD(), -1) - p0.beginD());
  // find which inQ faces shadow these vertices
  VecDH<glm::vec3> vertPosP(p0.size());
  thrust::gather(p0.beginD(), p0.endD(), inP.vertPos_.cbeginD(),
                 vertPosP.beginD());
  SparseIndices p0q2 = inQ.VertexCollisionsZ(vertPosP);
  VecDH<int> i02temp(p0q2.size());
  thrust::copy(p0q2.beginD(0), p0q2.endD(0), i02temp.beginD());
  thrust::gather(i02temp.beginD(), i02temp.endD(), p0.beginD(), p0q2.beginD(0));
  return p0q2;
}

struct CopyFaceEdges {
  // x can be either vert or edge (0 or 1).
  thrust::pair<int *, int *> pXq1;
  const int *facesQ;
  const Halfedge *halfedgesQ;

  __host__ __device__ void operator()(thrust::tuple<int, int, int> in) {
    int idx = thrust::get<0>(in);
    const int pX = thrust::get<1>(in);
    const int q2 = thrust::get<2>(in);

    int q1 = facesQ[q2];
    const int end = facesQ[q2 + 1];
    while (q1 < end) {
      pXq1.first[idx] = pX;
      const Halfedge edge = halfedgesQ[q1];
      pXq1.second[idx++] = edge.IsForward() ? q1 : edge.pairedHalfedge;
      ++q1;
    }
  }
};

SparseIndices Filter11(const Manifold::Impl &inP, const VecDH<int> &faceSizeP,
                       const Manifold::Impl &inQ, const VecDH<int> &faceSizeQ,
                       const SparseIndices &p1q2, const SparseIndices &p2q1) {
  VecDH<int> expandedIdxQ(p1q2.size() + 1);
  auto includedFaceSizeQ = perm(faceSizeQ.beginD() + 1, p1q2.beginD(1));
  thrust::inclusive_scan(includedFaceSizeQ, includedFaceSizeQ + p1q2.size(),
                         expandedIdxQ.beginD() + 1);
  const int secondStart = expandedIdxQ.H().back();

  VecDH<int> expandedIdxP(p2q1.size() + 1);
  auto includedFaceSizeP = perm(faceSizeP.beginD() + 1, p2q1.beginD(0));
  thrust::inclusive_scan(includedFaceSizeP, includedFaceSizeP + p2q1.size(),
                         expandedIdxP.beginD() + 1);

  SparseIndices p1q1(secondStart + expandedIdxP.H().back());
  thrust::for_each_n(zip(expandedIdxQ.beginD(), p1q2.beginD(0), p1q2.beginD(1)),
                     p1q2.size(),
                     CopyFaceEdges({p1q1.ptrDpq(), inQ.faceEdge_.cptrD(),
                                    inQ.halfedge_.cptrD()}));

  p1q1.SwapPQ();
  thrust::for_each_n(
      zip(expandedIdxP.beginD(), p2q1.beginD(1), p2q1.beginD(0)), p2q1.size(),
      CopyFaceEdges({p1q1.ptrDpq(secondStart), inP.faceEdge_.cptrD(),
                     inP.halfedge_.cptrD()}));
  p1q1.SwapPQ();
  p1q1.Unique();
  return p1q1;
}

struct CopyEdgeVerts {
  thrust::pair<int *, int *> p0q1;
  const Halfedge *halfedges;

  __host__ __device__ void operator()(thrust::tuple<int, int, int> in) {
    int idx = 2 * thrust::get<0>(in);
    const int p1 = thrust::get<1>(in);
    const int q1 = thrust::get<2>(in);

    p0q1.first[idx] = halfedges[p1].startVert;
    p0q1.second[idx] = q1;
    p0q1.first[idx + 1] = halfedges[p1].endVert;
    p0q1.second[idx + 1] = q1;
  }
};

SparseIndices Filter01(const Manifold::Impl &inP, const Manifold::Impl &inQ,
                       const VecDH<int> &faceSizeQ, const SparseIndices &p0q2,
                       const SparseIndices &p1q1) {
  VecDH<int> expandedIdxQ(p0q2.size() + 1);
  auto includedFaceSizeQ = perm(faceSizeQ.beginD() + 1, p0q2.beginD(1));
  thrust::inclusive_scan(includedFaceSizeQ, includedFaceSizeQ + p0q2.size(),
                         expandedIdxQ.beginD() + 1);
  const int secondStart = expandedIdxQ.H().back();

  SparseIndices p0q1(secondStart + 2 * p1q1.size());

  thrust::for_each_n(zip(expandedIdxQ.beginD(), p0q2.beginD(0), p0q2.beginD(1)),
                     p0q2.size(),
                     CopyFaceEdges({p0q1.ptrDpq(), inQ.faceEdge_.cptrD(),
                                    inQ.halfedge_.cptrD()}));

  thrust::for_each_n(
      zip(thrust::make_counting_iterator(0), p1q1.beginD(0), p1q1.beginD(1)),
      p1q1.size(),
      CopyEdgeVerts({p0q1.ptrDpq(secondStart), inP.halfedge_.cptrD()}));
  return p0q1;
}

struct AbsSum : public thrust::binary_function<int, int, int> {
  __host__ __device__ int operator()(int a, int b) { return abs(a) + abs(b); }
};

__host__ __device__ bool Shadows(float p, float q, float dir) {
  return p == q ? dir < 0 : p < q;
}

struct ShadowKernel01 {
  const bool reverse;
  const glm::vec3 *vertPosP;
  const glm::vec3 *vertPosQ;
  const Halfedge *halfedgeQ;
  const float expandP;
  const glm::vec3 *normalP;

  __host__ __device__ void operator()(thrust::tuple<int &, int, int> inout) {
    int &s01 = thrust::get<0>(inout);
    const int p0 = thrust::get<1>(inout);
    const int q1 = thrust::get<2>(inout);

    const int q1s = halfedgeQ[q1].startVert;
    const int q1e = halfedgeQ[q1].endVert;
    const float p0x = vertPosP[p0].x;
    const float q1sx = vertPosQ[q1s].x;
    const float q1ex = vertPosQ[q1e].x;
    s01 = reverse ? Shadows(q1sx, p0x, expandP * normalP[q1s].x) -
                        Shadows(q1ex, p0x, expandP * normalP[q1e].x)
                  : Shadows(p0x, q1ex, expandP * normalP[p0].x) -
                        Shadows(p0x, q1sx, expandP * normalP[p0].x);
  }
};

struct Kernel01 {
  const bool reverse;
  const glm::vec3 *vertPosP;
  const glm::vec3 *vertPosQ;
  const Halfedge *halfedgeQ;
  const float expandP;
  const glm::vec3 *normalP;

  __host__ __device__ void operator()(
      thrust::tuple<glm::vec2 &, int &, int, int> inout) {
    glm::vec2 &yz01 = thrust::get<0>(inout);
    int &s01 = thrust::get<1>(inout);
    const int p0 = thrust::get<2>(inout);
    const int q1 = thrust::get<3>(inout);

    const int q1s = halfedgeQ[q1].startVert;
    const int q1e = halfedgeQ[q1].endVert;
    yz01 = Interpolate(vertPosQ[q1s], vertPosQ[q1e], vertPosP[p0].x);
    if (reverse) {
      glm::vec3 diff = vertPosQ[q1s] - vertPosP[p0];
      const float start2 = glm::dot(diff, diff);
      diff = vertPosQ[q1e] - vertPosP[p0];
      const float end2 = glm::dot(diff, diff);
      const float dir = start2 < end2 ? normalP[q1s].y : normalP[q1e].y;
      if (!Shadows(yz01[0], vertPosP[p0].y, expandP * dir)) s01 = 0;
    } else {
      if (!Shadows(vertPosP[p0].y, yz01[0], expandP * normalP[p0].y)) s01 = 0;
    }
  }
};

std::tuple<VecDH<int>, VecDH<glm::vec2>> Shadow01(SparseIndices &p0q1,
                                                  const Manifold::Impl &inP,
                                                  const Manifold::Impl &inQ,
                                                  bool reverse, float expandP) {
  VecDH<int> s01(p0q1.size());
  if (reverse) p0q1.SwapPQ();
  auto normalP = reverse ? inQ.vertNormal_.cptrD() : inP.vertNormal_.cptrD();
  thrust::for_each_n(
      zip(s01.beginD(), p0q1.beginD(0), p0q1.beginD(1)), p0q1.size(),
      ShadowKernel01({reverse, inP.vertPos_.cptrD(), inQ.vertPos_.cptrD(),
                      inQ.halfedge_.cptrD(), expandP, normalP}));
  size_t size = p0q1.RemoveZeros(s01);
  VecDH<glm::vec2> yz01(size);

  normalP = reverse ? inQ.vertNormal_.cptrD() : inP.vertNormal_.cptrD();
  thrust::for_each_n(
      zip(yz01.beginD(), s01.beginD(), p0q1.beginD(0), p0q1.beginD(1)), size,
      Kernel01({reverse, inP.vertPos_.cptrD(), inQ.vertPos_.cptrD(),
                inQ.halfedge_.cptrD(), expandP, normalP}));
  if (reverse) p0q1.SwapPQ();
  return std::make_tuple(s01, yz01);
}

template <typename Val>
__host__ __device__ Val BinarySearchByKey(
    const thrust::pair<const int *, const int *> keys, const Val *vals,
    const int size, const thrust::pair<int, int> key, const Val missingVal) {
  if (size <= 0) return missingVal;
  int left = 0;
  int right = size - 1;
  int m;
  thrust::pair<int, int> keyM;
  while (1) {
    m = right - (right - left) / 2;
    keyM = thrust::make_pair(keys.first[m], keys.second[m]);
    if (left == right) break;
    if (keyM > key)
      right = m - 1;
    else
      left = m;
  }
  if (keyM == key)
    return vals[m];
  else
    return missingVal;
}

struct Gather11 {
  const thrust::pair<const int *, const int *> p0q1;
  const int *s01;
  const int size;
  const Halfedge *halfedgeP;
  const bool reverse;

  __host__ __device__ void operator()(thrust::tuple<int &, int, int> inout) {
    int &s11 = thrust::get<0>(inout);
    const int p1 = thrust::get<1>(inout);
    const int q1 = thrust::get<2>(inout);

    int p0 = halfedgeP[p1].endVert;
    auto key = reverse ? thrust::make_pair(q1, p0) : thrust::make_pair(p0, q1);
    s11 += BinarySearchByKey(p0q1, s01, size, key, 0);
    p0 = halfedgeP[p1].startVert;
    key = reverse ? thrust::make_pair(q1, p0) : thrust::make_pair(p0, q1);
    s11 -= BinarySearchByKey(p0q1, s01, size, key, 0);
  }
};

struct Kernel11 {
  const glm::vec3 *vertPosP;
  const glm::vec3 *vertPosQ;
  const Halfedge *halfedgeP;
  const Halfedge *halfedgeQ;
  thrust::pair<const int *, const int *> p0q1;
  const glm::vec2 *yz01;
  int size01;
  thrust::pair<const int *, const int *> p1q0;
  const glm::vec2 *yz10;
  int size10;
  float expandP;
  const glm::vec3 *normalP;

  __host__ __device__ void operator()(
      thrust::tuple<glm::vec4 &, int &, int, int> inout) {
    glm::vec4 &xyzz11 = thrust::get<0>(inout);
    int &s11 = thrust::get<1>(inout);
    const int p1 = thrust::get<2>(inout);
    const int q1 = thrust::get<3>(inout);

    glm::vec3 p2[2], q2[2];
    int k = 0;
    thrust::pair<int, int> key2[2];

    key2[0] = thrust::make_pair(halfedgeP[p1].startVert, q1);
    key2[1] = thrust::make_pair(halfedgeP[p1].endVert, q1);
    for (int i : {0, 1}) {
      p2[k] = vertPosP[key2[i].first];
      q2[k] = glm::vec3(p2[k].x, BinarySearchByKey(p0q1, yz01, size01, key2[i],
                                                   glm::vec2(0.0f / 0.0f)));
      if (!isnan(q2[k].y)) k++;
    }

    key2[0] = thrust::make_pair(p1, halfedgeQ[q1].startVert);
    key2[1] = thrust::make_pair(p1, halfedgeQ[q1].endVert);
    for (int i : {0, 1}) {
      if (k > 1) break;
      q2[k] = vertPosQ[key2[i].second];
      p2[k] = glm::vec3(q2[k].x, BinarySearchByKey(p1q0, yz10, size10, key2[i],
                                                   glm::vec2(0.0f / 0.0f)));
      if (!isnan(p2[k].y)) k++;
    }

    // assert two of these four were found
    if (k != 2) printf("k = %d\n", k);

    xyzz11 = Intersect(p2[0], p2[1], q2[0], q2[1]);

    const int p1s = halfedgeP[p1].startVert;
    const int p1e = halfedgeP[p1].endVert;
    glm::vec3 diff = vertPosP[p1s] - glm::vec3(xyzz11);
    const float start2 = glm::dot(diff, diff);
    diff = vertPosP[p1e] - glm::vec3(xyzz11);
    const float end2 = glm::dot(diff, diff);
    const float dir = start2 < end2 ? normalP[p1s].z : normalP[p1e].z;

    if (!Shadows(xyzz11.z, xyzz11.w, expandP * dir)) s11 = 0;
  }
};

std::tuple<VecDH<int>, VecDH<glm::vec4>> Shadow11(
    SparseIndices &p1q1, const Manifold::Impl &inP, const Manifold::Impl &inQ,
    const SparseIndices &p0q1, const VecDH<int> &s01,
    const VecDH<glm::vec2> &yz01, const SparseIndices &p1q0,
    const VecDH<int> &s10, const VecDH<glm::vec2> &yz10, float expandP) {
  VecDH<int> s11(p1q1.size(), 0);

  thrust::for_each_n(zip(s11.beginD(), p1q1.beginD(0), p1q1.beginD(1)),
                     p1q1.size(),
                     Gather11({p0q1.ptrDpq(), s01.cptrD(), p0q1.size(),
                               inP.halfedge_.cptrD(), false}));
  thrust::for_each_n(zip(s11.beginD(), p1q1.beginD(1), p1q1.beginD(0)),
                     p1q1.size(),
                     Gather11({p1q0.ptrDpq(), s10.cptrD(), p1q0.size(),
                               inQ.halfedge_.cptrD(), true}));

  size_t size = p1q1.RemoveZeros(s11);
  VecDH<glm::vec4> xyzz11(size);

  thrust::for_each_n(
      zip(xyzz11.beginD(), s11.beginD(), p1q1.beginD(0), p1q1.beginD(1)),
      p1q1.size(),
      Kernel11({inP.vertPos_.cptrD(), inQ.vertPos_.cptrD(),
                inP.halfedge_.cptrD(), inQ.halfedge_.cptrD(), p0q1.ptrDpq(),
                yz01.cptrD(), p0q1.size(), p1q0.ptrDpq(), yz10.cptrD(),
                p1q0.size(), expandP, inP.vertNormal_.cptrD()}));

  return std::make_tuple(s11, xyzz11);
};

struct Gather02 {
  const thrust::pair<const int *, const int *> p0q1;
  const int *s01;
  const int size;
  const int *facesQ;
  const Halfedge *halfedgesQ;
  const bool forward;

  __host__ __device__ void operator()(thrust::tuple<int &, int, int> inout) {
    int &s02 = thrust::get<0>(inout);
    const int p0 = thrust::get<1>(inout);
    const int q2 = thrust::get<2>(inout);

    int q1 = facesQ[q2];
    const int lastEdge = facesQ[q2 + 1];
    while (q1 < lastEdge) {
      const Halfedge edge = halfedgesQ[q1];
      const int q1F = edge.IsForward() ? q1 : edge.pairedHalfedge;
      const auto key =
          forward ? thrust::make_pair(p0, q1F) : thrust::make_pair(q1F, p0);
      s02 += (forward == edge.IsForward() ? -1 : 1) *
             BinarySearchByKey(p0q1, s01, size, key, 0);
      ++q1;
    }
  }
};

struct Kernel02 {
  const glm::vec3 *vertPosP;
  const thrust::pair<const int *, const int *> p0q1;
  const glm::vec2 *yz01;
  const int size;
  const int *facesQ;
  const Halfedge *halfedgesQ;
  const bool forward;
  const float expandP;
  const glm::vec3 *normalP;

  __host__ __device__ void operator()(
      thrust::tuple<float &, int &, int, int> inout) {
    float &z02 = thrust::get<0>(inout);
    int &s02 = thrust::get<1>(inout);
    const int p0 = thrust::get<2>(inout);
    const int q2 = thrust::get<3>(inout);

    int q1 = facesQ[q2];
    const int lastEdge = facesQ[q2 + 1];
    glm::vec3 yzz2[2];
    int k = 0;
    while (q1 < lastEdge) {
      const Halfedge edge = halfedgesQ[q1];
      const int q1F = edge.IsForward() ? q1 : edge.pairedHalfedge;
      const auto key =
          forward ? thrust::make_pair(p0, q1F) : thrust::make_pair(q1F, p0);
      const glm::vec2 yz =
          BinarySearchByKey(p0q1, yz01, size, key, glm::vec2(0.0f / 0.0f));
      if (!isnan(yz[0])) yzz2[k++] = glm::vec3(yz[0], yz[1], yz[1]);
      if (k > 1) break;
      ++q1;
    }
    // assert two of these were found
    if (k != 2) printf("k = %d\n", k);

    glm::vec3 vertPos = vertPosP[p0];
    z02 = Interpolate(yzz2[0], yzz2[1], vertPos.y)[1];
    if (forward) {
      if (!Shadows(vertPos.z, z02, expandP * normalP[p0].z)) s02 = 0;
    } else {
      if (!Shadows(z02, vertPos.z, expandP * normalP[q2].z)) s02 = 0;
    }
  }
};

std::tuple<VecDH<int>, VecDH<float>> Shadow02(
    const Manifold::Impl &inP, const Manifold::Impl &inQ, const VecDH<int> &s01,
    const SparseIndices &p0q1, const VecDH<glm::vec2> &yz01,
    SparseIndices &p0q2, bool forward, float expandP) {
  VecDH<int> s02(p0q2.size(), 0);

  thrust::for_each_n(
      zip(s02.beginD(), p0q2.beginD(!forward), p0q2.beginD(forward)),
      p0q2.size(),
      Gather02({p0q1.ptrDpq(), s01.cptrD(), p0q1.size(), inQ.faceEdge_.cptrD(),
                inQ.halfedge_.cptrD(), forward}));

  size_t size = p0q2.RemoveZeros(s02);
  VecDH<float> z02(size);

  auto normalP = forward ? inP.vertNormal_.cptrD() : inQ.faceNormal_.cptrD();
  thrust::for_each_n(
      zip(z02.beginD(), s02.beginD(), p0q2.beginD(!forward),
          p0q2.beginD(forward)),
      size,
      Kernel02({inP.vertPos_.cptrD(), p0q1.ptrDpq(), yz01.cptrD(), p0q1.size(),
                inQ.faceEdge_.cptrD(), inQ.halfedge_.cptrD(), forward, expandP,
                normalP}));

  return std::make_tuple(s02, z02);
};

struct Gather12 {
  const thrust::pair<const int *, const int *> p0q2;
  const int *s02;
  const int size02;
  const thrust::pair<const int *, const int *> p1q1;
  const int *s11;
  const int size11;
  const Halfedge *halfedgesP;
  const int *facesQ;
  const Halfedge *halfedgesQ;
  const bool forward;

  __host__ __device__ void operator()(thrust::tuple<int &, int, int> inout) {
    int &x12 = thrust::get<0>(inout);
    const int p1 = thrust::get<1>(inout);
    const int q2 = thrust::get<2>(inout);

    const Halfedge edge = halfedgesP[p1];
    auto key = forward ? thrust::make_pair(edge.startVert, q2)
                       : thrust::make_pair(q2, edge.endVert);
    x12 = BinarySearchByKey(p0q2, s02, size02, key, 0);
    key = forward ? thrust::make_pair(edge.endVert, q2)
                  : thrust::make_pair(q2, edge.startVert);
    x12 -= BinarySearchByKey(p0q2, s02, size02, key, 0);

    int q1 = facesQ[q2];
    const int lastEdge = facesQ[q2 + 1];
    while (q1 < lastEdge) {
      const Halfedge edge = halfedgesQ[q1];
      const int q1F = edge.IsForward() ? q1 : edge.pairedHalfedge;
      key = forward ? thrust::make_pair(p1, q1F) : thrust::make_pair(q1F, p1);
      x12 -= (edge.IsForward() ? 1 : -1) *
             BinarySearchByKey(p1q1, s11, size11, key, 0);
      ++q1;
    }
  }
};

struct Kernel12 {
  const thrust::pair<const int *, const int *> p0q2;
  const float *z02;
  const int size02;
  const thrust::pair<const int *, const int *> p1q1;
  const glm::vec4 *xyzz11;
  const int size11;
  const Halfedge *halfedgesP;
  const int *facesQ;
  const Halfedge *halfedgesQ;
  const glm::vec3 *vertPosP;
  const bool forward;

  __host__ __device__ void operator()(
      thrust::tuple<glm::vec3 &, int, int> inout) {
    glm::vec3 &v12 = thrust::get<0>(inout);
    const int p1 = thrust::get<1>(inout);
    const int q2 = thrust::get<2>(inout);

    const Halfedge edge = halfedgesP[p1];
    auto key = forward ? thrust::make_pair(edge.startVert, q2)
                       : thrust::make_pair(q2, edge.startVert);
    const float z0 = BinarySearchByKey(p0q2, z02, size02, key, 0.0f / 0.0f);
    key = forward ? thrust::make_pair(edge.endVert, q2)
                  : thrust::make_pair(q2, edge.endVert);
    const float z1 = BinarySearchByKey(p0q2, z02, size02, key, 0.0f / 0.0f);

    glm::vec3 xzyLR0[2];
    glm::vec3 xzyLR1[2];
    int k = 0;
    if (!isnan(z0)) {
      xzyLR0[k] = vertPosP[edge.startVert];
      thrust::swap(xzyLR0[k].y, xzyLR0[k].z);
      xzyLR1[k] = xzyLR0[k];
      xzyLR1[k][1] = z0;
      k++;
    }
    if (!isnan(z1)) {
      xzyLR0[k] = vertPosP[edge.endVert];
      thrust::swap(xzyLR0[k].y, xzyLR0[k].z);
      xzyLR1[k] = xzyLR0[k];
      xzyLR1[k][1] = z1;
      k++;
    }

    int q1 = facesQ[q2];
    const int lastEdge = facesQ[q2 + 1];
    while (q1 < lastEdge) {
      if (k > 1) break;
      const Halfedge edge = halfedgesQ[q1];
      const int q1F = edge.IsForward() ? q1 : edge.pairedHalfedge;
      key = forward ? thrust::make_pair(p1, q1F) : thrust::make_pair(q1F, p1);
      const glm::vec4 xyzz =
          BinarySearchByKey(p1q1, xyzz11, size11, key, glm::vec4(0.0f / 0.0f));

      if (!isnan(xyzz.x)) {
        xzyLR0[k][0] = xyzz.x;
        xzyLR0[k][1] = xyzz.z;
        xzyLR0[k][2] = xyzz.y;
        xzyLR1[k] = xzyLR0[k];
        xzyLR1[k][1] = xyzz.w;
        if (!forward) thrust::swap(xzyLR0[k][1], xzyLR1[k][1]);
        k++;
      }
      ++q1;
    }

    // assert two of these five were found
    if (k != 2) printf("k = %d\n", k);

    const glm::vec4 xzyy =
        Intersect(xzyLR0[0], xzyLR0[1], xzyLR1[0], xzyLR1[1]);
    v12.x = xzyy[0];
    v12.y = xzyy[2];
    v12.z = xzyy[1];
  }
};

std::tuple<VecDH<int>, VecDH<glm::vec3>> Intersect12(
    const Manifold::Impl &inP, const Manifold::Impl &inQ, const VecDH<int> &s02,
    const SparseIndices &p0q2, const VecDH<int> &s11, const SparseIndices &p1q1,
    const VecDH<float> &z02, const VecDH<glm::vec4> &xyzz11,
    SparseIndices &p1q2, bool forward) {
  VecDH<int> x12(p1q2.size());
  VecDH<glm::vec3> v12;

  const auto halfedgesP =
      forward ? inP.halfedge_.cptrD() : inQ.halfedge_.cptrD();
  const auto halfedgesQ =
      forward ? inQ.halfedge_.cptrD() : inP.halfedge_.cptrD();
  const auto facesQ = forward ? inQ.faceEdge_.cptrD() : inP.faceEdge_.cptrD();

  thrust::for_each_n(
      zip(x12.beginD(), p1q2.beginD(!forward), p1q2.beginD(forward)),
      p1q2.size(),
      Gather12({p0q2.ptrDpq(), s02.ptrD(), p0q2.size(), p1q1.ptrDpq(),
                s11.ptrD(), p1q1.size(), halfedgesP, facesQ, halfedgesQ,
                forward}));

  size_t size = p1q2.RemoveZeros(x12);
  v12.resize(size);

  const auto vertPosPtr = forward ? inP.vertPos_.cptrD() : inQ.vertPos_.cptrD();
  thrust::for_each_n(
      zip(v12.beginD(), p1q2.beginD(!forward), p1q2.beginD(forward)),
      p1q2.size(),
      Kernel12({p0q2.ptrDpq(), z02.cptrD(), p0q2.size(), p1q1.ptrDpq(),
                xyzz11.cptrD(), p1q1.size(), halfedgesP, facesQ, halfedgesQ,
                vertPosPtr, forward}));
  return std::make_tuple(x12, v12);
};

VecDH<int> Winding03(const Manifold::Impl &inP, SparseIndices &p0q2,
                     VecDH<int> &s02, const SparseIndices &p1q2, bool reverse) {
  VecDH<int> w03(inP.NumVert(), kInvalidInt);
  // keepEdgesP is the set of edges that connect regions of the manifold with
  // the same winding number, so we remove any edges associated with
  // intersections.
  VecDH<bool> keepEdgesP(inP.halfedge_.size(), true);
  thrust::scatter(thrust::make_constant_iterator(false, 0),
                  thrust::make_constant_iterator(false, p1q2.size()),
                  p1q2.beginD(reverse), keepEdgesP.beginD());

  if (!thrust::is_sorted(p0q2.beginD(reverse), p0q2.endD(reverse)))
    thrust::sort_by_key(p0q2.beginD(reverse), p0q2.endD(reverse), s02.beginD());
  VecDH<int> w03val(w03.size());
  VecDH<int> w03vert(w03.size());
  // sum known s02 values into w03 (winding number)
  auto endPair =
      thrust::reduce_by_key(p0q2.beginD(reverse), p0q2.endD(reverse),
                            s02.beginD(), w03vert.beginD(), w03val.beginD());
  thrust::scatter(w03val.beginD(), endPair.second, w03vert.beginD(),
                  w03.beginD());

  // find connected regions (separated by intersections)
  VecDH<int> vertLabels;
  int n_comp =
      ConnectedComponents(vertLabels, inP.NumVert(), inP.halfedge_, keepEdgesP);
  // flood the w03 values throughout their connected components (they are
  // consistent)
  FloodComponents(w03, vertLabels, n_comp);

  if (kVerbose) std::cout << n_comp << " components" << std::endl;

  if (reverse)
    thrust::transform(w03.beginD(), w03.endD(), w03.beginD(),
                      thrust::negate<int>());
  return w03;
};

struct DuplicateVerts {
  glm::vec3 *vertPosR;

  __host__ __device__ void operator()(thrust::tuple<int, int, glm::vec3> in) {
    int inclusion = abs(thrust::get<0>(in));
    int vertR = thrust::get<1>(in);
    glm::vec3 vertPosP = thrust::get<2>(in);

    for (int i = 0; i < inclusion; ++i) {
      vertPosR[vertR + i] = vertPosP;
    }
  }
};

__host__ __device__ int AtomicAddInt(int &target, int add) {
#ifdef __CUDA_ARCH__
  return atomicAdd(&target, add);
#else
  int out;
#pragma omp atomic
  {
    out = target;
    target += add;
  }
  return out;
#endif
}

struct CountVerts {
  int *count;
  const int *inclusion;

  __host__ __device__ void operator()(const Halfedge &edge) {
    AtomicAddInt(count[edge.face], glm::abs(inclusion[edge.startVert]));
  }
};

struct CountNewVerts {
  int *countP;
  int *countQ;
  const Halfedge *halfedges;

  __host__ __device__ void operator()(thrust::tuple<int, int, int> in) {
    int edgeP = thrust::get<0>(in);
    int faceQ = thrust::get<1>(in);
    int inclusion = glm::abs(thrust::get<2>(in));

    AtomicAddInt(countQ[faceQ], inclusion);
    const Halfedge half = halfedges[edgeP];
    AtomicAddInt(countP[half.face], inclusion);
    AtomicAddInt(countP[halfedges[half.pairedHalfedge].face], inclusion);
  }
};

struct NotZero : public thrust::unary_function<int, int> {
  __host__ __device__ int operator()(int x) const { return x > 0 ? 1 : 0; }
};

VecDH<int> SizeOutput(Manifold::Impl &outR, const Manifold::Impl &inP,
                      const Manifold::Impl &inQ, const VecDH<int> &i03,
                      const VecDH<int> &i30, const VecDH<int> &i12,
                      const VecDH<int> &i21, const SparseIndices &p1q2,
                      const SparseIndices &p2q1, bool invertQ) {
  VecDH<int> sidesPerFacePQ(inP.NumFace() + inQ.NumFace());
  auto sidesPerFaceP = sidesPerFacePQ.ptrD();
  auto sidesPerFaceQ = sidesPerFacePQ.ptrD() + inP.NumFace();

  thrust::for_each(inP.halfedge_.beginD(), inP.halfedge_.endD(),
                   CountVerts({sidesPerFaceP, i03.cptrD()}));
  thrust::for_each(inQ.halfedge_.beginD(), inQ.halfedge_.endD(),
                   CountVerts({sidesPerFaceQ, i30.cptrD()}));
  thrust::for_each_n(
      zip(p1q2.beginD(0), p1q2.beginD(1), i12.beginD()), i12.size(),
      CountNewVerts({sidesPerFaceP, sidesPerFaceQ, inP.halfedge_.cptrD()}));
  thrust::for_each_n(
      zip(p2q1.beginD(1), p2q1.beginD(0), i21.beginD()), i21.size(),
      CountNewVerts({sidesPerFaceQ, sidesPerFaceP, inQ.halfedge_.cptrD()}));

  VecDH<int> facePQ2R(inP.NumFace() + inQ.NumFace() + 1);
  auto keepFace =
      thrust::make_transform_iterator(sidesPerFacePQ.beginD(), NotZero());
  thrust::inclusive_scan(keepFace, keepFace + sidesPerFacePQ.size(),
                         facePQ2R.beginD() + 1);
  int numFaceR = facePQ2R.H().back();
  facePQ2R.resize(inP.NumFace() + inQ.NumFace());

  outR.faceNormal_.resize(numFaceR);
  auto next = thrust::copy_if(inP.faceNormal_.beginD(), inP.faceNormal_.endD(),
                              keepFace, outR.faceNormal_.beginD(),
                              thrust::identity<bool>());
  if (invertQ) {
    auto start = thrust::make_transform_iterator(inQ.faceNormal_.beginD(),
                                                 thrust::negate<glm::vec3>());
    auto end = thrust::make_transform_iterator(inQ.faceNormal_.endD(),
                                               thrust::negate<glm::vec3>());
    thrust::copy_if(start, end, keepFace + inP.NumFace(), next,
                    thrust::identity<bool>());
  } else {
    thrust::copy_if(inQ.faceNormal_.beginD(), inQ.faceNormal_.endD(),
                    keepFace + inP.NumFace(), next, thrust::identity<bool>());
  }

  auto newEnd =
      thrust::remove(sidesPerFacePQ.beginD(), sidesPerFacePQ.endD(), 0);
  outR.faceEdge_.resize(newEnd - sidesPerFacePQ.beginD() + 1);
  thrust::inclusive_scan(sidesPerFacePQ.beginD(), newEnd,
                         outR.faceEdge_.beginD() + 1);
  outR.halfedge_.resize(outR.faceEdge_.H().back());

  return facePQ2R;
}

struct DuplicateHalfedges {
  Halfedge *halfedgesR;
  int *facePtr;
  const Halfedge *halfedgesP;
  const int *i03;
  const int *vP2R;
  const int *faceP2R;

  __host__ __device__ void operator()(thrust::tuple<bool, Halfedge> in) {
    if (!thrust::get<0>(in)) return;
    Halfedge halfedge = thrust::get<1>(in);
    if (!halfedge.IsForward()) return;

    const int inclusion = i03[halfedge.startVert];
    if (inclusion == 0) return;
    if (inclusion < 0) {  // reverse
      int tmp = halfedge.startVert;
      halfedge.startVert = halfedge.endVert;
      halfedge.endVert = tmp;
    }
    halfedge.startVert = vP2R[halfedge.startVert];
    halfedge.endVert = vP2R[halfedge.endVert];
    halfedge.face = faceP2R[halfedge.face];
    int faceRight = faceP2R[halfedgesP[halfedge.pairedHalfedge].face];

    Halfedge backward = {halfedge.endVert, halfedge.startVert, -1, faceRight};

    for (int i = 0; i < glm::abs(inclusion); ++i) {
      int forwardIdx = AtomicAddInt(facePtr[halfedge.face], 1);
      int backwardIdx = AtomicAddInt(facePtr[faceRight], 1);
      halfedge.pairedHalfedge = backwardIdx;
      backward.pairedHalfedge = forwardIdx;
      halfedgesR[forwardIdx] = halfedge;
      halfedgesR[backwardIdx] = backward;
    }
  }
};

struct EdgePos {
  int vert;
  float edgePos;
  bool isStart;
};

void AddNewEdgeVerts(
    std::map<int, std::vector<EdgePos>> &edgesP,
    std::map<std::pair<int, int>, std::vector<EdgePos>> &edgesNew,
    const SparseIndices &p1q2, const VecH<int> &i12, const VecH<int> &v12R,
    const VecH<Halfedge> &halfedgeP, bool forward) {
  // For each edge of P that intersects a face of Q (p1q2), add this vertex to
  // P's corresponding edge vector and to the two new edges, which are
  // intersections between the face of Q and the two faces of P attached to the
  // edge. The direction and duplicity are given by i12, while v12R remaps to
  // the output vert index. When forward is false, all is reversed.
  const VecH<int> &p1 = p1q2.Get(!forward).H();
  const VecH<int> &q2 = p1q2.Get(forward).H();
  for (int i = 0; i < p1q2.size(); ++i) {
    const int edgeP = p1[i];
    const int faceQ = q2[i];
    const int vert = v12R[i];
    const int inclusion = i12[i];

    auto &edgePosP = edgesP[edgeP];

    Halfedge halfedge = halfedgeP[edgeP];
    std::pair<int, int> key = {halfedgeP[halfedge.pairedHalfedge].face, faceQ};
    if (!forward) std::swap(key.first, key.second);
    auto &edgePosRight = edgesNew[key];

    key = {halfedge.face, faceQ};
    if (!forward) std::swap(key.first, key.second);
    auto &edgePosLeft = edgesNew[key];

    EdgePos edgePos = {vert, 0.0f, inclusion < 0};
    EdgePos edgePosRev = edgePos;
    edgePosRev.isStart = !edgePos.isStart;

    for (int j = 0; j < glm::abs(inclusion); ++j) {
      edgePosP.push_back(edgePos);
      edgePosRight.push_back(forward ? edgePos : edgePosRev);
      edgePosLeft.push_back(forward ? edgePosRev : edgePos);
      ++edgePos.vert;
      ++edgePosRev.vert;
    }
  }
}

std::vector<Halfedge> PairUp(std::vector<EdgePos> &edgePos) {
  // Pair start vertices with end vertices to form edges. The choice of pairing
  // is arbitrary for the manifoldness guarantee, but must be ordered to be
  // geometrically valid. If the order does not go start-end-start-end... then
  // the input and output are not geometrically valid and this algorithm becomes
  // a heuristic.
  ALWAYS_ASSERT(edgePos.size() % 2 == 0, logicErr,
                "Non-manifold edge! Not an even number of points.");
  int nEdges = edgePos.size() / 2;
  auto middle = std::partition(edgePos.begin(), edgePos.end(),
                               [](EdgePos x) { return x.isStart; });
  ALWAYS_ASSERT(middle - edgePos.begin() == nEdges, logicErr,
                "Non-manifold edge!");
  auto cmp = [](EdgePos a, EdgePos b) { return a.edgePos < b.edgePos; };
  std::sort(edgePos.begin(), middle, cmp);
  std::sort(middle, edgePos.end(), cmp);
  std::vector<Halfedge> edges;
  for (int i = 0; i < nEdges; ++i)
    edges.push_back({edgePos[i].vert, edgePos[i + nEdges].vert, -1, -1});
  return edges;
}

void AppendPartialEdges(
    Manifold::Impl &outR, VecH<bool> &wholeHalfedgeP, VecH<int> &facePtrR,
    std::map<int, std::vector<EdgePos>> &edgesP, const Manifold::Impl &inP,
    const VecH<int> &i03, const VecH<int> &vP2R,
    const thrust::host_vector<int>::const_iterator faceP2R) {
  // Each edge in the map is partially retained; for each of these, look up
  // their original verts and include them based on their winding number (i03),
  // while remaping them to the output using vP2R. Use the verts position
  // projected along the edge vector to pair them up, then distribute these
  // edges to their faces. Copy any original edges of each face in that are not
  // in the retained edge map.
  VecH<Halfedge> &halfedgeR = outR.halfedge_.H();
  const VecH<glm::vec3> &vertPosP = inP.vertPos_.H();
  const VecH<Halfedge> &halfedgeP = inP.halfedge_.H();

  for (auto &value : edgesP) {
    const int edgeP = value.first;
    std::vector<EdgePos> &edgePosP = value.second;

    const Halfedge &halfedge = halfedgeP[edgeP];
    wholeHalfedgeP[edgeP] = false;
    wholeHalfedgeP[halfedge.pairedHalfedge] = false;

    const int vStart = halfedge.startVert;
    const int vEnd = halfedge.endVert;
    const glm::vec3 edgeVec = vertPosP[vEnd] - vertPosP[vStart];
    // Fill in the edge positions of the old points.
    for (EdgePos &edge : edgePosP) {
      edge.edgePos = glm::dot(outR.vertPos_.H()[edge.vert], edgeVec);
    }

    int inclusion = i03[vStart];
    EdgePos edgePos = {vP2R[vStart], -1.0f / 0.0f, inclusion > 0};
    for (int j = 0; j < glm::abs(inclusion); ++j) {
      edgePosP.push_back(edgePos);
      ++edgePos.vert;
    }

    inclusion = i03[vEnd];
    edgePos = {vP2R[vEnd], 1.0f / 0.0f, inclusion < 0};
    for (int j = 0; j < glm::abs(inclusion); ++j) {
      edgePosP.push_back(edgePos);
      ++edgePos.vert;
    }

    // sort edges into start/end pairs along length
    std::vector<Halfedge> edges = PairUp(edgePosP);

    // add halfedges to result
    const int faceLeft = faceP2R[halfedge.face];
    const int faceRight = faceP2R[halfedgeP[halfedge.pairedHalfedge].face];
    for (Halfedge e : edges) {
      const int forwardEdge = facePtrR[faceLeft]++;
      const int backwardEdge = facePtrR[faceRight]++;

      e.face = faceLeft;
      e.pairedHalfedge = backwardEdge;
      halfedgeR[forwardEdge] = e;

      std::swap(e.startVert, e.endVert);
      e.face = faceRight;
      e.pairedHalfedge = forwardEdge;
      halfedgeR[backwardEdge] = e;
    }
  }
}

void AppendNewEdges(
    Manifold::Impl &outR, VecH<int> &facePtrR,
    std::map<std::pair<int, int>, std::vector<EdgePos>> &edgesNew,
    const VecH<int> &facePQ2R, const int numFaceP) {
  // Pair up each edge's verts and distribute to faces based on indices in key.
  // Usually only two verts are in each edge, and if not, they are degenerate
  // anyway, so pair arbitrarily without bothering with vertex projections.
  VecH<Halfedge> &halfedgeR = outR.halfedge_.H();

  for (auto &value : edgesNew) {
    const int faceP = value.first.first;
    const int faceQ = value.first.second;
    std::vector<EdgePos> &edgePos = value.second;

    // sort edges into start/end pairs along length.
    std::vector<Halfedge> edges = PairUp(edgePos);

    // add halfedges to result
    const int faceLeft = facePQ2R[faceP];
    const int faceRight = facePQ2R[numFaceP + faceQ];
    for (Halfedge e : edges) {
      const int forwardEdge = facePtrR[faceLeft]++;
      const int backwardEdge = facePtrR[faceRight]++;

      e.face = faceLeft;
      e.pairedHalfedge = backwardEdge;
      halfedgeR[forwardEdge] = e;

      std::swap(e.startVert, e.endVert);
      e.face = faceRight;
      e.pairedHalfedge = forwardEdge;
      halfedgeR[backwardEdge] = e;
    }
  }
}

void AppendWholeEdges(Manifold::Impl &outR, VecDH<int> &facePtrR,
                      const Manifold::Impl &inP, VecDH<bool> wholeHalfedgeP,
                      const VecDH<int> &i03, const VecDH<int> &vP2R,
                      const int *faceP2R) {
  thrust::for_each_n(zip(wholeHalfedgeP.beginD(), inP.halfedge_.beginD()),
                     inP.halfedge_.size(),
                     DuplicateHalfedges({outR.halfedge_.ptrD(), facePtrR.ptrD(),
                                         inP.halfedge_.cptrD(), i03.cptrD(),
                                         vP2R.cptrD(), faceP2R}));
}
}  // namespace

namespace manifold {
Boolean3::Boolean3(const Manifold::Impl &inP, const Manifold::Impl &inQ,
                   Manifold::OpType op)
    : inP_(inP), inQ_(inQ), expandP_(op == Manifold::OpType::ADD ? 1.0 : -1.0) {
  // Symbolic perturbation:
  // Union -> expand inP
  // Difference, Intersection -> contract inP

  inP_.Tri2Face();
  inQ_.Tri2Face();

  VecDH<int> faceSizeP = inP_.FaceSize();
  VecDH<int> faceSizeQ = inQ_.FaceSize();

  Time t0 = NOW();
  Time t1;
  // Level 3
  // Find edge-triangle overlaps (broad phase)
  p1q2_ = inQ_.EdgeCollisions(inP_);
  p1q2_.Sort();
  if (kVerbose) std::cout << "p1q2 size = " << p1q2_.size() << std::endl;

  p2q1_ = inP_.EdgeCollisions(inQ_);
  p2q1_.SwapPQ();
  p2q1_.Sort();
  if (kVerbose) std::cout << "p2q1 size = " << p2q1_.size() << std::endl;

  // Level 2
  // Find vertices from Level 3 that overlap faces in XY-projection
  SparseIndices p0q2 = Filter02(inP_, inQ_, p1q2_.Get(0), p2q1_.Get(0));
  p0q2.Sort();
  if (kVerbose) std::cout << "p0q2 size = " << p0q2.size() << std::endl;

  SparseIndices p2q0 = Filter02(inQ_, inP_, p2q1_.Get(1), p1q2_.Get(1));
  p2q0.SwapPQ();
  p2q0.Sort();
  if (kVerbose) std::cout << "p2q0 size = " << p2q0.size() << std::endl;

  // Find involved edge pairs from Level 3
  SparseIndices p1q1 = Filter11(inP_, faceSizeP, inQ_, faceSizeQ, p1q2_, p2q1_);
  if (kVerbose) std::cout << "p1q1 size = " << p1q1.size() << std::endl;

  // Level 1
  // Find involved vertex-edge pairs from Level 2
  SparseIndices p0q1 = Filter01(inP_, inQ_, faceSizeQ, p0q2, p1q1);
  p0q1.Unique();
  if (kVerbose) std::cout << "p0q1 size = " << p0q1.size() << std::endl;

  p2q0.SwapPQ();
  p1q1.SwapPQ();
  SparseIndices p1q0 = Filter01(inQ_, inP_, faceSizeP, p2q0, p1q1);
  p2q0.SwapPQ();
  p1q1.SwapPQ();
  p1q0.SwapPQ();
  p1q0.Unique();
  if (kVerbose) std::cout << "p1q0 size = " << p1q0.size() << std::endl;

  if (kVerbose) {
    std::cout << "Time for Filter";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }

  // Level 1
  // Find X-projections of vertices onto edges, keeping only those that actually
  // fall inside the edge.
  VecDH<int> s01;
  VecDH<glm::vec2> yz01;
  std::tie(s01, yz01) = Shadow01(p0q1, inP, inQ, false, expandP_);
  if (kVerbose) std::cout << "s01 size = " << s01.size() << std::endl;

  VecDH<int> s10;
  VecDH<glm::vec2> yz10;
  std::tie(s10, yz10) = Shadow01(p1q0, inQ, inP, true, expandP_);
  if (kVerbose) std::cout << "s10 size = " << s10.size() << std::endl;

  // Level 2
  // Build up XY-projection intersection of two edges, including the z-value for
  // each edge, keeping only those whose intersection exists.
  VecDH<int> s11;
  VecDH<glm::vec4> xyzz11;
  std::tie(s11, xyzz11) =
      Shadow11(p1q1, inP, inQ, p0q1, s01, yz01, p1q0, s10, yz10, expandP_);
  if (kVerbose) std::cout << "s11 size = " << s11.size() << std::endl;

  // Build up Z-projection of vertices onto triangles, keeping only those that
  // fall inside the triangle.
  VecDH<int> s02;
  VecDH<float> z02;
  std::tie(s02, z02) =
      Shadow02(inP, inQ, s01, p0q1, yz01, p0q2, true, expandP_);
  if (kVerbose) std::cout << "s02 size = " << s02.size() << std::endl;

  VecDH<int> s20;
  VecDH<float> z20;
  std::tie(s20, z20) =
      Shadow02(inQ, inP, s10, p1q0, yz10, p2q0, false, expandP_);
  if (kVerbose) std::cout << "s20 size = " << s20.size() << std::endl;

  // Level 3
  // Build up the intersection of the edges and triangles, keeping only those
  // that intersect, and record the direction the edge is passing through the
  // triangle.
  std::tie(x12_, v12_) =
      Intersect12(inP, inQ, s02, p0q2, s11, p1q1, z02, xyzz11, p1q2_, true);
  if (kVerbose) std::cout << "dir12 size = " << x12_.size() << std::endl;

  std::tie(x21_, v21_) =
      Intersect12(inP, inQ, s20, p2q0, s11, p1q1, z20, xyzz11, p2q1_, false);
  if (kVerbose) std::cout << "dir21 size = " << x21_.size() << std::endl;

  if (kVerbose) {
    std::cout << "Time for Levels 1-3";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }

  // Build up the winding numbers of all vertices. The involved vertices are
  // calculated from Level 2, while the rest are assigned consistently with
  // connected-components flooding.
  w03_ = Winding03(inP, p0q2, s02, p1q2_, false);

  w30_ = Winding03(inQ, p2q0, s20, p2q1_, true);

  if (kVerbose) {
    std::cout << "Time for rest of first stage";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }
}

Manifold::Impl Boolean3::Result(Manifold::OpType op) const {
  if ((expandP_ > 0) != (op == Manifold::OpType::ADD))
    std::cout << "Warning! Result op type not compatible with constructor op "
                 "type: coplanar faces may have incorrect results."
              << std::endl;
  int c1, c2, c3;
  switch (op) {
    case Manifold::OpType::ADD:
      c1 = 1;
      c2 = 1;
      c3 = -1;
      if (kVerbose) std::cout << "ADD" << std::endl;
      break;
    case Manifold::OpType::SUBTRACT:
      c1 = 1;
      c2 = 0;
      c3 = -1;
      if (kVerbose) std::cout << "SUBTRACT" << std::endl;
      break;
    case Manifold::OpType::INTERSECT:
      c1 = 0;
      c2 = 0;
      c3 = 1;
      if (kVerbose) std::cout << "INTERSECT" << std::endl;
      break;
    default:
      throw std::invalid_argument("invalid enum: OpType.");
  }

  Time t0 = NOW();
  Time t1;

  // Convert winding numbers to inclusion values based on operation type.
  VecDH<int> i12(x12_.size());
  VecDH<int> i21(x21_.size());
  VecDH<int> i03(w03_.size());
  VecDH<int> i30(w30_.size());
  thrust::transform(x12_.beginD(), x12_.endD(), i12.beginD(), c3 * _1);
  thrust::transform(x21_.beginD(), x21_.endD(), i21.beginD(), c3 * _1);
  thrust::transform(w03_.beginD(), w03_.endD(), i03.beginD(), c1 + c3 * _1);
  thrust::transform(w30_.beginD(), w30_.endD(), i30.beginD(), c2 + c3 * _1);

  VecDH<int> vP2R(inP_.NumVert());
  thrust::exclusive_scan(i03.beginD(), i03.endD(), vP2R.beginD(), 0, AbsSum());
  int numVertR = AbsSum()(vP2R.H().back(), i03.H().back());
  const int nPv = numVertR;

  VecDH<int> vQ2R(inQ_.NumVert());
  thrust::exclusive_scan(i30.beginD(), i30.endD(), vQ2R.beginD(), numVertR,
                         AbsSum());
  numVertR = AbsSum()(vQ2R.H().back(), i30.H().back());
  const int nQv = numVertR - nPv;

  VecDH<int> v12R(v12_.size());
  if (v12_.size() > 0) {
    thrust::exclusive_scan(i12.beginD(), i12.endD(), v12R.beginD(), numVertR,
                           AbsSum());
    numVertR = AbsSum()(v12R.H().back(), i12.H().back());
  }
  const int n12 = numVertR - nPv - nQv;

  VecDH<int> v21R(v21_.size());
  if (v21_.size() > 0) {
    thrust::exclusive_scan(i21.beginD(), i21.endD(), v21R.beginD(), numVertR,
                           AbsSum());
    numVertR = AbsSum()(v21R.H().back(), i21.H().back());
  }
  const int n21 = numVertR - nPv - nQv - n12;

  // Create the output Manifold
  Manifold::Impl outR;

  if (numVertR == 0) return outR;

  outR.vertPos_.resize(numVertR);
  // Add vertices, duplicating for inclusion numbers not in [-1, 1].
  // Retained vertices from P and Q:
  thrust::for_each_n(zip(i03.beginD(), vP2R.beginD(), inP_.vertPos_.beginD()),
                     inP_.NumVert(), DuplicateVerts({outR.vertPos_.ptrD()}));
  thrust::for_each_n(zip(i30.beginD(), vQ2R.beginD(), inQ_.vertPos_.beginD()),
                     inQ_.NumVert(), DuplicateVerts({outR.vertPos_.ptrD()}));
  // New vertices created from intersections:
  thrust::for_each_n(zip(i12.beginD(), v12R.beginD(), v12_.beginD()),
                     i12.size(), DuplicateVerts({outR.vertPos_.ptrD()}));
  thrust::for_each_n(zip(i21.beginD(), v21R.beginD(), v21_.beginD()),
                     i21.size(), DuplicateVerts({outR.vertPos_.ptrD()}));

  if (kVerbose) {
    std::cout << nPv << " verts from inP" << std::endl;
    std::cout << nQv << " verts from inQ" << std::endl;
    std::cout << n12 << " new verts from edgesP -> facesQ" << std::endl;
    std::cout << n21 << " new verts from facesP -> edgesQ" << std::endl;
  }

  if (kVerbose) {
    std::cout << "Time for GPU part of result";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }

  // Build up new polygonal faces from triangle intersections. At this point the
  // calculation switches from parallel to serial.

  // Level 3

  // This key is the forward halfedge index of P or Q. Only includes intersected
  // edges.
  std::map<int, std::vector<EdgePos>> edgesP, edgesQ;
  // This key is the face index of <P, Q>
  std::map<std::pair<int, int>, std::vector<EdgePos>> edgesNew;

  AddNewEdgeVerts(edgesP, edgesNew, p1q2_, i12.H(), v12R.H(),
                  inP_.halfedge_.H(), true);
  AddNewEdgeVerts(edgesQ, edgesNew, p2q1_, i21.H(), v21R.H(),
                  inQ_.halfedge_.H(), false);

  // Level 4

  VecDH<int> facePQ2R = SizeOutput(outR, inP_, inQ_, i03, i30, i12, i21, p1q2_,
                                   p2q1_, op == Manifold::OpType::SUBTRACT);

  // This gets incremented for each halfedge that's added to a face so that the
  // next one knows where to slot in.
  VecDH<int> facePtrR = outR.faceEdge_;
  // Intersected halfedges are marked false.
  VecDH<bool> wholeHalfedgeP(inP_.halfedge_.size(), true);
  VecDH<bool> wholeHalfedgeQ(inQ_.halfedge_.size(), true);

  AppendPartialEdges(outR, wholeHalfedgeP.H(), facePtrR.H(), edgesP, inP_,
                     i03.H(), vP2R.H(), facePQ2R.begin());
  AppendPartialEdges(outR, wholeHalfedgeQ.H(), facePtrR.H(), edgesQ, inQ_,
                     i30.H(), vQ2R.H(), facePQ2R.begin() + inP_.NumFace());

  AppendNewEdges(outR, facePtrR.H(), edgesNew, facePQ2R.H(), inP_.NumFace());

  AppendWholeEdges(outR, facePtrR, inP_, wholeHalfedgeP, i03, vP2R,
                   facePQ2R.cptrD());
  AppendWholeEdges(outR, facePtrR, inQ_, wholeHalfedgeQ, i30, vQ2R,
                   facePQ2R.cptrD() + inP_.NumFace());

  if (kVerbose) {
    std::cout << "Time for CPU part of result";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }

  // Level 6

  // Create the manifold's data structures and verify manifoldness.
  outR.LabelVerts();
  outR.Finish();

  if (kVerbose) {
    std::cout << "Time for manifold finishing";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }

  return outR;
}

}  // namespace manifold