#include "hip/hip_runtime.h"
// Copyright 2019 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_math_constants.h>
#include <thrust/binary_search.h>
#include <thrust/count.h>
#include <thrust/gather.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/logical.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>
#include <thrust/unique.h>

#include <algorithm>
#include <map>

#include "boolean3.cuh"
#include "polygon.h"

/**
 * The notation in this file is abbreviated due to the complexity of the
 * functions involved. The key is that the input manifolds are P and Q, while
 * the output is R, and these letters in both upper and lower case refer to
 * these objects. Operations are based on dimensionality: vert: 0, edge: 1,
 * face: 2, solid: 3. X denotes a winding-number type quantity from the source
 * paper of this algorithm, while S is closely related but includes only the
 * subset of X values which "shadow" (are on the correct side of).
 *
 * Nearly everything here are sparse arrays, where for instance each pair in
 * p2q1 refers to a face index of P interacting with a halfedge index of Q.
 * Adjacent arrays like x21 refer to the values of X corresponding to each
 * sparse index pair.
 *
 * Note many functions are designed to work symmetrically, for instance for both
 * p2q1 and p1q2. Inside of these functions P and Q are marked as though the
 * funtion is forwards, but it may include a Boolean "reverse" that indicates P
 * and Q have been swapped.
 */

// TODO: make this runtime configurable for quicker debug
constexpr bool kVerbose = false;

using namespace thrust::placeholders;

namespace {
using namespace manifold;

// These two functions (Interpolate and Intersect) are the only places where
// floating-point operations take place in the whole Boolean function. These are
// carefully designed to minimize rounding error and to eliminate it at edge
// cases to ensure consistency.

__host__ __device__ glm::vec2 Interpolate(glm::vec3 pL, glm::vec3 pR, float x) {
  float dxL = x - pL.x;
  float dxR = x - pR.x;
  if (dxL * dxR > 0) printf("Not in domain!\n");
  bool useL = fabs(dxL) < fabs(dxR);
  float lambda = (useL ? dxL : dxR) / (pR.x - pL.x);
  if (!isfinite(lambda)) return glm::vec2(pL.y, pL.z);
  glm::vec2 yz;
  yz[0] = (useL ? pL.y : pR.y) + lambda * (pR.y - pL.y);
  yz[1] = (useL ? pL.z : pR.z) + lambda * (pR.z - pL.z);
  return yz;
}

__host__ __device__ glm::vec4 Intersect(const glm::vec3 &pL,
                                        const glm::vec3 &pR,
                                        const glm::vec3 &qL,
                                        const glm::vec3 &qR) {
  float dyL = qL.y - pL.y;
  float dyR = qR.y - pR.y;
  if (dyL * dyR > 0) printf("No intersection!\n");
  bool useL = fabs(dyL) < fabs(dyR);
  float dx = pR.x - pL.x;
  float lambda = (useL ? dyL : dyR) / (dyL - dyR);
  if (!isfinite(lambda)) lambda = 0.0f;
  glm::vec4 xyzz;
  xyzz.x = (useL ? pL.x : pR.x) + lambda * dx;
  float pDy = pR.y - pL.y;
  float qDy = qR.y - qL.y;
  bool useP = fabs(pDy) < fabs(qDy);
  xyzz.y = (useL ? (useP ? pL.y : qL.y) : (useP ? pR.y : qR.y)) +
           lambda * (useP ? pDy : qDy);
  xyzz.z = (useL ? pL.z : pR.z) + lambda * (pR.z - pL.z);
  xyzz.w = (useL ? qL.z : qR.z) + lambda * (qR.z - qL.z);
  return xyzz;
}

struct CopyFaceEdges {
  // x can be either vert or edge (0 or 1).
  thrust::pair<int *, int *> pXq1;
  const Halfedge *halfedgesQ;

  __host__ __device__ void operator()(thrust::tuple<int, int, int> in) {
    int idx = 3 * thrust::get<0>(in);
    const int pX = thrust::get<1>(in);
    const int q2 = thrust::get<2>(in);

    for (const int i : {0, 1, 2}) {
      pXq1.first[idx + i] = pX;
      const int q1 = 3 * q2 + i;
      const Halfedge edge = halfedgesQ[q1];
      pXq1.second[idx + i] = edge.IsForward() ? q1 : edge.pairedHalfedge;
    }
  }
};

SparseIndices Filter11(const Manifold::Impl &inP, const Manifold::Impl &inQ,
                       const SparseIndices &p1q2, const SparseIndices &p2q1) {
  SparseIndices p1q1(3 * p1q2.size() + 3 * p2q1.size());
  thrust::for_each_n(zip(countAt(0), p1q2.beginD(0), p1q2.beginD(1)),
                     p1q2.size(),
                     CopyFaceEdges({p1q1.ptrDpq(), inQ.halfedge_.cptrD()}));

  p1q1.SwapPQ();
  thrust::for_each_n(zip(countAt(p1q2.size()), p2q1.beginD(1), p2q1.beginD(0)),
                     p2q1.size(),
                     CopyFaceEdges({p1q1.ptrDpq(), inP.halfedge_.cptrD()}));
  p1q1.SwapPQ();
  p1q1.Unique();
  return p1q1;
}

struct AbsSum : public thrust::binary_function<int, int, int> {
  __host__ __device__ int operator()(int a, int b) { return abs(a) + abs(b); }
};

__host__ __device__ bool Shadows(float p, float q, float dir) {
  return p == q ? dir < 0 : p < q;
}

__host__ __device__ thrust::pair<int, glm::vec2> Shadow01(
    const int p0, const int q1, const glm::vec3 *vertPosP,
    const glm::vec3 *vertPosQ, const Halfedge *halfedgeQ, const float expandP,
    const glm::vec3 *normalP, const bool reverse) {
  const int q1s = halfedgeQ[q1].startVert;
  const int q1e = halfedgeQ[q1].endVert;
  const float p0x = vertPosP[p0].x;
  const float q1sx = vertPosQ[q1s].x;
  const float q1ex = vertPosQ[q1e].x;
  int s01 = reverse ? Shadows(q1sx, p0x, expandP * normalP[q1s].x) -
                          Shadows(q1ex, p0x, expandP * normalP[q1e].x)
                    : Shadows(p0x, q1ex, expandP * normalP[p0].x) -
                          Shadows(p0x, q1sx, expandP * normalP[p0].x);
  glm::vec2 yz01(0.0f / 0.0f);

  if (s01 != 0) {
    yz01 = Interpolate(vertPosQ[q1s], vertPosQ[q1e], vertPosP[p0].x);
    if (reverse) {
      glm::vec3 diff = vertPosQ[q1s] - vertPosP[p0];
      const float start2 = glm::dot(diff, diff);
      diff = vertPosQ[q1e] - vertPosP[p0];
      const float end2 = glm::dot(diff, diff);
      const float dir = start2 < end2 ? normalP[q1s].y : normalP[q1e].y;
      if (!Shadows(yz01[0], vertPosP[p0].y, expandP * dir)) s01 = 0;
    } else {
      if (!Shadows(vertPosP[p0].y, yz01[0], expandP * normalP[p0].y)) s01 = 0;
    }
  }
  return thrust::make_pair(s01, yz01);
}

__host__ __device__ int BinarySearch(
    const thrust::pair<const int *, const int *> keys, const int size,
    const thrust::pair<int, int> key) {
  if (size <= 0) return -1;
  int left = 0;
  int right = size - 1;
  int m;
  thrust::pair<int, int> keyM;
  while (1) {
    m = right - (right - left) / 2;
    keyM = thrust::make_pair(keys.first[m], keys.second[m]);
    if (left == right) break;
    if (keyM > key)
      right = m - 1;
    else
      left = m;
  }
  if (keyM == key)
    return m;
  else
    return -1;
}

struct Kernel11 {
  const glm::vec3 *vertPosP;
  const glm::vec3 *vertPosQ;
  const Halfedge *halfedgeP;
  const Halfedge *halfedgeQ;
  float expandP;
  const glm::vec3 *normalP;

  __host__ __device__ void operator()(
      thrust::tuple<glm::vec4 &, int &, int, int> inout) {
    glm::vec4 &xyzz11 = thrust::get<0>(inout);
    int &s11 = thrust::get<1>(inout);
    const int p1 = thrust::get<2>(inout);
    const int q1 = thrust::get<3>(inout);

    // For pRL[k], qRL[k], k==0 is the left and k==1 is the right.
    int k = 0;
    glm::vec3 pRL[2], qRL[2];
    // Either the left or right must shadow, but not both. This ensures the
    // intersection is between the left and right.
    bool shadows;
    s11 = 0;

    const int p0[2] = {halfedgeP[p1].startVert, halfedgeP[p1].endVert};
    for (int i : {0, 1}) {
      const auto syz01 = Shadow01(p0[i], q1, vertPosP, vertPosQ, halfedgeQ,
                                  expandP, normalP, false);
      const int s01 = syz01.first;
      const glm::vec2 yz01 = syz01.second;
      // If the value is NaN, then these do not overlap.
      if (isfinite(yz01[0])) {
        s11 += s01 * (i == 0 ? -1 : 1);
        if (k < 2 && (k == 0 || (s01 != 0) != shadows)) {
          shadows = s01 != 0;
          pRL[k] = vertPosP[p0[i]];
          qRL[k] = glm::vec3(pRL[k].x, yz01);
          ++k;
        }
      }
    }

    const int q0[2] = {halfedgeQ[q1].startVert, halfedgeQ[q1].endVert};
    for (int i : {0, 1}) {
      const auto syz10 = Shadow01(q0[i], p1, vertPosQ, vertPosP, halfedgeP,
                                  expandP, normalP, true);
      const int s10 = syz10.first;
      const glm::vec2 yz10 = syz10.second;
      // If the value is NaN, then these do not overlap.
      if (isfinite(yz10[0])) {
        s11 += s10 * (i == 0 ? -1 : 1);
        if (k < 2 && (k == 0 || (s10 != 0) != shadows)) {
          shadows = s10 != 0;
          qRL[k] = vertPosQ[q0[i]];
          pRL[k] = glm::vec3(qRL[k].x, yz10);
          ++k;
        }
      }
    }

    if (s11 == 0) {  // No intersection
      xyzz11 = glm::vec4(0.0f / 0.0f);
    } else {
      // Assert left and right were both found
      if (k != 2) {
        printf("k = %d\n", k);
      }

      xyzz11 = Intersect(pRL[0], pRL[1], qRL[0], qRL[1]);

      const int p1s = halfedgeP[p1].startVert;
      const int p1e = halfedgeP[p1].endVert;
      glm::vec3 diff = vertPosP[p1s] - glm::vec3(xyzz11);
      const float start2 = glm::dot(diff, diff);
      diff = vertPosP[p1e] - glm::vec3(xyzz11);
      const float end2 = glm::dot(diff, diff);
      const float dir = start2 < end2 ? normalP[p1s].z : normalP[p1e].z;

      if (!Shadows(xyzz11.z, xyzz11.w, expandP * dir)) s11 = 0;
    }
  }
};

std::tuple<VecDH<int>, VecDH<glm::vec4>> Shadow11(SparseIndices &p1q1,
                                                  const Manifold::Impl &inP,
                                                  const Manifold::Impl &inQ,
                                                  float expandP) {
  VecDH<int> s11(p1q1.size());
  VecDH<glm::vec4> xyzz11(p1q1.size());

  thrust::for_each_n(
      zip(xyzz11.beginD(), s11.beginD(), p1q1.beginD(0), p1q1.beginD(1)),
      p1q1.size(),
      Kernel11({inP.vertPos_.cptrD(), inQ.vertPos_.cptrD(),
                inP.halfedge_.cptrD(), inQ.halfedge_.cptrD(), expandP,
                inP.vertNormal_.cptrD()}));

  p1q1.KeepFinite(xyzz11, s11);

  return std::make_tuple(s11, xyzz11);
};

struct Kernel02 {
  const glm::vec3 *vertPosP;
  const Halfedge *halfedgeQ;
  const glm::vec3 *vertPosQ;
  const bool forward;
  const float expandP;
  const glm::vec3 *vertNormalP;
  const glm::vec3 *faceNormalP;

  __host__ __device__ void operator()(
      thrust::tuple<int &, float &, int, int> inout) {
    int &s02 = thrust::get<0>(inout);
    float &z02 = thrust::get<1>(inout);
    const int p0 = thrust::get<2>(inout);
    const int q2 = thrust::get<3>(inout);

    // For yzzLR[k], k==0 is the left and k==1 is the right.
    int k = 0;
    glm::vec3 yzzRL[2];
    // Either the left or right must shadow, but not both. This ensures the
    // intersection is between the left and right.
    bool shadows;
    int closestVert;
    float minMetric = 1.0f / 0.0f;
    s02 = 0;

    const glm::vec3 posP = vertPosP[p0];
    for (const int i : {0, 1, 2}) {
      const int q1 = 3 * q2 + i;
      const Halfedge edge = halfedgeQ[q1];
      const int q1F = edge.IsForward() ? q1 : edge.pairedHalfedge;

      if (!forward) {
        const int qVert = halfedgeQ[q1F].startVert;
        const glm::vec3 diff = posP - vertPosQ[qVert];
        const float metric = glm::dot(diff, diff);
        if (metric < minMetric) {
          minMetric = metric;
          closestVert = qVert;
        }
      }

      const auto syz01 = Shadow01(p0, q1F, vertPosP, vertPosQ, halfedgeQ,
                                  expandP, vertNormalP, !forward);
      const int s01 = syz01.first;
      const glm::vec2 yz01 = syz01.second;
      // If the value is NaN, then these do not overlap.
      if (isfinite(yz01[0])) {
        s02 += s01 * (forward == edge.IsForward() ? -1 : 1);
        if (k < 2 && (k == 0 || (s01 != 0) != shadows)) {
          shadows = s01 != 0;
          yzzRL[k++] = glm::vec3(yz01[0], yz01[1], yz01[1]);
        }
      }
    }

    if (s02 == 0) {  // No intersection
      z02 = 0.0f / 0.0f;
    } else {
      // Assert left and right were both found
      if (k != 2) {
        printf("k = %d\n", k);
      }

      glm::vec3 vertPos = vertPosP[p0];
      z02 = Interpolate(yzzRL[0], yzzRL[1], vertPos.y)[1];
      if (forward) {
        if (!Shadows(vertPos.z, z02, expandP * vertNormalP[p0].z)) s02 = 0;
      } else {
        if (!Shadows(z02, vertPos.z, expandP * vertNormalP[closestVert].z))
          s02 = 0;
      }
    }
  }
};

std::tuple<VecDH<int>, VecDH<float>> Shadow02(const Manifold::Impl &inP,
                                              const Manifold::Impl &inQ,
                                              SparseIndices &p0q2, bool forward,
                                              float expandP) {
  VecDH<int> s02(p0q2.size());
  VecDH<float> z02(p0q2.size());

  auto vertNormalP =
      forward ? inP.vertNormal_.cptrD() : inQ.vertNormal_.cptrD();
  auto faceNormalP =
      forward ? inP.faceNormal_.cptrD() : inQ.faceNormal_.cptrD();
  thrust::for_each_n(zip(s02.beginD(), z02.beginD(), p0q2.beginD(!forward),
                         p0q2.beginD(forward)),
                     p0q2.size(),
                     Kernel02({inP.vertPos_.cptrD(), inQ.halfedge_.cptrD(),
                               inQ.vertPos_.cptrD(), forward, expandP,
                               vertNormalP, faceNormalP}));

  p0q2.KeepFinite(z02, s02);

  return std::make_tuple(s02, z02);
};

struct Kernel12 {
  const thrust::pair<const int *, const int *> p0q2;
  const int *s02;
  const float *z02;
  const int size02;
  const thrust::pair<const int *, const int *> p1q1;
  const int *s11;
  const glm::vec4 *xyzz11;
  const int size11;
  const Halfedge *halfedgesP;
  const Halfedge *halfedgesQ;
  const glm::vec3 *vertPosP;
  const bool forward;

  __host__ __device__ void operator()(
      thrust::tuple<int &, glm::vec3 &, int, int> inout) {
    int &x12 = thrust::get<0>(inout);
    glm::vec3 &v12 = thrust::get<1>(inout);
    const int p1 = thrust::get<2>(inout);
    const int q2 = thrust::get<3>(inout);

    // For xzyLR-[k], k==0 is the left and k==1 is the right.
    int k = 0;
    glm::vec3 xzyLR0[2];
    glm::vec3 xzyLR1[2];
    // Either the left or right must shadow, but not both. This ensures the
    // intersection is between the left and right.
    bool shadows;
    x12 = 0;

    const Halfedge edge = halfedgesP[p1];

    for (int vert : {edge.startVert, edge.endVert}) {
      const auto key =
          forward ? thrust::make_pair(vert, q2) : thrust::make_pair(q2, vert);
      const int idx = BinarySearch(p0q2, size02, key);
      if (idx != -1) {
        const int s = s02[idx];
        x12 += s * ((vert == edge.startVert) == forward ? 1 : -1);
        if (k < 2 && (k == 0 || (s != 0) != shadows)) {
          shadows = s != 0;
          xzyLR0[k] = vertPosP[vert];
          thrust::swap(xzyLR0[k].y, xzyLR0[k].z);
          xzyLR1[k] = xzyLR0[k];
          xzyLR1[k][1] = z02[idx];
          k++;
        }
      }
    }

    for (const int i : {0, 1, 2}) {
      const int q1 = 3 * q2 + i;
      const Halfedge edge = halfedgesQ[q1];
      const int q1F = edge.IsForward() ? q1 : edge.pairedHalfedge;
      const auto key =
          forward ? thrust::make_pair(p1, q1F) : thrust::make_pair(q1F, p1);
      const int idx = BinarySearch(p1q1, size11, key);
      if (idx != -1) {  // s is implicitly zero for anything not found
        const int s = s11[idx];
        x12 -= s * (edge.IsForward() ? 1 : -1);
        if (k < 2 && (k == 0 || (s != 0) != shadows)) {
          shadows = s != 0;
          const glm::vec4 xyzz = xyzz11[idx];
          xzyLR0[k][0] = xyzz.x;
          xzyLR0[k][1] = xyzz.z;
          xzyLR0[k][2] = xyzz.y;
          xzyLR1[k] = xzyLR0[k];
          xzyLR1[k][1] = xyzz.w;
          if (!forward) thrust::swap(xzyLR0[k][1], xzyLR1[k][1]);
          k++;
        }
      }
    }

    if (x12 == 0) {  // No intersection
      v12 = glm::vec3(0.0f / 0.0f);
    } else {
      // Assert left and right were both found
      if (k != 2) {
        printf("k = %d\n", k);
      }
      const glm::vec4 xzyy =
          Intersect(xzyLR0[0], xzyLR0[1], xzyLR1[0], xzyLR1[1]);
      v12.x = xzyy[0];
      v12.y = xzyy[2];
      v12.z = xzyy[1];
    }
  }
};

std::tuple<VecDH<int>, VecDH<glm::vec3>> Intersect12(
    const Manifold::Impl &inP, const Manifold::Impl &inQ, const VecDH<int> &s02,
    const SparseIndices &p0q2, const VecDH<int> &s11, const SparseIndices &p1q1,
    const VecDH<float> &z02, const VecDH<glm::vec4> &xyzz11,
    SparseIndices &p1q2, bool forward) {
  VecDH<int> x12(p1q2.size());
  VecDH<glm::vec3> v12(p1q2.size());

  thrust::for_each_n(
      zip(x12.beginD(), v12.beginD(), p1q2.beginD(!forward),
          p1q2.beginD(forward)),
      p1q2.size(),
      Kernel12({p0q2.ptrDpq(), s02.ptrD(), z02.cptrD(), p0q2.size(),
                p1q1.ptrDpq(), s11.ptrD(), xyzz11.cptrD(), p1q1.size(),
                inP.halfedge_.cptrD(), inQ.halfedge_.cptrD(),
                inP.vertPos_.cptrD(), forward}));

  p1q2.KeepFinite(v12, x12);

  return std::make_tuple(x12, v12);
};

VecDH<int> Winding03(const Manifold::Impl &inP, SparseIndices &p0q2,
                     VecDH<int> &s02, const SparseIndices &p1q2, bool reverse) {
  // verts that are not shadowed (not in p0q2) have winding number zero.
  VecDH<int> w03(inP.NumVert(), 0);

  if (!thrust::is_sorted(p0q2.beginD(reverse), p0q2.endD(reverse)))
    thrust::sort_by_key(p0q2.beginD(reverse), p0q2.endD(reverse), s02.beginD());
  VecDH<int> w03val(w03.size());
  VecDH<int> w03vert(w03.size());
  // sum known s02 values into w03 (winding number)
  auto endPair =
      thrust::reduce_by_key(p0q2.beginD(reverse), p0q2.endD(reverse),
                            s02.beginD(), w03vert.beginD(), w03val.beginD());
  thrust::scatter(w03val.beginD(), endPair.second, w03vert.beginD(),
                  w03.beginD());

  if (reverse)
    thrust::transform(w03.beginD(), w03.endD(), w03.beginD(),
                      thrust::negate<int>());
  return w03;
};

struct DuplicateVerts {
  glm::vec3 *vertPosR;

  __host__ __device__ void operator()(thrust::tuple<int, int, glm::vec3> in) {
    int inclusion = abs(thrust::get<0>(in));
    int vertR = thrust::get<1>(in);
    glm::vec3 vertPosP = thrust::get<2>(in);

    for (int i = 0; i < inclusion; ++i) {
      vertPosR[vertR + i] = vertPosP;
    }
  }
};

__host__ __device__ int AtomicAddInt(int &target, int add) {
#ifdef __CUDA_ARCH__
  return atomicAdd(&target, add);
#else
  int out;
#pragma omp atomic capture
  {
    out = target;
    target += add;
  }
  return out;
#endif
}

struct CountVerts {
  int *count;
  const int *inclusion;

  __host__ __device__ void operator()(const Halfedge &edge) {
    AtomicAddInt(count[edge.face], glm::abs(inclusion[edge.startVert]));
  }
};

struct CountNewVerts {
  int *countP;
  int *countQ;
  const Halfedge *halfedges;

  __host__ __device__ void operator()(thrust::tuple<int, int, int> in) {
    int edgeP = thrust::get<0>(in);
    int faceQ = thrust::get<1>(in);
    int inclusion = glm::abs(thrust::get<2>(in));

    AtomicAddInt(countQ[faceQ], inclusion);
    const Halfedge half = halfedges[edgeP];
    AtomicAddInt(countP[half.face], inclusion);
    AtomicAddInt(countP[halfedges[half.pairedHalfedge].face], inclusion);
  }
};

struct NotZero : public thrust::unary_function<int, int> {
  __host__ __device__ int operator()(int x) const { return x > 0 ? 1 : 0; }
};

std::tuple<VecDH<int>, VecDH<int>> SizeOutput(
    Manifold::Impl &outR, const Manifold::Impl &inP, const Manifold::Impl &inQ,
    const VecDH<int> &i03, const VecDH<int> &i30, const VecDH<int> &i12,
    const VecDH<int> &i21, const SparseIndices &p1q2, const SparseIndices &p2q1,
    bool invertQ) {
  VecDH<int> sidesPerFacePQ(inP.NumTri() + inQ.NumTri());
  auto sidesPerFaceP = sidesPerFacePQ.ptrD();
  auto sidesPerFaceQ = sidesPerFacePQ.ptrD() + inP.NumTri();

  thrust::for_each(inP.halfedge_.beginD(), inP.halfedge_.endD(),
                   CountVerts({sidesPerFaceP, i03.cptrD()}));
  thrust::for_each(inQ.halfedge_.beginD(), inQ.halfedge_.endD(),
                   CountVerts({sidesPerFaceQ, i30.cptrD()}));
  thrust::for_each_n(
      zip(p1q2.beginD(0), p1q2.beginD(1), i12.beginD()), i12.size(),
      CountNewVerts({sidesPerFaceP, sidesPerFaceQ, inP.halfedge_.cptrD()}));
  thrust::for_each_n(
      zip(p2q1.beginD(1), p2q1.beginD(0), i21.beginD()), i21.size(),
      CountNewVerts({sidesPerFaceQ, sidesPerFaceP, inQ.halfedge_.cptrD()}));

  VecDH<int> facePQ2R(inP.NumTri() + inQ.NumTri() + 1);
  auto keepFace =
      thrust::make_transform_iterator(sidesPerFacePQ.beginD(), NotZero());
  thrust::inclusive_scan(keepFace, keepFace + sidesPerFacePQ.size(),
                         facePQ2R.beginD() + 1);
  int numFaceR = facePQ2R.H().back();
  facePQ2R.resize(inP.NumTri() + inQ.NumTri());

  outR.faceNormal_.resize(numFaceR);
  auto next = thrust::copy_if(inP.faceNormal_.beginD(), inP.faceNormal_.endD(),
                              keepFace, outR.faceNormal_.beginD(),
                              thrust::identity<bool>());
  if (invertQ) {
    auto start = thrust::make_transform_iterator(inQ.faceNormal_.beginD(),
                                                 thrust::negate<glm::vec3>());
    auto end = thrust::make_transform_iterator(inQ.faceNormal_.endD(),
                                               thrust::negate<glm::vec3>());
    thrust::copy_if(start, end, keepFace + inP.NumTri(), next,
                    thrust::identity<bool>());
  } else {
    thrust::copy_if(inQ.faceNormal_.beginD(), inQ.faceNormal_.endD(),
                    keepFace + inP.NumTri(), next, thrust::identity<bool>());
  }

  auto newEnd =
      thrust::remove(sidesPerFacePQ.beginD(), sidesPerFacePQ.endD(), 0);
  VecDH<int> faceEdge(newEnd - sidesPerFacePQ.beginD() + 1);
  thrust::inclusive_scan(sidesPerFacePQ.beginD(), newEnd,
                         faceEdge.beginD() + 1);
  outR.halfedge_.resize(faceEdge.H().back());

  return std::make_tuple(faceEdge, facePQ2R);
}

struct DuplicateHalfedges {
  Halfedge *halfedgesR;
  int *facePtr;
  const Halfedge *halfedgesP;
  const int *i03;
  const int *vP2R;
  const int *faceP2R;

  __host__ __device__ void operator()(thrust::tuple<bool, Halfedge> in) {
    if (!thrust::get<0>(in)) return;
    Halfedge halfedge = thrust::get<1>(in);
    if (!halfedge.IsForward()) return;

    const int inclusion = i03[halfedge.startVert];
    if (inclusion == 0) return;
    if (inclusion < 0) {  // reverse
      int tmp = halfedge.startVert;
      halfedge.startVert = halfedge.endVert;
      halfedge.endVert = tmp;
    }
    halfedge.startVert = vP2R[halfedge.startVert];
    halfedge.endVert = vP2R[halfedge.endVert];
    halfedge.face = faceP2R[halfedge.face];
    int faceRight = faceP2R[halfedgesP[halfedge.pairedHalfedge].face];

    for (int i = 0; i < glm::abs(inclusion); ++i) {
      int forwardIdx = AtomicAddInt(facePtr[halfedge.face], 1);
      int backwardIdx = AtomicAddInt(facePtr[faceRight], 1);
      halfedge.pairedHalfedge = backwardIdx;

      halfedgesR[forwardIdx] = halfedge;
      halfedgesR[backwardIdx] = {halfedge.endVert, halfedge.startVert,
                                 forwardIdx, faceRight};

      ++halfedge.startVert;
      ++halfedge.endVert;
    }
  }
};

struct EdgePos {
  int vert;
  float edgePos;
  bool isStart;
};

void AddNewEdgeVerts(
    std::map<int, std::vector<EdgePos>> &edgesP,
    std::map<std::pair<int, int>, std::vector<EdgePos>> &edgesNew,
    const SparseIndices &p1q2, const VecH<int> &i12, const VecH<int> &v12R,
    const VecH<Halfedge> &halfedgeP, bool forward) {
  // For each edge of P that intersects a face of Q (p1q2), add this vertex to
  // P's corresponding edge vector and to the two new edges, which are
  // intersections between the face of Q and the two faces of P attached to the
  // edge. The direction and duplicity are given by i12, while v12R remaps to
  // the output vert index. When forward is false, all is reversed.
  const VecH<int> &p1 = p1q2.Get(!forward).H();
  const VecH<int> &q2 = p1q2.Get(forward).H();
  for (int i = 0; i < p1q2.size(); ++i) {
    const int edgeP = p1[i];
    const int faceQ = q2[i];
    const int vert = v12R[i];
    const int inclusion = i12[i];

    auto &edgePosP = edgesP[edgeP];

    Halfedge halfedge = halfedgeP[edgeP];
    std::pair<int, int> key = {halfedgeP[halfedge.pairedHalfedge].face, faceQ};
    if (!forward) std::swap(key.first, key.second);
    auto &edgePosRight = edgesNew[key];

    key = {halfedge.face, faceQ};
    if (!forward) std::swap(key.first, key.second);
    auto &edgePosLeft = edgesNew[key];

    EdgePos edgePos = {vert, 0.0f, inclusion < 0};
    EdgePos edgePosRev = edgePos;
    edgePosRev.isStart = !edgePos.isStart;

    for (int j = 0; j < glm::abs(inclusion); ++j) {
      edgePosP.push_back(edgePos);
      edgePosRight.push_back(forward ? edgePos : edgePosRev);
      edgePosLeft.push_back(forward ? edgePosRev : edgePos);
      ++edgePos.vert;
      ++edgePosRev.vert;
    }
  }
}

std::vector<Halfedge> PairUp(std::vector<EdgePos> &edgePos) {
  // Pair start vertices with end vertices to form edges. The choice of pairing
  // is arbitrary for the manifoldness guarantee, but must be ordered to be
  // geometrically valid. If the order does not go start-end-start-end... then
  // the input and output are not geometrically valid and this algorithm becomes
  // a heuristic.
  ALWAYS_ASSERT(edgePos.size() % 2 == 0, topologyErr,
                "Non-manifold edge! Not an even number of points.");
  int nEdges = edgePos.size() / 2;
  auto middle = std::partition(edgePos.begin(), edgePos.end(),
                               [](EdgePos x) { return x.isStart; });
  ALWAYS_ASSERT(middle - edgePos.begin() == nEdges, topologyErr,
                "Non-manifold edge!");
  auto cmp = [](EdgePos a, EdgePos b) { return a.edgePos < b.edgePos; };
  std::sort(edgePos.begin(), middle, cmp);
  std::sort(middle, edgePos.end(), cmp);
  std::vector<Halfedge> edges;
  for (int i = 0; i < nEdges; ++i)
    edges.push_back({edgePos[i].vert, edgePos[i + nEdges].vert, -1, -1});
  return edges;
}

void AppendPartialEdges(
    Manifold::Impl &outR, VecH<bool> &wholeHalfedgeP, VecH<int> &facePtrR,
    std::map<int, std::vector<EdgePos>> &edgesP, const Manifold::Impl &inP,
    const VecH<int> &i03, const VecH<int> &vP2R,
    const thrust::host_vector<int>::const_iterator faceP2R) {
  // Each edge in the map is partially retained; for each of these, look up
  // their original verts and include them based on their winding number (i03),
  // while remaping them to the output using vP2R. Use the verts position
  // projected along the edge vector to pair them up, then distribute these
  // edges to their faces.
  VecH<Halfedge> &halfedgeR = outR.halfedge_.H();
  const VecH<glm::vec3> &vertPosP = inP.vertPos_.H();
  const VecH<Halfedge> &halfedgeP = inP.halfedge_.H();

  for (auto &value : edgesP) {
    const int edgeP = value.first;
    std::vector<EdgePos> &edgePosP = value.second;

    const Halfedge &halfedge = halfedgeP[edgeP];
    wholeHalfedgeP[edgeP] = false;
    wholeHalfedgeP[halfedge.pairedHalfedge] = false;

    const int vStart = halfedge.startVert;
    const int vEnd = halfedge.endVert;
    const glm::vec3 edgeVec = vertPosP[vEnd] - vertPosP[vStart];
    // Fill in the edge positions of the old points.
    for (EdgePos &edge : edgePosP) {
      edge.edgePos = glm::dot(outR.vertPos_.H()[edge.vert], edgeVec);
    }

    int inclusion = i03[vStart];
    EdgePos edgePos = {vP2R[vStart],
                       glm::dot(outR.vertPos_.H()[vP2R[vStart]], edgeVec),
                       inclusion > 0};
    for (int j = 0; j < glm::abs(inclusion); ++j) {
      edgePosP.push_back(edgePos);
      ++edgePos.vert;
    }

    inclusion = i03[vEnd];
    edgePos = {vP2R[vEnd], glm::dot(outR.vertPos_.H()[vP2R[vEnd]], edgeVec),
               inclusion < 0};
    for (int j = 0; j < glm::abs(inclusion); ++j) {
      edgePosP.push_back(edgePos);
      ++edgePos.vert;
    }

    // sort edges into start/end pairs along length
    std::vector<Halfedge> edges = PairUp(edgePosP);

    // add halfedges to result
    const int faceLeft = faceP2R[halfedge.face];
    const int faceRight = faceP2R[halfedgeP[halfedge.pairedHalfedge].face];
    for (Halfedge e : edges) {
      const int forwardEdge = facePtrR[faceLeft]++;
      const int backwardEdge = facePtrR[faceRight]++;

      e.face = faceLeft;
      e.pairedHalfedge = backwardEdge;
      halfedgeR[forwardEdge] = e;

      std::swap(e.startVert, e.endVert);
      e.face = faceRight;
      e.pairedHalfedge = forwardEdge;
      halfedgeR[backwardEdge] = e;
    }
  }
}

void AppendNewEdges(
    Manifold::Impl &outR, VecH<int> &facePtrR,
    std::map<std::pair<int, int>, std::vector<EdgePos>> &edgesNew,
    const VecH<int> &facePQ2R, const int numFaceP) {
  // Pair up each edge's verts and distribute to faces based on indices in key.
  VecH<Halfedge> &halfedgeR = outR.halfedge_.H();
  VecH<glm::vec3> &vertPosR = outR.vertPos_.H();

  for (auto &value : edgesNew) {
    const int faceP = value.first.first;
    const int faceQ = value.first.second;
    std::vector<EdgePos> &edgePos = value.second;

    Box bbox;
    for (auto edge : edgePos) {
      bbox.Union(vertPosR[edge.vert]);
    }
    const glm::vec3 size = bbox.Size();
    // Order the points along their longest dimension.
    const int i = (size.x > size.y && size.x > size.z) ? 0
                  : size.y > size.z                    ? 1
                                                       : 2;
    for (auto &edge : edgePos) {
      edge.edgePos = vertPosR[edge.vert][i];
    }

    // sort edges into start/end pairs along length.
    std::vector<Halfedge> edges = PairUp(edgePos);

    // add halfedges to result
    const int faceLeft = facePQ2R[faceP];
    const int faceRight = facePQ2R[numFaceP + faceQ];
    for (Halfedge e : edges) {
      const int forwardEdge = facePtrR[faceLeft]++;
      const int backwardEdge = facePtrR[faceRight]++;

      e.face = faceLeft;
      e.pairedHalfedge = backwardEdge;
      halfedgeR[forwardEdge] = e;

      std::swap(e.startVert, e.endVert);
      e.face = faceRight;
      e.pairedHalfedge = forwardEdge;
      halfedgeR[backwardEdge] = e;
    }
  }
}

void AppendWholeEdges(Manifold::Impl &outR, VecDH<int> &facePtrR,
                      const Manifold::Impl &inP, VecDH<bool> wholeHalfedgeP,
                      const VecDH<int> &i03, const VecDH<int> &vP2R,
                      const int *faceP2R) {
  thrust::for_each_n(zip(wholeHalfedgeP.beginD(), inP.halfedge_.beginD()),
                     inP.halfedge_.size(),
                     DuplicateHalfedges({outR.halfedge_.ptrD(), facePtrR.ptrD(),
                                         inP.halfedge_.cptrD(), i03.cptrD(),
                                         vP2R.cptrD(), faceP2R}));
}
}  // namespace

namespace manifold {
Boolean3::Boolean3(const Manifold::Impl &inP, const Manifold::Impl &inQ,
                   Manifold::OpType op)
    : inP_(inP), inQ_(inQ), expandP_(op == Manifold::OpType::ADD ? 1.0 : -1.0) {
  // Symbolic perturbation:
  // Union -> expand inP
  // Difference, Intersection -> contract inP

  Timer filter;
  filter.Start();

  if (inP.IsEmpty() || inQ.IsEmpty() || !inP.bBox_.DoesOverlap(inQ.bBox_)) {
    if (kVerbose) std::cout << "No overlap, early out" << std::endl;
    w03_.resize(inP.NumVert(), 0);
    w30_.resize(inQ.NumVert(), 0);
    return;
  }

  // Level 3
  // Find edge-triangle overlaps (broad phase)
  p1q2_ = inQ_.EdgeCollisions(inP_);
  p1q2_.Sort();
  if (kVerbose) std::cout << "p1q2 size = " << p1q2_.size() << std::endl;

  p2q1_ = inP_.EdgeCollisions(inQ_);
  p2q1_.SwapPQ();
  p2q1_.Sort();
  if (kVerbose) std::cout << "p2q1 size = " << p2q1_.size() << std::endl;

  // Level 2
  // Find vertices that overlap faces in XY-projection
  SparseIndices p0q2 = inQ.VertexCollisionsZ(inP.vertPos_);
  p0q2.Sort();
  if (kVerbose) std::cout << "p0q2 size = " << p0q2.size() << std::endl;

  SparseIndices p2q0 = inP.VertexCollisionsZ(inQ.vertPos_);
  p2q0.SwapPQ();
  p2q0.Sort();
  if (kVerbose) std::cout << "p2q0 size = " << p2q0.size() << std::endl;

  // Find involved edge pairs from Level 3
  SparseIndices p1q1 = Filter11(inP_, inQ_, p1q2_, p2q1_);
  if (kVerbose) std::cout << "p1q1 size = " << p1q1.size() << std::endl;

  filter.Stop();
  Timer levels;
  levels.Start();

  // Level 2
  // Build up XY-projection intersection of two edges, including the z-value for
  // each edge, keeping only those whose intersection exists.
  VecDH<int> s11;
  VecDH<glm::vec4> xyzz11;
  std::tie(s11, xyzz11) = Shadow11(p1q1, inP, inQ, expandP_);
  if (kVerbose) std::cout << "s11 size = " << s11.size() << std::endl;

  // Build up Z-projection of vertices onto triangles, keeping only those that
  // fall inside the triangle.
  VecDH<int> s02;
  VecDH<float> z02;
  std::tie(s02, z02) = Shadow02(inP, inQ, p0q2, true, expandP_);
  if (kVerbose) std::cout << "s02 size = " << s02.size() << std::endl;

  VecDH<int> s20;
  VecDH<float> z20;
  std::tie(s20, z20) = Shadow02(inQ, inP, p2q0, false, expandP_);
  if (kVerbose) std::cout << "s20 size = " << s20.size() << std::endl;

  // Level 3
  // Build up the intersection of the edges and triangles, keeping only those
  // that intersect, and record the direction the edge is passing through the
  // triangle.
  std::tie(x12_, v12_) =
      Intersect12(inP, inQ, s02, p0q2, s11, p1q1, z02, xyzz11, p1q2_, true);
  if (kVerbose) std::cout << "x12 size = " << x12_.size() << std::endl;

  std::tie(x21_, v21_) =
      Intersect12(inQ, inP, s20, p2q0, s11, p1q1, z20, xyzz11, p2q1_, false);
  if (kVerbose) std::cout << "x21 size = " << x21_.size() << std::endl;

  // Sum up the winding numbers of all vertices.
  w03_ = Winding03(inP, p0q2, s02, p1q2_, false);

  w30_ = Winding03(inQ, p2q0, s20, p2q1_, true);

  levels.Stop();

  if (kVerbose) {
    filter.Print("Filter");
    levels.Print("Levels 1-3");
    MemUsage();
  }
}

Manifold::Impl Boolean3::Result(Manifold::OpType op) const {
  Timer assemble;
  assemble.Start();

  if ((expandP_ > 0) != (op == Manifold::OpType::ADD))
    std::cout << "Warning! Result op type not compatible with constructor op "
                 "type: coplanar faces may have incorrect results."
              << std::endl;
  int c1, c2, c3;
  switch (op) {
    case Manifold::OpType::ADD:
      c1 = 1;
      c2 = 1;
      c3 = -1;
      if (kVerbose) std::cout << "ADD" << std::endl;
      break;
    case Manifold::OpType::SUBTRACT:
      c1 = 1;
      c2 = 0;
      c3 = -1;
      if (kVerbose) std::cout << "SUBTRACT" << std::endl;
      break;
    case Manifold::OpType::INTERSECT:
      c1 = 0;
      c2 = 0;
      c3 = 1;
      if (kVerbose) std::cout << "INTERSECT" << std::endl;
      break;
    default:
      throw std::invalid_argument("invalid enum: OpType.");
  }

  if (w03_.size() == 0) {
    if (w30_.size() != 0 && op == Manifold::OpType::ADD) {
      return inQ_;
    }
    return Manifold::Impl();
  } else if (w30_.size() == 0) {
    if (op == Manifold::OpType::INTERSECT) {
      return Manifold::Impl();
    }
    return inP_;
  }

  // Convert winding numbers to inclusion values based on operation type.
  VecDH<int> i12(x12_.size());
  VecDH<int> i21(x21_.size());
  VecDH<int> i03(w03_.size());
  VecDH<int> i30(w30_.size());
  thrust::transform(x12_.beginD(), x12_.endD(), i12.beginD(), c3 * _1);
  thrust::transform(x21_.beginD(), x21_.endD(), i21.beginD(), c3 * _1);
  thrust::transform(w03_.beginD(), w03_.endD(), i03.beginD(), c1 + c3 * _1);
  thrust::transform(w30_.beginD(), w30_.endD(), i30.beginD(), c2 + c3 * _1);

  VecDH<int> vP2R(inP_.NumVert());
  thrust::exclusive_scan(i03.beginD(), i03.endD(), vP2R.beginD(), 0, AbsSum());
  int numVertR = AbsSum()(vP2R.H().back(), i03.H().back());
  const int nPv = numVertR;

  VecDH<int> vQ2R(inQ_.NumVert());
  thrust::exclusive_scan(i30.beginD(), i30.endD(), vQ2R.beginD(), numVertR,
                         AbsSum());
  numVertR = AbsSum()(vQ2R.H().back(), i30.H().back());
  const int nQv = numVertR - nPv;

  VecDH<int> v12R(v12_.size());
  if (v12_.size() > 0) {
    thrust::exclusive_scan(i12.beginD(), i12.endD(), v12R.beginD(), numVertR,
                           AbsSum());
    numVertR = AbsSum()(v12R.H().back(), i12.H().back());
  }
  const int n12 = numVertR - nPv - nQv;

  VecDH<int> v21R(v21_.size());
  if (v21_.size() > 0) {
    thrust::exclusive_scan(i21.beginD(), i21.endD(), v21R.beginD(), numVertR,
                           AbsSum());
    numVertR = AbsSum()(v21R.H().back(), i21.H().back());
  }
  const int n21 = numVertR - nPv - nQv - n12;

  // Create the output Manifold
  Manifold::Impl outR;

  if (numVertR == 0) return outR;

  outR.vertPos_.resize(numVertR);
  // Add vertices, duplicating for inclusion numbers not in [-1, 1].
  // Retained vertices from P and Q:
  thrust::for_each_n(zip(i03.beginD(), vP2R.beginD(), inP_.vertPos_.beginD()),
                     inP_.NumVert(), DuplicateVerts({outR.vertPos_.ptrD()}));
  thrust::for_each_n(zip(i30.beginD(), vQ2R.beginD(), inQ_.vertPos_.beginD()),
                     inQ_.NumVert(), DuplicateVerts({outR.vertPos_.ptrD()}));
  // New vertices created from intersections:
  thrust::for_each_n(zip(i12.beginD(), v12R.beginD(), v12_.beginD()),
                     i12.size(), DuplicateVerts({outR.vertPos_.ptrD()}));
  thrust::for_each_n(zip(i21.beginD(), v21R.beginD(), v21_.beginD()),
                     i21.size(), DuplicateVerts({outR.vertPos_.ptrD()}));

  if (kVerbose) {
    std::cout << nPv << " verts from inP" << std::endl;
    std::cout << nQv << " verts from inQ" << std::endl;
    std::cout << n12 << " new verts from edgesP -> facesQ" << std::endl;
    std::cout << n21 << " new verts from facesP -> edgesQ" << std::endl;
  }

  // Build up new polygonal faces from triangle intersections. At this point the
  // calculation switches from parallel to serial.

  // Level 3

  // This key is the forward halfedge index of P or Q. Only includes intersected
  // edges.
  std::map<int, std::vector<EdgePos>> edgesP, edgesQ;
  // This key is the face index of <P, Q>
  std::map<std::pair<int, int>, std::vector<EdgePos>> edgesNew;

  AddNewEdgeVerts(edgesP, edgesNew, p1q2_, i12.H(), v12R.H(),
                  inP_.halfedge_.H(), true);
  AddNewEdgeVerts(edgesQ, edgesNew, p2q1_, i21.H(), v21R.H(),
                  inQ_.halfedge_.H(), false);

  // Level 4
  VecDH<int> faceEdge;
  VecDH<int> facePQ2R;
  std::tie(faceEdge, facePQ2R) =
      SizeOutput(outR, inP_, inQ_, i03, i30, i12, i21, p1q2_, p2q1_,
                 op == Manifold::OpType::SUBTRACT);

  // This gets incremented for each halfedge that's added to a face so that the
  // next one knows where to slot in.
  VecDH<int> facePtrR = faceEdge;
  // Intersected halfedges are marked false.
  VecDH<bool> wholeHalfedgeP(inP_.halfedge_.size(), true);
  VecDH<bool> wholeHalfedgeQ(inQ_.halfedge_.size(), true);

  AppendPartialEdges(outR, wholeHalfedgeP.H(), facePtrR.H(), edgesP, inP_,
                     i03.H(), vP2R.H(), facePQ2R.begin());
  AppendPartialEdges(outR, wholeHalfedgeQ.H(), facePtrR.H(), edgesQ, inQ_,
                     i30.H(), vQ2R.H(), facePQ2R.begin() + inP_.NumTri());

  AppendNewEdges(outR, facePtrR.H(), edgesNew, facePQ2R.H(), inP_.NumTri());

  AppendWholeEdges(outR, facePtrR, inP_, wholeHalfedgeP, i03, vP2R,
                   facePQ2R.cptrD());
  AppendWholeEdges(outR, facePtrR, inQ_, wholeHalfedgeQ, i30, vQ2R,
                   facePQ2R.cptrD() + inP_.NumTri());

  assemble.Stop();
  Timer triangulate;
  triangulate.Start();

  // Level 6

  // Create the manifold's data structures.
  outR.precision_ = glm::max(inP_.precision_, inQ_.precision_);

  outR.Face2Tri(faceEdge);

  // int chi = outR.NumVert() - outR.NumEdge() + outR.NumTri();
  // std::cout << "triangle Genus = " << 1 - chi / 2 << std::endl;

  // outR.SplitNonmanifoldVerts();

  outR.CollapseDegenerates();

  triangulate.Stop();
  Timer finish;
  finish.Start();

  outR.Finish();

  finish.Stop();
  if (kVerbose) {
    assemble.Print("Assembly");
    triangulate.Print("Triangulation");
    finish.Print("Finishing the manifold");
  }

  return outR;
}

}  // namespace manifold