#include "hip/hip_runtime.h"
// Copyright 2019 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "boolean3.cuh"
#include "connected_components.cuh"
#include "polygon.h"

#include <hip/hip_math_constants.h>
#include <thrust/adjacent_difference.h>
#include <thrust/binary_search.h>
#include <thrust/count.h>
#include <thrust/gather.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/logical.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>
#include <thrust/unique.h>
#include <algorithm>
#include <map>

constexpr bool kVerbose = false;

using namespace thrust::placeholders;

namespace {
using namespace manifold;

// These two functions (Interpolate and Intersect) are the only places where
// floating-point operations take place in the whole Boolean function. These are
// carefully designed to minimize rounding error and to eliminate it at edge
// cases to ensure consistency.

__host__ __device__ glm::vec2 Interpolate(glm::vec3 pL, glm::vec3 pR, float x) {
  float dxL = x - pL.x;
  float dxR = x - pR.x;
  bool useL = fabs(dxL) < fabs(dxR);
  float lambda = (useL ? dxL : dxR) / (pR.x - pL.x);
  if (isnan(lambda)) return glm::vec2(pL.y, pL.z);
  glm::vec2 yz;
  yz[0] = (useL ? pL.y : pR.y) + lambda * (pR.y - pL.y);
  yz[1] = (useL ? pL.z : pR.z) + lambda * (pR.z - pL.z);
  return yz;
}

__host__ __device__ glm::vec4 Intersect(const glm::vec3 &pL,
                                        const glm::vec3 &pR,
                                        const glm::vec3 &qL,
                                        const glm::vec3 &qR) {
  float dyL = qL.y - pL.y;
  float dyR = qR.y - pR.y;
  bool useL = fabs(dyL) < fabs(dyR);
  float dx = pR.x - pL.x;
  float lambda = (useL ? dyL : dyR) / (dyL - dyR);
  if (isnan(lambda)) lambda = 0.0f;
  glm::vec4 xyzz;
  xyzz.x = (useL ? pL.x : pR.x) + lambda * dx;
  float pDy = pR.y - pL.y;
  float qDy = qR.y - qL.y;
  bool useP = fabs(pDy) < fabs(qDy);
  xyzz.y = (useL ? (useP ? pL.y : qL.y) : (useP ? pR.y : qR.y)) +
           lambda * (useP ? pDy : qDy);
  xyzz.z = (useL ? pL.z : pR.z) + lambda * (pR.z - pL.z);
  xyzz.w = (useL ? qL.z : qR.z) + lambda * (qR.z - qL.z);
  return xyzz;
}

struct MarkEdgeVerts {
  int *verts;
  const Halfedge *halfedges;

  __host__ __device__ void operator()(int edge) {
    int vert = halfedges[edge].startVert;
    verts[vert] = vert;
    vert = halfedges[edge].endVert;
    verts[vert] = vert;
  }
};

struct MarkFaceVerts {
  int *verts;
  const Halfedge *halfedges;
  const int *faces;

  __host__ __device__ void operator()(int face) {
    int edge = faces[face];
    const int lastEdge = faces[face + 1];
    while (edge < lastEdge) {
      int vert = halfedges[edge++].startVert;
      verts[vert] = vert;
    }
  }
};

SparseIndices Filter02(const Manifold::Impl &inP, const Manifold::Impl &inQ,
                       const VecDH<int> &edges, const VecDH<int> &faces) {
  // find inP's involved vertices from edges & faces
  VecDH<int> p0(inP.NumVert(), -1);
  // We keep the verts unique by marking the ones we want to keep
  // with their own index in parallel (collisions don't matter because any given
  // element is always being written with the same value). Any that are still
  // initialized to -1 are not involved and can be removed.
  thrust::for_each_n(edges.beginD(), edges.size(),
                     MarkEdgeVerts({p0.ptrD(), inP.halfedge_.cptrD()}));

  thrust::for_each_n(
      faces.beginD(), faces.size(),
      MarkFaceVerts({p0.ptrD(), inP.halfedge_.cptrD(), inP.face_.cptrD()}));

  // find one vertex from each connected component of inP (in case it has no
  // intersections)
  VecDH<int> compVerts(inP.numLabel_);
  for (int i = 0; i < inP.numLabel_; ++i) {
    compVerts.H()[i] =
        thrust::find(inP.vertLabel_.beginD(), inP.vertLabel_.endD(), i) -
        inP.vertLabel_.beginD();
  }
  thrust::scatter(compVerts.beginD(), compVerts.endD(), compVerts.beginD(),
                  p0.beginD());

  p0.resize(thrust::remove(p0.beginD(), p0.endD(), -1) - p0.beginD());
  // find which inQ faces shadow these vertices
  VecDH<glm::vec3> vertPosP(p0.size());
  thrust::gather(p0.beginD(), p0.endD(), inP.vertPos_.cbeginD(),
                 vertPosP.beginD());
  SparseIndices p0q2 = inQ.VertexCollisionsZ(vertPosP);
  VecDH<int> i02temp(p0q2.size());
  thrust::copy(p0q2.beginD(0), p0q2.endD(0), i02temp.beginD());
  thrust::gather(i02temp.beginD(), i02temp.endD(), p0.beginD(), p0q2.beginD(0));
  return p0q2;
}

struct CopyFaceEdges {
  // x can be either vert or edge (0 or 1).
  thrust::pair<int *, int *> pXq1;
  const int *facesQ;
  const Halfedge *halfedgesQ;

  __host__ __device__ void operator()(thrust::tuple<int, int, int> in) {
    int idx = thrust::get<0>(in);
    const int pX = thrust::get<1>(in);
    const int q2 = thrust::get<2>(in);

    int q1 = facesQ[q2];
    const int end = facesQ[q2 + 1];
    while (q1 < end) {
      pXq1.first[idx] = pX;
      const Halfedge edge = halfedgesQ[q1];
      pXq1.second[idx++] = edge.IsForward() ? q1 : edge.pairedHalfedge;
      ++q1;
    }
  }
};

SparseIndices Filter11(const Manifold::Impl &inP, const VecDH<int> &faceSizeP,
                       const Manifold::Impl &inQ, const VecDH<int> &faceSizeQ,
                       const SparseIndices &p1q2, const SparseIndices &p2q1) {
  VecDH<int> expandedIdxQ(p1q2.size() + 1);
  auto includedFaceSizeQ = perm(faceSizeQ.beginD(), p1q2.beginD(1));
  thrust::inclusive_scan(includedFaceSizeQ, includedFaceSizeQ + p1q2.size(),
                         expandedIdxQ.beginD() + 1);
  const int secondStart = expandedIdxQ.H().back();

  VecDH<int> expandedIdxP(p2q1.size() + 1);
  auto includedFaceSizeP = perm(faceSizeP.beginD(), p2q1.beginD(0));
  thrust::inclusive_scan(includedFaceSizeP, includedFaceSizeP + p2q1.size(),
                         expandedIdxP.beginD() + 1);

  SparseIndices p1q1(secondStart + expandedIdxP.H().back());
  thrust::for_each_n(
      zip(expandedIdxQ.beginD(), p1q2.beginD(0), p1q2.beginD(1)), p1q2.size(),
      CopyFaceEdges({p1q1.ptrDpq(), inQ.face_.cptrD(), inQ.halfedge_.cptrD()}));

  p1q1.SwapPQ();
  thrust::for_each_n(zip(expandedIdxP.beginD(), p2q1.beginD(1), p2q1.beginD(0)),
                     p2q1.size(),
                     CopyFaceEdges({p1q1.ptrDpq(secondStart), inP.face_.cptrD(),
                                    inP.halfedge_.cptrD()}));
  p1q1.SwapPQ();
  p1q1.Unique();
  return p1q1;
}

struct CopyEdgeVerts {
  thrust::pair<int *, int *> p0q1;
  const Halfedge *halfedges;

  __host__ __device__ void operator()(thrust::tuple<int, int, int> in) {
    int idx = 2 * thrust::get<0>(in);
    const int p1 = thrust::get<1>(in);
    const int q1 = thrust::get<2>(in);

    p0q1.first[idx] = halfedges[p1].startVert;
    p0q1.second[idx] = q1;
    p0q1.first[idx + 1] = halfedges[p1].endVert;
    p0q1.second[idx + 1] = q1;
  }
};

SparseIndices Filter01(const Manifold::Impl &inP, const Manifold::Impl &inQ,
                       const VecDH<int> &faceSizeQ, const SparseIndices &p0q2,
                       const SparseIndices &p1q1) {
  VecDH<int> expandedIdxQ(p0q2.size() + 1);
  auto includedFaceSizeQ = perm(faceSizeQ.beginD(), p0q2.beginD(1));
  thrust::inclusive_scan(includedFaceSizeQ, includedFaceSizeQ + p0q2.size(),
                         expandedIdxQ.beginD() + 1);
  const int secondStart = expandedIdxQ.H().back();

  SparseIndices p0q1(secondStart + 2 * p1q1.size());

  thrust::for_each_n(
      zip(expandedIdxQ.beginD(), p0q2.beginD(0), p0q2.beginD(1)), p0q2.size(),
      CopyFaceEdges({p0q1.ptrDpq(), inQ.face_.cptrD(), inQ.halfedge_.cptrD()}));

  thrust::for_each_n(
      zip(thrust::make_counting_iterator(0), p1q1.beginD(0), p1q1.beginD(1)),
      p1q1.size(),
      CopyEdgeVerts({p0q1.ptrDpq(secondStart), inP.halfedge_.cptrD()}));
  return p0q1;
}

struct Not_zero {
  __host__ __device__ bool operator()(const int x) { return x != 0; }
};

struct Right : public thrust::unary_function<EdgeTrisD, int> {
  __host__ __device__ int operator()(EdgeTrisD edge) { return edge.right; }
};

struct Left : public thrust::unary_function<EdgeTrisD, int> {
  __host__ __device__ int operator()(EdgeTrisD edge) { return edge.left; }
};

struct AbsSum : public thrust::binary_function<int, int, int> {
  __host__ __device__ int operator()(int a, int b) { return abs(a) + abs(b); }
};

__host__ __device__ bool Shadows(float p, float q, float dir) {
  return p == q ? dir < 0 : p < q;
}

struct ShadowKernel01 {
  const bool reverse;
  const glm::vec3 *vertPosP;
  const glm::vec3 *vertPosQ;
  const Halfedge *halfedgeQ;
  const float expandP;
  const glm::vec3 *normalP;

  __host__ __device__ void operator()(thrust::tuple<int &, int, int> inout) {
    int &s01 = thrust::get<0>(inout);
    const int p0 = thrust::get<1>(inout);
    const int q1 = thrust::get<2>(inout);

    const int q1s = halfedgeQ[q1].startVert;
    const int q1e = halfedgeQ[q1].endVert;
    const float p0x = vertPosP[p0].x;
    const float q1sx = vertPosQ[q1s].x;
    const float q1ex = vertPosQ[q1e].x;
    s01 = reverse
              ? Shadows(q1sx, p0x, expandP * normalP[q1s].x) -
                    Shadows(q1ex, p0x, expandP * normalP[q1e].x)
              : Shadows(p0x, q1ex, expandP * normalP[p0].x) -
                    Shadows(p0x, q1sx, expandP * normalP[p0].x);
  }
};

struct Kernel01 {
  const bool reverse;
  const glm::vec3 *vertPosP;
  const glm::vec3 *vertPosQ;
  const Halfedge *halfedgeQ;
  const float expandP;
  const glm::vec3 *normalP;

  __host__ __device__ void operator()(
      thrust::tuple<glm::vec2 &, int &, int, int> inout) {
    glm::vec2 &yz01 = thrust::get<0>(inout);
    int &s01 = thrust::get<1>(inout);
    const int p0 = thrust::get<2>(inout);
    const int q1 = thrust::get<3>(inout);

    const int q1s = halfedgeQ[q1].startVert;
    const int q1e = halfedgeQ[q1].endVert;
    yz01 = Interpolate(vertPosQ[q1s], vertPosQ[q1e], vertPosP[p0].x);
    if (reverse) {
      if (!Shadows(yz01[0], vertPosP[p0].y, expandP * normalP[q1s].y)) s01 = 0;
    } else {
      if (!Shadows(vertPosP[p0].y, yz01[0], expandP * normalP[p0].y)) s01 = 0;
    }
  }
};

std::tuple<VecDH<int>, VecDH<glm::vec2>> Shadow01(SparseIndices &p0q1,
                                                  const Manifold::Impl &inP,
                                                  const Manifold::Impl &inQ,
                                                  bool reverse, float expandP) {
  VecDH<int> s01(p0q1.size());
  if (reverse) p0q1.SwapPQ();
  auto normalP = reverse ? inQ.vertNormal_.cptrD() : inP.vertNormal_.cptrD();
  thrust::for_each_n(
      zip(s01.beginD(), p0q1.beginD(0), p0q1.beginD(1)), p0q1.size(),
      ShadowKernel01({reverse, inP.vertPos_.cptrD(), inQ.vertPos_.cptrD(),
                      inQ.halfedge_.cptrD(), expandP, normalP}));
  size_t size = p0q1.RemoveZeros(s01);
  VecDH<glm::vec2> yz01(size);

  normalP = reverse ? inQ.vertNormal_.cptrD() : inP.vertNormal_.cptrD();
  thrust::for_each_n(
      zip(yz01.beginD(), s01.beginD(), p0q1.beginD(0), p0q1.beginD(1)), size,
      Kernel01({reverse, inP.vertPos_.cptrD(), inQ.vertPos_.cptrD(),
                inQ.halfedge_.cptrD(), expandP, normalP}));
  if (reverse) p0q1.SwapPQ();
  return std::make_tuple(s01, yz01);
}

template <typename Val>
__host__ __device__ Val BinarySearchByKey(
    const thrust::pair<const int *, const int *> keys, const Val *vals,
    const int size, const thrust::pair<int, int> key, const Val missingVal) {
  if (size <= 0) return missingVal;
  int left = 0;
  int right = size - 1;
  int m;
  thrust::pair<int, int> keyM;
  while (1) {
    m = right - (right - left) / 2;
    keyM = thrust::make_pair(keys.first[m], keys.second[m]);
    if (left == right) break;
    if (keyM > key)
      right = m - 1;
    else
      left = m;
  }
  if (keyM == key)
    return vals[m];
  else
    return missingVal;
}

struct Gather11 {
  const thrust::pair<const int *, const int *> p0q1;
  const int *s01;
  const int size;
  const Halfedge *halfedgeP;
  const bool reverse;

  __host__ __device__ void operator()(thrust::tuple<int &, int, int> inout) {
    int &s11 = thrust::get<0>(inout);
    const int p1 = thrust::get<1>(inout);
    const int q1 = thrust::get<2>(inout);

    int p0 = halfedgeP[p1].endVert;
    auto key = reverse ? thrust::make_pair(q1, p0) : thrust::make_pair(p0, q1);
    s11 += BinarySearchByKey(p0q1, s01, size, key, 0);
    p0 = halfedgeP[p1].startVert;
    key = reverse ? thrust::make_pair(q1, p0) : thrust::make_pair(p0, q1);
    s11 -= BinarySearchByKey(p0q1, s01, size, key, 0);
  }
};

struct Kernel11 {
  const glm::vec3 *vertPosP;
  const glm::vec3 *vertPosQ;
  const Halfedge *halfedgeP;
  const Halfedge *halfedgeQ;
  thrust::pair<const int *, const int *> p0q1;
  const glm::vec2 *yz01;
  int size01;
  thrust::pair<const int *, const int *> p1q0;
  const glm::vec2 *yz10;
  int size10;
  float expandP;
  const glm::vec3 *normalP;

  __host__ __device__ void operator()(
      thrust::tuple<glm::vec4 &, int &, int, int> inout) {
    glm::vec4 &xyzz11 = thrust::get<0>(inout);
    int &s11 = thrust::get<1>(inout);
    const int p1 = thrust::get<2>(inout);
    const int q1 = thrust::get<3>(inout);

    glm::vec3 p2[2], q2[2];
    int k = 0;
    thrust::pair<int, int> key2[2];

    key2[0] = thrust::make_pair(halfedgeP[p1].startVert, q1);
    key2[1] = thrust::make_pair(halfedgeP[p1].endVert, q1);
    for (int i : {0, 1}) {
      p2[k] = vertPosP[key2[i].first];
      q2[k] = glm::vec3(p2[k].x, BinarySearchByKey(p0q1, yz01, size01, key2[i],
                                                   glm::vec2(0.0f / 0.0f)));
      if (!isnan(q2[k].y)) k++;
    }

    key2[0] = thrust::make_pair(p1, halfedgeQ[q1].startVert);
    key2[1] = thrust::make_pair(p1, halfedgeQ[q1].endVert);
    for (int i : {0, 1}) {
      if (k > 1) break;
      q2[k] = vertPosQ[key2[i].second];
      p2[k] = glm::vec3(q2[k].x, BinarySearchByKey(p1q0, yz10, size10, key2[i],
                                                   glm::vec2(0.0f / 0.0f)));
      if (!isnan(p2[k].y)) k++;
    }

    // assert two of these four were found
    if (k != 2) printf("k = %d\n", k);

    xyzz11 = Intersect(p2[0], p2[1], q2[0], q2[1]);
    if (!Shadows(xyzz11.z, xyzz11.w,
                 expandP * normalP[halfedgeP[p1].startVert].z))
      s11 = 0;
  }
};

std::tuple<VecDH<int>, VecDH<glm::vec4>> Shadow11(
    SparseIndices &p1q1, const Manifold::Impl &inP, const Manifold::Impl &inQ,
    const SparseIndices &p0q1, const VecDH<int> &s01,
    const VecDH<glm::vec2> &yz01, const SparseIndices &p1q0,
    const VecDH<int> &s10, const VecDH<glm::vec2> &yz10, float expandP) {
  VecDH<int> s11(p1q1.size(), 0);

  thrust::for_each_n(zip(s11.beginD(), p1q1.beginD(0), p1q1.beginD(1)),
                     p1q1.size(),
                     Gather11({p0q1.ptrDpq(), s01.cptrD(), p0q1.size(),
                               inP.halfedge_.cptrD(), false}));
  thrust::for_each_n(zip(s11.beginD(), p1q1.beginD(1), p1q1.beginD(0)),
                     p1q1.size(),
                     Gather11({p1q0.ptrDpq(), s10.cptrD(), p1q0.size(),
                               inQ.halfedge_.cptrD(), true}));

  size_t size = p1q1.RemoveZeros(s11);
  VecDH<glm::vec4> xyzz11(size);

  thrust::for_each_n(
      zip(xyzz11.beginD(), s11.beginD(), p1q1.beginD(0), p1q1.beginD(1)),
      p1q1.size(),
      Kernel11({inP.vertPos_.cptrD(), inQ.vertPos_.cptrD(),
                inP.halfedge_.cptrD(), inQ.halfedge_.cptrD(), p0q1.ptrDpq(),
                yz01.cptrD(), p0q1.size(), p1q0.ptrDpq(), yz10.cptrD(),
                p1q0.size(), expandP, inP.vertNormal_.cptrD()}));

  return std::make_tuple(s11, xyzz11);
};

struct Gather02 {
  const thrust::pair<const int *, const int *> p0q1;
  const int *s01;
  const int size;
  const int *facesQ;
  const Halfedge *halfedgesQ;
  const bool forward;

  __host__ __device__ void operator()(thrust::tuple<int &, int, int> inout) {
    int &s02 = thrust::get<0>(inout);
    const int p0 = thrust::get<1>(inout);
    const int q2 = thrust::get<2>(inout);

    int q1 = facesQ[q2];
    const int lastEdge = facesQ[q2 + 1];
    while (q1 < lastEdge) {
      const Halfedge edge = halfedgesQ[q1];
      const int q1F = edge.IsForward() ? q1 : edge.pairedHalfedge;
      const auto key =
          forward ? thrust::make_pair(p0, q1F) : thrust::make_pair(q1F, p0);
      s02 += (forward == edge.IsForward() ? -1 : 1) *
             BinarySearchByKey(p0q1, s01, size, key, 0);
      ++q1;
    }
  }
};

struct Kernel02 {
  const glm::vec3 *vertPosP;
  const thrust::pair<const int *, const int *> p0q1;
  const glm::vec2 *yz01;
  const int size;
  const int *facesQ;
  const Halfedge *halfedgesQ;
  const bool forward;
  const float expandP;
  const glm::vec3 *normalP;

  __host__ __device__ void operator()(
      thrust::tuple<float &, int &, int, int> inout) {
    float &z02 = thrust::get<0>(inout);
    int &s02 = thrust::get<1>(inout);
    const int p0 = thrust::get<2>(inout);
    const int q2 = thrust::get<3>(inout);

    int q1 = facesQ[q2];
    const int lastEdge = facesQ[q2 + 1];
    glm::vec3 yzz2[2];
    int k = 0;
    while (q1 < lastEdge) {
      const Halfedge edge = halfedgesQ[q1];
      const int q1F = edge.IsForward() ? q1 : edge.pairedHalfedge;
      const auto key =
          forward ? thrust::make_pair(p0, q1F) : thrust::make_pair(q1F, p0);
      const glm::vec2 yz =
          BinarySearchByKey(p0q1, yz01, size, key, glm::vec2(0.0f / 0.0f));
      if (!isnan(yz[0])) yzz2[k++] = glm::vec3(yz[0], yz[1], yz[1]);
      if (k > 1) break;
      ++q1;
    }
    // assert two of these were found
    if (k != 2) printf("k = %d\n", k);

    glm::vec3 vertPos = vertPosP[p0];
    z02 = Interpolate(yzz2[0], yzz2[1], vertPos.y)[1];
    if (forward) {
      if (!Shadows(vertPos.z, z02, expandP * normalP[p0].z)) s02 = 0;
    } else {
      if (!Shadows(z02, vertPos.z, expandP * normalP[q2].z)) s02 = 0;
    }
  }
};

std::tuple<VecDH<int>, VecDH<float>> Shadow02(
    const Manifold::Impl &inP, const Manifold::Impl &inQ, const VecDH<int> &s01,
    const SparseIndices &p0q1, const VecDH<glm::vec2> &yz01,
    SparseIndices &p0q2, bool forward, float expandP) {
  VecDH<int> s02(p0q2.size(), 0);

  thrust::for_each_n(
      zip(s02.beginD(), p0q2.beginD(!forward), p0q2.beginD(forward)),
      p0q2.size(),
      Gather02({p0q1.ptrDpq(), s01.cptrD(), p0q1.size(), inQ.face_.cptrD(),
                inQ.halfedge_.cptrD(), forward}));

  size_t size = p0q2.RemoveZeros(s02);
  VecDH<float> z02(size);

  auto normalP = forward ? inP.vertNormal_.cptrD() : inQ.triNormal_.cptrD();
  thrust::for_each_n(
      zip(z02.beginD(), s02.beginD(), p0q2.beginD(!forward),
          p0q2.beginD(forward)),
      size, Kernel02({inP.vertPos_.cptrD(), p0q1.ptrDpq(), yz01.cptrD(),
                      p0q1.size(), inQ.face_.cptrD(), inQ.halfedge_.cptrD(),
                      forward, expandP, normalP}));

  return std::make_tuple(s02, z02);
};

struct Gather12 {
  const thrust::pair<const int *, const int *> p0q2;
  const int *s02;
  const int size02;
  const thrust::pair<const int *, const int *> p1q1;
  const int *s11;
  const int size11;
  const Halfedge *halfedgesP;
  const int *facesQ;
  const Halfedge *halfedgesQ;
  const bool forward;

  __host__ __device__ void operator()(thrust::tuple<int &, int, int> inout) {
    int &x12 = thrust::get<0>(inout);
    const int p1 = thrust::get<1>(inout);
    const int q2 = thrust::get<2>(inout);

    const Halfedge edge = halfedgesP[p1];
    auto key = forward ? thrust::make_pair(edge.startVert, q2)
                       : thrust::make_pair(q2, edge.endVert);
    x12 = BinarySearchByKey(p0q2, s02, size02, key, 0);
    key = forward ? thrust::make_pair(edge.endVert, q2)
                  : thrust::make_pair(q2, edge.startVert);
    x12 -= BinarySearchByKey(p0q2, s02, size02, key, 0);

    int q1 = facesQ[q2];
    const int lastEdge = facesQ[q2 + 1];
    while (q1 < lastEdge) {
      const Halfedge edge = halfedgesQ[q1];
      const int q1F = edge.IsForward() ? q1 : edge.pairedHalfedge;
      key = forward ? thrust::make_pair(p1, q1F) : thrust::make_pair(q1F, p1);
      x12 -= (edge.IsForward() ? 1 : -1) *
             BinarySearchByKey(p1q1, s11, size11, key, 0);
      ++q1;
    }
  }
};

struct Kernel12 {
  const thrust::pair<const int *, const int *> p0q2;
  const float *z02;
  const int size02;
  const thrust::pair<const int *, const int *> p1q1;
  const glm::vec4 *xyzz11;
  const int size11;
  const Halfedge *halfedgesP;
  const int *facesQ;
  const Halfedge *halfedgesQ;
  const glm::vec3 *vertPosP;
  const bool forward;

  __host__ __device__ void operator()(
      thrust::tuple<glm::vec3 &, int, int> inout) {
    glm::vec3 &v12 = thrust::get<0>(inout);
    const int p1 = thrust::get<1>(inout);
    const int q2 = thrust::get<2>(inout);

    const Halfedge edge = halfedgesP[p1];
    auto key = forward ? thrust::make_pair(edge.startVert, q2)
                       : thrust::make_pair(q2, edge.endVert);
    const float z0 = BinarySearchByKey(p0q2, z02, size02, key, 0.0f / 0.0f);
    key = forward ? thrust::make_pair(edge.endVert, q2)
                  : thrust::make_pair(q2, edge.startVert);
    const float z1 = BinarySearchByKey(p0q2, z02, size02, key, 0.0f / 0.0f);

    glm::vec3 xzyLR0[2];
    glm::vec3 xzyLR1[2];
    int k = 0;
    if (!isnan(z0)) {
      xzyLR0[k] = vertPosP[edge.startVert];
      thrust::swap(xzyLR0[k].y, xzyLR0[k].z);
      xzyLR1[k] = xzyLR0[k];
      xzyLR1[k][1] = z0;
      k++;
    }
    if (!isnan(z1)) {
      xzyLR0[k] = vertPosP[edge.endVert];
      thrust::swap(xzyLR0[k].y, xzyLR0[k].z);
      xzyLR1[k] = xzyLR0[k];
      xzyLR1[k][1] = z1;
      k++;
    }

    int q1 = facesQ[q2];
    const int lastEdge = facesQ[q2 + 1];
    while (q1 < lastEdge) {
      if (k > 1) break;
      const Halfedge edge = halfedgesQ[q1];
      const int q1F = edge.IsForward() ? q1 : edge.pairedHalfedge;
      key = forward ? thrust::make_pair(p1, q1F) : thrust::make_pair(q1F, p1);
      const glm::vec4 xyzz =
          BinarySearchByKey(p1q1, xyzz11, size11, key, glm::vec4(0.0f / 0.0f));

      if (!isnan(xyzz.x)) {
        xzyLR0[k][0] = xyzz.x;
        xzyLR0[k][1] = xyzz.z;
        xzyLR0[k][2] = xyzz.y;
        xzyLR1[k] = xzyLR0[k];
        xzyLR1[k][1] = xyzz.w;
        if (!forward) thrust::swap(xzyLR0[k][1], xzyLR1[k][1]);
        k++;
      }
      ++q1;
    }

    // assert two of these five were found
    if (k != 2) printf("k = %d\n", k);

    const glm::vec4 xzyy =
        Intersect(xzyLR0[0], xzyLR0[1], xzyLR1[0], xzyLR1[1]);
    v12.x = xzyy[0];
    v12.y = xzyy[2];
    v12.z = xzyy[1];
  }
};

std::tuple<VecDH<int>, VecDH<glm::vec3>> Intersect12(
    const Manifold::Impl &inP, const Manifold::Impl &inQ, const VecDH<int> &s02,
    const SparseIndices &p0q2, const VecDH<int> &s11, const SparseIndices &p1q1,
    const VecDH<float> &z02, const VecDH<glm::vec4> &xyzz11,
    SparseIndices &p1q2, bool forward) {
  VecDH<int> x12(p1q2.size());
  VecDH<glm::vec3> v12;

  const auto halfedgesP =
      forward ? inP.halfedge_.cptrD() : inQ.halfedge_.cptrD();
  const auto halfedgesQ =
      forward ? inQ.halfedge_.cptrD() : inP.halfedge_.cptrD();
  const auto facesQ = forward ? inQ.face_.cptrD() : inP.face_.cptrD();

  thrust::for_each_n(
      zip(x12.beginD(), p1q2.beginD(!forward), p1q2.beginD(forward)),
      p1q2.size(), Gather12({p0q2.ptrDpq(), s02.ptrD(), p0q2.size(),
                             p1q1.ptrDpq(), s11.ptrD(), p1q1.size(), halfedgesP,
                             facesQ, halfedgesQ, forward}));

  size_t size = p1q2.RemoveZeros(x12);
  v12.resize(size);

  const auto vertPosPtr = forward ? inP.vertPos_.cptrD() : inQ.vertPos_.cptrD();
  thrust::for_each_n(
      zip(v12.beginD(), p1q2.beginD(!forward), p1q2.beginD(forward)),
      p1q2.size(),
      Kernel12({p0q2.ptrDpq(), z02.cptrD(), p0q2.size(), p1q1.ptrDpq(),
                xyzz11.cptrD(), p1q1.size(), halfedgesP, facesQ, halfedgesQ,
                vertPosPtr, forward}));
  return std::make_tuple(x12, v12);
};

VecDH<int> Winding03(const Manifold::Impl &inP, SparseIndices &p0q2,
                     VecDH<int> &s02, const SparseIndices &p1q2, bool reverse) {
  VecDH<int> w03(inP.NumVert(), kInvalidInt);
  // keepEdgesP is the set of edges that connect regions of the manifold with
  // the same winding number, so we remove any edges associated with
  // intersections.
  VecDH<bool> keepEdgesP(inP.halfedge_.size(), true);
  thrust::scatter(thrust::make_constant_iterator(false, 0),
                  thrust::make_constant_iterator(false, p1q2.size()),
                  p1q2.beginD(reverse), keepEdgesP.beginD());

  if (!thrust::is_sorted(p0q2.beginD(reverse), p0q2.endD(reverse)))
    thrust::sort_by_key(p0q2.beginD(reverse), p0q2.endD(reverse), s02.beginD());
  VecDH<int> w03val(w03.size());
  VecDH<int> w03vert(w03.size());
  // sum known s02 values into w03 (winding number)
  auto endPair =
      thrust::reduce_by_key(p0q2.beginD(reverse), p0q2.endD(reverse),
                            s02.beginD(), w03vert.beginD(), w03val.beginD());
  thrust::scatter(w03val.beginD(), endPair.second, w03vert.beginD(),
                  w03.beginD());

  // find connected regions (separated by intersections)
  VecDH<int> vertLabels;
  int n_comp =
      ConnectedComponents(vertLabels, inP.NumVert(), inP.halfedge_, keepEdgesP);
  // flood the w03 values throughout their connected components (they are
  // consistent)
  FloodComponents(w03, vertLabels, n_comp);

  if (kVerbose) std::cout << n_comp << " components" << std::endl;

  if (reverse)
    thrust::transform(w03.beginD(), w03.endD(), w03.beginD(),
                      thrust::negate<int>());
  return w03;
};

struct DuplicateVerts {
  glm::vec3 *vertPosR;

  __host__ __device__ void operator()(thrust::tuple<int, int, glm::vec3> in) {
    int inclusion = abs(thrust::get<0>(in));
    int vertR = thrust::get<1>(in);
    glm::vec3 vertPosP = thrust::get<2>(in);

    for (int i = 0; i < inclusion; ++i) {
      vertPosR[vertR + i] = vertPosP;
    }
  }
};

struct EdgePos {
  int vert;
  float edgePos;
  bool isStart;
};

void AddNewEdgeVerts(
    std::map<int, std::vector<EdgePos>> &edgesP,
    std::map<std::pair<int, int>, std::vector<EdgePos>> &edgesNew,
    const SparseIndices &p1q2, const VecH<int> &i12, const VecH<int> &v12R,
    const VecH<Halfedge> &halfedgeP, bool forward) {
  // For each edge of P that intersects a face of Q (p1q2), add this vertex to
  // P's corresponding edge vector and to the two new edges, which are
  // intersections between the face of Q and the two faces of P attached to the
  // edge. The direction and duplicity are given by i12, while v12R remaps to
  // the output vert index. When forward is false, all is reversed.
  const VecH<int> &p1 = p1q2.Get(!forward).H();
  const VecH<int> &q2 = p1q2.Get(forward).H();
  for (int i = 0; i < p1q2.size(); ++i) {
    const int edgeP = p1[i];
    const int faceQ = q2[i];
    const int vert = v12R[i];
    const int inclusion = i12[i];

    const auto edgePosP = edgesP.insert({edgeP, {}});

    Halfedge halfedge = halfedgeP[edgeP];
    std::pair<int, int> key = {halfedgeP[halfedge.pairedHalfedge].face, faceQ};
    if (!forward) std::swap(key.first, key.second);
    const auto edgePosRight = edgesNew.insert({key, {}});

    key = {halfedge.face, faceQ};
    if (!forward) std::swap(key.first, key.second);
    const auto edgePosLeft = edgesNew.insert({key, {}});

    EdgePos edgePos = {vert, 0.0f, inclusion < 0};
    EdgePos edgePosRev = edgePos;
    edgePosRev.isStart = !edgePos.isStart;

    for (int j = 0; j < glm::abs(inclusion); ++j) {
      edgePosP.first->second.push_back(edgePos);
      edgePosRight.first->second.push_back(forward ? edgePos : edgePosRev);
      edgePosLeft.first->second.push_back(forward ? edgePosRev : edgePos);
      ++edgePos.vert;
      ++edgePosRev.vert;
    }
  }
}

std::vector<EdgeVerts> PairUp(std::vector<EdgePos> &edgePos, int edge) {
  // Pair start vertices with end vertices to form edges. The choice of pairing
  // is arbitrary for the manifoldness guarantee, but must be ordered to be
  // geometrically valid. If the order does not go start-end-start-end... then
  // the input and output are not geometrically valid and this algorithm becomes
  // a heuristic.
  ALWAYS_ASSERT(edgePos.size() % 2 == 0, logicErr,
                "Non-manifold edge! Not an even number of points.");
  int nEdges = edgePos.size() / 2;
  auto middle = std::partition(edgePos.begin(), edgePos.end(),
                               [](EdgePos x) { return x.isStart; });
  ALWAYS_ASSERT(middle - edgePos.begin() == nEdges, logicErr,
                "Non-manifold edge!");
  auto cmp = [](EdgePos a, EdgePos b) { return a.edgePos < b.edgePos; };
  std::sort(edgePos.begin(), middle, cmp);
  std::sort(middle, edgePos.end(), cmp);
  std::vector<EdgeVerts> edges;
  for (int i = 0; i < nEdges; ++i)
    edges.push_back({edgePos[i].vert, edgePos[i + nEdges].vert, edge});
  return edges;
}

void AppendRetainedEdges(std::map<int, std::vector<EdgeVerts>> &facesP,
                         std::map<int, std::vector<EdgePos>> &edgesP,
                         const Manifold::Impl &inP, const VecH<int> &i03,
                         const VecH<int> &vP2R,
                         const VecH<glm::vec3> &vertPos) {
  // Each edge in the map is partially retained; for each of these, look up
  // their original verts and include them based on their winding number (i03),
  // while remaping them to the output using vP2R. Use the verts position
  // projected along the edge vector to pair them up, then distribute these
  // edges to their faces. Copy any original edges of each face in that are not
  // in the retained edge map.
  const VecH<glm::vec3> &vertPosP = inP.vertPos_.H();
  const VecH<Halfedge> &halfedgeP = inP.halfedge_.H();

  for (auto &value : edgesP) {
    const int edgeP = value.first;
    std::vector<EdgePos> &edgePosP = value.second;

    const Halfedge &halfedge = halfedgeP[edgeP];
    const int vStart = halfedge.startVert;
    const int vEnd = halfedge.endVert;
    const glm::vec3 edgeVec = vertPosP[vEnd] - vertPosP[vStart];
    // Fill in the edge positions of the old points.
    for (EdgePos &edge : edgePosP) {
      edge.edgePos = glm::dot(vertPos[edge.vert], edgeVec);
    }

    int inclusion = i03[vStart];
    EdgePos edgePos = {vP2R[vStart], -1.0f / 0.0f, inclusion > 0};
    for (int j = 0; j < glm::abs(inclusion); ++j) {
      edgePosP.push_back(edgePos);
      ++edgePos.vert;
    }

    inclusion = i03[vEnd];
    edgePos = {vP2R[vEnd], 1.0f / 0.0f, inclusion < 0};
    for (int j = 0; j < glm::abs(inclusion); ++j) {
      edgePosP.push_back(edgePos);
      ++edgePos.vert;
    }

    // sort edges into start/end pairs along length
    std::vector<EdgeVerts> edges = PairUp(edgePosP, edgeP);

    // add edges to left face
    const int faceLeft = halfedge.face;
    auto result = facesP.insert({faceLeft, edges});
    if (!result.second) {
      auto &vec = result.first->second;
      vec.insert(vec.end(), edges.begin(), edges.end());
    }
    // reverse edges and add to right face
    for (auto &e : edges) std::swap(e.first, e.second);
    const int faceRight = halfedgeP[halfedge.pairedHalfedge].face;
    result = facesP.insert({faceRight, edges});
    if (!result.second) {
      auto &vec = result.first->second;
      vec.insert(vec.end(), edges.begin(), edges.end());
    }
  }
}

void AppendNewEdges(
    std::map<int, std::vector<EdgeVerts>> &facesP,
    std::map<int, std::vector<EdgeVerts>> &facesQ,
    std::map<std::pair<int, int>, std::vector<EdgePos>> &edgesNew) {
  // Pair up each edge's verts and distribute to faces based on indices in key.
  // Usually only two verts are in each edge, and if not, they are degenerate
  // anyway, so pair arbitrarily without bothering with vertex projections.
  int edgeID = std::numeric_limits<int>::max();
  for (auto &value : edgesNew) {
    const int faceP = value.first.first;
    const int faceQ = value.first.second;
    std::vector<EdgePos> &edgePos = value.second;

    // sort edges into start/end pairs along length
    // Since these are not input edges, their index is undefined.
    std::vector<EdgeVerts> edges = PairUp(edgePos, edgeID--);

    auto result = facesP.insert({faceP, edges});
    if (!result.second) {
      auto &vec = result.first->second;
      vec.insert(vec.end(), edges.begin(), edges.end());
    }
    // reverse edges and add to right face
    for (auto &e : edges) std::swap(e.first, e.second);
    result = facesQ.insert({faceQ, edges});
    if (!result.second) {
      auto &vec = result.first->second;
      vec.insert(vec.end(), edges.begin(), edges.end());
    }
  }
}

glm::mat3x2 GetAxisAlignedProjection(glm::vec3 normal) {
  glm::vec3 absNormal = glm::abs(normal);
  float xyzMax;
  glm::mat2x3 projection;
  if (absNormal.z > absNormal.x && absNormal.z > absNormal.y) {
    projection = glm::mat2x3(1.0f, 0.0f, 0.0f,  //
                             0.0f, 1.0f, 0.0f);
    xyzMax = normal.z;
  } else if (absNormal.y > absNormal.x) {
    projection = glm::mat2x3(0.0f, 0.0f, 1.0f,  //
                             1.0f, 0.0f, 0.0f);
    xyzMax = normal.y;
  } else {
    projection = glm::mat2x3(0.0f, 1.0f, 0.0f,  //
                             0.0f, 0.0f, 1.0f);
    xyzMax = normal.x;
  }
  if (xyzMax < 0) projection[0] *= -1.0f;
  return glm::transpose(projection);
}

void AppendFaces(Manifold::Impl &outR,
                 std::map<int, std::vector<EdgeVerts>> &facesP,
                 const std::map<int, std::vector<EdgePos>> &edgesP,
                 const VecH<int> &i03, const Manifold::Impl &inP,
                 const VecH<int> &vP2R, bool invertNormals) {
  // Proceed through the map, triangulating each face into the result. For each
  // face not included as a map index, copy it from the original mesh,
  // duplicating according to its inclusion number (i03).
  const VecH<glm::ivec3> &triVertsP = inP.triVerts_.H();
  const VecH<TriEdges> &triEdgesP = inP.triEdges_.H();
  const VecH<glm::vec3> &triNormalP = inP.triNormal_.H();
  const VecH<EdgeVertsD> &edgeVertsP = inP.edgeVerts_.H();
  VecH<glm::ivec3> &triVertsR = outR.triVerts_.H();
  VecH<glm::vec3> &triNormalR = outR.triNormal_.H();
  VecH<glm::vec3> &vertPosR = outR.vertPos_.H();

  auto nextIntersectedFace = facesP.begin();
  for (int triP = 0; triP < inP.NumTri(); ++triP) {
    const int faceP = facesP.empty() ? -1 : nextIntersectedFace->first;
    if (faceP != triP) {  // Non-intersecting face
      // Copy triangle from inP
      glm::ivec3 triVerts = triVertsP[triP];
      glm::vec3 normal = triNormalP[triP];
      // Check the inclusion number of a single vertex of a triangle, since
      // non-intersecting triangles must have all identical inclusion numbers.
      int inclusion = i03[triVerts[0]];
      glm::ivec3 outTri(vP2R[triVerts[0]], vP2R[triVerts[1]],
                        vP2R[triVerts[2]]);
      if (inclusion < 0) {
        std::swap(outTri[1], outTri[2]);
        normal *= -1.0f;
      }
      for (int j = 0; j < abs(inclusion); ++j) {
        triVertsR.push_back(outTri + j);
        triNormalR.push_back(normal);
      }
    } else {  // intersecting face
      std::vector<EdgeVerts> &faceEdges = nextIntersectedFace->second;
      if (std::next(nextIntersectedFace) != facesP.end()) ++nextIntersectedFace;

      // Copy in non-intersecting edges of intersected face
      for (int i : {0, 1, 2}) {
        EdgeIdx edge = triEdgesP[faceP][i];
        if (edgesP.find(edge.Idx()) == edgesP.end()) {
          EdgeVertsD oldEdgeVerts = edgeVertsP[edge.Idx()];
          // Non-intersecting edge has the same inclusion number at both ends.
          const int inclusion = i03[oldEdgeVerts.first];
          int vStart = vP2R[oldEdgeVerts.first];
          int vEnd = vP2R[oldEdgeVerts.second];
          if ((inclusion > 0) != (edge.Dir() > 0)) std::swap(vStart, vEnd);
          for (int j = 0; j < std::abs(inclusion); ++j) {
            faceEdges.push_back({vStart + j, vEnd + j, edge.Idx()});
          }
        }
      }

      // Triangulate intersected face
      ALWAYS_ASSERT(faceEdges.size() >= 3, logicErr,
                    "face has less than three edges.");
      const glm::vec3 normal =
          (invertNormals ? -1.0f : 1.0f) * triNormalP[faceP];

      if (faceEdges.size() == 3) {  // Special case to increase performance
        auto tri = faceEdges;
        if (tri[0].second == tri[2].first) std::swap(tri[1], tri[2]);
        ALWAYS_ASSERT(tri[0].second == tri[1].first &&
                          tri[1].second == tri[2].first &&
                          tri[2].second == tri[0].first,
                      runtimeErr, "These 3 edges do not form a triangle!");
        glm::ivec3 triangle(tri[0].first, tri[1].first, tri[2].first);
        triVertsR.push_back(triangle);
        triNormalR.push_back(normal);
      } else {  // General triangulation
        const glm::mat3x2 projection = GetAxisAlignedProjection(normal);
        Polygons polys =
            Assemble(faceEdges, [&vertPosR, &projection](int vert) {
              return projection * vertPosR[vert];
            });
        std::vector<glm::ivec3> newTris;
        try {
          newTris = Triangulate(polys);
        } catch (const runtimeErr &e) {
          if (PolygonParams().checkGeometry) throw;
          /**
          To ensure the triangulation maintains the mesh as 2-manifold, we
          require it to not create edges connecting non-neighboring vertices
          from the same input edge. This is because if two neighboring
          polygons were to create an edge like this between two of their
          shared vertices, this would create a 4-manifold edge, which is not
          allowed.

          For some self-overlapping polygons, there exists no triangulation
          that adheres to this constraint. In this case, we create an extra
          vertex for each polygon and triangulate them like a wagon wheel,
          which is guaranteed to be manifold. This is very rare and only
          occurs when the input manifolds are self-overlapping.
           */
          for (const auto &poly : polys) {
            glm::vec3 centroid = thrust::transform_reduce(
                poly.begin(), poly.end(),
                [&vertPosR](PolyVert v) { return vertPosR[v.idx]; },
                glm::vec3(0.0f),
                [](glm::vec3 a, glm::vec3 b) { return a + b; });
            centroid /= poly.size();
            int newVert = vertPosR.size();
            vertPosR.push_back(centroid);
            newTris.push_back({poly.back().idx, poly.front().idx, newVert});
            for (int j = 1; j < poly.size(); ++j)
              newTris.push_back({poly[j - 1].idx, poly[j].idx, newVert});
          }
        }
        for (auto tri : newTris) {
          triVertsR.push_back(tri);
          triNormalR.push_back(normal);
        }
      }
    }
  }
}
}  // namespace

namespace manifold {
Boolean3::Boolean3(const Manifold::Impl &inP, const Manifold::Impl &inQ,
                   Manifold::OpType op)
    : inP_(inP), inQ_(inQ), expandP_(op == Manifold::OpType::ADD ? 1.0 : -1.0) {
  // Symbolic perturbation:
  // Union -> expand inP
  // Difference, Intersection -> contract inP

  inP_.Tri2Face();
  inQ_.Tri2Face();

  VecDH<int> faceSizeP(inP_.face_.size());
  thrust::adjacent_difference(inP_.face_.beginD(), inP_.face_.endD(),
                              faceSizeP.beginD());
  VecDH<int> faceSizeQ(inQ_.face_.size());
  thrust::adjacent_difference(inQ_.face_.beginD(), inQ_.face_.endD(),
                              faceSizeQ.beginD());

  Time t0 = NOW();
  Time t1;
  // Level 3
  // Find edge-triangle overlaps (broad phase)
  p1q2_ = inQ_.EdgeCollisions(inP_);
  p1q2_.Sort();
  if (kVerbose) std::cout << "p1q2 size = " << p1q2_.size() << std::endl;

  p2q1_ = inP_.EdgeCollisions(inQ_);
  p2q1_.SwapPQ();
  p2q1_.Sort();
  if (kVerbose) std::cout << "p2q1 size = " << p2q1_.size() << std::endl;

  // Level 2
  // Find vertices from Level 3 that overlap faces in XY-projection
  SparseIndices p0q2 = Filter02(inP_, inQ_, p1q2_.Get(0), p2q1_.Get(0));
  p0q2.Sort();
  if (kVerbose) std::cout << "p0q2 size = " << p0q2.size() << std::endl;

  SparseIndices p2q0 = Filter02(inQ_, inP_, p2q1_.Get(1), p1q2_.Get(1));
  p2q0.SwapPQ();
  p2q0.Sort();
  if (kVerbose) std::cout << "p2q0 size = " << p2q0.size() << std::endl;

  // Find involved edge pairs from Level 3
  SparseIndices p1q1 = Filter11(inP_, faceSizeP, inQ_, faceSizeQ, p1q2_, p2q1_);
  if (kVerbose) std::cout << "p1q1 size = " << p1q1.size() << std::endl;

  // Level 1
  // Find involved vertex-edge pairs from Level 2
  SparseIndices p0q1 = Filter01(inP_, inQ_, faceSizeQ, p0q2, p1q1);
  p0q1.Unique();
  if (kVerbose) std::cout << "p0q1 size = " << p0q1.size() << std::endl;

  p2q0.SwapPQ();
  p1q1.SwapPQ();
  SparseIndices p1q0 = Filter01(inQ_, inP_, faceSizeP, p2q0, p1q1);
  p2q0.SwapPQ();
  p1q1.SwapPQ();
  p1q0.SwapPQ();
  p1q0.Unique();
  if (kVerbose) std::cout << "p1q0 size = " << p1q0.size() << std::endl;

  if (kVerbose) {
    std::cout << "Time for Filter";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }

  // Level 1
  // Find X-projections of vertices onto edges, keeping only those that actually
  // fall inside the edge.
  VecDH<int> s01;
  VecDH<glm::vec2> yz01;
  std::tie(s01, yz01) = Shadow01(p0q1, inP, inQ, false, expandP_);
  if (kVerbose) std::cout << "s01 size = " << s01.size() << std::endl;

  VecDH<int> s10;
  VecDH<glm::vec2> yz10;
  std::tie(s10, yz10) = Shadow01(p1q0, inQ, inP, true, expandP_);
  if (kVerbose) std::cout << "s10 size = " << s10.size() << std::endl;

  // Level 2
  // Build up XY-projection intersection of two edges, including the z-value for
  // each edge, keeping only those whose intersection exists.
  VecDH<int> s11;
  VecDH<glm::vec4> xyzz11;
  std::tie(s11, xyzz11) =
      Shadow11(p1q1, inP, inQ, p0q1, s01, yz01, p1q0, s10, yz10, expandP_);
  if (kVerbose) std::cout << "s11 size = " << s11.size() << std::endl;

  // Build up Z-projection of vertices onto triangles, keeping only those that
  // fall inside the triangle.
  VecDH<int> s02;
  VecDH<float> z02;
  std::tie(s02, z02) =
      Shadow02(inP, inQ, s01, p0q1, yz01, p0q2, true, expandP_);
  if (kVerbose) std::cout << "s02 size = " << s02.size() << std::endl;

  VecDH<int> s20;
  VecDH<float> z20;
  std::tie(s20, z20) =
      Shadow02(inQ, inP, s10, p1q0, yz10, p2q0, false, expandP_);
  if (kVerbose) std::cout << "s20 size = " << s20.size() << std::endl;

  // Level 3
  // Build up the intersection of the edges and triangles, keeping only those
  // that intersect, and record the direction the edge is passing through the
  // triangle.
  std::tie(x12_, v12_) =
      Intersect12(inP, inQ, s02, p0q2, s11, p1q1, z02, xyzz11, p1q2_, true);
  if (kVerbose) std::cout << "dir12 size = " << x12_.size() << std::endl;

  std::tie(x21_, v21_) =
      Intersect12(inP, inQ, s20, p2q0, s11, p1q1, z20, xyzz11, p2q1_, false);
  if (kVerbose) std::cout << "dir21 size = " << x21_.size() << std::endl;

  if (kVerbose) {
    std::cout << "Time for Levels 1-3";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }

  // Build up the winding numbers of all vertices. The involved vertices are
  // calculated from Level 2, while the rest are assigned consistently with
  // connected-components flooding.
  w03_ = Winding03(inP, p0q2, s02, p1q2_, false);

  w30_ = Winding03(inQ, p2q0, s20, p2q1_, true);

  if (kVerbose) {
    std::cout << "Time for rest of first stage";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }
}

Manifold::Impl Boolean3::Result(Manifold::OpType op) const {
  if ((expandP_ > 0) != (op == Manifold::OpType::ADD))
    std::cout << "Warning! Result op type not compatible with constructor op "
                 "type: coplanar faces may have incorrect results."
              << std::endl;
  int c1, c2, c3;
  switch (op) {
    case Manifold::OpType::ADD:
      c1 = 1;
      c2 = 1;
      c3 = -1;
      if (kVerbose) std::cout << "ADD" << std::endl;
      break;
    case Manifold::OpType::SUBTRACT:
      c1 = 1;
      c2 = 0;
      c3 = -1;
      if (kVerbose) std::cout << "SUBTRACT" << std::endl;
      break;
    case Manifold::OpType::INTERSECT:
      c1 = 0;
      c2 = 0;
      c3 = 1;
      if (kVerbose) std::cout << "INTERSECT" << std::endl;
      break;
    default:
      throw std::invalid_argument("invalid enum: OpType.");
  }

  Time t0 = NOW();
  Time t1;

  // Convert winding numbers to inclusion values based on operation type.
  VecDH<int> i12(x12_.size());
  VecDH<int> i21(x21_.size());
  VecDH<int> i03(w03_.size());
  VecDH<int> i30(w30_.size());
  thrust::transform(x12_.beginD(), x12_.endD(), i12.beginD(), c3 * _1);
  thrust::transform(x21_.beginD(), x21_.endD(), i21.beginD(), c3 * _1);
  thrust::transform(w03_.beginD(), w03_.endD(), i03.beginD(), c1 + c3 * _1);
  thrust::transform(w30_.beginD(), w30_.endD(), i30.beginD(), c2 + c3 * _1);

  VecDH<int> vP2R(inP_.NumVert());
  thrust::exclusive_scan(i03.beginD(), i03.endD(), vP2R.beginD(), 0, AbsSum());
  int numVertR = AbsSum()(vP2R.H().back(), i03.H().back());
  const int nPv = numVertR;

  VecDH<int> vQ2R(inQ_.NumVert());
  thrust::exclusive_scan(i30.beginD(), i30.endD(), vQ2R.beginD(), numVertR,
                         AbsSum());
  numVertR = AbsSum()(vQ2R.H().back(), i30.H().back());
  const int nQv = numVertR - nPv;

  VecDH<int> v12R(v12_.size());
  if (v12_.size() > 0) {
    thrust::exclusive_scan(i12.beginD(), i12.endD(), v12R.beginD(), numVertR,
                           AbsSum());
    numVertR = AbsSum()(v12R.H().back(), i12.H().back());
  }
  const int n12 = numVertR - nPv - nQv;

  VecDH<int> v21R(v21_.size());
  if (v21_.size() > 0) {
    thrust::exclusive_scan(i21.beginD(), i21.endD(), v21R.beginD(), numVertR,
                           AbsSum());
    numVertR = AbsSum()(v21R.H().back(), i21.H().back());
  }
  const int n21 = numVertR - nPv - nQv - n12;

  // Create the output Manifold
  Manifold::Impl outR;

  if (numVertR == 0) return outR;

  outR.vertPos_.resize(numVertR);
  // Add vertices, duplicating for inclusion numbers not in [-1, 1].
  // Retained vertices from P and Q:
  thrust::for_each_n(zip(i03.beginD(), vP2R.beginD(), inP_.vertPos_.beginD()),
                     inP_.NumVert(), DuplicateVerts({outR.vertPos_.ptrD()}));
  thrust::for_each_n(zip(i30.beginD(), vQ2R.beginD(), inQ_.vertPos_.beginD()),
                     inQ_.NumVert(), DuplicateVerts({outR.vertPos_.ptrD()}));
  // New vertices created from intersections:
  thrust::for_each_n(zip(i12.beginD(), v12R.beginD(), v12_.beginD()),
                     i12.size(), DuplicateVerts({outR.vertPos_.ptrD()}));
  thrust::for_each_n(zip(i21.beginD(), v21R.beginD(), v21_.beginD()),
                     i21.size(), DuplicateVerts({outR.vertPos_.ptrD()}));

  if (kVerbose) {
    std::cout << nPv << " verts from inP" << std::endl;
    std::cout << nQv << " verts from inQ" << std::endl;
    std::cout << n12 << " new verts from edgesP -> facesQ" << std::endl;
    std::cout << n21 << " new verts from facesP -> edgesQ" << std::endl;
  }

  if (kVerbose) {
    std::cout << "Time for GPU part of result";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }

  // Build up new polygonal faces from triangle intersections. At this point the
  // calculation switches from parallel to serial.

  // Level 3

  // This key is the edge index of P or Q. Only includes intersected edges.
  std::map<int, std::vector<EdgePos>> edgesP, edgesQ;
  // This key is the tri index of <P, Q>
  std::map<std::pair<int, int>, std::vector<EdgePos>> edgesNew;

  AddNewEdgeVerts(edgesP, edgesNew, p1q2_, i12.H(), v12R.H(),
                  inP_.halfedge_.H(), true);
  AddNewEdgeVerts(edgesQ, edgesNew, p2q1_, i21.H(), v21R.H(),
                  inQ_.halfedge_.H(), false);

  // Level 4

  // This key is the tri index of P or Q. Only includes intersected faces.
  std::map<int, std::vector<EdgeVerts>> facesP, facesQ;

  AppendRetainedEdges(facesP, edgesP, inP_, i03.H(), vP2R.H(),
                      outR.vertPos_.H());
  AppendRetainedEdges(facesQ, edgesQ, inQ_, i30.H(), vQ2R.H(),
                      outR.vertPos_.H());
  AppendNewEdges(facesP, facesQ, edgesNew);

  if (kVerbose) {
    std::cout << "Time for CPU part of result";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }

  // Level 5

  // Copy retained triangles and triangulate the intersected faces and add them
  // to the manifold.
  if (kVerbose) std::cout << "Adding faces of inP" << std::endl;
  AppendFaces(outR, facesP, edgesP, i03.H(), inP_, vP2R.H(), false);
  if (kVerbose) std::cout << "Adding faces of inQ" << std::endl;
  AppendFaces(outR, facesQ, edgesQ, i30.H(), inQ_, vQ2R.H(),
              op == Manifold::OpType::SUBTRACT);

  // outR.triVerts_.Dump();

  if (kVerbose) {
    std::cout << "Time for triangulation";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }

  // Level 6

  // Create the manifold's data structures and verify manifoldness.
  outR.Finish();
  outR.RemoveChaff();
  outR.Finish();

  if (kVerbose) {
    std::cout << "Time for manifold finishing";
    t1 = NOW();
    PrintDuration(t1 - t0);
    t0 = t1;
  }

  return outR;
}

}  // namespace manifold