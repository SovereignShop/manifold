#include "hip/hip_runtime.h"
// Copyright 2019 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// #include <nvgraph.h>
#include <thrust/iterator/constant_iterator.h>

#include <boost/config.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/connected_components.hpp>

#include "connected_components.cuh"

// 2^31-1 is nvgraph's unreachable flag
// constexpr int kNvgraphInvalid = std::numeric_limits<int>::max();

namespace {
using namespace manifold;

// void CheckStatus(nvgraphStatus_t status) {
//   ALWAYS_ASSERT((int)status == 0, runtimeErr, "nvGraph error: " + status);
// }

// struct DuplicateEdges {
//   int* source;
//   int* sink;

//   __host__ __device__ void operator()(thrust::tuple<int, Halfedge> in) {
//     int idx = thrust::get<0>(in);
//     Halfedge halfedges = thrust::get<1>(in);
//     source[idx] = halfedges.startVert;
//     sink[idx] = halfedges.endVert;
//   }
// };

// struct DuplicateKeep {
//   int* edgeMask;

//   __host__ __device__ void operator()(thrust::tuple<int, bool, Halfedge> in)
//   {
//     int idx = thrust::get<0>(in);
//     bool keep = thrust::get<1>(in);
//     Halfedge halfedge = thrust::get<2>(in);
//     edgeMask[idx] = keep;
//     edgeMask[halfedge.pairedHalfedge] = keep;
//   }
// };

// void Edges2CSR(VecDH<int>& rowOffsets, VecDH<int>& sink, VecDH<int>&
// edgeMask,
//                const VecDH<Halfedge>& halfedges, const VecDH<bool>& keep,
//                int numVert) {
//   // Duplicate undirected graph edges
//   int numHalfedge = halfedges.size();
//   VecDH<int> source(numHalfedge);
//   sink.resize(numHalfedge);
//   thrust::for_each_n(
//       zip(thrust::make_counting_iterator(0), halfedges.cbeginD()),
//       numHalfedge, DuplicateEdges({source.ptrD(), sink.ptrD()}));
//   // Build symmetric CSR adjacency matrix
//   VecDH<int> degree(numVert, 0);
//   VecDH<int> vid(numVert);
//   VecDH<int> temp(numVert);
//   if (keep.size() > 0) {
//     edgeMask.resize(numHalfedge);
//     thrust::for_each_n(zip(thrust::make_counting_iterator(0), keep.cbeginD(),
//                            halfedges.cbeginD()),
//                        numHalfedge, DuplicateKeep({edgeMask.ptrD()}));
//     thrust::sort_by_key(zip(source.beginD(), sink.beginD()),
//                         zip(source.endD(), sink.endD()), edgeMask.beginD());
//   } else {
//     thrust::sort(zip(source.beginD(), sink.beginD()),
//                  zip(source.endD(), sink.endD()));
//   }
//   auto endPair = thrust::reduce_by_key(source.beginD(), source.endD(),
//                                        thrust::constant_iterator<int>(1),
//                                        vid.beginD(), temp.beginD());
//   thrust::scatter(temp.beginD(), endPair.second, vid.beginD(),
//   degree.beginD()); rowOffsets.resize(numVert + 1, 0);
//   thrust::inclusive_scan(degree.beginD(), degree.endD(),
//                          rowOffsets.beginD() + 1);
// }

// struct Reachable {
//   __host__ __device__ bool operator()(int x) { return x != kNvgraphInvalid; }
// };

struct NextStart : public thrust::binary_function<int, int, bool> {
  __host__ __device__ bool operator()(int value, int component) {
    // mismatch finds the point where this is false, so this is inverted.
    return !(component >= 0 && value != kInvalidInt);
  }
};

struct NextLabel {
  __host__ __device__ bool operator()(int component) { return component >= 0; }
};

struct FloodComponent {
  int value;
  int label;

  __host__ __device__ void operator()(thrust::tuple<int&, int&> inOut) {
    int& valueOut = thrust::get<0>(inOut);
    int& labelOut = thrust::get<1>(inOut);

    if (labelOut == label) {
      labelOut = -1;
      valueOut = value;
    }
  }
};

}  // namespace

namespace manifold {

int ConnectedComponents(VecDH<int>& components, int numVert,
                        const VecDH<Halfedge>& halfedges,
                        const VecDH<bool>& keep) {
#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
  // Using the CPU version of connected components even when GPU is available,
  // because it is dramatically faster.
  return ConnectedComponentsCPU(components, numVert, halfedges, keep);
#else
  return ConnectedComponentsCPU(components, numVert, halfedges, keep);
#endif
}

// int ConnectedComponentsGPU(VecDH<int>& components, int numVert,
//                            const VecDH<Halfedge>& halfedges,
//                            const VecDH<bool>& keep) {
//   VecDH<int> rowOffsets, sink, edgeMask;
//   Edges2CSR(rowOffsets, sink, edgeMask, halfedges, keep, numVert);
//   // Set up graph
//   nvgraphHandle_t handle;
//   nvgraphGraphDescr_t graph;
//   constexpr size_t vertDataSize = 2;
//   hipDataType vertDataTypes[vertDataSize];
//   constexpr size_t distIdx = 0;
//   constexpr size_t predIdx = 1;
//   vertDataTypes[distIdx] = HIP_R_32I;
//   vertDataTypes[predIdx] = HIP_R_32I;
//   CheckStatus(nvgraphCreate(&handle));
//   CheckStatus(nvgraphCreateGraphDescr(handle, &graph));
//   // Set up traversal
//   nvgraphTraversalParameter_t traversal;
//   nvgraphTraversalParameterInit(&traversal);
//   nvgraphTraversalSetDistancesIndex(&traversal, distIdx);
//   nvgraphTraversalSetPredecessorsIndex(&traversal, predIdx);
//   nvgraphTraversalSetUndirectedFlag(&traversal, false);
//   // Fill graph
//   nvgraphCSRTopology32I_st adjacencyCSR;
//   adjacencyCSR.nvertices = numVert;
//   adjacencyCSR.nedges = sink.size();
//   adjacencyCSR.source_offsets = rowOffsets.ptrD();
//   adjacencyCSR.destination_indices = sink.ptrD();
//   CheckStatus(nvgraphSetGraphStructure(
//       handle, graph, static_cast<void*>(&adjacencyCSR), NVGRAPH_CSR_32));
//   CheckStatus(
//       nvgraphAllocateVertexData(handle, graph, vertDataSize, vertDataTypes));
//   // Apply mask if given
//   if (keep.size() > 0) {
//     constexpr size_t edgeDataSize = 1, maskIdx = 0;
//     hipDataType edgeDataTypes[edgeDataSize];
//     edgeDataTypes[maskIdx] = HIP_R_32I;
//     CheckStatus(
//         nvgraphAllocateEdgeData(handle, graph, edgeDataSize, edgeDataTypes));
//     CheckStatus(nvgraphSetEdgeData(
//         handle, graph, static_cast<void*>(edgeMask.ptrD()), maskIdx));
//     CheckStatus(nvgraphTraversalSetEdgeMaskIndex(&traversal, maskIdx));
//   }
//   components.resize(numVert);
//   thrust::fill(components.beginD(), components.endD(), -1);
//   VecDH<int> distBFS(numVert);
//   int numComponent = 0;
//   while (1) {
//     // Find the first vertex that hasn't been visited
//     int sourceVert = thrust::find(components.beginD(), components.endD(), -1)
//     -
//                      components.beginD();
//     if (sourceVert >= components.size()) break;
//     // Find the sourceVert connected component using breadth-first search
//     CheckStatus(nvgraphTraversal(handle, graph, NVGRAPH_TRAVERSAL_BFS,
//                                  &sourceVert, traversal));
//     CheckStatus(nvgraphGetVertexData(
//         handle, graph, static_cast<void*>(distBFS.ptrD()), distIdx));
//     // Use numComponent as the component label
//     thrust::replace_if(components.beginD(), components.endD(),
//     distBFS.beginD(),
//                        Reachable(), numComponent++);
//   }

//   CheckStatus(nvgraphDestroyGraphDescr(handle, graph));
//   CheckStatus(nvgraphDestroy(handle));

//   return numComponent;
// }

int ConnectedComponentsCPU(VecDH<int>& components, int numVert,
                           const VecDH<Halfedge>& halfedges,
                           const VecDH<bool>& keep) {
  boost::adjacency_list<boost::vecS, boost::vecS, boost::undirectedS> graph(
      numVert);
  for (int i = 0; i < halfedges.size(); ++i) {
    const Halfedge halfedge = halfedges.H()[i];
    if (halfedge.IsForward() && (keep.size() == 0 || keep.H()[i])) {
      boost::add_edge(halfedge.startVert, halfedge.endVert, graph);
    }
  }
  components.resize(numVert);
  int numComponent = boost::connected_components(graph, components.H().data());
  return numComponent;
}

void FloodComponents(VecDH<int>& valuesInOut, VecDH<int>& componentLabels,
                     int numComponent) {
  // componentLabels will be replaced entirely with -1
  ALWAYS_ASSERT(valuesInOut.size() == componentLabels.size(), logicErr,
                "These vectors must both be NumVert long.");
  for (int comp = 0; comp < numComponent; ++comp) {
    // find first vertex in component that is also has a value
    int sourceVert = thrust::mismatch(valuesInOut.begin(), valuesInOut.end(),
                                      componentLabels.begin(), NextStart())
                         .first -
                     valuesInOut.begin();
    int label, value;
    if (sourceVert < valuesInOut.size()) {
      label = componentLabels.H()[sourceVert];
      value = valuesInOut.H()[sourceVert];
    } else {
      // If no vertices in a component have a value, then their value must be
      // zero, because zeros are removed from the sparse representation.
      sourceVert = thrust::find_if(componentLabels.begin(),
                                   componentLabels.end(), NextLabel()) -
                   componentLabels.begin();
      label = componentLabels.H()[sourceVert];
      value = 0;
      ALWAYS_ASSERT(sourceVert < componentLabels.size(), logicErr,
                    "Failed to find component!");
    }
    thrust::for_each_n(zip(valuesInOut.beginD(), componentLabels.beginD()),
                       valuesInOut.size(), FloodComponent({value, label}));
  }
}

}  // namespace manifold