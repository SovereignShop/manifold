#include "hip/hip_runtime.h"
// Copyright 2019 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <boost/config.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/connected_components.hpp>

#include "connected_components.cuh"

namespace {
using namespace manifold;

struct InitTriLabels {
  const Halfedge* halfedge;

  __host__ __device__ void operator()(thrust::tuple<int, int&> inOut) {
    const int edge = 3 * thrust::get<0>(inOut);
    int& label = thrust::get<1>(inOut);

    label = glm::min(
        glm::min(halfedge[edge].startVert, halfedge[edge + 1].startVert),
        halfedge[edge + 2].startVert);
  }
};

struct UpdateTriLabels {
  int* triLabel;
  bool* isFinished;
  const Halfedge* halfedge;
  const bool* keep;

  __host__ __device__ void operator()(int tri) {
    const int edge = 3 * tri;
    if (keep[edge] && keep[edge + 1] && keep[edge + 2]) {
      int label = triLabel[tri];
      label = glm::min(
          glm::min(glm::min(label, triLabel[halfedge[edge].pairedHalfedge / 3]),
                   triLabel[halfedge[edge + 1].pairedHalfedge / 3]),
          triLabel[halfedge[edge + 2].pairedHalfedge / 3]);

      if (label != triLabel[tri]) {
        triLabel[tri] = label;
        *isFinished = false;
      }
    }
  }
};

struct TriLabelsToVert {
  int* vertLabel;
  const Halfedge* halfedge;
  const bool* keep;

  __host__ __device__ void operator()(thrust::tuple<int, int> in) {
    const int edge = 3 * thrust::get<0>(in);
    const int label = thrust::get<1>(in);

    if (keep[edge] && keep[edge + 1] && keep[edge + 2]) {
      for (const int i : {0, 1, 2}) {
        vertLabel[halfedge[edge + i].startVert] = label;
      }
    }
  }
};

struct NextStart : public thrust::binary_function<int, int, bool> {
  __host__ __device__ bool operator()(int value, int component) {
    // mismatch finds the point where this is false, so this is inverted.
    return !(component >= 0 && value != kInvalidInt);
  }
};

struct NextLabel {
  __host__ __device__ bool operator()(int component) { return component >= 0; }
};

struct FloodComponent {
  int value;
  int label;

  __host__ __device__ void operator()(thrust::tuple<int&, int&> inOut) {
    int& valueOut = thrust::get<0>(inOut);
    int& labelOut = thrust::get<1>(inOut);

    if (labelOut == label) {
      labelOut = -1;
      valueOut = value;
    }
  }
};

}  // namespace

namespace manifold {

int ConnectedComponents(VecDH<int>& components, int numVert,
                        const VecDH<Halfedge>& halfedges,
                        const VecDH<bool>& keep) {
#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
  // Using the CPU version of connected components even when GPU is available,
  // because it is dramatically faster.
  return ConnectedComponentsCPU(components, numVert, halfedges, keep);
#else
  return ConnectedComponentsCPU(components, numVert, halfedges, keep);
#endif
}

int ConnectedComponentsGPU(VecDH<int>& components, int numVert,
                           const VecDH<Halfedge>& halfedges,
                           const VecDH<bool>& keep) {
  const int numTri = halfedges.size() / 3;
  VecDH<int> triLabel(numTri);
  thrust::for_each_n(zip(countAt(0), triLabel.beginD()), numTri,
                     InitTriLabels({halfedges.cptrD()}));

  VecDH<bool> isFinished(1, false);
  while (!isFinished.H()[0]) {
    isFinished.H()[0] = true;
    thrust::for_each_n(countAt(0), numTri,
                       UpdateTriLabels({triLabel.ptrD(), isFinished.ptrD(),
                                        halfedges.cptrD(), keep.cptrD()}));
  }

  components.resize(numVert);
  thrust::for_each_n(
      zip(countAt(0), triLabel.beginD()), numTri,
      TriLabelsToVert({components.ptrD(), halfedges.cptrD(), keep.cptrD()}));

  VecDH<int> minVerts = components;
  thrust::sort(minVerts.beginD(), minVerts.endD());
  int numComponent =
      thrust::unique(minVerts.beginD(), minVerts.endD()) - minVerts.beginD();
  return numComponent;
}

int ConnectedComponentsCPU(VecDH<int>& components, int numVert,
                           const VecDH<Halfedge>& halfedges,
                           const VecDH<bool>& keep) {
  boost::adjacency_list<boost::vecS, boost::vecS, boost::undirectedS> graph(
      numVert);
  for (int i = 0; i < halfedges.size(); ++i) {
    const Halfedge halfedge = halfedges.H()[i];
    if (halfedge.IsForward() && (keep.size() == 0 || keep.H()[i])) {
      boost::add_edge(halfedge.startVert, halfedge.endVert, graph);
    }
  }
  components.resize(numVert);
  int numComponent = boost::connected_components(graph, components.H().data());
  return numComponent;
}

void FloodComponents(VecDH<int>& valuesInOut, VecDH<int>& componentLabels,
                     int numComponent) {
  // componentLabels will be replaced entirely with -1
  ALWAYS_ASSERT(valuesInOut.size() == componentLabels.size(), logicErr,
                "These vectors must both be NumVert long.");
  for (int comp = 0; comp < numComponent; ++comp) {
    // find first vertex in component that is also has a value
    int sourceVert = thrust::mismatch(valuesInOut.begin(), valuesInOut.end(),
                                      componentLabels.begin(), NextStart())
                         .first -
                     valuesInOut.begin();
    int label, value;
    if (sourceVert < valuesInOut.size()) {
      label = componentLabels.H()[sourceVert];
      value = valuesInOut.H()[sourceVert];
    } else {
      // If no vertices in a component have a value, then their value must be
      // zero, because zeros are removed from the sparse representation.
      sourceVert = thrust::find_if(componentLabels.begin(),
                                   componentLabels.end(), NextLabel()) -
                   componentLabels.begin();
      label = componentLabels.H()[sourceVert];
      value = 0;
      ALWAYS_ASSERT(sourceVert < componentLabels.size(), logicErr,
                    "Failed to find component!");
    }
    thrust::for_each_n(zip(valuesInOut.beginD(), componentLabels.beginD()),
                       valuesInOut.size(), FloodComponent({value, label}));
  }
}

}  // namespace manifold