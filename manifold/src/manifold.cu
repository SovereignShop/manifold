#include "hip/hip_runtime.h"
// Copyright 2020 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/sequence.h>
#include <thrust/transform_reduce.h>

#include "boolean3.cuh"
#include "connected_components.cuh"
#include "manifold_impl.cuh"
#include "polygon.h"

namespace {
using namespace manifold;
using namespace thrust::placeholders;

struct ToSphere {
  float length;
  __host__ __device__ void operator()(glm::vec3& v) {
    v = glm::cos(glm::half_pi<float>() * (1.0f - v));
    v = length * glm::normalize(v);
    if (isnan(v.x)) v = glm::vec3(0.0);
  }
};

struct UpdateHalfedge {
  const int nextVert;
  const int nextEdge;
  const int nextFace;

  __host__ __device__ Halfedge operator()(Halfedge edge) {
    edge.startVert += nextVert;
    edge.endVert += nextVert;
    edge.pairedHalfedge += nextEdge;
    edge.face += nextFace;
    return edge;
  }
};

struct Equals {
  int val;
  __host__ __device__ bool operator()(int x) { return x == val; }
};

struct RemoveFace {
  const Halfedge* halfedge;
  const int* vertLabel;
  const int keepLabel;

  __host__ __device__ bool operator()(int face) {
    return vertLabel[halfedge[3 * face].startVert] != keepLabel;
  }
};

struct MakeTri {
  const Halfedge* halfedges;

  __host__ __device__ void operator()(thrust::tuple<glm::ivec3&, int> inOut) {
    glm::ivec3& tri = thrust::get<0>(inOut);
    const int face = 3 * thrust::get<1>(inOut);

    for (int i : {0, 1, 2}) {
      tri[i] = halfedges[face + i].startVert;
    }
  }
};
}  // namespace

namespace manifold {

Manifold::Manifold() : pImpl_{std::make_unique<Impl>()} {}
Manifold::Manifold(const Mesh& manifold)
    : pImpl_{std::make_unique<Impl>(manifold)} {}
Manifold::~Manifold() = default;
Manifold::Manifold(Manifold&&) noexcept = default;
Manifold& Manifold::operator=(Manifold&&) noexcept = default;

Manifold::Manifold(const Manifold& other) : pImpl_(new Impl(*other.pImpl_)) {}

Manifold& Manifold::operator=(const Manifold& other) {
  if (this != &other) {
    pImpl_.reset(new Impl(*other.pImpl_));
  }
  return *this;
}

/**
 * Constructs a tetrahedron centered at the origin with one vertex at (1,1,1)
 * and the rest at similarly symmetric points.
 */
Manifold Manifold::Tetrahedron() {
  Manifold tetrahedron;
  tetrahedron.pImpl_ = std::make_unique<Impl>(Impl::Shape::TETRAHEDRON);
  return tetrahedron;
}

/**
 * Constructs an octahedron centered at the origin with vertices one unit out
 * along each axis.
 */
Manifold Manifold::Octahedron() {
  Manifold octahedron;
  octahedron.pImpl_ = std::make_unique<Impl>(Impl::Shape::OCTAHEDRON);
  return octahedron;
}

/**
 * Constructs a unit cube (edge lengths all one), by default in the first
 * octant, touching the origin. Set center to true to shift the center to the
 * origin.
 */
Manifold Manifold::Cube(glm::vec3 size, bool center) {
  Manifold cube;
  cube.pImpl_ = std::make_unique<Impl>(Impl::Shape::CUBE);
  cube.Scale(size);
  if (center) cube.Translate(-size / 2.0f);
  return cube;
}

/**
 * A convenience constructor for the common case of extruding a circle. Can also
 * form cones if both radii are specified. Set center to true to center the
 * manifold vertically on the origin (default places the bottom on the origin).
 */
Manifold Manifold::Cylinder(float height, float radiusLow, float radiusHigh,
                            int circularSegments, bool center) {
  float scale = radiusHigh >= 0.0f ? radiusHigh / radiusLow : 1.0f;
  float radius = max(radiusLow, radiusHigh);
  int n = circularSegments > 2 ? circularSegments : GetCircularSegments(radius);
  Polygons circle(1);
  float dPhi = 360.0f / n;
  for (int i = 0; i < n; ++i) {
    circle[0].push_back(
        {radiusLow * glm::vec2(cosd(dPhi * i), sind(dPhi * i)), 0});
  }
  Manifold cylinder =
      Manifold::Extrude(circle, height, 0, 0.0f, glm::vec2(scale));
  if (center) cylinder.Translate(glm::vec3(0.0f, 0.0f, -height / 2.0f));
  return cylinder;
}

/**
 * Constructs a sphere of a given radius and number of segments along its
 * diameter. This number will always be rounded up to the nearest factor of
 * four, as this sphere is constructed by refining an octahedron. This means
 * there are a circle of vertices on all three of the axis planes.
 */
Manifold Manifold::Sphere(float radius, int circularSegments) {
  int n = circularSegments > 0 ? (circularSegments + 3) / 4
                               : GetCircularSegments(radius) / 4;
  Manifold sphere;
  sphere.pImpl_ = std::make_unique<Impl>(Impl::Shape::OCTAHEDRON);
  sphere.pImpl_->Refine(n);
  thrust::for_each_n(sphere.pImpl_->vertPos_.beginD(), sphere.NumVert(),
                     ToSphere({radius}));
  sphere.pImpl_->Finish();
  return sphere;
}

/**
 * Constructs a manifold from a set of polygons by extruding them along the
 * Z-axis. The overall height and the scale at the top (X and Y independently)
 * can be specified, as can a twist, to be applied linearly. In the case of
 * twist, it can also be helpful to specify nDivisions, which specifies the
 * quantization of the triangles vertically. If the scale is {0,0}, a pure cone
 * is formed with only a single vertex at the top.
 */
Manifold Manifold::Extrude(Polygons crossSection, float height, int nDivisions,
                           float twistDegrees, glm::vec2 scaleTop) {
  ALWAYS_ASSERT(scaleTop.x >= 0 && scaleTop.y >= 0, userErr,
                "scale values cannot be negative");
  Manifold extrusion;
  ++nDivisions;
  auto& vertPos = extrusion.pImpl_->vertPos_.H();
  VecDH<glm::ivec3> triVertsDH;
  auto& triVerts = triVertsDH.H();
  int nCrossSection = 0;
  bool isCone = scaleTop.x == 0.0 && scaleTop.y == 0.0;
  int idx = 0;
  for (auto& poly : crossSection) {
    nCrossSection += poly.size();
    for (PolyVert& polyVert : poly) {
      vertPos.push_back({polyVert.pos.x, polyVert.pos.y, 0.0f});
      polyVert.idx = idx++;
    }
  }
  for (int i = 1; i < nDivisions + 1; ++i) {
    float alpha = i / float(nDivisions);
    float phi = alpha * twistDegrees;
    glm::mat2 transform(cosd(phi), sind(phi), -sind(phi), cosd(phi));
    glm::vec2 scale = glm::mix(glm::vec2(1.0f), scaleTop, alpha);
    transform = transform * glm::mat2(scale.x, 0.0f, 0.0f, scale.y);
    int j = 0;
    int idx = 0;
    for (const auto& poly : crossSection) {
      for (int vert = 0; vert < poly.size(); ++vert) {
        int offset = idx + nCrossSection * i;
        int thisVert = vert + offset;
        int lastVert = (vert == 0 ? poly.size() : vert) - 1 + offset;
        if (i == nDivisions && isCone) {
          triVerts.push_back({nCrossSection * i + j, lastVert - nCrossSection,
                              thisVert - nCrossSection});
        } else {
          glm::vec2 pos = transform * poly[vert].pos;
          vertPos.push_back({pos.x, pos.y, height * alpha});
          triVerts.push_back({thisVert, lastVert, thisVert - nCrossSection});
          triVerts.push_back(
              {lastVert, lastVert - nCrossSection, thisVert - nCrossSection});
        }
      }
      ++j;
      idx += poly.size();
    }
  }
  if (isCone)
    for (int j = 0; j < crossSection.size(); ++j)  // Duplicate vertex for Genus
      vertPos.push_back({0.0f, 0.0f, height});
  std::vector<glm::ivec3> top = Triangulate(crossSection);
  for (const glm::ivec3& tri : top) {
    triVerts.push_back({tri[0], tri[2], tri[1]});
    if (!isCone) triVerts.push_back(tri + nCrossSection * nDivisions);
  }

  extrusion.pImpl_->CreateHalfedges(triVertsDH);
  extrusion.pImpl_->Finish();
  return extrusion;
}

/**
 * Constructs a manifold from a set of polygons by revolving this cross-section
 * around its Y-axis and then setting this as the Z-axis of the resulting
 * manifold. If the polygons cross the Y-axis, only the part on the positive X
 * side is used. Geometrically valid input will result in geometrically valid
 * output.
 */
Manifold Manifold::Revolve(const Polygons& crossSection, int circularSegments) {
  float radius = 0.0f;
  for (const auto& poly : crossSection) {
    for (const auto& vert : poly) {
      radius = max(radius, vert.pos.x);
    }
  }
  int nDivisions =
      circularSegments > 2 ? circularSegments : GetCircularSegments(radius);
  Manifold revoloid;
  auto& vertPos = revoloid.pImpl_->vertPos_.H();
  VecDH<glm::ivec3> triVertsDH;
  auto& triVerts = triVertsDH.H();
  float dPhi = 360.0f / nDivisions;
  for (const auto& poly : crossSection) {
    int start = -1;
    for (int polyVert = 0; polyVert < poly.size(); ++polyVert) {
      if (poly[polyVert].pos.x <= 0) {
        start = polyVert;
        break;
      }
    }
    if (start == -1) {  // poly all positive
      for (int polyVert = 0; polyVert < poly.size(); ++polyVert) {
        int startVert = vertPos.size();
        int lastStart =
            startVert +
            (polyVert == 0 ? nDivisions * (poly.size() - 1) : -nDivisions);
        for (int slice = 0; slice < nDivisions; ++slice) {
          int lastSlice = (slice == 0 ? nDivisions : slice) - 1;
          float phi = slice * dPhi;
          glm::vec2 pos = poly[polyVert].pos;
          vertPos.push_back({pos.x * cosd(phi), pos.x * sind(phi), pos.y});
          triVerts.push_back({startVert + slice, startVert + lastSlice,
                              lastStart + lastSlice});
          triVerts.push_back(
              {lastStart + lastSlice, lastStart + slice, startVert + slice});
        }
      }
    } else {  // poly crosses zero
      int polyVert = start;
      glm::vec2 pos = poly[polyVert].pos;
      do {
        glm::vec2 lastPos = pos;
        polyVert = (polyVert + 1) % poly.size();
        pos = poly[polyVert].pos;
        if (pos.x > 0) {
          if (lastPos.x <= 0) {
            float a = pos.x / (pos.x - lastPos.x);
            vertPos.push_back({0.0f, 0.0f, glm::mix(pos.y, lastPos.y, a)});
          }
          int startVert = vertPos.size();
          for (int slice = 0; slice < nDivisions; ++slice) {
            int lastSlice = (slice == 0 ? nDivisions : slice) - 1;
            float phi = slice * dPhi;
            glm::vec2 pos = poly[polyVert].pos;
            vertPos.push_back({pos.x * cosd(phi), pos.x * sind(phi), pos.y});
            if (lastPos.x > 0) {
              triVerts.push_back({startVert + slice, startVert + lastSlice,
                                  startVert - nDivisions + lastSlice});
              triVerts.push_back({startVert - nDivisions + lastSlice,
                                  startVert - nDivisions + slice,
                                  startVert + slice});
            } else {
              triVerts.push_back(
                  {startVert - 1, startVert + slice, startVert + lastSlice});
            }
          }
        } else if (lastPos.x > 0) {
          int startVert = vertPos.size();
          float a = pos.x / (pos.x - lastPos.x);
          vertPos.push_back({0.0f, 0.0f, glm::mix(pos.y, lastPos.y, a)});
          for (int slice = 0; slice < nDivisions; ++slice) {
            int lastSlice = (slice == 0 ? nDivisions : slice) - 1;
            triVerts.push_back({startVert, startVert - nDivisions + lastSlice,
                                startVert - nDivisions + slice});
          }
        }
      } while (polyVert != start);
    }
  }

  revoloid.pImpl_->CreateHalfedges(triVertsDH);
  revoloid.pImpl_->Finish();
  return revoloid;
}

/**
 * Constructs a new manifold from a vector of other manifolds. This is a purely
 * topological operation, so care should be taken to avoid creating
 * geometrically-invalid results.
 */
Manifold Manifold::Compose(const std::vector<Manifold>& manifolds) {
  int numVert = 0;
  int numEdge = 0;
  int NumTri = 0;
  for (const Manifold& manifold : manifolds) {
    numVert += manifold.NumVert();
    numEdge += manifold.NumEdge();
    NumTri += manifold.NumTri();
  }

  Manifold out;
  Impl& combined = *(out.pImpl_);
  combined.vertPos_.resize(numVert);
  combined.halfedge_.resize(2 * numEdge);
  combined.faceNormal_.resize(NumTri);

  int nextVert = 0;
  int nextEdge = 0;
  int nextFace = 0;
  for (const Manifold& manifold : manifolds) {
    const Impl& impl = *(manifold.pImpl_);
    impl.ApplyTransform();

    thrust::copy(impl.vertPos_.beginD(), impl.vertPos_.endD(),
                 combined.vertPos_.beginD() + nextVert);
    thrust::copy(impl.faceNormal_.beginD(), impl.faceNormal_.endD(),
                 combined.faceNormal_.beginD() + nextFace);
    thrust::transform(impl.halfedge_.beginD(), impl.halfedge_.endD(),
                      combined.halfedge_.beginD() + nextEdge,
                      UpdateHalfedge({nextVert, nextEdge, nextFace}));

    nextVert += manifold.NumVert();
    nextEdge += 2 * manifold.NumEdge();
    nextFace += manifold.NumTri();
  }

  combined.Finish();
  return out;
}

/**
 * This operation returns a copy of this manifold, but as a vector of meshes
 * that are topologically disconnected.
 */
std::vector<Manifold> Manifold::Decompose() const {
  VecDH<int> vertLabel;
  int numLabel = ConnectedComponents(vertLabel, NumVert(), pImpl_->halfedge_);

  if (numLabel == 1) {
    std::vector<Manifold> meshes(1);
    meshes[0] = *this;
    return meshes;
  }

  std::vector<Manifold> meshes(numLabel);
  for (int i = 0; i < numLabel; ++i) {
    meshes[i].pImpl_->vertPos_.resize(NumVert());
    VecDH<int> vertNew2Old(NumVert());
    int nVert =
        thrust::copy_if(
            zip(pImpl_->vertPos_.beginD(), countAt(0)),
            zip(pImpl_->vertPos_.endD(), countAt(NumVert())),
            vertLabel.beginD(),
            zip(meshes[i].pImpl_->vertPos_.beginD(), vertNew2Old.beginD()),
            Equals({i})) -
        zip(meshes[i].pImpl_->vertPos_.beginD(), countAt(0));
    meshes[i].pImpl_->vertPos_.resize(nVert);

    VecDH<int> faceNew2Old(NumTri());
    thrust::sequence(faceNew2Old.beginD(), faceNew2Old.endD());

    int nFace =
        thrust::remove_if(
            faceNew2Old.beginD(), faceNew2Old.endD(),
            RemoveFace({pImpl_->halfedge_.cptrD(), vertLabel.cptrD(), i})) -
        faceNew2Old.beginD();
    faceNew2Old.resize(nFace);

    meshes[i].pImpl_->GatherFaces(pImpl_->halfedge_, faceNew2Old);
    meshes[i].pImpl_->ReindexVerts(vertNew2Old, pImpl_->NumVert());

    meshes[i].pImpl_->Finish();
    meshes[i].pImpl_->transform_ = pImpl_->transform_;
  }
  return meshes;
}

/**
 * This returns a Mesh of simple vectors of vertices and triangles suitable for
 * saving or other operations outside of the context of this library.
 */
Mesh Manifold::Extract(bool includeNormals) const {
  pImpl_->ApplyTransform();

  Mesh result;
  result.vertPos.insert(result.vertPos.end(), pImpl_->vertPos_.begin(),
                        pImpl_->vertPos_.end());
  if (includeNormals) {
    result.vertNormal.insert(result.vertNormal.end(),
                             pImpl_->vertNormal_.begin(),
                             pImpl_->vertNormal_.end());
  }

  result.triVerts.resize(NumTri());
  thrust::for_each_n(zip(result.triVerts.begin(), countAt(0)), NumTri(),
                     MakeTri({pImpl_->halfedge_.cptrH()}));

  return result;
}

/**
 * These static properties control how circular shapes are quantized by default
 * on construction. If circularSegments is specified, it takes precedence. If it
 * is zero, then instead the minimum is used of the segments calculated based on
 * edge length and angle, rounded up to the nearest multiple of four. To get
 * numbers not divisible by four, circularSegements must be specified.
 */
int Manifold::circularSegments = 0;
float Manifold::circularAngle = 10.0f;
float Manifold::circularEdgeLength = 1.0f;

void Manifold::SetMinCircularAngle(float angle) {
  ALWAYS_ASSERT(angle > 0.0f, userErr, "angle must be positive!");
  Manifold::circularAngle = angle;
}

void Manifold::SetMinCircularEdgeLength(float length) {
  ALWAYS_ASSERT(length > 0.0f, userErr, "length must be positive!");
  Manifold::circularEdgeLength = length;
}

void Manifold::SetCircularSegments(int number) {
  ALWAYS_ASSERT(number > 2 || number == 0, userErr,
                "must have at least three segments in circle!");
  Manifold::circularSegments = number;
}

int Manifold::GetCircularSegments(float radius) {
  if (Manifold::circularSegments > 0) return Manifold::circularSegments;
  int nSegA = 360.0f / Manifold::circularAngle;
  int nSegL = 2.0f * radius * glm::pi<float>() / Manifold::circularEdgeLength;
  int nSeg = min(nSegA, nSegL) + 3;
  nSeg -= nSeg % 4;
  return nSeg;
}

bool Manifold::IsEmpty() const { return pImpl_->IsEmpty(); }
int Manifold::NumVert() const { return pImpl_->NumVert(); }
int Manifold::NumEdge() const { return pImpl_->NumEdge(); }
int Manifold::NumTri() const { return pImpl_->NumTri(); }

Box Manifold::BoundingBox() const {
  return pImpl_->bBox_.Transform(pImpl_->transform_);
}

float Manifold::Precision() const {
  pImpl_->ApplyTransform();
  return pImpl_->precision_;
}

/**
 * The genus is a topological property of the manifold, representing the number
 * of "handles". A sphere is 0, torus 1, etc. It is only meaningful for a single
 * mesh, so it is best to call Decompose() first.
 */
int Manifold::Genus() const {
  int chi = NumVert() - NumEdge() + NumTri();
  return 1 - chi / 2;
}

Manifold::Properties Manifold::GetProperties() const {
  return pImpl_->GetProperties();
}

bool Manifold::IsManifold() const { return pImpl_->IsManifold(); }

bool Manifold::MatchesTriNormals() const { return pImpl_->MatchesTriNormals(); }

Manifold& Manifold::Translate(glm::vec3 v) {
  pImpl_->transform_[3] += v;
  return *this;
}

Manifold& Manifold::Scale(glm::vec3 v) {
  glm::mat3 s(1.0f);
  for (int i : {0, 1, 2}) s[i] *= v;
  pImpl_->transform_ = s * pImpl_->transform_;
  return *this;
}

/**
 * Applys an Euler angle rotation to the manifold, first about the X axis, then
 * Y, then Z, in degrees. We use degrees so that we can minimize rounding error,
 * and elimiate it completely for any multiples of 90 degrees. Addtionally, more
 * efficient code paths are used to update the manifold when the transforms only
 * rotate by multiples of 90 degrees.
 */
Manifold& Manifold::Rotate(float xDegrees, float yDegrees, float zDegrees) {
  glm::mat3 rX(1.0f, 0.0f, 0.0f,                      //
               0.0f, cosd(xDegrees), sind(xDegrees),  //
               0.0f, -sind(xDegrees), cosd(xDegrees));
  glm::mat3 rY(cosd(yDegrees), 0.0f, -sind(yDegrees),  //
               0.0f, 1.0f, 0.0f,                       //
               sind(yDegrees), 0.0f, cosd(yDegrees));
  glm::mat3 rZ(cosd(zDegrees), sind(zDegrees), 0.0f,   //
               -sind(zDegrees), cosd(zDegrees), 0.0f,  //
               0.0f, 0.0f, 1.0f);
  pImpl_->transform_ = rZ * rY * rX * pImpl_->transform_;
  return *this;
}

/**
 * This function does not change the topology, but allows the vertices to be
 * moved according to any arbitrary input function. It is easy to create a
 * function that warps a geometrically valid object into one with is not, but
 * that is not checked here, so it is up to the user to choose their function
 * with discretion.
 */
Manifold& Manifold::Warp(std::function<void(glm::vec3&)> warpFunc) {
  pImpl_->ApplyTransform();
  thrust::for_each_n(pImpl_->vertPos_.begin(), NumVert(), warpFunc);
  pImpl_->Update();
  pImpl_->faceNormal_.resize(0);  // force recalculation of triNormal
  pImpl_->CalculateNormals();
  pImpl_->SetPrecision();
  return *this;
}

/**
 * This is a checksum-style verification of the collider, simply returning the
 * total number of edge-face bounding box overlaps between this and other.
 */
int Manifold::NumOverlaps(const Manifold& other) const {
  pImpl_->ApplyTransform();
  other.pImpl_->ApplyTransform();

  SparseIndices overlaps = pImpl_->EdgeCollisions(*other.pImpl_);
  int num_overlaps = overlaps.size();

  overlaps = other.pImpl_->EdgeCollisions(*pImpl_);
  return num_overlaps += overlaps.size();
}

Manifold Manifold::Boolean(const Manifold& second, OpType op) const {
  pImpl_->ApplyTransform();
  second.pImpl_->ApplyTransform();
  Boolean3 boolean(*pImpl_, *second.pImpl_, op);
  Manifold result;
  result.pImpl_ = std::make_unique<Impl>(boolean.Result(op));
  return result;
}

Manifold Manifold::operator+(const Manifold& Q) const {
  return Boolean(Q, OpType::ADD);
}

Manifold& Manifold::operator+=(const Manifold& Q) {
  *this = *this + Q;
  return *this;
}

Manifold Manifold::operator-(const Manifold& Q) const {
  return Boolean(Q, OpType::SUBTRACT);
}

Manifold& Manifold::operator-=(const Manifold& Q) {
  *this = *this - Q;
  return *this;
}

Manifold Manifold::operator^(const Manifold& Q) const {
  return Boolean(Q, OpType::INTERSECT);
}

Manifold& Manifold::operator^=(const Manifold& Q) {
  *this = *this ^ Q;
  return *this;
}

std::pair<Manifold, Manifold> Manifold::Split(const Manifold& cutter) const {
  pImpl_->ApplyTransform();
  cutter.pImpl_->ApplyTransform();
  Boolean3 boolean(*pImpl_, *cutter.pImpl_, OpType::SUBTRACT);
  std::pair<Manifold, Manifold> result;
  result.first.pImpl_ =
      std::make_unique<Impl>(boolean.Result(OpType::INTERSECT));
  result.second.pImpl_ =
      std::make_unique<Impl>(boolean.Result(OpType::SUBTRACT));
  return result;
}

std::pair<Manifold, Manifold> Manifold::SplitByPlane(glm::vec3 normal,
                                                     float originOffset) const {
  normal = glm::normalize(normal);
  Manifold cutter =
      Manifold::Cube(glm::vec3(2.0f), true).Translate({1.0f, 0.0f, 0.0f});
  float size = glm::length(BoundingBox().Center() - normal * originOffset) +
               0.5f * glm::length(BoundingBox().Size());
  cutter.Scale(glm::vec3(size)).Translate({originOffset, 0.0f, 0.0f});
  float yDeg = glm::degrees(-glm::asin(normal.z));
  float zDeg = glm::degrees(glm::atan(normal.y, normal.x));
  cutter.Rotate(0.0f, yDeg, zDeg);
  return Split(cutter);
}
}  // namespace manifold