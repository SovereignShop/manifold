#include "hip/hip_runtime.h"
// Copyright 2021 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/adjacent_difference.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>
#include <thrust/gather.h>
#include <thrust/logical.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>

#include <algorithm>
#include <map>
#include <stack>

#include "impl.cuh"

namespace {
using namespace manifold;

constexpr uint32_t kNoCode = 0xFFFFFFFFu;

__host__ __device__ void AtomicAddVec3(glm::vec3& target,
                                       const glm::vec3& add) {
  for (int i : {0, 1, 2}) {
#ifdef __CUDA_ARCH__
    atomicAdd(&target[i], add[i]);
#else
#pragma omp atomic
    target[i] += add[i];
#endif
  }
}

struct Normalize {
  __host__ __device__ void operator()(glm::vec3& v) { v = SafeNormalize(v); }
};

struct FaceAreaVolume {
  const Halfedge* halfedges;
  const glm::vec3* vertPos;
  const float precision;

  __host__ __device__ thrust::pair<float, float> operator()(int face) {
    float perimeter = 0;
    glm::vec3 edge[3];
    for (int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      edge[i] = vertPos[halfedges[3 * face + j].startVert] -
                vertPos[halfedges[3 * face + i].startVert];
      perimeter += glm::length(edge[i]);
    }
    glm::vec3 crossP = glm::cross(edge[0], edge[1]);

    float area = glm::length(crossP);
    float volume = glm::dot(crossP, vertPos[halfedges[3 * face].startVert]);

    return area > perimeter * precision
               ? thrust::make_pair(area / 2.0f, volume / 6.0f)
               : thrust::make_pair(0.0f, 0.0f);
  }
};

struct Extrema : public thrust::binary_function<Halfedge, Halfedge, Halfedge> {
  __host__ __device__ void MakeForward(Halfedge& a) {
    if (!a.IsForward()) {
      int tmp = a.startVert;
      a.startVert = a.endVert;
      a.endVert = tmp;
    }
  }

  __host__ __device__ int MaxOrMinus(int a, int b) {
    return glm::min(a, b) < 0 ? -1 : glm::max(a, b);
  }

  __host__ __device__ Halfedge operator()(Halfedge a, Halfedge b) {
    MakeForward(a);
    MakeForward(b);
    a.startVert = glm::min(a.startVert, b.startVert);
    a.endVert = glm::max(a.endVert, b.endVert);
    a.face = MaxOrMinus(a.face, b.face);
    a.pairedHalfedge = MaxOrMinus(a.pairedHalfedge, b.pairedHalfedge);
    return a;
  }
};

struct PosMin
    : public thrust::binary_function<glm::vec3, glm::vec3, glm::vec3> {
  __host__ __device__ glm::vec3 operator()(glm::vec3 a, glm::vec3 b) {
    if (isnan(a.x)) return b;
    if (isnan(b.x)) return a;
    return glm::min(a, b);
  }
};

struct PosMax
    : public thrust::binary_function<glm::vec3, glm::vec3, glm::vec3> {
  __host__ __device__ glm::vec3 operator()(glm::vec3 a, glm::vec3 b) {
    if (isnan(a.x)) return b;
    if (isnan(b.x)) return a;
    return glm::max(a, b);
  }
};

struct SumPair : public thrust::binary_function<thrust::pair<float, float>,
                                                thrust::pair<float, float>,
                                                thrust::pair<float, float>> {
  __host__ __device__ thrust::pair<float, float> operator()(
      thrust::pair<float, float> a, thrust::pair<float, float> b) {
    a.first += b.first;
    a.second += b.second;
    return a;
  }
};

struct CurvatureAngles {
  float* meanCurvature;
  float* gaussianCurvature;
  float* area;
  float* degree;
  const Halfedge* halfedge;
  const glm::vec3* vertPos;
  const glm::vec3* triNormal;

  __host__ __device__ void operator()(int tri) {
    glm::vec3 edge[3];
    glm::vec3 edgeLength;
    for (int i : {0, 1, 2}) {
      const int startVert = halfedge[3 * tri + i].startVert;
      const int endVert = halfedge[3 * tri + i].endVert;
      edge[i] = vertPos[endVert] - vertPos[startVert];
      edgeLength[i] = glm::length(edge[i]);
      edge[i] /= edgeLength[i];
      const int neighborTri = halfedge[3 * tri + i].pairedHalfedge / 3;
      const float dihedral =
          0.25 * edgeLength[i] *
          glm::asin(glm::dot(glm::cross(triNormal[tri], triNormal[neighborTri]),
                             edge[i]));
      AtomicAdd(meanCurvature[startVert], dihedral);
      AtomicAdd(meanCurvature[endVert], dihedral);
      AtomicAdd(degree[startVert], 1.0f);
    }

    glm::vec3 phi;
    phi[0] = glm::acos(-glm::dot(edge[2], edge[0]));
    phi[1] = glm::acos(-glm::dot(edge[0], edge[1]));
    phi[2] = glm::pi<float>() - phi[0] - phi[1];
    const float area3 = edgeLength[0] * edgeLength[1] *
                        glm::length(glm::cross(edge[0], edge[1])) / 6;

    for (int i : {0, 1, 2}) {
      const int vert = halfedge[3 * tri + i].startVert;
      AtomicAdd(gaussianCurvature[vert], -phi[i]);
      AtomicAdd(area[vert], area3);
    }
  }
};

struct NormalizeCurvature {
  __host__ __device__ void operator()(
      thrust::tuple<float&, float&, float, float> inOut) {
    float& meanCurvature = thrust::get<0>(inOut);
    float& gaussianCurvature = thrust::get<1>(inOut);
    float area = thrust::get<2>(inOut);
    float degree = thrust::get<3>(inOut);
    float factor = degree / (6 * area);
    meanCurvature *= factor;
    gaussianCurvature *= factor;
  }
};

struct Transform4x3 {
  const glm::mat4x3 transform;

  __host__ __device__ void operator()(glm::vec3& position) {
    position = transform * glm::vec4(position, 1.0f);
  }
};

struct TransformNormals {
  const glm::mat3 transform;

  __host__ __device__ void operator()(glm::vec3& normal) {
    normal = glm::normalize(transform * normal);
    if (isnan(normal.x)) normal = glm::vec3(0.0f);
  }
};

__host__ __device__ uint32_t SpreadBits3(uint32_t v) {
  v = 0xFF0000FFu & (v * 0x00010001u);
  v = 0x0F00F00Fu & (v * 0x00000101u);
  v = 0xC30C30C3u & (v * 0x00000011u);
  v = 0x49249249u & (v * 0x00000005u);
  return v;
}

__host__ __device__ uint32_t MortonCode(glm::vec3 position, Box bBox) {
  // Unreferenced vertices are marked NaN, and this will sort them to the end
  // (the Morton code only uses the first 30 of 32 bits).
  if (isnan(position.x)) return kNoCode;

  glm::vec3 xyz = (position - bBox.min) / (bBox.max - bBox.min);
  xyz = glm::min(glm::vec3(1023.0f), glm::max(glm::vec3(0.0f), 1024.0f * xyz));
  uint32_t x = SpreadBits3(static_cast<uint32_t>(xyz.x));
  uint32_t y = SpreadBits3(static_cast<uint32_t>(xyz.y));
  uint32_t z = SpreadBits3(static_cast<uint32_t>(xyz.z));
  return x * 4 + y * 2 + z;
}

struct Morton {
  const Box bBox;

  __host__ __device__ void operator()(
      thrust::tuple<uint32_t&, const glm::vec3&> inout) {
    glm::vec3 position = thrust::get<1>(inout);
    thrust::get<0>(inout) = MortonCode(position, bBox);
  }
};

struct FaceMortonBox {
  const Halfedge* halfedge;
  const glm::vec3* vertPos;
  const Box bBox;

  __host__ __device__ void operator()(
      thrust::tuple<uint32_t&, Box&, int> inout) {
    uint32_t& mortonCode = thrust::get<0>(inout);
    Box& faceBox = thrust::get<1>(inout);
    int face = thrust::get<2>(inout);

    // Removed tris are marked by all halfedges having pairedHalfedge = -1, and
    // this will sort them to the end (the Morton code only uses the first 30 of
    // 32 bits).
    if (halfedge[3 * face].pairedHalfedge < 0) {
      mortonCode = kNoCode;
      return;
    }

    glm::vec3 center(0.0f);

    for (const int i : {0, 1, 2}) {
      const glm::vec3 pos = vertPos[halfedge[3 * face + i].startVert];
      center += pos;
      faceBox.Union(pos);
    }
    center /= 3;

    mortonCode = MortonCode(center, bBox);
  }
};

struct Reindex {
  const int* indexInv;

  __host__ __device__ void operator()(Halfedge& edge) {
    if (edge.startVert < 0) return;
    edge.startVert = indexInv[edge.startVert];
    edge.endVert = indexInv[edge.endVert];
  }
};

template <typename T>
void Permute(VecDH<T>& inOut, const VecDH<int>& new2Old) {
  VecDH<T> tmp(inOut);
  inOut.resize(new2Old.size());
  thrust::gather(new2Old.beginD(), new2Old.endD(), tmp.beginD(),
                 inOut.beginD());
}

template void Permute<BaryRef>(VecDH<BaryRef>&, const VecDH<int>&);
template void Permute<glm::vec3>(VecDH<glm::vec3>&, const VecDH<int>&);

struct ReindexFace {
  Halfedge* halfedge;
  glm::vec4* halfedgeTangent;
  const Halfedge* oldHalfedge;
  const glm::vec4* oldHalfedgeTangent;
  const int* faceNew2Old;
  const int* faceOld2New;

  __host__ __device__ void operator()(int newFace) {
    const int oldFace = faceNew2Old[newFace];
    for (const int i : {0, 1, 2}) {
      const int oldEdge = 3 * oldFace + i;
      Halfedge edge = oldHalfedge[oldEdge];
      edge.face = newFace;
      const int pairedFace = edge.pairedHalfedge / 3;
      const int offset = edge.pairedHalfedge - 3 * pairedFace;
      edge.pairedHalfedge = 3 * faceOld2New[pairedFace] + offset;
      const int newEdge = 3 * newFace + i;
      halfedge[newEdge] = edge;
      if (oldHalfedgeTangent != nullptr) {
        halfedgeTangent[newEdge] = oldHalfedgeTangent[oldEdge];
      }
    }
  }
};

struct AssignNormals {
  glm::vec3* vertNormal;
  const glm::vec3* vertPos;
  const Halfedge* halfedges;
  const float precision;
  const bool calculateTriNormal;

  __host__ __device__ void operator()(thrust::tuple<glm::vec3&, int> in) {
    glm::vec3& triNormal = thrust::get<0>(in);
    const int face = thrust::get<1>(in);

    glm::ivec3 triVerts;
    for (int i : {0, 1, 2}) triVerts[i] = halfedges[3 * face + i].startVert;

    glm::vec3 edge[3];
    for (int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      edge[i] = glm::normalize(vertPos[triVerts[j]] - vertPos[triVerts[i]]);
    }

    if (calculateTriNormal) {
      triNormal = glm::normalize(glm::cross(edge[0], edge[1]));
      if (isnan(triNormal.x)) triNormal = glm::vec3(0, 0, 1);
    }

    // corner angles
    glm::vec3 phi;
    float dot = -glm::dot(edge[2], edge[0]);
    phi[0] = dot >= 1 ? 0 : (dot <= -1 ? glm::pi<float>() : glm::acos(dot));
    dot = -glm::dot(edge[0], edge[1]);
    phi[1] = dot >= 1 ? 0 : (dot <= -1 ? glm::pi<float>() : glm::acos(dot));
    phi[2] = glm::pi<float>() - phi[0] - phi[1];

    // assign weighted sum
    for (int i : {0, 1, 2}) {
      AtomicAddVec3(vertNormal[triVerts[i]], phi[i] * triNormal);
    }
  }
};

struct Tri2Halfedges {
  Halfedge* halfedges;
  TmpEdge* edges;

  __host__ __device__ void operator()(
      thrust::tuple<int, const glm::ivec3&> in) {
    const int tri = thrust::get<0>(in);
    const glm::ivec3& triVerts = thrust::get<1>(in);
    for (const int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      const int edge = 3 * tri + i;
      halfedges[edge] = {triVerts[i], triVerts[j], -1, tri};
      edges[edge] = TmpEdge(triVerts[i], triVerts[j], edge);
    }
  }
};

struct LinkHalfedges {
  Halfedge* halfedges;
  const TmpEdge* edges;

  __host__ __device__ void operator()(int k) {
    const int i = 2 * k;
    const int j = i + 1;
    const int pair0 = edges[i].halfedgeIdx;
    const int pair1 = edges[j].halfedgeIdx;
    if (halfedges[pair0].startVert != halfedges[pair1].endVert ||
        halfedges[pair0].endVert != halfedges[pair1].startVert ||
        halfedges[pair0].face == halfedges[pair1].face)
      printf("Not manifold!\n");
    halfedges[pair0].pairedHalfedge = pair1;
    halfedges[pair1].pairedHalfedge = pair0;
  }
};

struct SwapHalfedges {
  Halfedge* halfedges;
  const TmpEdge* edges;

  __host__ void operator()(int k) {
    const int i = 2 * k;
    const int j = i - 2;
    const TmpEdge thisEdge = edges[i];
    const TmpEdge lastEdge = edges[j];
    if (thisEdge.first == lastEdge.first &&
        thisEdge.second == lastEdge.second) {
      const int swap0idx = thisEdge.halfedgeIdx;
      Halfedge& swap0 = halfedges[swap0idx];
      const int swap1idx = swap0.pairedHalfedge;
      Halfedge& swap1 = halfedges[swap1idx];

      const int next0idx = swap0idx + ((swap0idx + 1) % 3 == 0 ? -2 : 1);
      const int next1idx = swap1idx + ((swap1idx + 1) % 3 == 0 ? -2 : 1);
      Halfedge& next0 = halfedges[next0idx];
      Halfedge& next1 = halfedges[next1idx];

      next0.startVert = swap0.endVert = next1.endVert;
      swap0.pairedHalfedge = next1.pairedHalfedge;
      halfedges[swap0.pairedHalfedge].pairedHalfedge = swap0idx;

      next1.startVert = swap1.endVert = next0.endVert;
      swap1.pairedHalfedge = next0.pairedHalfedge;
      halfedges[swap1.pairedHalfedge].pairedHalfedge = swap1idx;

      next0.pairedHalfedge = next1idx;
      next1.pairedHalfedge = next0idx;
    }
  }
};

struct InitializeBaryRef {
  const int meshID;
  const Halfedge* halfedge;

  __host__ __device__ void operator()(thrust::tuple<BaryRef&, int> inOut) {
    BaryRef& baryRef = thrust::get<0>(inOut);
    int tri = thrust::get<1>(inOut);

    // Leave existing meshID if input is negative
    if (meshID >= 0) baryRef.meshID = meshID;
    baryRef.face = tri;
    glm::ivec3 triVerts(0.0f);
    for (int i : {0, 1, 2}) triVerts[i] = halfedge[3 * tri + i].startVert;
    baryRef.verts = triVerts;
    baryRef.vertBary = {-1, -1, -1};
  }
};

struct CoplanarEdge {
  BaryRef* triBary;
  const Halfedge* halfedge;
  const glm::vec3* vertPos;
  const float precision;

  __host__ __device__ void operator()(int edgeIdx) {
    const Halfedge edge = halfedge[edgeIdx];
    if (!edge.IsForward()) return;
    const Halfedge pair = halfedge[edge.pairedHalfedge];
    const glm::vec3 base = vertPos[edge.startVert];

    const glm::vec3 jointVec = vertPos[edge.endVert] - base;
    const glm::vec3 edgeVec =
        vertPos[halfedge[NextHalfedge(edgeIdx)].endVert] - base;
    const glm::vec3 pairVec =
        vertPos[halfedge[NextHalfedge(edge.pairedHalfedge)].endVert] - base;

    const glm::vec3 cross = glm::cross(jointVec, edgeVec);
    const float area = glm::length(cross);
    const float areaPair = glm::length(glm::cross(pairVec, jointVec));
    const float volume = glm::abs(glm::dot(cross, pairVec));
    const float height = volume / glm::max(area, areaPair);
    // Only operate on coplanar triangles
    if (height > precision) return;

    const float length = glm::max(glm::length(edgeVec), glm::length(jointVec));
    const float lengthPair =
        glm::max(glm::length(pairVec), glm::length(jointVec));
    const bool edgeColinear = area < length * precision;
    const bool pairColinear = areaPair < lengthPair * precision;

    int& edgeFace = triBary[edge.face].face;
    int& pairFace = triBary[pair.face].face;
    // Point toward non-degenerate triangle
    if (edgeColinear && !pairColinear)
      edgeFace = pairFace;
    else if (pairColinear && !edgeColinear)
      pairFace = edgeFace;
    else {
      // Point toward lower index
      if (edgeFace < pairFace)
        pairFace = edgeFace;
      else
        edgeFace = pairFace;
    }
  }
};

struct EdgeBox {
  const glm::vec3* vertPos;

  __host__ __device__ void operator()(
      thrust::tuple<Box&, const TmpEdge&> inout) {
    const TmpEdge& edge = thrust::get<1>(inout);
    thrust::get<0>(inout) = Box(vertPos[edge.first], vertPos[edge.second]);
  }
};

struct CheckManifold {
  const Halfedge* halfedges;

  __host__ __device__ bool operator()(int edge) {
    const Halfedge halfedge = halfedges[edge];
    if (halfedge.startVert == -1 && halfedge.endVert == -1 &&
        halfedge.pairedHalfedge == -1)
      return true;

    const Halfedge paired = halfedges[halfedge.pairedHalfedge];
    bool good = true;
    good &= paired.pairedHalfedge == edge;
    good &= halfedge.startVert != halfedge.endVert;
    good &= halfedge.startVert == paired.endVert;
    good &= halfedge.endVert == paired.startVert;
    return good;
  }
};

struct NoDuplicates {
  const Halfedge* halfedges;

  __host__ __device__ bool operator()(int edge) {
    const Halfedge halfedge = halfedges[edge];
    if (halfedge.startVert == -1 && halfedge.endVert == -1 &&
        halfedge.pairedHalfedge == -1)
      return true;
    return halfedge.startVert != halfedges[edge + 1].startVert ||
           halfedge.endVert != halfedges[edge + 1].endVert;
  }
};

struct CheckCCW {
  const Halfedge* halfedges;
  const glm::vec3* vertPos;
  const glm::vec3* triNormal;
  const float tol;

  __host__ __device__ bool operator()(int face) {
    if (halfedges[3 * face].pairedHalfedge < 0) return true;

    const glm::mat3x2 projection = GetAxisAlignedProjection(triNormal[face]);
    glm::vec2 v[3];
    for (int i : {0, 1, 2})
      v[i] = projection * vertPos[halfedges[3 * face + i].startVert];

    int ccw = CCW(v[0], v[1], v[2], glm::abs(tol));
    bool check = tol > 0 ? ccw >= 0 : ccw == 0;

    if (tol > 0 && !check) {
      glm::vec2 v1 = v[1] - v[0];
      glm::vec2 v2 = v[2] - v[0];
      float area = v1.x * v2.y - v1.y * v2.x;
      float base2 = glm::max(glm::dot(v1, v1), glm::dot(v2, v2));
      float base = glm::sqrt(base2);
      glm::vec3 V0 = vertPos[halfedges[3 * face].startVert];
      glm::vec3 V1 = vertPos[halfedges[3 * face + 1].startVert];
      glm::vec3 V2 = vertPos[halfedges[3 * face + 2].startVert];
      glm::vec3 norm = glm::cross(V1 - V0, V2 - V0);
      printf(
          "Tri %d does not match normal, approx height = %g, base = %g\n"
          "tol = %g, area2 = %g, base2*tol2 = %g\n"
          "normal = %g, %g, %g\n"
          "norm = %g, %g, %g\nverts: %d, %d, %d\n",
          face, area / base, base, tol, area * area, base2 * tol * tol,
          triNormal[face].x, triNormal[face].y, triNormal[face].z, norm.x,
          norm.y, norm.z, halfedges[3 * face].startVert,
          halfedges[3 * face + 1].startVert, halfedges[3 * face + 2].startVert);
    }
    return check;
  }
};

}  // namespace

namespace manifold {

std::vector<int> Manifold::Impl::meshID2Original_;

/**
 * Create a manifold from an input triangle Mesh. Will throw if the Mesh is not
 * manifold. TODO: update halfedgeTangent during CollapseDegenerates.
 */
Manifold::Impl::Impl(const Mesh& mesh)
    : vertPos_(mesh.vertPos), halfedgeTangent_(mesh.halfedgeTangent) {
  CheckDevice();
  CalculateBBox();
  SetPrecision();
  CreateAndFixHalfedges(mesh.triVerts);
  InitializeNewReference();
  CalculateNormals();
  CollapseDegenerates();
  Finish();
}

/**
 * Create eiter a unit tetrahedron, cube or octahedron. The cube is in the first
 * octant, while the others are symmetric about the origin.
 */
Manifold::Impl::Impl(Shape shape) {
  std::vector<glm::vec3> vertPos;
  std::vector<glm::ivec3> triVerts;
  switch (shape) {
    case Shape::TETRAHEDRON:
      vertPos = {{-1.0f, -1.0f, 1.0f},
                 {-1.0f, 1.0f, -1.0f},
                 {1.0f, -1.0f, -1.0f},
                 {1.0f, 1.0f, 1.0f}};
      triVerts = {{2, 0, 1}, {0, 3, 1}, {2, 3, 0}, {3, 2, 1}};
      break;
    case Shape::CUBE:
      vertPos = {{0.0f, 0.0f, 0.0f},  //
                 {1.0f, 0.0f, 0.0f},  //
                 {1.0f, 1.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},  //
                 {1.0f, 0.0f, 1.0f},  //
                 {1.0f, 1.0f, 1.0f},  //
                 {0.0f, 1.0f, 1.0f}};
      triVerts = {{0, 2, 1}, {0, 3, 2},  //
                  {4, 5, 6}, {4, 6, 7},  //
                  {0, 1, 5}, {0, 5, 4},  //
                  {1, 2, 6}, {1, 6, 5},  //
                  {2, 3, 7}, {2, 7, 6},  //
                  {3, 0, 4}, {3, 4, 7}};
      break;
    case Shape::OCTAHEDRON:
      vertPos = {{1.0f, 0.0f, 0.0f},   //
                 {-1.0f, 0.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},   //
                 {0.0f, -1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},   //
                 {0.0f, 0.0f, -1.0f}};
      triVerts = {{0, 2, 4}, {1, 5, 3},  //
                  {2, 1, 4}, {3, 5, 0},  //
                  {1, 3, 4}, {0, 5, 2},  //
                  {3, 0, 4}, {2, 5, 1}};
      break;
    default:
      throw userErr("Unrecognized shape!");
  }
  vertPos_ = vertPos;
  CreateHalfedges(triVerts);
  Finish();
  InitializeNewReference();
  MergeCoplanarRelations();
}

/**
 * When a manifold is copied, it is given a new unique set of mesh relation IDs,
 * identifying a particular instance of a copied input mesh. The original mesh
 * ID can be found using the meshID2Original mapping.
 */
void Manifold::Impl::DuplicateMeshIDs() {
  std::map<int, int> old2new;
  for (BaryRef& ref : meshRelation_.triBary) {
    if (old2new.find(ref.meshID) == old2new.end()) {
      old2new[ref.meshID] = meshID2Original_.size();
      meshID2Original_.push_back(meshID2Original_[ref.meshID]);
    }
    ref.meshID = old2new[ref.meshID];
  }
}

void Manifold::Impl::ReinitializeReference(int meshID) {
  thrust::for_each_n(zip(meshRelation_.triBary.beginD(), countAt(0)), NumTri(),
                     InitializeBaryRef({meshID, halfedge_.cptrD()}));
}

int Manifold::Impl::InitializeNewReference() {
  meshRelation_.triBary.resize(NumTri());
  const int nextMeshID = meshID2Original_.size();
  meshID2Original_.push_back(nextMeshID);
  ReinitializeReference(nextMeshID);
  return nextMeshID;
}

void Manifold::Impl::MergeCoplanarRelations() {
  thrust::for_each_n(
      countAt(0), halfedge_.size(),
      CoplanarEdge({meshRelation_.triBary.ptrD(), halfedge_.cptrD(),
                    vertPos_.cptrD(), precision_}));

  VecH<BaryRef>& triBary = meshRelation_.triBary.H();
  std::stack<int> stack;
  for (int tri = 0; tri < NumTri(); ++tri) {
    int thisTri = tri;
    while (triBary[thisTri].face != thisTri) {
      stack.push(thisTri);
      thisTri = triBary[thisTri].face;
    }
    while (!stack.empty()) {
      triBary[stack.top()].face = thisTri;
      stack.pop();
    }
  }
}

/**
 * Create the halfedge_ data structure from an input triVerts array like Mesh.
 */
void Manifold::Impl::CreateHalfedges(const VecDH<glm::ivec3>& triVerts) {
  const int numTri = triVerts.size();
  halfedge_.resize(3 * numTri);
  VecDH<TmpEdge> edge(3 * numTri);
  thrust::for_each_n(zip(countAt(0), triVerts.beginD()), numTri,
                     Tri2Halfedges({halfedge_.ptrD(), edge.ptrD()}));
  thrust::sort(edge.beginD(), edge.endD());
  thrust::for_each_n(countAt(0), halfedge_.size() / 2,
                     LinkHalfedges({halfedge_.ptrD(), edge.cptrD()}));
}

/**
 * Create the halfedge_ data structure from an input triVerts array like Mesh.
 * Check that the input is an even-manifold, and if it is not 2-manifold,
 * perform edge swaps until it is. This is a host function.
 */
void Manifold::Impl::CreateAndFixHalfedges(const VecDH<glm::ivec3>& triVerts) {
  const int numTri = triVerts.size();
  halfedge_.resize(3 * numTri);
  VecDH<TmpEdge> edge(3 * numTri);
  thrust::for_each_n(zip(countAt(0), triVerts.begin()), numTri,
                     Tri2Halfedges({halfedge_.ptrH(), edge.ptrH()}));
  // Stable sort is required here so that halfedges from the same face are
  // paired together (the triangles were created in face order). In some
  // degenerate situations the triangulator can add the same internal edge in
  // two different faces, causing this edge to not be 2-manifold. We detect this
  // and fix it by swapping one of the identical edges, so it is important that
  // we have the edges paired according to their face.
  std::stable_sort(edge.begin(), edge.end());
  thrust::for_each_n(thrust::host, countAt(0), halfedge_.size() / 2,
                     LinkHalfedges({halfedge_.ptrH(), edge.cptrH()}));
  thrust::for_each(thrust::host, countAt(1), countAt(halfedge_.size() / 2),
                   SwapHalfedges({halfedge_.ptrH(), edge.cptrH()}));
}

/**
 * Once halfedge_ has been filled in, this function can be called to create the
 * rest of the internal data structures. This function also removes the verts
 * and halfedges flagged for removal (NaN verts and -1 halfedges).
 */
void Manifold::Impl::Finish() {
  if (halfedge_.size() == 0) return;

  CalculateBBox();
  SetPrecision(precision_);
  if (!bBox_.isFinite()) {
    vertPos_.resize(0);
    halfedge_.resize(0);
    faceNormal_.resize(0);
    return;
  }

  SortVerts();
  VecDH<Box> faceBox;
  VecDH<uint32_t> faceMorton;
  GetFaceBoxMorton(faceBox, faceMorton);
  SortFaces(faceBox, faceMorton);
  if (halfedge_.size() == 0) return;

  ALWAYS_ASSERT(halfedge_.size() % 6 == 0, topologyErr,
                "Not an even number of faces after sorting faces!");
  Halfedge extrema = {0, 0, 0, 0};
  extrema =
      thrust::reduce(halfedge_.beginD(), halfedge_.endD(), extrema, Extrema());

  ALWAYS_ASSERT(extrema.startVert >= 0, topologyErr,
                "Vertex index is negative!");
  ALWAYS_ASSERT(extrema.endVert < NumVert(), topologyErr,
                "Vertex index exceeds number of verts!");
  ALWAYS_ASSERT(extrema.face >= 0, topologyErr, "Face index is negative!");
  ALWAYS_ASSERT(extrema.face < NumTri(), topologyErr,
                "Face index exceeds number of faces!");
  ALWAYS_ASSERT(extrema.pairedHalfedge >= 0, topologyErr,
                "Halfedge index is negative!");
  ALWAYS_ASSERT(extrema.pairedHalfedge < 2 * NumEdge(), topologyErr,
                "Halfedge index exceeds number of halfedges!");

  CalculateNormals();
  collider_ = Collider(faceBox, faceMorton);
}

/**
 * Does a full recalculation of the face bounding boxes, including updating the
 * collider, but does not resort the faces.
 */
void Manifold::Impl::Update() {
  CalculateBBox();
  VecDH<Box> faceBox;
  VecDH<uint32_t> faceMorton;
  GetFaceBoxMorton(faceBox, faceMorton);
  collider_.UpdateBoxes(faceBox);
}

void Manifold::Impl::ApplyTransform() const {
  // This const_cast is here because these operations cancel out, leaving the
  // state conceptually unchanged. This enables lazy transformation evaluation.
  const_cast<Impl*>(this)->ApplyTransform();
}

/**
 * Bake the manifold's transform into its vertices. This function allows lazy
 * evaluation, which is important because often several transforms are applied
 * between operations.
 */
void Manifold::Impl::ApplyTransform() {
  if (transform_ == glm::mat4x3(1.0f)) return;
  thrust::for_each(vertPos_.beginD(), vertPos_.endD(),
                   Transform4x3({transform_}));

  glm::mat3 normalTransform =
      glm::inverse(glm::transpose(glm::mat3(transform_)));
  thrust::for_each(faceNormal_.beginD(), faceNormal_.endD(),
                   TransformNormals({normalTransform}));
  thrust::for_each(vertNormal_.beginD(), vertNormal_.endD(),
                   TransformNormals({normalTransform}));
  // This optimization does a cheap collider update if the transform is
  // axis-aligned.
  if (!collider_.Transform(transform_)) Update();

  const float oldScale = bBox_.Scale();
  transform_ = glm::mat4x3(1.0f);
  CalculateBBox();

  const float newScale = bBox_.Scale();
  precision_ *= glm::max(1.0f, newScale / oldScale) *
                glm::max(glm::length(transform_[0]),
                         glm::max(glm::length(transform_[1]),
                                  glm::length(transform_[2])));

  // Maximum of inherited precision loss and translational precision loss.
  SetPrecision(precision_);
}

/**
 * Returns true if this manifold is in fact an oriented 2-manifold and all of
 * the data structures are consistent.
 */
bool Manifold::Impl::IsManifold() const {
  if (halfedge_.size() == 0) return true;
  bool isManifold = thrust::all_of(countAt(0), countAt(halfedge_.size()),
                                   CheckManifold({halfedge_.cptrD()}));

  VecDH<Halfedge> halfedge(halfedge_);
  thrust::sort(halfedge.beginD(), halfedge.endD());
  isManifold &= thrust::all_of(countAt(0), countAt(2 * NumEdge() - 1),
                               NoDuplicates({halfedge.cptrD()}));
  return isManifold;
}

/**
 * Returns true if all triangles are CCW relative to their triNormals_.
 */
bool Manifold::Impl::MatchesTriNormals() const {
  if (halfedge_.size() == 0 || faceNormal_.size() != NumTri()) return true;
  return thrust::all_of(thrust::device, countAt(0), countAt(NumTri()),
                        CheckCCW({halfedge_.cptrD(), vertPos_.cptrD(),
                                  faceNormal_.cptrD(), 2 * precision_}));
}

/**
 * Returns the number of triangles that are colinear within precision_.
 */
int Manifold::Impl::NumDegenerateTris() const {
  if (halfedge_.size() == 0 || faceNormal_.size() != NumTri()) return true;
  return thrust::count_if(thrust::device, countAt(0), countAt(NumTri()),
                          CheckCCW({halfedge_.cptrD(), vertPos_.cptrD(),
                                    faceNormal_.cptrD(), -1 * precision_ / 2}));
}

Properties Manifold::Impl::GetProperties() const {
  if (IsEmpty()) return {0, 0};
  ApplyTransform();
  thrust::pair<float, float> areaVolume = thrust::transform_reduce(
      countAt(0), countAt(NumTri()),
      FaceAreaVolume({halfedge_.cptrD(), vertPos_.cptrD(), precision_}),
      thrust::make_pair(0.0f, 0.0f), SumPair());
  return {areaVolume.first, areaVolume.second};
}

Curvature Manifold::Impl::GetCurvature() const {
  Curvature result;
  if (IsEmpty()) return result;
  ApplyTransform();
  VecDH<float> vertMeanCurvature(NumVert(), 0);
  VecDH<float> vertGaussianCurvature(NumVert(), glm::two_pi<float>());
  VecDH<float> vertArea(NumVert(), 0);
  VecDH<float> degree(NumVert(), 0);
  thrust::for_each(
      countAt(0), countAt(NumTri()),
      CurvatureAngles({vertMeanCurvature.ptrD(), vertGaussianCurvature.ptrD(),
                       vertArea.ptrD(), degree.ptrD(), halfedge_.cptrD(),
                       vertPos_.cptrD(), faceNormal_.cptrD()}));
  thrust::for_each_n(
      zip(vertMeanCurvature.beginD(), vertGaussianCurvature.beginD(),
          vertArea.beginD(), degree.beginD()),
      NumVert(), NormalizeCurvature());
  result.minMeanCurvature =
      thrust::reduce(vertMeanCurvature.beginD(), vertMeanCurvature.endD(),
                     1.0f / 0.0f, thrust::minimum<float>());
  result.maxMeanCurvature =
      thrust::reduce(vertMeanCurvature.beginD(), vertMeanCurvature.endD(),
                     -1.0f / 0.0f, thrust::maximum<float>());
  result.minGaussianCurvature = thrust::reduce(
      vertGaussianCurvature.beginD(), vertGaussianCurvature.endD(), 1.0f / 0.0f,
      thrust::minimum<float>());
  result.maxGaussianCurvature = thrust::reduce(
      vertGaussianCurvature.beginD(), vertGaussianCurvature.endD(),
      -1.0f / 0.0f, thrust::maximum<float>());
  result.vertMeanCurvature.insert(result.vertMeanCurvature.end(),
                                  vertMeanCurvature.begin(),
                                  vertMeanCurvature.end());
  result.vertGaussianCurvature.insert(result.vertGaussianCurvature.end(),
                                      vertGaussianCurvature.begin(),
                                      vertGaussianCurvature.end());
  return result;
}

/**
 * Calculates the bounding box of the entire manifold, which is stored
 * internally to short-cut Boolean operations and to serve as the precision
 * range for Morton code calculation.
 */
void Manifold::Impl::CalculateBBox() {
  bBox_.min = thrust::reduce(vertPos_.beginD(), vertPos_.endD(),
                             glm::vec3(1 / 0.0f), PosMin());
  bBox_.max = thrust::reduce(vertPos_.beginD(), vertPos_.endD(),
                             glm::vec3(-1 / 0.0f), PosMax());
}

/**
 * Sets the precision based on the bounding box, and limits its minimum value by
 * the optional input.
 */
void Manifold::Impl::SetPrecision(float minPrecision) {
  precision_ = glm::max(minPrecision, kTolerance * bBox_.Scale());
  if (!glm::isfinite(precision_)) precision_ = -1;
}

/**
 * Sorts the vertices according to their Morton code.
 */
void Manifold::Impl::SortVerts() {
  VecDH<uint32_t> vertMorton(NumVert());
  thrust::for_each_n(zip(vertMorton.beginD(), vertPos_.cbeginD()), NumVert(),
                     Morton({bBox_}));

  VecDH<int> vertNew2Old(NumVert());
  thrust::sequence(vertNew2Old.beginD(), vertNew2Old.endD());
  thrust::sort_by_key(vertMorton.beginD(), vertMorton.endD(),
                      zip(vertPos_.beginD(), vertNew2Old.beginD()));

  ReindexVerts(vertNew2Old, NumVert());

  // Verts were flagged for removal with NaNs and assigned kNoCode to sort them
  // to the end, which allows them to be removed.
  const int newNumVert =
      thrust::find(vertMorton.beginD(), vertMorton.endD(), kNoCode) -
      vertMorton.beginD();
  vertPos_.resize(newNumVert);
}

/**
 * Updates the halfedges to point to new vert indices based on a mapping,
 * vertNew2Old. This may be a subset, so the total number of original verts is
 * also given.
 */
void Manifold::Impl::ReindexVerts(const VecDH<int>& vertNew2Old,
                                  int oldNumVert) {
  VecDH<int> vertOld2New(oldNumVert);
  thrust::scatter(countAt(0), countAt(NumVert()), vertNew2Old.beginD(),
                  vertOld2New.beginD());
  thrust::for_each(halfedge_.beginD(), halfedge_.endD(),
                   Reindex({vertOld2New.cptrD()}));
}

/**
 * Fills the faceBox and faceMorton input with the bounding boxes and Morton
 * codes of the faces, respectively. The Morton code is based on the center of
 * the bounding box.
 */
void Manifold::Impl::GetFaceBoxMorton(VecDH<Box>& faceBox,
                                      VecDH<uint32_t>& faceMorton) const {
  faceBox.resize(NumTri());
  faceMorton.resize(NumTri());
  thrust::for_each_n(
      zip(faceMorton.beginD(), faceBox.beginD(), countAt(0)), NumTri(),
      FaceMortonBox({halfedge_.cptrD(), vertPos_.cptrD(), bBox_}));
}

/**
 * Sorts the faces of this manifold according to their input Morton code. The
 * bounding box and Morton code arrays are also sorted accordingly.
 */
void Manifold::Impl::SortFaces(VecDH<Box>& faceBox,
                               VecDH<uint32_t>& faceMorton) {
  VecDH<int> faceNew2Old(NumTri());
  thrust::sequence(faceNew2Old.beginD(), faceNew2Old.endD());

  thrust::sort_by_key(faceMorton.beginD(), faceMorton.endD(),
                      zip(faceBox.beginD(), faceNew2Old.beginD()));

  // Tris were flagged for removal with pairedHalfedge = -1 and assigned kNoCode
  // to sort them to the end, which allows them to be removed.
  const int newNumTri =
      thrust::find(faceMorton.beginD(), faceMorton.endD(), kNoCode) -
      faceMorton.beginD();
  faceBox.resize(newNumTri);
  faceMorton.resize(newNumTri);
  faceNew2Old.resize(newNumTri);

  GatherFaces(faceNew2Old);
}

/**
 * Creates the halfedge_ vector for this manifold by copying a set of faces from
 * another manifold, given by oldHalfedge. Input faceNew2Old defines the old
 * faces to gather into this.
 */
void Manifold::Impl::GatherFaces(const VecDH<int>& faceNew2Old) {
  const int numTri = faceNew2Old.size();
  if (meshRelation_.triBary.size() == NumTri())
    Permute(meshRelation_.triBary, faceNew2Old);

  if (faceNormal_.size() == NumTri()) Permute(faceNormal_, faceNew2Old);

  VecDH<Halfedge> oldHalfedge(halfedge_);
  VecDH<glm::vec4> oldHalfedgeTangent(halfedgeTangent_);
  VecDH<int> faceOld2New(oldHalfedge.size() / 3);
  thrust::scatter(countAt(0), countAt(numTri), faceNew2Old.beginD(),
                  faceOld2New.beginD());

  halfedge_.resize(3 * numTri);
  if (oldHalfedgeTangent.size() != 0) halfedgeTangent_.resize(3 * numTri);
  thrust::for_each_n(
      countAt(0), numTri,
      ReindexFace({halfedge_.ptrD(), halfedgeTangent_.ptrD(),
                   oldHalfedge.cptrD(), oldHalfedgeTangent.cptrD(),
                   faceNew2Old.cptrD(), faceOld2New.cptrD()}));
}

void Manifold::Impl::GatherFaces(const Impl& old,
                                 const VecDH<int>& faceNew2Old) {
  const int numTri = faceNew2Old.size();
  meshRelation_.triBary.resize(numTri);
  thrust::gather(faceNew2Old.beginD(), faceNew2Old.endD(),
                 old.meshRelation_.triBary.beginD(),
                 meshRelation_.triBary.beginD());
  meshRelation_.barycentric = old.meshRelation_.barycentric;
  DuplicateMeshIDs();

  if (old.faceNormal_.size() == old.NumTri()) {
    faceNormal_.resize(numTri);
    thrust::gather(faceNew2Old.beginD(), faceNew2Old.endD(),
                   old.faceNormal_.beginD(), faceNormal_.beginD());
  }

  VecDH<int> faceOld2New(old.NumTri());
  thrust::scatter(countAt(0), countAt(numTri), faceNew2Old.beginD(),
                  faceOld2New.beginD());

  halfedge_.resize(3 * numTri);
  if (old.halfedgeTangent_.size() != 0) halfedgeTangent_.resize(3 * numTri);
  thrust::for_each_n(
      countAt(0), numTri,
      ReindexFace({halfedge_.ptrD(), halfedgeTangent_.ptrD(),
                   old.halfedge_.cptrD(), old.halfedgeTangent_.cptrD(),
                   faceNew2Old.cptrD(), faceOld2New.cptrD()}));
}

/**
 * If face normals are already present, this function uses them to compute
 * vertex normals (angle-weighted pseudo-normals); otherwise it also computes
 * the face normals. Face normals are only calculated when needed because nearly
 * degenerate faces will accrue rounding error, while the Boolean can retain
 * their original normal, which is more accurate and can help with merging
 * coplanar faces.
 *
 * If the face normals have been invalidated by an operation like Warp(), ensure
 * you do faceNormal_.resize(0) before calling this function to force
 * recalculation.
 */
void Manifold::Impl::CalculateNormals() {
  vertNormal_.resize(NumVert());
  thrust::fill(vertNormal_.beginD(), vertNormal_.endD(), glm::vec3(0));
  bool calculateTriNormal = false;
  if (faceNormal_.size() != NumTri()) {
    faceNormal_.resize(NumTri());
    calculateTriNormal = true;
  }
  thrust::for_each_n(
      zip(faceNormal_.beginD(), countAt(0)), NumTri(),
      AssignNormals({vertNormal_.ptrD(), vertPos_.cptrD(), halfedge_.cptrD(),
                     precision_, calculateTriNormal}));
  thrust::for_each(vertNormal_.beginD(), vertNormal_.endD(), Normalize());
}

/**
 * Returns a sparse array of the bounding box overlaps between the edges of the
 * input manifold, Q and the faces of this manifold. Returned indices only
 * point to forward halfedges.
 */
SparseIndices Manifold::Impl::EdgeCollisions(const Impl& Q) const {
  VecDH<TmpEdge> edges = CreateTmpEdges(Q.halfedge_);
  const int numEdge = edges.size();
  VecDH<Box> QedgeBB(numEdge);
  thrust::for_each_n(zip(QedgeBB.beginD(), edges.cbeginD()), numEdge,
                     EdgeBox({Q.vertPos_.cptrD()}));

  SparseIndices q1p2 = collider_.Collisions(QedgeBB);

  thrust::for_each(q1p2.beginD(0), q1p2.endD(0), ReindexEdge({edges.cptrD()}));
  return q1p2;
}

/**
 * Returns a sparse array of the input vertices that project inside the XY
 * bounding boxes of the faces of this manifold.
 */
SparseIndices Manifold::Impl::VertexCollisionsZ(
    const VecDH<glm::vec3>& vertsIn) const {
  return collider_.Collisions(vertsIn);
}
}  // namespace manifold