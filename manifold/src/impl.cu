#include "hip/hip_runtime.h"
// Copyright 2021 Emmett Lalish
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/execution_policy.h>

#include <algorithm>
#include <map>
#include <stack>

#include "impl.cuh"

namespace {
using namespace manifold;

__host__ __device__ void AtomicAddVec3(glm::vec3& target,
                                       const glm::vec3& add) {
  for (int i : {0, 1, 2}) {
#ifdef __CUDA_ARCH__
    atomicAdd(&target[i], add[i]);
#else
#pragma omp atomic
    target[i] += add[i];
#endif
  }
}

struct Normalize {
  __host__ __device__ void operator()(glm::vec3& v) { v = SafeNormalize(v); }
};

struct Transform4x3 {
  const glm::mat4x3 transform;

  __host__ __device__ void operator()(glm::vec3& position) {
    position = transform * glm::vec4(position, 1.0f);
  }
};

struct TransformNormals {
  const glm::mat3 transform;

  __host__ __device__ void operator()(glm::vec3& normal) {
    normal = glm::normalize(transform * normal);
    if (isnan(normal.x)) normal = glm::vec3(0.0f);
  }
};

struct AssignNormals {
  glm::vec3* vertNormal;
  const glm::vec3* vertPos;
  const Halfedge* halfedges;
  const float precision;
  const bool calculateTriNormal;

  __host__ __device__ void operator()(thrust::tuple<glm::vec3&, int> in) {
    glm::vec3& triNormal = thrust::get<0>(in);
    const int face = thrust::get<1>(in);

    glm::ivec3 triVerts;
    for (int i : {0, 1, 2}) triVerts[i] = halfedges[3 * face + i].startVert;

    glm::vec3 edge[3];
    for (int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      edge[i] = glm::normalize(vertPos[triVerts[j]] - vertPos[triVerts[i]]);
    }

    if (calculateTriNormal) {
      triNormal = glm::normalize(glm::cross(edge[0], edge[1]));
      if (isnan(triNormal.x)) triNormal = glm::vec3(0, 0, 1);
    }

    // corner angles
    glm::vec3 phi;
    float dot = -glm::dot(edge[2], edge[0]);
    phi[0] = dot >= 1 ? 0 : (dot <= -1 ? glm::pi<float>() : glm::acos(dot));
    dot = -glm::dot(edge[0], edge[1]);
    phi[1] = dot >= 1 ? 0 : (dot <= -1 ? glm::pi<float>() : glm::acos(dot));
    phi[2] = glm::pi<float>() - phi[0] - phi[1];

    // assign weighted sum
    for (int i : {0, 1, 2}) {
      AtomicAddVec3(vertNormal[triVerts[i]], phi[i] * triNormal);
    }
  }
};

struct Tri2Halfedges {
  Halfedge* halfedges;
  TmpEdge* edges;

  __host__ __device__ void operator()(
      thrust::tuple<int, const glm::ivec3&> in) {
    const int tri = thrust::get<0>(in);
    const glm::ivec3& triVerts = thrust::get<1>(in);
    for (const int i : {0, 1, 2}) {
      const int j = (i + 1) % 3;
      const int edge = 3 * tri + i;
      halfedges[edge] = {triVerts[i], triVerts[j], -1, tri};
      edges[edge] = TmpEdge(triVerts[i], triVerts[j], edge);
    }
  }
};

struct LinkHalfedges {
  Halfedge* halfedges;
  const TmpEdge* edges;

  __host__ __device__ void operator()(int k) {
    const int i = 2 * k;
    const int j = i + 1;
    const int pair0 = edges[i].halfedgeIdx;
    const int pair1 = edges[j].halfedgeIdx;
    if (halfedges[pair0].startVert != halfedges[pair1].endVert ||
        halfedges[pair0].endVert != halfedges[pair1].startVert ||
        halfedges[pair0].face == halfedges[pair1].face)
      printf("Not manifold!\n");
    halfedges[pair0].pairedHalfedge = pair1;
    halfedges[pair1].pairedHalfedge = pair0;
  }
};

struct SwapHalfedges {
  Halfedge* halfedges;
  const TmpEdge* edges;

  __host__ void operator()(int k) {
    const int i = 2 * k;
    const int j = i - 2;
    const TmpEdge thisEdge = edges[i];
    const TmpEdge lastEdge = edges[j];
    if (thisEdge.first == lastEdge.first &&
        thisEdge.second == lastEdge.second) {
      const int swap0idx = thisEdge.halfedgeIdx;
      Halfedge& swap0 = halfedges[swap0idx];
      const int swap1idx = swap0.pairedHalfedge;
      Halfedge& swap1 = halfedges[swap1idx];

      const int next0idx = swap0idx + ((swap0idx + 1) % 3 == 0 ? -2 : 1);
      const int next1idx = swap1idx + ((swap1idx + 1) % 3 == 0 ? -2 : 1);
      Halfedge& next0 = halfedges[next0idx];
      Halfedge& next1 = halfedges[next1idx];

      next0.startVert = swap0.endVert = next1.endVert;
      swap0.pairedHalfedge = next1.pairedHalfedge;
      halfedges[swap0.pairedHalfedge].pairedHalfedge = swap0idx;

      next1.startVert = swap1.endVert = next0.endVert;
      swap1.pairedHalfedge = next0.pairedHalfedge;
      halfedges[swap1.pairedHalfedge].pairedHalfedge = swap1idx;

      next0.pairedHalfedge = next1idx;
      next1.pairedHalfedge = next0idx;
    }
  }
};

struct InitializeBaryRef {
  const int meshID;
  const Halfedge* halfedge;

  __host__ __device__ void operator()(thrust::tuple<BaryRef&, int> inOut) {
    BaryRef& baryRef = thrust::get<0>(inOut);
    int tri = thrust::get<1>(inOut);

    // Leave existing meshID if input is negative
    if (meshID >= 0) baryRef.meshID = meshID;
    baryRef.face = tri;
    glm::ivec3 triVerts(0.0f);
    for (int i : {0, 1, 2}) triVerts[i] = halfedge[3 * tri + i].startVert;
    baryRef.verts = triVerts;
    baryRef.vertBary = {-1, -1, -1};
  }
};

struct CoplanarEdge {
  BaryRef* triBary;
  const Halfedge* halfedge;
  const glm::vec3* vertPos;
  const float precision;

  __host__ __device__ void operator()(int edgeIdx) {
    const Halfedge edge = halfedge[edgeIdx];
    if (!edge.IsForward()) return;
    const Halfedge pair = halfedge[edge.pairedHalfedge];
    const glm::vec3 base = vertPos[edge.startVert];

    const glm::vec3 jointVec = vertPos[edge.endVert] - base;
    const glm::vec3 edgeVec =
        vertPos[halfedge[NextHalfedge(edgeIdx)].endVert] - base;
    const glm::vec3 pairVec =
        vertPos[halfedge[NextHalfedge(edge.pairedHalfedge)].endVert] - base;

    const glm::vec3 cross = glm::cross(jointVec, edgeVec);
    const float area = glm::length(cross);
    const float areaPair = glm::length(glm::cross(pairVec, jointVec));
    const float volume = glm::abs(glm::dot(cross, pairVec));
    const float height = volume / glm::max(area, areaPair);
    // Only operate on coplanar triangles
    if (height > precision) return;

    const float length = glm::max(glm::length(edgeVec), glm::length(jointVec));
    const float lengthPair =
        glm::max(glm::length(pairVec), glm::length(jointVec));
    const bool edgeColinear = area < length * precision;
    const bool pairColinear = areaPair < lengthPair * precision;

    int& edgeFace = triBary[edge.face].face;
    int& pairFace = triBary[pair.face].face;
    // Point toward non-degenerate triangle
    if (edgeColinear && !pairColinear)
      edgeFace = pairFace;
    else if (pairColinear && !edgeColinear)
      pairFace = edgeFace;
    else {
      // Point toward lower index
      if (edgeFace < pairFace)
        pairFace = edgeFace;
      else
        edgeFace = pairFace;
    }
  }
};

struct EdgeBox {
  const glm::vec3* vertPos;

  __host__ __device__ void operator()(
      thrust::tuple<Box&, const TmpEdge&> inout) {
    const TmpEdge& edge = thrust::get<1>(inout);
    thrust::get<0>(inout) = Box(vertPos[edge.first], vertPos[edge.second]);
  }
};
}  // namespace

namespace manifold {

std::vector<int> Manifold::Impl::meshID2Original_;

/**
 * Create a manifold from an input triangle Mesh. Will throw if the Mesh is not
 * manifold. TODO: update halfedgeTangent during CollapseDegenerates.
 */
Manifold::Impl::Impl(const Mesh& mesh)
    : vertPos_(mesh.vertPos), halfedgeTangent_(mesh.halfedgeTangent) {
  CheckDevice();
  CalculateBBox();
  SetPrecision();
  CreateAndFixHalfedges(mesh.triVerts);
  InitializeNewReference();
  CalculateNormals();
  CollapseDegenerates();
  Finish();
}

/**
 * Create eiter a unit tetrahedron, cube or octahedron. The cube is in the first
 * octant, while the others are symmetric about the origin.
 */
Manifold::Impl::Impl(Shape shape) {
  std::vector<glm::vec3> vertPos;
  std::vector<glm::ivec3> triVerts;
  switch (shape) {
    case Shape::TETRAHEDRON:
      vertPos = {{-1.0f, -1.0f, 1.0f},
                 {-1.0f, 1.0f, -1.0f},
                 {1.0f, -1.0f, -1.0f},
                 {1.0f, 1.0f, 1.0f}};
      triVerts = {{2, 0, 1}, {0, 3, 1}, {2, 3, 0}, {3, 2, 1}};
      break;
    case Shape::CUBE:
      vertPos = {{0.0f, 0.0f, 0.0f},  //
                 {1.0f, 0.0f, 0.0f},  //
                 {1.0f, 1.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},  //
                 {1.0f, 0.0f, 1.0f},  //
                 {1.0f, 1.0f, 1.0f},  //
                 {0.0f, 1.0f, 1.0f}};
      triVerts = {{0, 2, 1}, {0, 3, 2},  //
                  {4, 5, 6}, {4, 6, 7},  //
                  {0, 1, 5}, {0, 5, 4},  //
                  {1, 2, 6}, {1, 6, 5},  //
                  {2, 3, 7}, {2, 7, 6},  //
                  {3, 0, 4}, {3, 4, 7}};
      break;
    case Shape::OCTAHEDRON:
      vertPos = {{1.0f, 0.0f, 0.0f},   //
                 {-1.0f, 0.0f, 0.0f},  //
                 {0.0f, 1.0f, 0.0f},   //
                 {0.0f, -1.0f, 0.0f},  //
                 {0.0f, 0.0f, 1.0f},   //
                 {0.0f, 0.0f, -1.0f}};
      triVerts = {{0, 2, 4}, {1, 5, 3},  //
                  {2, 1, 4}, {3, 5, 0},  //
                  {1, 3, 4}, {0, 5, 2},  //
                  {3, 0, 4}, {2, 5, 1}};
      break;
    default:
      throw userErr("Unrecognized shape!");
  }
  vertPos_ = vertPos;
  CreateHalfedges(triVerts);
  Finish();
  InitializeNewReference();
  MergeCoplanarRelations();
}

/**
 * When a manifold is copied, it is given a new unique set of mesh relation IDs,
 * identifying a particular instance of a copied input mesh. The original mesh
 * ID can be found using the meshID2Original mapping.
 */
void Manifold::Impl::DuplicateMeshIDs() {
  std::map<int, int> old2new;
  for (BaryRef& ref : meshRelation_.triBary) {
    if (old2new.find(ref.meshID) == old2new.end()) {
      old2new[ref.meshID] = meshID2Original_.size();
      meshID2Original_.push_back(meshID2Original_[ref.meshID]);
    }
    ref.meshID = old2new[ref.meshID];
  }
}

void Manifold::Impl::ReinitializeReference(int meshID) {
  thrust::for_each_n(zip(meshRelation_.triBary.beginD(), countAt(0)), NumTri(),
                     InitializeBaryRef({meshID, halfedge_.cptrD()}));
}

int Manifold::Impl::InitializeNewReference() {
  meshRelation_.triBary.resize(NumTri());
  const int nextMeshID = meshID2Original_.size();
  meshID2Original_.push_back(nextMeshID);
  ReinitializeReference(nextMeshID);
  return nextMeshID;
}

void Manifold::Impl::MergeCoplanarRelations() {
  thrust::for_each_n(
      countAt(0), halfedge_.size(),
      CoplanarEdge({meshRelation_.triBary.ptrD(), halfedge_.cptrD(),
                    vertPos_.cptrD(), precision_}));

  VecH<BaryRef>& triBary = meshRelation_.triBary.H();
  std::stack<int> stack;
  for (int tri = 0; tri < NumTri(); ++tri) {
    int thisTri = tri;
    while (triBary[thisTri].face != thisTri) {
      stack.push(thisTri);
      thisTri = triBary[thisTri].face;
    }
    while (!stack.empty()) {
      triBary[stack.top()].face = thisTri;
      stack.pop();
    }
  }
}

/**
 * Create the halfedge_ data structure from an input triVerts array like Mesh.
 */
void Manifold::Impl::CreateHalfedges(const VecDH<glm::ivec3>& triVerts) {
  const int numTri = triVerts.size();
  halfedge_.resize(3 * numTri);
  VecDH<TmpEdge> edge(3 * numTri);
  thrust::for_each_n(zip(countAt(0), triVerts.beginD()), numTri,
                     Tri2Halfedges({halfedge_.ptrD(), edge.ptrD()}));
  thrust::sort(edge.beginD(), edge.endD());
  thrust::for_each_n(countAt(0), halfedge_.size() / 2,
                     LinkHalfedges({halfedge_.ptrD(), edge.cptrD()}));
}

/**
 * Create the halfedge_ data structure from an input triVerts array like Mesh.
 * Check that the input is an even-manifold, and if it is not 2-manifold,
 * perform edge swaps until it is. This is a host function.
 */
void Manifold::Impl::CreateAndFixHalfedges(const VecDH<glm::ivec3>& triVerts) {
  const int numTri = triVerts.size();
  halfedge_.resize(3 * numTri);
  VecDH<TmpEdge> edge(3 * numTri);
  thrust::for_each_n(zip(countAt(0), triVerts.begin()), numTri,
                     Tri2Halfedges({halfedge_.ptrH(), edge.ptrH()}));
  // Stable sort is required here so that halfedges from the same face are
  // paired together (the triangles were created in face order). In some
  // degenerate situations the triangulator can add the same internal edge in
  // two different faces, causing this edge to not be 2-manifold. We detect this
  // and fix it by swapping one of the identical edges, so it is important that
  // we have the edges paired according to their face.
  std::stable_sort(edge.begin(), edge.end());
  thrust::for_each_n(thrust::host, countAt(0), halfedge_.size() / 2,
                     LinkHalfedges({halfedge_.ptrH(), edge.cptrH()}));
  thrust::for_each(thrust::host, countAt(1), countAt(halfedge_.size() / 2),
                   SwapHalfedges({halfedge_.ptrH(), edge.cptrH()}));
}

/**
 * Does a full recalculation of the face bounding boxes, including updating the
 * collider, but does not resort the faces.
 */
void Manifold::Impl::Update() {
  CalculateBBox();
  VecDH<Box> faceBox;
  VecDH<uint32_t> faceMorton;
  GetFaceBoxMorton(faceBox, faceMorton);
  collider_.UpdateBoxes(faceBox);
}

void Manifold::Impl::ApplyTransform() const {
  // This const_cast is here because these operations cancel out, leaving the
  // state conceptually unchanged. This enables lazy transformation evaluation.
  const_cast<Impl*>(this)->ApplyTransform();
}

/**
 * Bake the manifold's transform into its vertices. This function allows lazy
 * evaluation, which is important because often several transforms are applied
 * between operations.
 */
void Manifold::Impl::ApplyTransform() {
  if (transform_ == glm::mat4x3(1.0f)) return;
  thrust::for_each(vertPos_.beginD(), vertPos_.endD(),
                   Transform4x3({transform_}));

  glm::mat3 normalTransform =
      glm::inverse(glm::transpose(glm::mat3(transform_)));
  thrust::for_each(faceNormal_.beginD(), faceNormal_.endD(),
                   TransformNormals({normalTransform}));
  thrust::for_each(vertNormal_.beginD(), vertNormal_.endD(),
                   TransformNormals({normalTransform}));
  // This optimization does a cheap collider update if the transform is
  // axis-aligned.
  if (!collider_.Transform(transform_)) Update();

  const float oldScale = bBox_.Scale();
  transform_ = glm::mat4x3(1.0f);
  CalculateBBox();

  const float newScale = bBox_.Scale();
  precision_ *= glm::max(1.0f, newScale / oldScale) *
                glm::max(glm::length(transform_[0]),
                         glm::max(glm::length(transform_[1]),
                                  glm::length(transform_[2])));

  // Maximum of inherited precision loss and translational precision loss.
  SetPrecision(precision_);
}

/**
 * Sets the precision based on the bounding box, and limits its minimum value by
 * the optional input.
 */
void Manifold::Impl::SetPrecision(float minPrecision) {
  precision_ = glm::max(minPrecision, kTolerance * bBox_.Scale());
  if (!glm::isfinite(precision_)) precision_ = -1;
}

/**
 * If face normals are already present, this function uses them to compute
 * vertex normals (angle-weighted pseudo-normals); otherwise it also computes
 * the face normals. Face normals are only calculated when needed because nearly
 * degenerate faces will accrue rounding error, while the Boolean can retain
 * their original normal, which is more accurate and can help with merging
 * coplanar faces.
 *
 * If the face normals have been invalidated by an operation like Warp(), ensure
 * you do faceNormal_.resize(0) before calling this function to force
 * recalculation.
 */
void Manifold::Impl::CalculateNormals() {
  vertNormal_.resize(NumVert());
  thrust::fill(vertNormal_.beginD(), vertNormal_.endD(), glm::vec3(0));
  bool calculateTriNormal = false;
  if (faceNormal_.size() != NumTri()) {
    faceNormal_.resize(NumTri());
    calculateTriNormal = true;
  }
  thrust::for_each_n(
      zip(faceNormal_.beginD(), countAt(0)), NumTri(),
      AssignNormals({vertNormal_.ptrD(), vertPos_.cptrD(), halfedge_.cptrD(),
                     precision_, calculateTriNormal}));
  thrust::for_each(vertNormal_.beginD(), vertNormal_.endD(), Normalize());
}

/**
 * Returns a sparse array of the bounding box overlaps between the edges of the
 * input manifold, Q and the faces of this manifold. Returned indices only
 * point to forward halfedges.
 */
SparseIndices Manifold::Impl::EdgeCollisions(const Impl& Q) const {
  VecDH<TmpEdge> edges = CreateTmpEdges(Q.halfedge_);
  const int numEdge = edges.size();
  VecDH<Box> QedgeBB(numEdge);
  thrust::for_each_n(zip(QedgeBB.beginD(), edges.cbeginD()), numEdge,
                     EdgeBox({Q.vertPos_.cptrD()}));

  SparseIndices q1p2 = collider_.Collisions(QedgeBB);

  thrust::for_each(q1p2.beginD(0), q1p2.endD(0), ReindexEdge({edges.cptrD()}));
  return q1p2;
}

/**
 * Returns a sparse array of the input vertices that project inside the XY
 * bounding boxes of the faces of this manifold.
 */
SparseIndices Manifold::Impl::VertexCollisionsZ(
    const VecDH<glm::vec3>& vertsIn) const {
  return collider_.Collisions(vertsIn);
}
}  // namespace manifold